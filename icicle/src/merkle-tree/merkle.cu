#include "hip/hip_runtime.h"
#include "hash/hash.cuh"
#include "merkle-tree/merkle.cuh"

namespace merkle_tree {
  /// Constructs merkle subtree without parallelization
  /// The digests are aligned sequentially per row
  /// Example:
  ///
  /// Big tree:
  ///
  ///        1      <- Root
  ///       / \     <- Arity = 2
  ///      2   3    <- Digests
  ///     / \ / \   <- Height = 2 (as the number of edges)
  ///    4  5 6  7  <- height^arity leaves
  ///    |  | |  |  <- Bottom layer hash 1 to 1
  ///    a  b c  d  <- Input vector 1x4
  ///
  /// Subtree 1    Subtree 2
  ///    2            3
  ///   / \          / \
  ///  4   5        6   7
  ///
  /// Digests array for subtree 1:
  /// [4 5 . . 2 . .]
  /// |   |    |
  /// -----    V
  ///   |    Segment (offset = 4, subtree_idx = 0)
  ///   v
  /// Segment (offset = 0, subtree_idx = 0)
  ///
  /// Digests array for subtree 2:
  /// [. . 6 7 . 3 .]
  ///     |   |
  ///     -----
  ///       |
  ///       v
  ///    Segment (offset = 0, subtree_idx = 1)
  ///
  /// Total digests array:
  /// [4 5 6 7 2 3 .]
  ///
  /// Example for custom config:
  ///
  /// arity = 2
  /// input_block_len = 2
  /// digest_elements = 2
  /// bottom_layer hash width = 4
  /// compression width = 4
  /// height = 2
  ///
  ///                    [a, b]    <- Root of the tree
  ///                     |  |
  ///                    [a, b, c, d]
  ///                     /  \  /  \ 
  ///                    [i, j, m, n]
  ///           ┌──┬──────┴──┴──┴──┴──────┬──┐
  ///           |  |                      |  |
  ///          [i, j, k, l]              [m, n, o, p]       <- compression states
  ///           /  \  /  \                /  \  /  \        <- Running permutation
  ///          [1, 2, 5, 6]              [9, 1, 4, 5]       <- compression states
  ///    ┌──┬───┴──┴──┼──┤         ┌──┬───┴──┴──┼──┤
  ///    |  |         |  |         |  |         |  |        <- digest_element * height^arity leaves
  ///   [1, 2, 3, 4] [5, 6, 7, 8] [9, 1, 2, 3] [4, 5, 6, 7] <- Permuted states
  ///    /  \  /  \   /  \  /  \   /  \  /  \   /  \  /  \  <- Running permutation
  ///   [a, b, 0, 0] [c, d, 0, 0] [e, f, 0, 0] [g, h, 0, 0] <- States of the bottom layer hash
  ///    |  |         |  |         |  |         |  |        <- Bottom layer hash 2 to 2
  ///    a  b         c  d         e  f         g  h        <- Input vector 2x4
  ///
  /// Input matrix:
  ///   ┌     ┐
  ///   | a b |
  ///   | c d |
  ///   | e f |
  ///   | g h |
  ///   └     ┘

  template <typename L, typename D>
  hipError_t build_merkle_subtree(
    const L* leaves,
    L* d_leaves,
    D* digests,
    size_t subtree_idx,
    size_t subtree_height,
    D* big_tree_digests,
    size_t start_segment_size,
    size_t start_segment_offset,
    uint64_t keep_rows,
    uint64_t input_block_len,
    const Hasher<L, D>& bottom_layer,
    const Hasher<L, D>& compression,
    const TreeBuilderConfig& tree_config,
    device_context::DeviceContext& ctx)
  {
    uint64_t arity = tree_config.arity;

    HashConfig hash_config = default_hash_config(ctx);
    hash_config.are_inputs_on_device = true;
    hash_config.are_outputs_on_device = true;
    hash_config.is_async = true;

    size_t bottom_layer_states = pow(arity, subtree_height);

    if (!tree_config.are_inputs_on_device) {
      CHK_IF_RETURN(hipMemcpyAsync(
        d_leaves, leaves, bottom_layer_states * input_block_len * sizeof(L), hipMemcpyHostToDevice, ctx.stream));
    }

    bottom_layer.hash_many(
      tree_config.are_inputs_on_device ? leaves : d_leaves, digests, bottom_layer_states, input_block_len,
      tree_config.digest_elements, hash_config);

    uint64_t number_of_states = bottom_layer_states / arity;
    size_t segment_size = start_segment_size;
    size_t segment_offset = start_segment_offset;

    if (!keep_rows || subtree_height < keep_rows) {
      D* digests_with_offset = big_tree_digests + segment_offset + subtree_idx * bottom_layer_states;
      CHK_IF_RETURN(hipMemcpyAsync(
        digests_with_offset, digests, bottom_layer_states * tree_config.digest_elements * sizeof(D),
        hipMemcpyDeviceToHost, ctx.stream));
      segment_offset += segment_size;
    }
    segment_size /= arity;
    subtree_height--;

    D* prev_layer = digests;
    D* next_layer = (D*)d_leaves;

    while (number_of_states > 0) {
      CHK_IF_RETURN(compression.run_hash_many_kernel(
        (L*)prev_layer, next_layer, number_of_states, tree_config.digest_elements * tree_config.arity,
        tree_config.digest_elements, hash_config.ctx));

      if (!keep_rows || subtree_height < keep_rows) {
        D* digests_with_offset =
          big_tree_digests + segment_offset + subtree_idx * number_of_states * tree_config.digest_elements;
        CHK_IF_RETURN(hipMemcpyAsync(
          digests_with_offset, next_layer, number_of_states * tree_config.digest_elements * sizeof(D),
          hipMemcpyDeviceToHost, ctx.stream));
        segment_offset += segment_size;
      }
      swap<D>(&prev_layer, &next_layer);
      segment_size /= arity;
      subtree_height--;
      number_of_states /= arity;
    }

    return CHK_LAST();
  }

  template <typename L, typename D>
  hipError_t build_merkle_tree(
    const L* leaves,
    D* digests,
    unsigned int height,
    unsigned int input_block_len,
    const Hasher<L, D>& compression,
    const Hasher<L, D>& bottom_layer,
    const TreeBuilderConfig& tree_config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = tree_config.ctx.stream;

    if (compression.preimage_max_length < tree_config.arity * tree_config.digest_elements)
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument,
        "Hash max preimage length does not match merkle tree arity multiplied by digest elements");

    uint64_t number_of_bottom_layer_states = pow(tree_config.arity, height);

    // This will determine how much splitting do we need to do
    // `number_of_streams` subtrees should fit in the device
    // This means each subtree should fit in `STREAM_CHUNK_SIZE` memory
    uint64_t number_of_subtrees = 1;
    uint64_t subtree_height = height;
    uint64_t subtree_bottom_layer_states = number_of_bottom_layer_states;
    uint64_t subtree_leaves_size = subtree_bottom_layer_states * input_block_len;
    uint64_t subtree_digests_size = subtree_bottom_layer_states * tree_config.digest_elements;

    size_t subtree_d_leaves_memory = std::max(
      tree_config.are_inputs_on_device ? 0 : (sizeof(L) * subtree_leaves_size),
      subtree_digests_size * sizeof(D) / tree_config.arity);
    size_t subtree_memory_required = sizeof(D) * subtree_digests_size + subtree_d_leaves_memory;
    while (subtree_memory_required > STREAM_CHUNK_SIZE) {
      number_of_subtrees *= tree_config.arity;
      subtree_height--;
      subtree_bottom_layer_states /= tree_config.arity;
      subtree_digests_size /= tree_config.arity;
      subtree_leaves_size /= tree_config.arity;
      subtree_d_leaves_memory /= tree_config.arity;
      subtree_memory_required = sizeof(D) * subtree_digests_size + subtree_d_leaves_memory;
    }
    int cap_height = height - subtree_height;
    size_t caps_len = pow(tree_config.arity, cap_height) * tree_config.digest_elements;

    size_t available_memory, _total_memory;
    CHK_IF_RETURN(hipMemGetInfo(&available_memory, &_total_memory));
    available_memory -= GIGA / 8; // Leave 128 MB just in case

    // We can effectively parallelize memory copy with streams
    // as long as they don't operate on more than `STREAM_CHUNK_SIZE` bytes
    const size_t number_of_streams = std::min((uint64_t)(available_memory / STREAM_CHUNK_SIZE), number_of_subtrees);
    hipStream_t* streams = static_cast<hipStream_t*>(malloc(sizeof(hipStream_t) * number_of_streams));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamCreate(&streams[i]));
    }

    bool caps_mode = tree_config.keep_rows && tree_config.keep_rows <= cap_height;
    D* caps;
    if (caps_mode) { caps = static_cast<D*>(malloc(caps_len * sizeof(D))); }

#ifdef MERKLE_DEBUG
    std::cout << "Available memory = " << available_memory / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Number of streams = " << number_of_streams << std::endl;
    std::cout << "Number of subtrees = " << number_of_subtrees << std::endl;
    std::cout << "Height of a subtree = " << subtree_height << std::endl;
    std::cout << "Cutoff height = " << height - subtree_height << std::endl;
    std::cout << "Number of leaves in a subtree = " << subtree_bottom_layer_states << std::endl;
    std::cout << "Digest elements for a subtree = " << subtree_digests_size << std::endl;
    std::cout << "Size of 1 subtree digests = " << subtree_digests_size * sizeof(D) / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Cap height = " << cap_height << std::endl;
    std::cout << "Enabling caps mode? " << caps_mode << std::endl;
    std::cout << "Allocated " << subtree_d_leaves_memory << " bytes for d_leaves" << std::endl;
#endif

    // Allocate memory for the leaves and digests
    // These are shared by streams in a pool
    L* d_leaves_ptr;
    D* digests_ptr;
    CHK_IF_RETURN(hipMallocAsync(&d_leaves_ptr, subtree_d_leaves_memory * number_of_streams, stream));
    CHK_IF_RETURN(hipMallocAsync(&digests_ptr, subtree_digests_size * number_of_streams * sizeof(D), stream));
    // Wait for these allocations to finish
    CHK_IF_RETURN(hipStreamSynchronize(stream));

    // Build subtrees in parallel. This for loop invokes kernels that can run in a pool of size `number_of_streams`
    for (size_t subtree_idx = 0; subtree_idx < number_of_subtrees; subtree_idx++) {
      size_t stream_idx = subtree_idx % number_of_streams;
      hipStream_t subtree_stream = streams[stream_idx];

      const L* subtree_leaves = leaves + subtree_idx * subtree_bottom_layer_states * input_block_len;
      L* subtree_d_leaves = (L*)((unsigned char*)d_leaves_ptr + stream_idx * subtree_d_leaves_memory);
      D* subtree_digests = digests_ptr + stream_idx * subtree_digests_size;

      int subtree_keep_rows = 0;
      if (tree_config.keep_rows) {
        int diff = tree_config.keep_rows - cap_height;
        subtree_keep_rows = std::max(1, diff);
      }
      device_context::DeviceContext subtree_context{subtree_stream, tree_config.ctx.device_id, tree_config.ctx.mempool};

      uint64_t start_segment_size = number_of_bottom_layer_states * tree_config.digest_elements;
      hipError_t subtree_result = build_merkle_subtree<L, D>(
        subtree_leaves,             // leaves
        subtree_d_leaves,           // d_leves
        subtree_digests,            // digests
        subtree_idx,                // subtree_idx
        subtree_height,             // subtree_height
        caps_mode ? caps : digests, // big_tree_digests
        start_segment_size,         // start_segment_size
        0,                          // start_segment_offset
        subtree_keep_rows,          // keep_rows
        input_block_len,            // input_block_len
        bottom_layer,               // bottom_layer
        compression,                // compression
        tree_config,                // tree_config
        subtree_context             // subtree_context
      );
      CHK_IF_RETURN(subtree_result);
    }

    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
    }

    // Finish the top-level tree if any
    if (cap_height > 0) {
      size_t start_segment_size = caps_len / tree_config.arity;
      size_t start_segment_offset = 0;
      if (!caps_mode) { // Calculate offset
        size_t keep_rows = tree_config.keep_rows ? tree_config.keep_rows : height + 1;
        size_t layer_size = pow(tree_config.arity, keep_rows - 1) * tree_config.digest_elements;
        for (int i = 0; i < keep_rows - cap_height; i++) {
          start_segment_offset += layer_size;
          layer_size /= tree_config.arity;
        }
      }
      CHK_IF_RETURN(hipMemcpyAsync(
        d_leaves_ptr, caps_mode ? caps : (digests + start_segment_offset - caps_len), caps_len * sizeof(D),
        (caps_mode || !tree_config.are_outputs_on_device) ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice, stream));

      uint64_t number_of_states = caps_len / tree_config.arity / tree_config.digest_elements;

      D* prev_layer = (D*)d_leaves_ptr;
      D* next_layer = digests_ptr;

      size_t segment_size = start_segment_size;
      size_t segment_offset = start_segment_offset;
      while (number_of_states > 0) {
        CHK_IF_RETURN(compression.run_hash_many_kernel(
          (L*)prev_layer, next_layer, number_of_states, tree_config.digest_elements * tree_config.arity,
          tree_config.digest_elements, tree_config.ctx));
        if (!tree_config.keep_rows || cap_height < tree_config.keep_rows + (int)caps_mode) {
          D* digests_with_offset = digests + segment_offset;
          CHK_IF_RETURN(hipMemcpyAsync(
            digests_with_offset, next_layer, number_of_states * tree_config.digest_elements * sizeof(D),
            hipMemcpyDeviceToHost, stream));
          segment_offset += segment_size;
        }

        swap<D>(&prev_layer, &next_layer);

        segment_size /= tree_config.arity;
        cap_height--;
        number_of_states /= tree_config.arity;
      }
      if (caps_mode) { free(caps); }
    }

    CHK_IF_RETURN(hipFreeAsync(d_leaves_ptr, stream));
    CHK_IF_RETURN(hipFreeAsync(digests_ptr, stream));
    if (!tree_config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
      CHK_IF_RETURN(hipStreamDestroy(streams[i]));
    }
    free(streams);
    return CHK_LAST();
  }

} // namespace merkle_tree
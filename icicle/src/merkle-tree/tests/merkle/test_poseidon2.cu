#include "hip/hip_runtime.h"
#ifndef __CUDA_ARCH__
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <math.h>

#include "merkle-tree/merkle.cuh"

#include "poseidon2/poseidon2.cuh"

#include "api/babybear.h"
using namespace babybear;

using FpMilliseconds = std::chrono::duration<float, std::chrono::milliseconds::period>;
#define START_TIMER(timer) auto timer##_start = std::chrono::high_resolution_clock::now();
#define END_TIMER(timer, msg)                                                                                          \
  printf("%s: %.0f ms\n", msg, FpMilliseconds(std::chrono::high_resolution_clock::now() - timer##_start).count());

int main(int argc, char* argv[])
{
  /// Tree of height N and arity A contains \sum{A^i} for i in 0..N elements
  uint32_t tree_arity = 2;
  uint32_t width = 16;
  uint32_t input_block_len = 8;
  uint32_t digest_elements = 8;
  uint64_t tree_height = argc > 1 ? atoi(argv[1]) : 3;
  uint64_t number_of_leaves = pow(tree_arity, tree_height);
  uint64_t total_number_of_leaves = number_of_leaves * input_block_len;

  // Load poseidon constants
  START_TIMER(timer_const);
  device_context::DeviceContext ctx = device_context::get_default_device_context();
  poseidon2::Poseidon2<scalar_t> poseidon(
    width, poseidon2::MdsType::DEFAULT_MDS, poseidon2::DiffusionStrategy::DEFAULT_DIFFUSION, ctx);
  END_TIMER(timer_const, "Load poseidon constants");

  /// Use keep_rows to specify how many rows do you want to store
  int keep_rows = argc > 2 ? atoi(argv[2]) : 3;
  size_t digests_len = merkle_tree::get_digests_len(keep_rows - 1, tree_arity, digest_elements);

  /// Fill leaves with scalars [0, 1, ... 2^tree_height - 1]
  START_TIMER(timer_allocation);
  scalar_t input = scalar_t::zero();
  size_t leaves_mem = total_number_of_leaves * sizeof(scalar_t);
  scalar_t* leaves = static_cast<scalar_t*>(malloc(leaves_mem));
  for (uint64_t i = 0; i < total_number_of_leaves; i++) {
    leaves[i] = input;
    input = input + scalar_t::one();
  }
  END_TIMER(timer_allocation, "Allocated memory for leaves: ");

  /// Allocate memory for digests of {keep_rows} rows of a tree
  START_TIMER(timer_digests);
  size_t digests_mem = digests_len * sizeof(scalar_t);
  scalar_t* digests = static_cast<scalar_t*>(malloc(digests_mem));
  END_TIMER(timer_digests, "Allocated memory for digests");

  std::cout << "Memory for leaves = " << leaves_mem / 1024 / 1024 << " MB; " << leaves_mem / 1024 / 1024 / 1024 << " GB"
            << std::endl;
  std::cout << "Number of leaves = " << number_of_leaves << std::endl;
  std::cout << "Total Number of leaves = " << total_number_of_leaves << std::endl;
  std::cout << "Memory for digests = " << digests_mem / 1024 / 1024 << " MB; " << digests_mem / 1024 / 1024 / 1024
            << " GB" << std::endl;
  std::cout << "Number of digest elements = " << digests_len << std::endl;

  std::cout << "Total RAM consumption = " << (digests_mem + leaves_mem) / 1024 / 1024 << " MB; "
            << (digests_mem + leaves_mem) / 1024 / 1024 / 1024 << " GB" << std::endl;

  merkle_tree::TreeBuilderConfig tree_config = merkle_tree::default_merkle_config();
  tree_config.arity = tree_arity;
  tree_config.keep_rows = keep_rows;
  tree_config.digest_elements = digest_elements;
  START_TIMER(timer_merkle);
  babybear_build_merkle_tree(leaves, digests, tree_height, input_block_len, &poseidon, &poseidon, tree_config);
  END_TIMER(timer_merkle, "Merkle tree built: ")

  for (int i = 0; i < digests_len; i++) {
    std::cout << digests[i] << std::endl;
  }

  // Use this to generate test vectors
  // for (int i = 0; i < digests_len; i++) {
  //   std::cout << "{";
  //   for (int j = 0; j < 8; j++) {
  //     std::cout << ((uint64_t*)&digests[i].limbs_storage)[j];
  //     if (j != 7) { std::cout << ", "; }
  //   }
  //   std::cout << "}," << std::endl;
  // }

  /// These scalars are digests of top-7 rows of a Merkle tree.
  /// Arity = 2, Tree height = 28, keep_rows = 7
  /// They are aligned in the following format:
  ///  L-7      L-6     L-5       L-4       L-3       L-2    L-1
  /// [0..63, 64..95, 96..111, 112..119, 120..123, 124..125, 126]
  scalar_t expected[0] = {};

  for (int i = 0; i < digests_len; i++) {
    scalar_t root = digests[i];
    assert(root == expected[i]);
  }
  free(digests);
  free(leaves);
}

#endif
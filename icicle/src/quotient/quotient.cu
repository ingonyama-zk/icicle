#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include "gpu-utils/device_context.cuh"
#include "vec_ops/vec_ops.cuh"
#include "fields/stark_fields/m31.cuh"
#include "fields/point.cuh"
#include "quotient/quotient.cuh"

namespace quotient {
    namespace {
        template <typename QF, typename CF>
        __device__ QF mul(QF q, CF c) {
            CF a = CF{q.real, q.im1} * c;
            CF b = CF{q.im2, q.im3} * c;
            return QF{a.real, a.imaginary, b.real, b.imaginary};
        }

        template <typename QF>
        __device__ QF pow(QF val, uint32_t scalar) {
            QF res = QF::one();
            while (scalar) {
                if (scalar & 1 == 1) {
                    res = res * val;
                }
                val = val * val;
                scalar >>= 1;
            }
            return res;
        }
    
        template <typename QF, typename F>
        __device__ QF complex_conjugate(QF point) {
            return QF{point.real, point.im1, F::neg(point.im2), F::neg(point.im3)};
        }

        template <typename QF>
        __device__ QF scalar_mul(QF val, uint32_t scalar) {
            QF res = QF::one();
            while (scalar) {
                if (scalar & 1 == 1) {
                    res = res + val;
                }
                val = val + val;
                scalar >>= 1;
            }
            return res;
        }

        template <typename QP, typename QF, typename F>
        __device__ void complex_conjugate_line_coeffs(QP point, QF value, QF alpha, QF* a_out, QF* b_out, QF* c_out) {
            QF a = complex_conjugate<QF, F>(value) - value; 
            QF c = complex_conjugate<QF, F>(point.y) - point.y;
            QF b = (value * c) - (a * point.y);  

            *a_out = alpha * a;
            *b_out = alpha * b;
            *c_out = alpha * c;
        }

        template <typename QP, typename QF, typename CF, typename P>
        __device__ void denominator_inverse(
            ColumnSampleBatch<QP, QF> *sample_batches,
            uint32_t sample_size,
            const P domain_point,
            CF *flat_denominators) {

            for (unsigned int i = 0; i < sample_size; i++) {
                CF prx = CF{sample_batches[i].point->x.real, sample_batches[i].point->x.im1};
                CF pry = CF{sample_batches[i].point->y.real, sample_batches[i].point->y.im1};
                CF pix = CF{sample_batches[i].point->x.im2, sample_batches[i].point->x.im3};
                CF piy = CF{sample_batches[i].point->y.im2, sample_batches[i].point->y.im3};

                CF first_substraction = CF{prx.real - domain_point.x, prx.imaginary};
                CF second_substraction = CF{pry.real - domain_point.y, pry.imaginary};
                CF result = (first_substraction * piy) - (second_substraction * pix);
                flat_denominators[i] = CF::inverse(result);
            }
        }
    }

    template <typename QP, typename QF>
    std::ostream& operator<<(std::ostream& os, const ColumnSampleBatch<QP, QF>& batch) {
        os << "ColumnSampleBatch {\n";
        os << "  point: " << batch.point << "\n";
        os << "  columns: [";
        for (uint32_t i = 0; i < batch.size; ++i) {
            os << batch.columns[i];
            if (i < batch.size - 1) os << ", ";
        }
        os << "]\n";
        os << "  values: [";
        for (uint32_t i = 0; i < batch.size; ++i) {
            os << batch.values[i];
            if (i < batch.size - 1) os << ", ";
        }
        os << "]\n";
        os << "  size: " << batch.size << "\n";
        os << "}\n";
        return os;
    }

    template <typename QP, typename QF, typename F>
    __global__ void column_line_and_batch_random_coeffs(
        ColumnSampleBatch<QP, QF> *sample_batches,
        uint32_t sample_size,
        QF random_coefficient,
        QF *flattened_line_coeffs,
        uint32_t *line_coeffs_sizes,
        QF *batch_random_coeffs) {
        int tid = threadIdx.x + blockDim.x * blockIdx.x;
        if(tid < sample_size) {
            batch_random_coeffs[tid] = pow<QF>(random_coefficient, sample_batches[tid].size); 

            // Calculate Column Line Coeffs
            line_coeffs_sizes[tid] = sample_batches[tid].size;
            size_t sample_batches_offset = 0;
            for (int i = 0; i < tid; ++i) {
                sample_batches_offset += line_coeffs_sizes[i];
            }
            sample_batches_offset *= 3;

            QF alpha = QF::one();

            for(size_t j = 0; j < sample_batches[tid].size; ++j) {
                QF sampled_value = sample_batches[tid].values[j];
                alpha = alpha * random_coefficient; 
                QP point = *sample_batches[tid].point;
                QF value = sampled_value; 

                size_t sampled_offset = sample_batches_offset + (j * 3);
                complex_conjugate_line_coeffs<QP, QF, F>(point, value, alpha, &flattened_line_coeffs[sampled_offset], &flattened_line_coeffs[sampled_offset + 1], &flattened_line_coeffs[sampled_offset + 2]); 
            }
        }
    }

    template <typename QP, typename QF, typename CF, typename F, typename P, typename D>
    __global__ void accumulate_quotients_kernel(
        D domain,
        uint32_t domain_size,
        F **columns,
        uint32_t number_of_columns,
        QF random_coefficient,
        ColumnSampleBatch<QP, QF> *samples,
        uint32_t sample_size,
        QF *flattened_line_coeffs,
        uint32_t *line_coeffs_sizes,
        QF *batch_random_coeffs,
        CF *denominator_inverses,
        F *result1, 
        F *result2, 
        F *result3, 
        F *result4 ) {
        int row = threadIdx.x + blockDim.x * blockIdx.x;
        if (row < domain_size) {
            CF *denominator_inverses_local = &denominator_inverses[row * sample_size];
            uint32_t index = __brev(row) >> (32 - domain.lg_size());
            P point = domain.at(index);
            denominator_inverse<QP, QF, CF>(
                samples,
                sample_size,
                point,
                denominator_inverses_local
            );
            QF accumulator = QF::zero();
            for(uint32_t i = 0, offset = 0; i < sample_size; ++i) {
                ColumnSampleBatch<QP, QF> sample_batch = samples[i];
                QF *line_coeffs = &flattened_line_coeffs[offset * 3];
                QF batch_coeff = batch_random_coeffs[i];
                uint32_t line_coeffs_size = line_coeffs_sizes[i];

                QF numerator = QF::zero();

                for(uint32_t j = 0; j < line_coeffs_size; ++j) {
                    QF a = line_coeffs[3 * j];
                    QF b = line_coeffs[3 * j + 1];
                    QF c = line_coeffs[3 * j + 2];

                    uint32_t column_index = samples[i].columns[j];
                    QF linear_term = scalar_mul<QF>(a, point.y.limbs_storage.limbs[0]) + b;
                    QF value = scalar_mul<QF>(c, columns[column_index][row].limbs_storage.limbs[0]);

                    numerator = numerator + (value - linear_term);
                }

                accumulator = (accumulator * batch_coeff) + mul<QF, CF>(numerator, denominator_inverses_local[i]);
                offset += line_coeffs_size;
            }
            result1[row] = accumulator.real;
            result2[row] = accumulator.im1;
            result3[row] = accumulator.im2;
            result4[row] = accumulator.im3;
        }
    }

    template <typename QP, typename QF>
    __global__ void set_columns_and_values_pointers(ColumnSampleBatch<QP, QF> *d_samples, uint32_t **d_columns_ptrs, QF **d_values_ptrs, QP **d_point_ptrs, int sample_size) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < sample_size) {

            d_samples[i].columns = d_columns_ptrs[i];
            d_samples[i].values = d_values_ptrs[i];
            d_samples[i].point = d_point_ptrs[i];
        }
    }

    template <typename QP, typename QF, typename CF, typename F, typename P, typename D>
    hipError_t accumulate_quotients(
        D &domain,
        F **columns, // array of ptrs (number_of_columns) to columns (domain_size)
        uint32_t number_of_columns,
        QF &random_coefficient,
        ColumnSampleBatch<QP, QF> *samples,
        uint32_t sample_size,
        uint32_t flattened_line_coeffs_size,
        QuotientConfig &cfg,
        F *result1,
        F *result2,
        F *result3,
        F *result4
    ) {
        CHK_INIT_IF_RETURN();

        hipStream_t stream = cfg.ctx.stream;

        uint32_t domain_size = domain.size();
        // assuming column elements are already on device
        F** d_columns;
        // F *d_columns;
        if (cfg.are_columns_on_device) {
            d_columns = columns;
        }
        else {
            CHK_IF_RETURN(hipMallocAsync(&d_columns, sizeof(F*) * number_of_columns, stream));
            CHK_IF_RETURN(
            hipMemcpyAsync(d_columns, columns, sizeof(F*) * number_of_columns, hipMemcpyHostToDevice, stream));
        }
        ColumnSampleBatch<QP, QF> *d_samples;
        uint32_t **d_columns_ptrs;
        QF **d_values_ptrs;
        QP **d_point_ptrs;
        uint32_t **h_columns_ptrs;
        QF **h_values_ptrs;
        QP **h_point_ptrs;
        if (cfg.are_sample_points_on_device) {
            d_samples = samples;
        }
        else {
            CHK_IF_RETURN(hipMallocAsync(&d_samples, sizeof(ColumnSampleBatch<QP, QF>) * sample_size, stream));
            h_columns_ptrs = new uint32_t*[sample_size];
            h_values_ptrs = new QF*[sample_size];
            h_point_ptrs = new QP*[sample_size];

            for (int i = 0; i < sample_size; ++i) {
                // Allocate device memory for columns and values for each struct
                if (samples[i].size > 0) {
                    hipMallocAsync(&h_columns_ptrs[i], sizeof(uint32_t) * samples[i].size, stream);
                    hipMemcpyAsync(h_columns_ptrs[i], samples[i].columns, sizeof(uint32_t) * samples[i].size, hipMemcpyHostToDevice, stream);

                    hipMallocAsync(&h_values_ptrs[i], sizeof(QF) * samples[i].size, stream);
                    hipMemcpyAsync(h_values_ptrs[i], samples[i].values, sizeof(QF) * samples[i].size, hipMemcpyHostToDevice, stream);
                } else {
                    h_columns_ptrs[i] = nullptr;
                    h_values_ptrs[i] = nullptr;
                }
                hipMallocAsync(&h_point_ptrs[i], sizeof(QP), stream);
                hipMemcpyAsync(h_point_ptrs[i], samples[i].point, sizeof(QP), hipMemcpyHostToDevice, stream);
            }
            // Allocate device memory to store the arrays of pointers for columns and values
            CHK_IF_RETURN(hipMallocAsync(&d_columns_ptrs, sizeof(uint32_t*) * sample_size, stream));
            CHK_IF_RETURN(hipMallocAsync(&d_values_ptrs, sizeof(QF*) * sample_size, stream));
            CHK_IF_RETURN(hipMallocAsync(&d_point_ptrs, sizeof(QP*) * sample_size, stream));

            // Copy the host arrays of pointers to device memory
            CHK_IF_RETURN(hipMemcpyAsync(d_columns_ptrs, h_columns_ptrs, sizeof(uint32_t*) * sample_size, hipMemcpyHostToDevice, stream));
            CHK_IF_RETURN(hipMemcpyAsync(d_values_ptrs, h_values_ptrs, sizeof(QF*) * sample_size, hipMemcpyHostToDevice, stream));
            CHK_IF_RETURN(hipMemcpyAsync(d_point_ptrs, h_point_ptrs, sizeof(QP*) * sample_size, hipMemcpyHostToDevice, stream));

            // Copy the struct array from host to device (with placeholder pointers)
            CHK_IF_RETURN(hipMemcpyAsync(d_samples, samples, sizeof(ColumnSampleBatch<QP, QF>) * sample_size, hipMemcpyHostToDevice, stream));

            // Kernel to set the `columns` and `values` pointers in the device struct array
            int sample_size_to_kernel = sample_size == 0 ? 1 : (sample_size + 255) / 256;
            set_columns_and_values_pointers<QP, QF><<<sample_size_to_kernel, 256, 0, stream>>>(d_samples, d_columns_ptrs, d_values_ptrs, d_point_ptrs, sample_size);
        }
        
        QF *d_batch_random_coeffs;
        CHK_IF_RETURN(hipMallocAsync(&d_batch_random_coeffs, sizeof(QF) * sample_size, stream));

        uint32_t *d_line_coeffs_sizes;
        CHK_IF_RETURN(hipMallocAsync(&d_line_coeffs_sizes, sizeof(uint32_t) * sample_size, stream));

        QF *d_flattened_line_coeffs;
        CHK_IF_RETURN(hipMallocAsync(&d_flattened_line_coeffs, sizeof(QF) * flattened_line_coeffs_size, stream));

        int block_dim = sample_size < 512 ? sample_size : 512;
        block_dim = block_dim == 0 ? 1 : block_dim; 
        int num_blocks = block_dim < 512 ? 1 : (sample_size + block_dim - 1) / block_dim;
        column_line_and_batch_random_coeffs<QP, QF, F><<<num_blocks, block_dim, 0, stream>>>(
            d_samples, 
            sample_size, 
            random_coefficient,
            d_flattened_line_coeffs, 
            d_line_coeffs_sizes,
            d_batch_random_coeffs
        );

        F *d_result1;
        F *d_result2;
        F *d_result3;
        F *d_result4;
        if (cfg.are_results_on_device) {
            d_result1 = result1;
            d_result2 = result2;
            d_result3 = result3;
            d_result4 = result4;
        }
        else {
            CHK_IF_RETURN(hipMallocAsync(&d_result1, sizeof(F) * domain_size, stream));
            CHK_IF_RETURN(hipMallocAsync(&d_result2, sizeof(F) * domain_size, stream));
            CHK_IF_RETURN(hipMallocAsync(&d_result3, sizeof(F) * domain_size, stream));
            CHK_IF_RETURN(hipMallocAsync(&d_result4, sizeof(F) * domain_size, stream));
        }
        
        CF *d_denominator_inverses;
        CHK_IF_RETURN(hipMallocAsync(&d_denominator_inverses, sizeof(CF) * sample_size * domain_size, stream));


        block_dim = 512;
        num_blocks = (domain_size + block_dim - 1) / block_dim;
        accumulate_quotients_kernel<QP, QF, CF, F, P, D><<<num_blocks, block_dim, 0, stream>>>(
                domain,
                domain_size,
                d_columns,
                number_of_columns,
                random_coefficient,
                d_samples,
                sample_size,
                d_flattened_line_coeffs,
                d_line_coeffs_sizes,
                d_batch_random_coeffs,
                d_denominator_inverses,
                d_result1,
                d_result2,
                d_result3,
                d_result4
        );

        hipStream_t stream_free;
        CHK_IF_RETURN(hipStreamCreate(&stream_free));

        if (!cfg.are_results_on_device) {
            CHK_IF_RETURN(hipMemcpyAsync(result1, d_result1, sizeof(F) * domain_size, hipMemcpyDeviceToHost, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_result1, stream_free));
            CHK_IF_RETURN(hipMemcpyAsync(result2, d_result2, sizeof(F) * domain_size, hipMemcpyDeviceToHost, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_result2, stream_free));
            CHK_IF_RETURN(hipMemcpyAsync(result3, d_result3, sizeof(F) * domain_size, hipMemcpyDeviceToHost, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_result3, stream_free));
            CHK_IF_RETURN(hipMemcpyAsync(result4, d_result4, sizeof(F) * domain_size, hipMemcpyDeviceToHost, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_result4, stream_free));
        }
        CHK_IF_RETURN(hipFreeAsync(d_denominator_inverses, stream_free));
        CHK_IF_RETURN(hipFreeAsync(d_flattened_line_coeffs, stream_free));
        CHK_IF_RETURN(hipFreeAsync(d_line_coeffs_sizes, stream_free));
        CHK_IF_RETURN(hipFreeAsync(d_batch_random_coeffs, stream_free));

        if (!cfg.are_sample_points_on_device) {
            for (int i = 0; i < sample_size; ++i) {
                CHK_IF_RETURN(hipFreeAsync(h_columns_ptrs[i], stream_free));
                CHK_IF_RETURN(hipFreeAsync(h_values_ptrs[i], stream_free));
                CHK_IF_RETURN(hipFreeAsync(h_point_ptrs[i], stream_free));
            }
            CHK_IF_RETURN(hipFreeAsync(d_columns_ptrs, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_point_ptrs, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_values_ptrs, stream_free));
            CHK_IF_RETURN(hipFreeAsync(d_samples, stream_free));
            delete[] h_columns_ptrs;
            delete[] h_values_ptrs;
            delete[] h_point_ptrs;
        }

        if (!cfg.are_columns_on_device) {
            CHK_IF_RETURN(hipFreeAsync(d_columns, stream_free));
        }

        if (!cfg.is_async) CHK_IF_RETURN(hipStreamSynchronize(stream));
        
        return CHK_LAST();
    }
} // namespace quotient
#include "fields/field_config.cuh"

using namespace field_config;

#include "utils/mont.cuh"
#include "utils/utils.h"
#include "gpu-utils/device_context.cuh"

extern "C" void CONCAT_EXPAND(FIELD, generate_scalars)(scalar_t* scalars, int size)
{
  scalar_t::rand_host_many(scalars, size);
}

extern "C" hipError_t CONCAT_EXPAND(FIELD, scalar_convert_montgomery)(
  scalar_t* d_inout, size_t n, bool is_into, device_context::DeviceContext& ctx)
{
  if (is_into) {
    return mont::to_montgomery(d_inout, n, ctx.stream, d_inout);
  } else {
    return mont::from_montgomery(d_inout, n, ctx.stream, d_inout);
  }
}

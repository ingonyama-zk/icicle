#include "fields/field_config.cuh"

using namespace field_config;

#include "utils/mont.cuh"
#include "utils/utils.h"
#include "gpu-utils/device_context.cuh"

extern "C" void CONCAT_EXPAND(FIELD, ExtensionGenerateScalars)(extension_t* scalars, int size)
{
  extension_t::RandHostMany(scalars, size);
}

extern "C" hipError_t CONCAT_EXPAND(FIELD, ExtensionScalarConvertMontgomery)(
  extension_t* d_inout, size_t n, bool is_into, device_context::DeviceContext& ctx)
{
  if (is_into) {
    return mont::ToMontgomery(d_inout, n, ctx.stream, d_inout);
  } else {
    return mont::FromMontgomery(d_inout, n, ctx.stream, d_inout);
  }
}

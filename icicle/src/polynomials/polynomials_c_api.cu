#include "hip/hip_runtime.h"
#include "polynomials/polynomials.h"
#include "fields/field_config.cuh"
#include "utils/utils.h"
#include "utils/integrity_pointer.h"
#include "polynomials/cuda_backend/polynomial_cuda_backend.cuh"

namespace polynomials {
  extern "C" {

  // Defines a polynomial instance based on the scalar type from the FIELD configuration.
  typedef Polynomial<scalar_t> PolynomialInst;

  bool CONCAT_EXPAND(FIELD, polynomial_init_cuda_backend)()
  {
    static auto cuda_factory = std::make_shared<CUDAPolynomialFactory<scalar_t>>();
    PolynomialInst::initialize(cuda_factory);
    return cuda_factory != nullptr;
  }

  // Constructs a polynomial from a set of coefficients.
  // coeffs: Array of coefficients.
  // size: Number of coefficients in the array.
  // Returns a pointer to the newly created polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_create_from_coefficients)(scalar_t* coeffs, size_t size)
  {
    auto result = new PolynomialInst(PolynomialInst::from_coefficients(coeffs, size));
    return result;
  }

  // Constructs a polynomial from evaluations at the roots of unity.
  // evals: Array of evaluations.
  // size: Number of evaluations in the array.
  // Returns a pointer to the newly created polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_create_from_rou_evaluations)(scalar_t* evals, size_t size)
  {
    auto result = new PolynomialInst(PolynomialInst::from_rou_evaluations(evals, size));
    return result;
  }

  // Clones an existing polynomial instance.
  // p: Pointer to the polynomial instance to clone.
  // Returns a pointer to the cloned polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_clone)(const PolynomialInst* p)
  {
    auto result = new PolynomialInst(p->clone());
    return result;
  }

  // Deletes a polynomial instance, freeing its memory.
  // instance: Pointer to the polynomial instance to delete.
  void CONCAT_EXPAND(FIELD, polynomial_delete)(PolynomialInst* instance) { delete instance; }

  // Prints a polynomial to stdout
  void CONCAT_EXPAND(FIELD, polynomial_print(PolynomialInst* p)) { std::cout << *p << std::endl; }

  // Adds two polynomials.
  // a, b: Pointers to the polynomial instances to add.
  // Returns a pointer to the resulting polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_add)(const PolynomialInst* a, const PolynomialInst* b)
  {
    auto result = new PolynomialInst(std::move(*a + *b));
    return result;
  }

  // Adds a polynomial to another in place.
  // a: Pointer to the polynomial to add to.
  // b: Pointer to the polynomial to add.
  void CONCAT_EXPAND(FIELD, polynomial_add_inplace)(PolynomialInst* a, const PolynomialInst* b) { *a += *b; }

  // Subtracts one polynomial from another.
  // a, b: Pointers to the polynomial instances (minuend and subtrahend, respectively).
  // Returns a pointer to the resulting polynomial instance.

  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_subtract)(const PolynomialInst* a, const PolynomialInst* b)
  {
    auto result = new PolynomialInst(std::move(*a - *b));
    return result;
  }

  // Multiplies two polynomials.
  // a, b: Pointers to the polynomial instances to multiply.
  // Returns a pointer to the resulting polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_multiply)(const PolynomialInst* a, const PolynomialInst* b)
  {
    auto result = new PolynomialInst(std::move(*a * *b));
    return result;
  }

  // Multiplies a polynomial by scalar.
  // a: Pointer to the polynomial instance.
  // scalar: Scalar to multiply by.
  // Returns a pointer to the resulting polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_multiply_by_scalar)(const PolynomialInst* a, const scalar_t& scalar)
  {
    auto result = new PolynomialInst(std::move(*a * scalar));
    return result;
  }

  // Divides one polynomial by another, returning both quotient and remainder.
  // a, b: Pointers to the polynomial instances (dividend and divisor, respectively).
  // q: Output parameter for the quotient.
  // r: Output parameter for the remainder.
  void CONCAT_EXPAND(FIELD, polynomial_division)(
    const PolynomialInst* a, const PolynomialInst* b, PolynomialInst** q /*OUT*/, PolynomialInst** r /*OUT*/)
  {
    auto [_q, _r] = a->divide(*b);
    *q = new PolynomialInst(std::move(_q));
    *r = new PolynomialInst(std::move(_r));
  }

  // Calculates the quotient of dividing one polynomial by another.
  // a, b: Pointers to the polynomial instances (dividend and divisor, respectively).
  // Returns a pointer to the resulting quotient polynomial instance.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_quotient)(const PolynomialInst* a, const PolynomialInst* b)
  {
    auto result = new PolynomialInst(std::move(*a / *b));
    return result;
  }

  // Calculates the remainder of dividing one polynomial by another.
  // a, b: Pointers to the polynomial instances (dividend and divisor, respectively).
  // Returns a pointer to the resulting remainder polynomial instance.

  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_remainder)(const PolynomialInst* a, const PolynomialInst* b)
  {
    auto result = new PolynomialInst(std::move(*a % *b));
    return result;
  }

  // Divides a polynomial by a vanishing polynomial of a given degree, over rou domain.
  // p: Pointer to the polynomial instance.
  // vanishing_poly_degree: Degree of the vanishing polynomial.
  // Returns a pointer to the resulting polynomial instance.
  PolynomialInst*
  CONCAT_EXPAND(FIELD, polynomial_divide_by_vanishing)(const PolynomialInst* p, uint64_t vanishing_poly_degree)
  {
    auto result = new PolynomialInst(std::move(p->divide_by_vanishing_polynomial(vanishing_poly_degree)));
    return result;
  }

  // Adds a monomial to a polynomial in place.
  // p: Pointer to the polynomial instance.
  // monomial_coeff: Coefficient of the monomial to add.
  // monomial: Degree of the monomial to add.
  void CONCAT_EXPAND(FIELD, polynomial_add_monomial_inplace)(
    PolynomialInst* p, const scalar_t& monomial_coeff, uint64_t monomial)
  {
    p->add_monomial_inplace(monomial_coeff, monomial);
  }

  // Subtracts a monomial from a polynomial in place.
  // p: Pointer to the polynomial instance.
  // monomial_coeff: Coefficient of the monomial to subtract.
  // monomial: Degree of the monomial to subtract.
  void CONCAT_EXPAND(FIELD, polynomial_sub_monomial_inplace)(
    PolynomialInst* p, const scalar_t& monomial_coeff, uint64_t monomial)
  {
    p->sub_monomial_inplace(monomial_coeff, monomial);
  }

  // Creates a new polynomial instance by slicing an existing polynomial.
  // p: Pointer to the original polynomial instance to be sliced.
  // offset: Starting index for the slice.
  // stride: Interval between elements in the slice.
  // size: Number of elements in the slice.
  // Returns: Pointer to the new polynomial instance containing the slice.
  PolynomialInst*
  CONCAT_EXPAND(FIELD, polynomial_slice)(PolynomialInst* p, uint64_t offset, uint64_t stride, uint64_t size)
  {
    auto result = new PolynomialInst(std::move(p->slice(offset, stride, size)));
    return result;
  }

  // Creates a new polynomial instance containing only the even-powered terms of the original polynomial.
  // p: Pointer to the original polynomial instance.
  // Returns: Pointer to the new polynomial instance containing only even-powered terms.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_even)(PolynomialInst* p)
  {
    auto result = new PolynomialInst(std::move(p->even()));
    return result;
  }

  // Creates a new polynomial instance containing only the odd-powered terms of the original polynomial.
  // p: Pointer to the original polynomial instance.
  // Returns: Pointer to the new polynomial instance containing only odd-powered terms.
  PolynomialInst* CONCAT_EXPAND(FIELD, polynomial_odd)(PolynomialInst* p)
  {
    auto result = new PolynomialInst(std::move(p->odd()));
    return result;
  }

  // Evaluates a polynomial on a domain of points.
  // p: Pointer to the polynomial instance.
  // domain: Array of points constituting the domain.
  // domain_size: Number of points in the domain.
  // evals: Output array for the evaluations.
  void CONCAT_EXPAND(FIELD, polynomial_evaluate_on_domain)(
    const PolynomialInst* p, scalar_t* domain, uint64_t domain_size, scalar_t* evals /*OUT*/)
  {
    return p->evaluate_on_domain(domain, domain_size, evals);
  }

  // Returns the degree of a polynomial.
  // p: Pointer to the polynomial instance.
  // Returns the degree of the polynomial.
  int64_t CONCAT_EXPAND(FIELD, polynomial_degree)(PolynomialInst* p) { return p->degree(); }

  // Copies a range of polynomial coefficients to host/device memory.
  // p: Pointer to the polynomial instance.
  // host_memory: Array to copy the coefficients into. If NULL, not copying.
  // start_idx: Start index of the range to copy.
  // end_idx: End index of the range to copy.
  // Returns the number of coefficients copied. if memory is NULL, returns number of coefficients.
  uint64_t CONCAT_EXPAND(FIELD, polynomial_copy_coeffs_range)(
    PolynomialInst* p, scalar_t* memory, uint64_t start_idx, uint64_t end_idx)
  {
    return p->copy_coeffs(memory, start_idx, end_idx);
  }

  // Retrieves a device-memory raw-ptr of the polynomial coefficients.
  // p: Pointer to the polynomial instance.
  // size: Output parameter for the size of the view.
  // device_id: Output parameter for the device ID.
  // Returns a raw mutable pointer to the coefficients.
  scalar_t* CONCAT_EXPAND(FIELD, polynomial_get_coeffs_raw_ptr)(
    PolynomialInst* p, uint64_t* size /*OUT*/, uint64_t* device_id /*OUT*/)
  {
    auto [coeffs, _size, _device_id] = p->get_coefficients_view();
    *size = _size;
    *device_id = _device_id;
    return const_cast<scalar_t*>(coeffs.get());
  }

  // Retrieves a device-memory view of the polynomial coefficients.
  // p: Pointer to the polynomial instance.
  // size: Output parameter for the size of the view.
  // device_id: Output parameter for the device ID.
  // Returns a pointer to an integrity pointer encapsulating the coefficients view.
  IntegrityPointer<scalar_t>* CONCAT_EXPAND(FIELD, polynomial_get_coeff_view)(
    PolynomialInst* p, uint64_t* size /*OUT*/, uint64_t* device_id /*OUT*/)
  {
    auto [coeffs, _size, _device_id] = p->get_coefficients_view();
    *size = _size;
    *device_id = _device_id;
    return new IntegrityPointer<scalar_t>(std::move(coeffs));
  }

  // Retrieves a device-memory view of the polynomial's evaluations on the roots of unity.
  // p: Pointer to the polynomial instance.
  // nof_evals: Number of evaluations.
  // is_reversed: Whether the evaluations are in reversed order.
  // size: Output parameter for the size of the view.
  // device_id: Output parameter for the device ID.
  // Returns a pointer to an integrity pointer encapsulating the evaluations view.
  IntegrityPointer<scalar_t>* CONCAT_EXPAND(FIELD, polynomial_get_rou_evaluations_view)(
    PolynomialInst* p, uint64_t nof_evals, bool is_reversed, uint64_t* size /*OUT*/, uint64_t* device_id /*OUT*/)
  {
    auto [rou_evals, _size, _device_id] = p->get_rou_evaluations_view(nof_evals, is_reversed);
    *size = _size;
    *device_id = _device_id;
    return new IntegrityPointer<scalar_t>(std::move(rou_evals));
  }

  // Reads the pointer from an integrity pointer.
  // p: Pointer to the integrity pointer.
  // Returns the raw pointer if still valid, otherwise NULL.
  const scalar_t* CONCAT_EXPAND(FIELD, polynomial_intergrity_ptr_get)(IntegrityPointer<scalar_t>* p)
  {
    return p->get();
  }

  // Checks if an integrity pointer is still valid.
  // p: Pointer to the integrity pointer.
  // Returns true if the pointer is valid, false otherwise.
  bool CONCAT_EXPAND(FIELD, polynomial_intergrity_ptr_is_valid)(IntegrityPointer<scalar_t>* p) { return p->isValid(); }

  // Destroys an integrity pointer, freeing its resources.
  // p: Pointer to the integrity pointer to destroy.
  void CONCAT_EXPAND(FIELD, polynomial_intergrity_ptr_destroy)(IntegrityPointer<scalar_t>* p) { delete p; }

  } // extern "C"

} // namespace polynomials

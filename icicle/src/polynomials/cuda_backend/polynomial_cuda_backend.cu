#include "hip/hip_runtime.h"

#include "polynomials/polynomials.h"
#include "polynomials/cuda_backend/polynomial_cuda_backend.cuh"

#include "gpu-utils/device_context.cuh"
#include "hip/hip_runtime.h"
#include "ntt/ntt.cuh"
#include "kernels.cuh"

using device_context::DeviceContext;

namespace polynomials {

  static uint64_t ceil_to_power_of_two(uint64_t x) { return 1ULL << uint64_t(ceil(log2(x))); }
  /*============================== Polynomial CUDA-context ==============================*/

  // checking whether a pointer is on host or device and asserts device matches the polynmoial device
  static bool is_host_ptr(const void* p, int device_id)
  {
    // Note: device memory can me managed or not. Host memory can be registered or not. No distinction here.
    hipPointerAttribute_t attributes;
    CHK_STICKY(hipPointerGetAttributes(&attributes, p));
    const bool is_on_host = attributes.type == hipMemoryTypeHost ||
                            attributes.type == cudaMemoryTypeUnregistered; // unregistered is host memory
    const bool is_on_cur_device = !is_on_host && attributes.device == device_id;
    const bool is_valid_ptr = is_on_host || is_on_cur_device;
    if (!is_valid_ptr) { THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "Invalid ptr for polynomial"); }

    return is_on_host;
  }

  template <typename C = scalar_t, typename D = C, typename I = C>
  class CUDAPolynomialContext : public IPolynomialContext<C, D, I>
  {
    typedef IPolynomialContext<C, D, I> PolyContext;
    using typename IPolynomialContext<C, D, I>::State;
    using IPolynomialContext<C, D, I>::ElementSize;

  protected:
    State m_state = State::Invalid; // Current state of the polynomial context.
    uint64_t m_nof_elements = 0;    // Number of elements managed by the context.

  public:
    const DeviceContext& m_device_context;

    CUDAPolynomialContext(const DeviceContext& dev_context) : m_device_context{dev_context}
    {
      m_integrity_counter = std::make_shared<int>(0);
    }
    ~CUDAPolynomialContext() { release(); }

    void allocate(uint64_t nof_elements, State init_state, bool is_memset_zeros) override
    {
      const bool is_already_allocated = this->m_nof_elements >= nof_elements;
      this->set_state(init_state);

      if (is_already_allocated) {
        // zero the extra elements, if exist
        memset_zeros(this->m_storage, nof_elements, this->m_nof_elements);
        return;
      }

      release(); // in case allocated mem is too small and need to reallocate
      this->m_nof_elements = allocate_mem(nof_elements, &this->m_storage, is_memset_zeros);
      modified();
    }

    void memset_zeros(void* storage, uint64_t element_start_idx, uint64_t element_end_idx)
    {
      const uint64_t size = (element_end_idx - element_start_idx) * ElementSize;
      if (0 == size) { return; }

      modified();

      const auto offset = (void*)((uint64_t)storage + element_start_idx * ElementSize);
      CHK_STICKY(hipMemsetAsync(offset, 0, size, m_device_context.stream));
    }

    uint64_t allocate_mem(uint64_t nof_elements, void** storage /*OUT*/, bool is_memset_zeros)
    {
      const uint64_t nof_elements_nearset_power_of_two = ceil_to_power_of_two(nof_elements);
      const uint64_t mem_size = nof_elements_nearset_power_of_two * ElementSize;

      CHK_STICKY(hipMallocAsync(storage, mem_size, m_device_context.stream));

      if (is_memset_zeros) {
        memset_zeros(*storage, 0, nof_elements_nearset_power_of_two);
      } else {
        // if allocating more memory than requested, memset only the pad area to avoid higher invalid coefficients
        memset_zeros(*storage, nof_elements, nof_elements_nearset_power_of_two);
      }

      return nof_elements_nearset_power_of_two;
    }

    void set_storage(void* storage, uint64_t nof_elements)
    {
      release();
      m_storage = storage;
      this->m_nof_elements = nof_elements;

      modified();
    }

    // Note: this is protected and only backend can call
    void* get_storage_mutable() override
    {
      // since giving access to internal memory, cannot know if modified or not
      // backend should not take it mutable if not mutating
      modified();
      return m_storage;
    }
    const void* get_storage_immutable() override { return m_storage; }

    void extend_mem_and_pad(uint64_t nof_elements)
    {
      void* new_storage = nullptr;
      const uint64_t new_nof_elements = allocate_mem(nof_elements, &new_storage, true /*=memset zeros*/);
      const uint64_t old_mem_size = this->m_nof_elements * ElementSize;

      CHK_STICKY(
        hipMemcpyAsync(new_storage, m_storage, old_mem_size, hipMemcpyDeviceToDevice, m_device_context.stream));

      set_storage(new_storage, new_nof_elements);
    }

    void release() override
    {
      if (m_storage != nullptr) { CHK_STICKY(hipFreeAsync(m_storage, m_device_context.stream)); }

      m_storage = nullptr;
      this->m_nof_elements = 0;

      modified();
    }

    State get_state() const override { return m_state; }
    void set_state(State state) { m_state = state; }
    uint64_t get_nof_elements() const override { return m_nof_elements; }

    void from_coefficients(uint64_t nof_coefficients, const C* coefficients) override
    {
      const bool is_memset_zeros = coefficients == nullptr;
      allocate(nof_coefficients, State::Coefficients, is_memset_zeros);
      if (coefficients) {
        const bool is_ptr_on_host = is_host_ptr(coefficients, m_device_context.device_id);

        CHK_STICKY(hipMemcpyAsync(
          m_storage, coefficients, nof_coefficients * sizeof(C),
          is_ptr_on_host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice, m_device_context.stream));
        CHK_STICKY(
          hipStreamSynchronize(m_device_context.stream)); // protect against coefficients being released too soon
      }
    }

    void from_rou_evaluations(uint64_t nof_evaluations, const I* evaluations) override
    {
      const bool is_memset_zeros = evaluations == nullptr;
      allocate(nof_evaluations, State::EvaluationsOnRou_Natural, is_memset_zeros);
      if (evaluations) {
        const bool is_ptr_on_host = is_host_ptr(evaluations, m_device_context.device_id);

        CHK_STICKY(hipMemcpyAsync(
          m_storage, evaluations, nof_evaluations * sizeof(C),
          is_ptr_on_host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice, m_device_context.stream));
        CHK_STICKY(
          hipStreamSynchronize(m_device_context.stream)); // protect against evaluations being released too soon
      }
    }

    void clone(IPolynomialContext<C, D, I>& from) override
    {
      switch (from.get_state()) {
      case State::Coefficients: {
        auto [coeffs, N_coeffs] = from.get_coefficients();
        from_coefficients(N_coeffs, coeffs);
      } break;
      case State::EvaluationsOnRou_Natural: {
        auto [evals, N_evals] = from.get_rou_evaluations();
        from_rou_evaluations(N_evals, evals);
      } break;
      default:
        THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "clone() from non implemented state");
      }

      this->set_state(from.get_state()); // to handle both reversed evaluations case
    }

    std::pair<const C*, uint64_t> get_coefficients() override
    {
      transform_to_coefficients();
      return std::make_pair(static_cast<const C*>(m_storage), this->m_nof_elements);
    }

    std::tuple<IntegrityPointer<C>, uint64_t, uint64_t> get_coefficients_view() override
    {
      auto [coeffs, N] = get_coefficients();
      // when reading the pointer, if the counter was modified, the pointer is invalid
      IntegrityPointer<C> integrity_pointer(coeffs, m_integrity_counter, *m_integrity_counter);
      CHK_STICKY(hipStreamSynchronize(m_device_context.stream));
      return {std::move(integrity_pointer), N, m_device_context.device_id};
    }

    std::tuple<IntegrityPointer<I>, uint64_t, uint64_t>
    get_rou_evaluations_view(uint64_t nof_evaluations, bool is_reversed)
    {
      if (nof_evaluations != 0 && nof_evaluations < get_nof_elements()) {
        THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "get_rou_evaluations_view() can only expand #evals");
      }
      transform_to_evaluations(nof_evaluations, is_reversed);
      auto [evals, N] = get_rou_evaluations();
      // when reading the pointer, if the counter was modified, the pointer is invalid
      IntegrityPointer<I> integrity_pointer(evals, m_integrity_counter, *m_integrity_counter);
      CHK_STICKY(hipStreamSynchronize(m_device_context.stream));
      return {std::move(integrity_pointer), N, m_device_context.device_id};
    }

    std::pair<const I*, uint64_t> get_rou_evaluations() override
    {
      const bool is_reversed = this->m_state == State::EvaluationsOnRou_Reversed;
      transform_to_evaluations(0, is_reversed);
      return std::make_pair(static_cast<const I*>(m_storage), this->m_nof_elements);
    }

    void transform_to_coefficients(uint64_t nof_coefficients = 0) override
    {
      // cannot really get more coefficients but sometimes want to pad for NTT. In that case
      // nof_coefficients>m_nof_elements
      nof_coefficients = (nof_coefficients == 0) ? this->m_nof_elements : ceil_to_power_of_two(nof_coefficients);
      const bool is_same_nof_coefficients = this->m_nof_elements == nof_coefficients;
      const bool is_already_in_state = this->m_state == State::Coefficients && is_same_nof_coefficients;
      if (is_already_in_state) { return; }

      if (nof_coefficients < this->m_nof_elements) {
        THROW_ICICLE_ERR(
          IcicleError_t::InvalidArgument, "polynomial shrinking not supported. Probably encountered a bug");
      }

      modified();

      const bool is_already_in_coeffs = this->m_state == State::Coefficients;
      // case 1: already in coefficients. Need to allocate larger memory and zero pad
      if (is_already_in_coeffs) {
        extend_mem_and_pad(nof_coefficients);
        return;
      }

      // case 2: transform from evaluations. May need to allocate larger memory
      I* evals = static_cast<I*>(m_storage);
      C* coeffs = static_cast<C*>(m_storage);
      const bool is_allocate_new_mem = nof_coefficients > this->m_nof_elements;
      if (is_allocate_new_mem) {
        void* new_mem = nullptr;
        nof_coefficients = allocate_mem(nof_coefficients, &new_mem, true /*=memset zeros*/);
        coeffs = static_cast<C*>(new_mem);
      }

      // transform from evaluations to coefficients
      auto ntt_config = ntt::default_ntt_config<C>(m_device_context);
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;

      ntt_config.ordering =
        (this->m_state == State::EvaluationsOnRou_Natural) ? ntt::Ordering::kNN : ntt::Ordering::kRN;
      // Note: it is important to do the NTT with old size because padding in evaluations form is computing another
      // (higher order) polynomial
      CHK_STICKY(ntt::ntt(evals, this->m_nof_elements, ntt::NTTDir::kInverse, ntt_config, coeffs));
      this->set_state(State::Coefficients);

      if (is_allocate_new_mem) { set_storage(coeffs, nof_coefficients); } // release old memory and use new
    }

    void transform_to_evaluations(uint64_t nof_evaluations = 0, bool is_reversed = false) override
    {
      // TODO Yuval: can maybe optimize this
      nof_evaluations = (nof_evaluations == 0) ? this->m_nof_elements : ceil_to_power_of_two(nof_evaluations);
      const bool is_same_nof_evaluations = nof_evaluations == this->m_nof_elements;
      const bool is_same_order = is_reversed && this->m_state == State::EvaluationsOnRou_Reversed ||
                                 (!is_reversed && State::EvaluationsOnRou_Natural);
      const bool is_already_in_state = is_same_nof_evaluations && is_same_order;
      if (is_already_in_state) { return; }

      if (nof_evaluations < this->m_nof_elements) {
        THROW_ICICLE_ERR(
          IcicleError_t::InvalidArgument, "polynomial shrinking not supported. Probably encountered a bug");
      }

      modified();

      // TODO Yuval: evaluations->evaluations with different ordering can be implemented via inplace reorder more
      // efficiently than it is now

      // There are 3 cases:
      // (1) coefficients to evaluations
      //    (1a) same size -> NTT (NR or NN)
      //    (1b) different_size -> alloc new mem, copy coeffs and NTT inplace
      // (2) evaluations to evaluations (interpolation)
      //     transform to coefficients, extend memory, then NTT back to evals (NR or NN)

      const bool is_eval_to_eval = this->m_state != State::Coefficients;
      // interpolating more points requires going back to coefficients first. Note that it muse be done with the
      // original size. INTT after padding computes a higher degree polynomial
      if (is_eval_to_eval) { transform_to_coefficients(); }

      // reaching this point means polynomial is in coefficient form
      const bool is_allocate_new_mem = nof_evaluations > this->m_nof_elements;
      // allocate more memory and copy+pad
      if (is_allocate_new_mem) { extend_mem_and_pad(nof_evaluations); }

      C* coeffs = static_cast<C*>(m_storage);
      I* evals = static_cast<I*>(m_storage);
      auto ntt_config = ntt::default_ntt_config<C>(m_device_context);
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      // already copied the coefficients with padding. Now computing evaluations.
      ntt_config.ordering = is_reversed ? ntt::Ordering::kNR : ntt::Ordering::kNN;
      CHK_STICKY(ntt::ntt(coeffs, nof_evaluations, ntt::NTTDir::kForward, ntt_config, evals));

      this->set_state(is_reversed ? State::EvaluationsOnRou_Reversed : State::EvaluationsOnRou_Natural);
    }

    void print(std::ostream& os) override
    {
      if (this->get_state() == State::Coefficients) {
        print_coeffs(os);
      } else {
        print_evals(os);
      }
    }

    void print_coeffs(std::ostream& os)
    {
      transform_to_coefficients();
      auto host_coeffs = std::make_unique<C[]>(this->m_nof_elements);
      // using stream since previous ops may still be in progress. Sync stream before reading CPU mem
      CHK_STICKY(hipMemcpyAsync(
        host_coeffs.get(), m_storage, this->m_nof_elements * sizeof(C), hipMemcpyDeviceToHost,
        m_device_context.stream));
      CHK_STICKY(hipStreamSynchronize(m_device_context.stream));

      os << "(id=" << PolyContext::m_id << ")[";
      for (size_t i = 0; i < this->m_nof_elements; ++i) {
        os << host_coeffs[i];
        if (i < this->m_nof_elements - 1) { os << ", "; }
      }
      os << "] (state=coefficients)" << std::endl;
    }

    void print_evals(std::ostream& os)
    {
      transform_to_evaluations();
      auto host_evals = std::make_unique<I[]>(this->m_nof_elements);
      // using stream since previous ops may still be in progress. Sync stream before reading CPU mem
      CHK_STICKY(hipMemcpyAsync(
        host_evals.get(), m_storage, this->m_nof_elements * sizeof(I), hipMemcpyDeviceToHost,
        m_device_context.stream));
      CHK_STICKY(hipStreamSynchronize(m_device_context.stream));

      os << "(id=" << PolyContext::m_id << ")[";
      for (size_t i = 0; i < this->m_nof_elements; ++i) {
        os << host_evals[i];
        if (i < this->m_nof_elements - 1) { os << ", "; }
      }

      if (this->get_state() == State::EvaluationsOnRou_Reversed) {
        os << "] (state=rou evaluations Reversed)" << std::endl;
      } else {
        os << "] (state=rou evaluations )" << std::endl;
      }
    }

  private:
    // Members
    void* m_storage = nullptr;
    std::shared_ptr<int> m_integrity_counter; // used to implement integrity of coefficients pointer

    void modified() { (*m_integrity_counter)++; }
  };

  /*============================== Polynomial CUDA-backend ==============================*/

  template <typename C = scalar_t, typename D = C, typename I = C>
  class CUDAPolynomialBackend : public IPolynomialBackend<C, D, I>
  {
    typedef std::shared_ptr<IPolynomialContext<C, D, I>> PolyContext;
    typedef typename IPolynomialContext<C, D, I>::State State;

    int64_t* d_degree = nullptr; // used to avoid alloc/release every time

  public:
    const DeviceContext& m_device_context;
    CUDAPolynomialBackend(const DeviceContext& dev_context) : m_device_context{dev_context}
    {
      CHK_STICKY(hipMallocAsync(&d_degree, sizeof(int64_t), m_device_context.stream));
    }
    ~CUDAPolynomialBackend() { CHK_STICKY(hipFreeAsync(d_degree, m_device_context.stream)); }

    void from_coefficients(PolyContext p, uint64_t nof_coefficients, const C* coefficients) override
    {
      p->from_coefficients(nof_coefficients, coefficients);
    }

    void from_rou_evaluations(PolyContext p, uint64_t nof_evaluations, const I* evaluations) override
    {
      p->from_rou_evaluations(nof_evaluations, evaluations);
    }

    void clone(PolyContext out, PolyContext in) override { out->clone(*in); }

    template <typename T = C>
    T* get_context_storage_mutable(PolyContext p)
    {
      return static_cast<T*>(IPolynomialBackend<C, D, I>::get_context_storage_mutable(p));
    }

    template <typename T = C>
    const T* get_context_storage_immutable(PolyContext& p)
    {
      return static_cast<const T*>(IPolynomialBackend<C, D, I>::get_context_storage_immutable(p));
    }

    void slice(PolyContext out, PolyContext in, uint64_t offset, uint64_t stride, uint64_t size) override
    {
      assert_device_compatability(out, in);
      auto [in_coeffs, in_size] = in->get_coefficients();
      // size=0 means take as much as elements as there are to take
      uint64_t out_size = (size > 0) ? size : (1 + (in_size - 1 - offset) / stride);

      out->allocate(out_size, State::Coefficients, false /*=memset zeros*/);
      auto out_coeffs = get_context_storage_mutable(out);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (out_size + NOF_THREADS - 1) / NOF_THREADS;
      slice_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(
        in_coeffs, out_coeffs, offset, stride, out_size);

      CHK_LAST();
    }

    void add_sub(PolyContext& res, PolyContext a, PolyContext b, bool add1_sub0)
    {
      assert_device_compatability(a, b);
      assert_device_compatability(a, res);

      // add/sub can be done in both coefficients or evaluations, but operands must be in the same state.
      // For evaluations, same state also means same number of evaluations (and on same domain).
      // If not same state, compute in coefficients since computing in evaluations may require to interpolate a large
      // size. Consider a+b where a is degree 128 and b degree 4. In coefficients b has 4 elements but in evaluations
      // need 128.
      const bool is_same_size = a->get_nof_elements() == b->get_nof_elements();
      const bool is_same_state = a->get_state() == b->get_state();
      const auto output_state = (is_same_size && is_same_state) ? a->get_state() : State::Coefficients;
      const auto output_size = max(a->get_nof_elements(), b->get_nof_elements());

      if (State::Coefficients == output_state) {
        a->transform_to_coefficients();
        b->transform_to_coefficients();
      }
      const auto a_mem_p = get_context_storage_immutable(a);
      const auto b_mem_p = get_context_storage_immutable(b);

      res->allocate(output_size, output_state);
      auto res_mem_p = get_context_storage_mutable(res);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (output_size + NOF_THREADS - 1) / NOF_THREADS;
      add_sub_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(
        a_mem_p, b_mem_p, a->get_nof_elements(), b->get_nof_elements(), add1_sub0, res_mem_p);

      CHK_LAST();
    }

    void add(PolyContext& res, PolyContext a, PolyContext b) override { add_sub(res, a, b, true /*=add*/); }
    void subtract(PolyContext res, PolyContext a, PolyContext b) override { add_sub(res, a, b, false /*=sub*/); }

    void multiply(PolyContext c, PolyContext a, PolyContext b) override
    {
      assert_device_compatability(a, b);
      assert_device_compatability(a, c);

      const bool is_a_scalar = a->get_nof_elements() == 1;
      const bool is_b_scalar = b->get_nof_elements() == 1;

      // TODO: can add kernel that takes the scalar as device memory
      if (is_a_scalar) {
        return multiply(c, b, get_coeff(a, 0));
      } else if (is_b_scalar) {
        return multiply(c, a, get_coeff(b, 0));
      }

      const bool is_multiply_with_cosets = true; // TODO  Yuval: check when faster to do so.
      if (is_multiply_with_cosets) { return multiply_with_cosets(c, a, b); }
      return multiply_with_padding(c, a, b);
    }

    void multiply(PolyContext out, PolyContext p, D scalar) override
    {
      assert_device_compatability(out, p);

      // element wise multiplication is similar both in coefficients and evaluations (regardless of order too)
      const auto state = p->get_state();
      const auto N = p->get_nof_elements();

      auto p_elements_p =
        state == State::Coefficients ? get_context_storage_immutable<C>(p) : get_context_storage_immutable<I>(p);

      out->allocate(N, state, false /*=memset zeros*/);
      auto out_evals_p =
        state == State::Coefficients ? get_context_storage_mutable<C>(out) : get_context_storage_mutable<I>(out);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      mul_scalar_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(p_elements_p, scalar, N, out_evals_p);

      CHK_LAST();
    }

    void multiply_with_padding(PolyContext c, PolyContext a, PolyContext b)
    {
      // TODO Yuval: by using the degree I can optimize the memory size and avoid redundant computations too
      const uint64_t a_N_orig = a->get_nof_elements();
      const uint64_t b_N_orig = b->get_nof_elements();
      const uint64_t N = max(a_N_orig, b_N_orig);
      const uint64_t c_N = 2 * N;

      // (1) transform a,b to 2N evaluations
      a->transform_to_evaluations(c_N, true /*=reversed*/);
      b->transform_to_evaluations(c_N, true /*=reversed*/);
      auto [a_evals_p, a_N] = a->get_rou_evaluations();
      auto [b_evals_p, b_N] = b->get_rou_evaluations();

      // (2) allocate c (c=a*b) and compute element-wise multiplication on evaluations
      c->allocate(c_N, State::EvaluationsOnRou_Reversed, false /*=memset zeros*/);
      auto c_evals_p = get_context_storage_mutable<I>(c);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (c_N + NOF_THREADS - 1) / NOF_THREADS;
      mul_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(a_evals_p, b_evals_p, c_N, c_evals_p);

      CHK_LAST();
    }

    void multiply_with_cosets(PolyContext c, PolyContext a, PolyContext b)
    {
      const uint64_t a_N = a->get_nof_elements();
      const uint64_t b_N = b->get_nof_elements();
      const uint64_t N = max(a_N, b_N);

      // (1) transform a,b to coefficients such that both have N coefficients
      a->transform_to_coefficients(N);
      b->transform_to_coefficients(N);
      auto [a_coeff_p, _] = a->get_coefficients();
      auto [b_coeff_p, __] = b->get_coefficients();
      // (2) allocate c (c=a*b)
      const uint64_t c_N = 2 * N;
      c->allocate(c_N, State::EvaluationsOnRou_Reversed, false /*=memset zeros*/);
      auto c_evals_low_p = get_context_storage_mutable<I>(c);
      I* c_evals_high_p = c_evals_low_p + N;

      // (3) compute NTT of a,b on coset and write to c
      auto ntt_config = ntt::default_ntt_config<C>(m_device_context);
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = ntt::Ordering::kNR;
      ntt_config.coset_gen = ntt::get_root_of_unity_from_domain<C>((uint64_t)log2(c_N), ntt_config.ctx);

      CHK_STICKY(ntt::ntt(a_coeff_p, N, ntt::NTTDir::kForward, ntt_config, c_evals_low_p));  // a_H1
      CHK_STICKY(ntt::ntt(b_coeff_p, N, ntt::NTTDir::kForward, ntt_config, c_evals_high_p)); // b_H1

      // (4) compute a_H1 * b_H1 inplace
      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      mul_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(
        c_evals_low_p, c_evals_high_p, N, c_evals_high_p);
      // (5) transform a,b to evaluations
      a->transform_to_evaluations(N, true /*=reversed*/);
      b->transform_to_evaluations(N, true /*=reversed*/);
      auto [a_evals_p, a_nof_evals] = a->get_rou_evaluations();
      auto [b_evals_p, b_nof_evals] = b->get_rou_evaluations();

      // (6) compute a_H0 * b_H0
      mul_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(a_evals_p, b_evals_p, N, c_evals_low_p);

      CHK_LAST();
    }

    void divide(PolyContext Q /*OUT*/, PolyContext R /*OUT*/, PolyContext a, PolyContext b) override
    {
      assert_device_compatability(a, b);
      assert_device_compatability(a, Q);
      assert_device_compatability(a, R);

      auto [a_coeffs, a_N] = a->get_coefficients();
      auto [b_coeffs, b_N] = b->get_coefficients();

      const int64_t deg_a = degree(a);
      const int64_t deg_b = degree(b);
      if (deg_a < deg_b || deg_b < 0) {
        THROW_ICICLE_ERR(
          IcicleError_t::InvalidArgument, "Polynomial division (CUDA backend): numerator degree must be "
                                          "greater-or-equal to denumerator degree and denumerator must not be zero");
      }

      // init: Q=0, R=a
      Q->allocate(deg_a - deg_b + 1, State::Coefficients, true /*=memset zeros*/);
      auto Q_coeffs = get_context_storage_mutable(Q);

      //    TODO Yuval: Can do better in terms of memory allocation? deg(R) <= deg(b) by definition but it starts as
      R->allocate(a_N, State::Coefficients, false /*=memset_zeros*/);
      auto R_coeffs = get_context_storage_mutable(R);
      CHK_STICKY(
        hipMemcpyAsync(R_coeffs, a_coeffs, a_N * sizeof(C), hipMemcpyDeviceToDevice, m_device_context.stream));

      const C& lc_b_inv = C::inverse(get_coeff(b, deg_b)); // largest coeff of b

      int64_t deg_r = deg_a;
      while (deg_r >= deg_b) {
        // each iteration is removing the largest monomial in r until deg(r)<deg(b)
        const int NOF_THREADS = 128;
        const int NOF_BLOCKS = ((deg_r + 1) + NOF_THREADS - 1) / NOF_THREADS; // 'deg_r+1' is number of elements in R
        school_book_division_step<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(
          R_coeffs, Q_coeffs, b_coeffs, deg_r, deg_b, lc_b_inv);

        // faster than degree(R) based on the fact that degree is decreasing
        deg_r = degree_internal(R, deg_r + 1 /*size of R*/);
      }

      CHK_LAST();
    }

    void quotient(PolyContext Q, PolyContext op_a, PolyContext op_b) override
    {
      // TODO: can implement more efficiently?
      auto R = std::make_shared<CUDAPolynomialContext<C, D, I>>(m_device_context);
      divide(Q, R, op_a, op_b);
    }

    void remainder(PolyContext R, PolyContext op_a, PolyContext op_b) override
    {
      // TODO: can implement more efficiently?
      auto Q = std::make_shared<CUDAPolynomialContext<C, D, I>>(m_device_context);
      divide(Q, R, op_a, op_b);
    }

    void divide_by_vanishing_polynomial(PolyContext out, PolyContext numerator, uint64_t vanishing_poly_degree) override
    {
      assert_device_compatability(numerator, out);

      // TODO Yuval: vanishing polynomial x^n-1 evaluates to zero on ROU
      // Therefore constant on coset with u as coset generator ((wu)^n-1 = w^n*u^n-1 = u^n-1)
      // This is true for a coset of size n but if numerator is of size >n, then I need a larger coset and it
      // doesn't hold. Need to use this fact to optimize division

      // (1) allocate vanishing polynomial in coefficients form
      // TODO Yuval: maybe instead of taking numerator memory and modiyfing it diretcly add a state for evaluations
      // on coset of rou. In that case I can remain in this state and also won't need to access input memory
      // directly
      numerator->transform_to_coefficients();
      auto numerator_coeffs = get_context_storage_mutable(numerator);
      const auto N = numerator->get_nof_elements();
      if (vanishing_poly_degree > N) {
        THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "divide_by_vanishing_polynomial(): degree is too large");
      }
      out->allocate(N, State::Coefficients, true /*=set zeros*/);
      add_monomial_inplace(out, C::zero() - C::one(), 0);         //-1
      add_monomial_inplace(out, C::one(), vanishing_poly_degree); //+x^n

      // (2) NTT on coset. Note that NTT on ROU evaluates to zeros for vanihsing polynomials by definition.
      // Therefore evaluation on coset is required to compute non-zero evaluations, which make element-wise division
      // possible
      auto out_coeffs = get_context_storage_mutable(out);
      auto ntt_config = ntt::default_ntt_config<C>(m_device_context);
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = ntt::Ordering::kNM;
      ntt_config.coset_gen = ntt::get_root_of_unity_from_domain<C>((uint64_t)log2(2 * N), ntt_config.ctx);

      CHK_STICKY(ntt::ntt(out_coeffs, N, ntt::NTTDir::kForward, ntt_config, out_coeffs));
      CHK_STICKY(ntt::ntt(numerator_coeffs, N, ntt::NTTDir::kForward, ntt_config, numerator_coeffs));

      // (3) element wise division
      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      div_element_wise_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(
        numerator_coeffs, out_coeffs, N, out_coeffs);

      // (4) INTT back both a and out
      ntt_config.ordering = ntt::Ordering::kMN;
      CHK_STICKY(ntt::ntt(out_coeffs, N, ntt::NTTDir::kInverse, ntt_config, out_coeffs));
      CHK_STICKY(ntt::ntt(numerator_coeffs, N, ntt::NTTDir::kInverse, ntt_config, numerator_coeffs));
    }

    // arithmetic with monomials
    void add_monomial_inplace(PolyContext& poly, C monomial_coeff, uint64_t monomial) override
    {
      const uint64_t new_nof_elements = max(poly->get_nof_elements(), monomial + 1);
      poly->transform_to_coefficients(new_nof_elements);
      auto coeffs = get_context_storage_mutable(poly);
      add_single_element_inplace<<<1, 1, 0, m_device_context.stream>>>(coeffs + monomial, monomial_coeff);

      CHK_LAST();
    }

    void sub_monomial_inplace(PolyContext& poly, C monomial_coeff, uint64_t monomial) override
    {
      add_monomial_inplace(poly, C::zero() - monomial_coeff, monomial);
    }

    int64_t degree(PolyContext p) override { return degree_internal(p, p->get_nof_elements()); }

    // search degree starting from len, searching down (towards coeff0)
    int64_t degree_internal(PolyContext p, uint64_t len)
    {
      // TODO: parallelize kernel? Note that typically the largest coefficient is expected in the higher half since
      // memory is allocate based on #coefficients

      auto [coeff, _] = p->get_coefficients();

      int64_t h_degree;
      highest_non_zero_idx<<<1, 1, 0, m_device_context.stream>>>(coeff, len, d_degree);
      CHK_STICKY(
        hipMemcpyAsync(&h_degree, d_degree, sizeof(int64_t), hipMemcpyDeviceToHost, m_device_context.stream));
      CHK_STICKY(hipStreamSynchronize(m_device_context.stream)); // sync to make sure return value is copied to host

      return h_degree;
    }

  public:
    void evaluate(PolyContext p, const D* x, I* eval) override
    {
      // TODO Yuval: maybe use Horner's rule and just evaluate each domain point per thread. Alternatively Need to
      // reduce in parallel.

      auto [coeff, nof_coeff] = p->get_coefficients();

      const bool is_x_on_host = is_host_ptr(x, m_device_context.device_id);
      const bool is_eval_on_host = is_host_ptr(eval, m_device_context.device_id);

      const D* d_x = x;
      D* allocated_x = nullptr;
      if (is_x_on_host) {
        CHK_STICKY(hipMallocAsync(&allocated_x, sizeof(I), m_device_context.stream));
        CHK_STICKY(hipMemcpyAsync(allocated_x, x, sizeof(I), hipMemcpyHostToDevice, m_device_context.stream));
        d_x = allocated_x;
      }
      I* d_eval = eval;
      if (is_eval_on_host) { CHK_STICKY(hipMallocAsync(&d_eval, sizeof(I), m_device_context.stream)); }

      // TODO Yuval: other methods can avoid this allocation. Also for eval_on_domain() no need to reallocate every time
      I* d_tmp = nullptr;
      CHK_STICKY(hipMallocAsync(&d_tmp, sizeof(I) * nof_coeff, m_device_context.stream));
      const int NOF_THREADS = 32;
      const int NOF_BLOCKS = (nof_coeff + NOF_THREADS - 1) / NOF_THREADS;
      evaluate_polynomial_without_reduction<<<NOF_BLOCKS, NOF_THREADS, 0, m_device_context.stream>>>(
        d_x, coeff, nof_coeff, d_tmp); // TODO Yuval: parallelize kernel
      dummy_reduce<<<1, 1, 0, m_device_context.stream>>>(d_tmp, nof_coeff, d_eval);

      if (is_eval_on_host) {
        CHK_STICKY(hipMemcpyAsync(eval, d_eval, sizeof(I), hipMemcpyDeviceToHost, m_device_context.stream));
        CHK_STICKY(hipStreamSynchronize(m_device_context.stream)); // sync to make sure return value is copied to host
        CHK_STICKY(hipFreeAsync(d_eval, m_device_context.stream));
      }
      if (allocated_x) { CHK_STICKY(hipFreeAsync(allocated_x, m_device_context.stream)); }
      CHK_STICKY(hipFreeAsync(d_tmp, m_device_context.stream));
    }

    void evaluate_on_domain(PolyContext p, const D* domain, uint64_t size, I* evaluations /*OUT*/) override
    {
      // TODO Yuval: implement more efficiently ??
      for (uint64_t i = 0; i < size; ++i) {
        evaluate(p, &domain[i], &evaluations[i]);
      }
    }

    uint64_t copy_coeffs(PolyContext op, C* out_coeffs, uint64_t start_idx, uint64_t end_idx) override
    {
      const uint64_t nof_coeffs = op->get_nof_elements();
      if (nullptr == out_coeffs) { return nof_coeffs; } // no allocated memory

      const bool is_valid_start_idx = start_idx < nof_coeffs;
      const bool is_valid_end_idx = end_idx < nof_coeffs && end_idx >= start_idx;
      const bool is_valid_indices = is_valid_start_idx && is_valid_end_idx;
      if (!is_valid_indices) {
        // return -1 instead? I could but 'get_coeff()' cannot with its current declaration
        THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "copy_coeffs() invalid indices");
      }

      op->transform_to_coefficients();
      auto [device_coeffs, _] = op->get_coefficients();
      const size_t nof_coeffs_to_copy = end_idx - start_idx + 1;
      const bool is_copy_to_host = is_host_ptr(out_coeffs, m_device_context.device_id);
      CHK_STICKY(hipMemcpyAsync(
        out_coeffs, device_coeffs + start_idx, nof_coeffs_to_copy * sizeof(C),
        is_copy_to_host ? hipMemcpyDeviceToHost : hipMemcpyDeviceToDevice, m_device_context.stream));
      CHK_STICKY(hipStreamSynchronize(m_device_context.stream)); // sync to make sure return value is copied

      return nof_coeffs_to_copy;
    }

    // read coefficients to host
    C get_coeff(PolyContext op, uint64_t coeff_idx) override
    {
      C host_coeff;
      copy_coeffs(op, &host_coeff, coeff_idx, coeff_idx);
      return host_coeff;
    }

    std::tuple<IntegrityPointer<C>, uint64_t /*size*/, uint64_t /*device_id*/>
    get_coefficients_view(PolyContext p) override
    {
      return p->get_coefficients_view();
    }

    std::tuple<IntegrityPointer<I>, uint64_t /*size*/, uint64_t /*device_id*/>
    get_rou_evaluations_view(PolyContext p, uint64_t nof_evaluations, bool is_reversed) override
    {
      return p->get_rou_evaluations_view(nof_evaluations, is_reversed);
    }

    inline void assert_device_compatability(PolyContext a, PolyContext b) const
    {
      CUDAPolynomialContext<C, D, I>* a_cuda = static_cast<CUDAPolynomialContext<C, D, I>*>(a.get());
      CUDAPolynomialContext<C, D, I>* b_cuda = static_cast<CUDAPolynomialContext<C, D, I>*>(b.get());

      const bool is_same_device = a_cuda->m_device_context.device_id == b_cuda->m_device_context.device_id;
      if (!is_same_device) {
        THROW_ICICLE_ERR(
          IcicleError_t::InvalidArgument, "CUDA backend: incompatible polynomials, on different devices");
      }
    }
  };

  /*============================== Polynomial CUDA-factory ==============================*/
  template <typename C, typename D, typename I>
  CUDAPolynomialFactory<C, D, I>::CUDAPolynomialFactory()
  {
    int nof_cuda_devices = -1;
    CHK_STICKY(hipGetDeviceCount(&nof_cuda_devices));
    int orig_device = -1;

    CHK_STICKY(hipGetDevice(&orig_device));
    m_device_streams.resize(nof_cuda_devices, nullptr);

    for (int dev_id = 0; dev_id < nof_cuda_devices; ++dev_id) {
      CHK_STICKY(hipSetDevice(dev_id));
      CHK_STICKY(hipStreamCreate(&m_device_streams[dev_id]));
      DeviceContext context = {m_device_streams[dev_id], (size_t)dev_id, 0x0 /*mempool*/};
      m_device_contexts.push_back(context);
    }
    CHK_STICKY(hipSetDevice(orig_device)); // setting back original device
  }

  template <typename C, typename D, typename I>
  CUDAPolynomialFactory<C, D, I>::~CUDAPolynomialFactory()
  {
    for (auto stream_it : m_device_streams) {
      CHK_STICKY(hipStreamDestroy(stream_it)); // TODO Yuval: why does it fail?
    }
  }

  template <typename C, typename D, typename I>
  std::shared_ptr<IPolynomialContext<C, D, I>> CUDAPolynomialFactory<C, D, I>::create_context()
  {
    int cuda_device_id = -1;
    CHK_STICKY(hipGetDevice(&cuda_device_id));
    return std::make_shared<CUDAPolynomialContext<C, D, I>>(m_device_contexts[cuda_device_id]);
  }

  template <typename C, typename D, typename I>
  std::shared_ptr<IPolynomialBackend<C, D, I>> CUDAPolynomialFactory<C, D, I>::create_backend()
  {
    int cuda_device_id = -1;
    CHK_STICKY(hipGetDevice(&cuda_device_id));
    return std::make_shared<CUDAPolynomialBackend<C, D, I>>(m_device_contexts[cuda_device_id]);
  }

  // explicit instantiation for default type (scalar)
  template class CUDAPolynomialContext<>;
  template class CUDAPolynomialBackend<>;
  template class CUDAPolynomialFactory<>;

} // namespace polynomials
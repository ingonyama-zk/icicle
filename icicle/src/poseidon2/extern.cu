#include "utils/utils.h"

#include "fields/field_config.cuh"
using namespace field_config;

#include "poseidon.cu"

namespace poseidon2 {
  extern "C" hipError_t CONCAT_EXPAND(FIELD, create_optimized_poseidon2_constants_cuda)(
    int width,
    int alpha,
    int internal_rounds,
    int external_rounds,
    const scalar_t* round_constants,
    const scalar_t* internal_matrix_diag,
    MdsType mds_type,
    DiffusionStrategy diffusion,
    device_context::DeviceContext& ctx,
    Poseidon2Constants<scalar_t>* poseidon_constants)
  {
    return create_optimized_poseidon2_constants<scalar_t>(
      width, alpha, internal_rounds, external_rounds, round_constants, internal_matrix_diag, mds_type, diffusion, ctx,
      poseidon_constants);
  }

  extern "C" hipError_t CONCAT_EXPAND(FIELD, init_optimized_poseidon2_constants_cuda)(
    int width,
    MdsType mds_type,
    DiffusionStrategy diffusion,
    device_context::DeviceContext& ctx,
    Poseidon2Constants<scalar_t>* constants)
  {
    return init_optimized_poseidon2_constants<scalar_t>(width, mds_type, diffusion, ctx, constants);
  }

  extern "C" hipError_t CONCAT_EXPAND(FIELD, poseidon2_hash_cuda)(
    scalar_t* input,
    scalar_t* output,
    int number_of_states,
    int width,
    const Poseidon2Constants<scalar_t>* constants,
    Poseidon2Config* config)
  {
#define P2_HASH_T(width)                                                                                               \
  case width:                                                                                                          \
    return poseidon2_hash<scalar_t, width>(input, output, number_of_states, *constants, *config);

    switch (width) {
      P2_HASH_T(2)
      P2_HASH_T(3)
      P2_HASH_T(4)
      P2_HASH_T(8)
      P2_HASH_T(12)
      P2_HASH_T(16)
      P2_HASH_T(20)
      P2_HASH_T(24)
    default:
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument, "PoseidonHash: #arity must be one of [2, 3, 4, 8, 12, 16, 20, 24]");
    }
    return CHK_LAST();
  }
} // namespace poseidon2
#include "poseidon2/poseidon2.cuh"
#include "constants.cu"
#include "kernels.cu"

namespace poseidon2 {
  static int poseidon_block_size = 128;

  template <typename S, int T>
  int poseidon_number_of_blocks(size_t number_of_states)
  {
    return number_of_states / poseidon_block_size + static_cast<bool>(number_of_states % poseidon_block_size);
  }

  template <typename S, int T>
  hipError_t permute_many(
    S* states, S* states_out, size_t number_of_states, const Poseidon2Constants<S>& constants, hipStream_t& stream)
  {
    poseidon2_permutation_kernel<S, T>
      <<<poseidon_number_of_blocks<S, T>(number_of_states), poseidon_block_size, 0, stream>>>(
        states, states_out, number_of_states, constants);
    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  template <typename S, int T>
  hipError_t poseidon2_hash(
    S* states,
    S* output,
    size_t number_of_states,
    const Poseidon2Constants<S>& constants,
    const Poseidon2Config& config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = config.ctx.stream;
    S* d_states;
    if (config.are_states_on_device) {
      d_states = states;
    } else {
      // allocate memory for {number_of_states} states of {t} scalars each
      CHK_IF_RETURN(hipMallocAsync(&d_states, number_of_states * T * sizeof(S), stream))
      CHK_IF_RETURN(hipMemcpyAsync(d_states, states, number_of_states * T * sizeof(S), hipMemcpyHostToDevice, stream))
    }

    hipError_t hash_error = permute_many<S, T>(d_states, d_states, number_of_states, constants, stream);
    CHK_IF_RETURN(hash_error);

    if (config.mode == PoseidonMode::COMPRESSION) {
      S* output_device;
      if (config.are_outputs_on_device) {
        output_device = output;
      } else {
        CHK_IF_RETURN(hipMallocAsync(&output_device, number_of_states * sizeof(S), stream))
      }

      get_hash_results<S, T><<<poseidon_number_of_blocks<S, T>(number_of_states), poseidon_block_size, 0, stream>>>(
        d_states, number_of_states, config.output_index, output_device);
      CHK_IF_RETURN(hipPeekAtLastError());

      if (!config.are_outputs_on_device) {
        CHK_IF_RETURN(
          hipMemcpyAsync(output, output_device, number_of_states * sizeof(S), hipMemcpyDeviceToHost, stream));
        CHK_IF_RETURN(hipFreeAsync(output_device, stream));
      }
    } else {
      if (!config.are_states_on_device || !config.are_outputs_on_device) {
        CHK_IF_RETURN(
          hipMemcpyAsync(output, d_states, number_of_states * T * sizeof(S), hipMemcpyDeviceToHost, stream));
      }
    }

    if (!config.are_states_on_device) CHK_IF_RETURN(hipFreeAsync(d_states, stream));

    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    return CHK_LAST();
  }
} // namespace poseidon2
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

// include list of test files
// Ensure the device_error_test.cu is last to prevent aborting mid-test run
#include "primitives_test.cu"
#include "error_handler_test.cu"
#include "device_error_test.cu"

int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  printf("running gtests...\n");
  return RUN_ALL_TESTS();
}
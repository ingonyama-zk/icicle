#include <benchmark/benchmark.h>
#include "utils/test_functions.cuh"
#include "fields/field_config.cuh"

using namespace field_config;

static void BM_FieldAdd(benchmark::State& state)
{
  constexpr int N = 256;
  int n = state.range(0) / N;
  scalar_t* scalars1;
  scalar_t* scalars2;
  assert(!hipMalloc(&scalars1, n * sizeof(scalar_t)));
  assert(!hipMalloc(&scalars2, n * sizeof(scalar_t)));

  assert(device_populate_random<scalar_t>(scalars1, n) == hipSuccess);
  assert(device_populate_random<scalar_t>(scalars2, n) == hipSuccess);

  for (auto _ : state) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    assert((vec_add<scalar_t, scalar_t, N>(scalars1, scalars2, scalars1, n)) == hipSuccess);
    assert(hipStreamSynchronize(0) == hipSuccess);
    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    state.SetIterationTime((double)(milliseconds / 1000));
  }
  hipFree(scalars1);
  hipFree(scalars2);
}

static void BM_FieldMul(benchmark::State& state)
{
  constexpr int N = 128;
  int n = state.range(0) / N;
  scalar_t* scalars1;
  scalar_t* scalars2;
  assert(!hipMalloc(&scalars1, n * sizeof(scalar_t)));
  assert(!hipMalloc(&scalars2, n * sizeof(scalar_t)));

  assert(device_populate_random<scalar_t>(scalars1, n) == hipSuccess);
  assert(device_populate_random<scalar_t>(scalars2, n) == hipSuccess);

  for (auto _ : state) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    assert((vec_mul<scalar_t, scalar_t, N>(scalars1, scalars2, scalars1, n)) == hipSuccess);
    assert(hipStreamSynchronize(0) == hipSuccess);
    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    state.SetIterationTime((double)(milliseconds / 1000));
  }
  hipFree(scalars1);
  hipFree(scalars2);
}

static void BM_FieldSqr(benchmark::State& state)
{
  constexpr int N = 128;
  int n = state.range(0) / N;
  scalar_t* scalars;
  assert(!hipMalloc(&scalars, n * sizeof(scalar_t)));

  assert(device_populate_random<scalar_t>(scalars, n) == hipSuccess);

  for (auto _ : state) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    assert((field_vec_sqr<scalar_t, N>(scalars, scalars, n)) == hipSuccess);
    assert(hipStreamSynchronize(0) == hipSuccess);
    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    state.SetIterationTime((double)(milliseconds / 1000));
  }
  hipFree(scalars);
}

BENCHMARK(BM_FieldAdd)->Range(1 << 28, 1 << 28)->Unit(benchmark::kMicrosecond);
BENCHMARK(BM_FieldMul)->Range(1 << 27, 1 << 27)->Unit(benchmark::kMicrosecond);
BENCHMARK(BM_FieldSqr)->Range(1 << 27, 1 << 27)->Unit(benchmark::kMicrosecond);

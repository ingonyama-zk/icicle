#include <benchmark/benchmark.h>
#include "utils/test_functions.cuh"
#include "curves/curve_config.cuh"

using namespace curve_config;
using namespace benchmark;

static void BM_MixedECAdd(State& state)
{
  constexpr int N = 128;
  int n = state.range(0) / N;
  projective_t* points1;
  affine_t* points2;
  assert(!hipMalloc(&points1, n * sizeof(projective_t)));
  assert(!hipMalloc(&points2, n * sizeof(affine_t)));

  projective_t* h_points1 = (projective_t*)malloc(n * sizeof(projective_t));
  affine_t* h_points2 = (affine_t*)malloc(n * sizeof(affine_t));
  projective_t::rand_host_many(h_points1, n);
  projective_t::rand_host_many_affine(h_points2, n);
  hipMemcpy(points1, h_points1, sizeof(projective_t) * n, hipMemcpyHostToDevice);
  hipMemcpy(points2, h_points2, sizeof(affine_t) * n, hipMemcpyHostToDevice);

  for (auto _ : state) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    assert((vec_add<projective_t, affine_t, N>(points1, points2, points1, n)) == hipSuccess);
    assert(hipStreamSynchronize(0) == hipSuccess);
    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    state.SetIterationTime((double)(milliseconds / 1000));
  }
  state.counters["Throughput"] = Counter(state.range(0), Counter::kIsRate | Counter::kIsIterationInvariant);
  hipFree(points1);
  hipFree(points2);
}

static void BM_FullECAdd(benchmark::State& state)
{
  constexpr int N = 128;
  int n = state.range(0) / N;
  projective_t* points1;
  projective_t* points2;
  assert(!hipMalloc(&points1, n * sizeof(projective_t)));
  assert(!hipMalloc(&points2, n * sizeof(projective_t)));

  projective_t* h_points1 = (projective_t*)malloc(n * sizeof(projective_t));
  projective_t* h_points2 = (projective_t*)malloc(n * sizeof(projective_t));
  projective_t::rand_host_many(h_points1, n);
  projective_t::rand_host_many(h_points2, n);
  hipMemcpy(points1, h_points1, sizeof(projective_t) * n, hipMemcpyHostToDevice);
  hipMemcpy(points2, h_points2, sizeof(projective_t) * n, hipMemcpyHostToDevice);

  for (auto _ : state) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    assert((vec_add<projective_t, projective_t, N>(points1, points2, points1, n)) == hipSuccess);
    assert(hipStreamSynchronize(0) == hipSuccess);
    hipEventRecord(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    state.SetIterationTime((double)(milliseconds / 1000));
  }
  state.counters["Throughput"] = Counter(state.range(0), Counter::kIsRate | Counter::kIsIterationInvariant);
  hipFree(points1);
  hipFree(points2);
}

BENCHMARK(BM_FullECAdd)->Range(1 << 27, 1 << 27)->Unit(benchmark::kMillisecond);
BENCHMARK(BM_MixedECAdd)->Range(1 << 27, 1 << 27)->Unit(benchmark::kMillisecond);

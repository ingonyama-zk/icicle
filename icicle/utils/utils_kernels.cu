#include "hip/hip_runtime.h"
#include "utils_kernels.cuh"

namespace utils_internal {
  // TODO: weird linking issue - only works in headers
  // template <typename E, typename S>
  // __global__ void NormalizeKernel(E* arr, S scalar, unsigned n)
  // {
  //   int tid = blockIdx.x * blockDim.x + threadIdx.x;
  //   if (tid < n) { arr[tid] = scalar * arr[tid]; }
  // }

  template <typename E, typename S>
  __global__ void NormalizeKernel(E* arr, S scalar, int n)
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) { arr[tid] = scalar * arr[tid]; }
  }

  template <typename E, typename S>
  __global__ void BatchMulKernel(E* element_vec, S* scalar_vec, int n_scalars, int batch_size)
  {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_scalars * batch_size) {
      int scalar_id = tid % n_scalars;
      element_vec[tid] = scalar_vec[scalar_id] * element_vec[tid];
    }
  }

} // namespace utils_internal

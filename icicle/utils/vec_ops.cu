#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

#include "vec_ops.cuh"
#include "curves/curve_config.cuh"
#include "device_context.cuh"
#include "mont.cuh"
#include "utils/utils.h"

namespace vec_ops {

  namespace {

#define MAX_THREADS_PER_BLOCK 256

    template <typename E>
    __global__ void MulKernel(const E* scalar_vec, const E* element_vec, int n, E* result)
    {
      int tid = blockDim.x * blockIdx.x + threadIdx.x;
      if (tid < n) { result[tid] = scalar_vec[tid] * element_vec[tid]; }
    }

    template <typename E>
    __global__ void MulScalarKernel(const E* element_vec, const E* scalar, int n, E* result)
    {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n) { result[tid] = element_vec[tid] * (*scalar); }
    }

    template <typename E>
    __global__ void DivElementWiseKernel(const E* element_vec1, const E* element_vec2, int n, E* result)
    {
      // TODO:implement better based on https://eprint.iacr.org/2008/199
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n) { result[tid] = element_vec1[tid] * E::inverse(element_vec2[tid]); }
    }

    template <typename E>
    __global__ void AddKernel(const E* element_vec1, const E* element_vec2, int n, E* result)
    {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n) { result[tid] = element_vec1[tid] + element_vec2[tid]; }
    }

    template <typename E>
    __global__ void SubKernel(const E* element_vec1, const E* element_vec2, int n, E* result)
    {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n) { result[tid] = element_vec1[tid] - element_vec2[tid]; }
    }
  } // namespace

  template <typename E, void (*Kernel)(const E*, const E*, int, E*)>
  hipError_t VecOp(const E* vec_a, const E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    CHK_INIT_IF_RETURN();

    // Set the grid and block dimensions
    int num_threads = MAX_THREADS_PER_BLOCK;
    int num_blocks = (n + num_threads - 1) / num_threads;

    E *d_result, *d_alloc_vec_a, *d_alloc_vec_b;
    const E *d_vec_a, *d_vec_b;
    if (!config.is_a_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_alloc_vec_a, n * sizeof(E), config.ctx.stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_alloc_vec_a, vec_a, n * sizeof(E), hipMemcpyHostToDevice, config.ctx.stream));
      d_vec_a = d_alloc_vec_a;
    } else {
      d_vec_a = vec_a;
    }

    if (!config.is_b_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_alloc_vec_b, n * sizeof(E), config.ctx.stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_alloc_vec_b, vec_b, n * sizeof(E), hipMemcpyHostToDevice, config.ctx.stream));
      d_vec_b = d_alloc_vec_b;
    } else {
      d_vec_b = vec_b;
    }

    if (!config.is_result_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_result, n * sizeof(E), config.ctx.stream));
    } else {
      d_result = result;
    }

    // Call the kernel to perform element-wise operation
    Kernel<<<num_blocks, num_threads, 0, config.ctx.stream>>>(d_vec_a, d_vec_b, n, d_result);
    if (config.is_result_montgomery_form) CHK_IF_RETURN(mont::FromMontgomery(d_result, n, config.ctx.stream, d_result));

    if (!config.is_a_on_device) { CHK_IF_RETURN(hipFreeAsync(d_alloc_vec_a, config.ctx.stream)); }
    if (!config.is_b_on_device) { CHK_IF_RETURN(hipFreeAsync(d_alloc_vec_b, config.ctx.stream)); }

    if (!config.is_result_on_device) {
      CHK_IF_RETURN(hipMemcpyAsync(result, d_result, n * sizeof(E), hipMemcpyDeviceToHost, config.ctx.stream));
      CHK_IF_RETURN(hipFreeAsync(d_result, config.ctx.stream));
    }

    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(config.ctx.stream));

    return CHK_LAST();
  }

  template <typename E>
  hipError_t Mul(const E* vec_a, const E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    return VecOp<E, MulKernel>(vec_a, vec_b, n, config, result);
  }

  template <typename E>
  hipError_t Add(const E* vec_a, const E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    return VecOp<E, AddKernel>(vec_a, vec_b, n, config, result);
  }

  template <typename E>
  hipError_t Sub(const E* vec_a, const E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    return VecOp<E, SubKernel>(vec_a, vec_b, n, config, result);
  }

  /**
   * Extern version of [Mul](@ref Mul) function with the template parameters
   * `S` and `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, MulCuda)(
    const curve_config::scalar_t* vec_a,
    const curve_config::scalar_t* vec_b,
    int n,
    VecOpsConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* result)
  {
    return Mul<curve_config::scalar_t>(vec_a, vec_b, n, config, result);
  }

  /**
   * Extern version of [Add](@ref Add) function with the template parameter
   * `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, AddCuda)(
    const curve_config::scalar_t* vec_a,
    const curve_config::scalar_t* vec_b,
    int n,
    VecOpsConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* result)
  {
    return Add<curve_config::scalar_t>(vec_a, vec_b, n, config, result);
  }

  /**
   * Extern version of [Sub](@ref Sub) function with the template parameter
   * `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, SubCuda)(
    const curve_config::scalar_t* vec_a,
    const curve_config::scalar_t* vec_b,
    int n,
    VecOpsConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* result)
  {
    return Sub<curve_config::scalar_t>(vec_a, vec_b, n, config, result);
  }

} // namespace vec_ops
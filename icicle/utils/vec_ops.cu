#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

#include "vec_ops.cuh"
#include "curves/curve_config.cuh"
#include "device_context.cuh"
#include "mont.cuh"
#include "utils/utils.h"

namespace vec_ops {

  namespace {

#define MAX_THREADS_PER_BLOCK 256

    template <typename E>
    __global__ void MulKernel(E* scalar_vec, E* element_vec, int n, E* result)
    {
      int tid = blockDim.x * blockIdx.x + threadIdx.x;
      if (tid < n) { result[tid] = scalar_vec[tid] * element_vec[tid]; }
    }

    template <typename E>
    __global__ void AddKernel(E* element_vec1, E* element_vec2, int n, E* result)
    {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n) { result[tid] = element_vec1[tid] + element_vec2[tid]; }
    }

    template <typename E>
    __global__ void SubKernel(E* element_vec1, E* element_vec2, int n, E* result)
    {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n) { result[tid] = element_vec1[tid] - element_vec2[tid]; }
    }

    template <typename E>
    __global__ void transpose_kernel(const E* in, E* out, uint32_t row_size, uint32_t column_size)
    {
      int tid = blockDim.x * blockIdx.x + threadIdx.x;
      if (tid >= row_size * column_size) return;
      out[(tid % row_size) * column_size + (tid / row_size)] = in[tid];
    }
  } // namespace

  template <typename E, void (*Kernel)(E*, E*, int, E*)>
  hipError_t VecOp(E* vec_a, E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    CHK_INIT_IF_RETURN();

    // Set the grid and block dimensions
    int num_threads = MAX_THREADS_PER_BLOCK;
    int num_blocks = (n + num_threads - 1) / num_threads;

    E *d_vec_a, *d_vec_b, *d_result;
    if (!config.is_a_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_vec_a, n * sizeof(E), config.ctx.stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_vec_a, vec_a, n * sizeof(E), hipMemcpyHostToDevice, config.ctx.stream));
    } else {
      d_vec_a = vec_a;
    }

    if (!config.is_b_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_vec_b, n * sizeof(E), config.ctx.stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_vec_b, vec_b, n * sizeof(E), hipMemcpyHostToDevice, config.ctx.stream));
    } else {
      d_vec_b = vec_b;
    }

    if (!config.is_result_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_result, n * sizeof(E), config.ctx.stream));
    } else {
      d_result = result;
    }

    // Call the kernel to perform element-wise operation
    Kernel<<<num_blocks, num_threads, 0, config.ctx.stream>>>(d_vec_a, d_vec_b, n, d_result);
    if (config.is_result_montgomery_form) CHK_IF_RETURN(mont::FromMontgomery(d_result, n, config.ctx.stream, d_result));

    if (!config.is_a_on_device) { CHK_IF_RETURN(hipFreeAsync(d_vec_a, config.ctx.stream)); }

    if (!config.is_b_on_device) { CHK_IF_RETURN(hipFreeAsync(d_vec_b, config.ctx.stream)); }

    if (!config.is_result_on_device) {
      CHK_IF_RETURN(hipMemcpyAsync(result, d_result, n * sizeof(E), hipMemcpyDeviceToHost, config.ctx.stream));
      CHK_IF_RETURN(hipFreeAsync(d_result, config.ctx.stream));
    }

    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(config.ctx.stream));

    return CHK_LAST();
  }

  template <typename E>
  hipError_t Mul(E* vec_a, E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    return VecOp<E, MulKernel>(vec_a, vec_b, n, config, result);
  }

  template <typename E>
  hipError_t Add(E* vec_a, E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    return VecOp<E, AddKernel>(vec_a, vec_b, n, config, result);
  }

  template <typename E>
  hipError_t Sub(E* vec_a, E* vec_b, int n, VecOpsConfig<E>& config, E* result)
  {
    return VecOp<E, SubKernel>(vec_a, vec_b, n, config, result);
  }

  template <typename E>
  hipError_t transpose_matrix(
    const E* mat_in,
    E* mat_out,
    uint32_t row_size,
    uint32_t column_size,
    device_context::DeviceContext& ctx,
    bool on_device,
    bool is_async)
  {
    int number_of_threads = MAX_THREADS_PER_BLOCK;
    int number_of_blocks = (row_size * column_size + number_of_threads - 1) / number_of_threads;
    hipStream_t stream = ctx.stream;

    const E* d_mat_in;
    E* d_allocated_input = nullptr;
    E* d_mat_out;
    if (!on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_allocated_input, row_size * column_size * sizeof(E), ctx.stream));
      CHK_IF_RETURN(hipMemcpyAsync(
        d_allocated_input, mat_in, row_size * column_size * sizeof(E), hipMemcpyHostToDevice, ctx.stream));

      CHK_IF_RETURN(hipMallocAsync(&d_mat_out, row_size * column_size * sizeof(E), ctx.stream));
      d_mat_in = d_allocated_input;
    } else {
      d_mat_in = mat_in;
      d_mat_out = mat_out;
    }

    transpose_kernel<<<number_of_blocks, number_of_threads, 0, stream>>>(d_mat_in, d_mat_out, row_size, column_size);

    if (!on_device) {
      CHK_IF_RETURN(
        hipMemcpyAsync(mat_out, d_mat_out, row_size * column_size * sizeof(E), hipMemcpyDeviceToHost, ctx.stream));
      CHK_IF_RETURN(hipFreeAsync(d_mat_out, ctx.stream));
      CHK_IF_RETURN(hipFreeAsync(d_allocated_input, ctx.stream));
    }
    if (!is_async) return CHK_STICKY(hipStreamSynchronize(ctx.stream));

    return CHK_LAST();
  }

  /**
   * Extern version of [Mul](@ref Mul) function with the template parameters
   * `S` and `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, MulCuda)(
    curve_config::scalar_t* vec_a,
    curve_config::scalar_t* vec_b,
    int n,
    VecOpsConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* result)
  {
    return Mul<curve_config::scalar_t>(vec_a, vec_b, n, config, result);
  }

  /**
   * Extern version of [Add](@ref Add) function with the template parameter
   * `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, AddCuda)(
    curve_config::scalar_t* vec_a,
    curve_config::scalar_t* vec_b,
    int n,
    VecOpsConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* result)
  {
    return Add<curve_config::scalar_t>(vec_a, vec_b, n, config, result);
  }

  /**
   * Extern version of [Sub](@ref Sub) function with the template parameter
   * `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, SubCuda)(
    curve_config::scalar_t* vec_a,
    curve_config::scalar_t* vec_b,
    int n,
    VecOpsConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* result)
  {
    return Sub<curve_config::scalar_t>(vec_a, vec_b, n, config, result);
  }

  /**
   * Extern version of transpose_batch function with the template parameter
   * `E` being the [scalar field](@ref scalar_t) of the curve given by `-DCURVE` env variable during build.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, TransposeMatrix)(
    const curve_config::scalar_t* input,
    uint32_t row_size,
    uint32_t column_size,
    curve_config::scalar_t* output,
    device_context::DeviceContext& ctx,
    bool on_device,
    bool is_async)
  {
    return transpose_matrix<curve_config::scalar_t>(input, output, row_size, column_size, ctx, on_device, is_async);
  }

} // namespace vec_ops
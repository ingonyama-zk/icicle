#include "../curves/curve_config.cuh"
#include "device_context.cuh"
#include "mont.cuh"

namespace mont {
  extern "C" hipError_t
  ScalarConvertMontgomery(curve_config::scalar_t* d_inout, size_t n, bool is_into, device_context::DeviceContext& ctx)
  {
    if (is_into) {
      return ToMontgomery(d_inout, n, ctx.stream, d_inout);
    } else {
      return FromMontgomery(d_inout, n, ctx.stream, d_inout);
    }
  }

  extern "C" hipError_t
  AffineConvertMontgomery(curve_config::affine_t* d_inout, size_t n, bool is_into, device_context::DeviceContext& ctx)
  {
    if (is_into) {
      return ToMontgomery(d_inout, n, ctx.stream, d_inout);
    } else {
      return FromMontgomery(d_inout, n, ctx.stream, d_inout);
    }
  }

  extern "C" hipError_t ProjectiveConvertMontgomery(
    curve_config::projective_t* d_inout, size_t n, bool is_into, device_context::DeviceContext& ctx)
  {
    if (is_into) {
      return ToMontgomery(d_inout, n, ctx.stream, d_inout);
    } else {
      return FromMontgomery(d_inout, n, ctx.stream, d_inout);
    }
  }
} // namespace mont
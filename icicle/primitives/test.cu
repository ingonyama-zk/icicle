#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

// TODO: change the curve depending on env variable
#include "../curves/bls12_381.cuh"
#include "projective.cuh"
#include "field.cuh"
#include "test_kernels.cuh"


typedef Field<fp_config> scalar_field;
typedef Field<fq_config> base_field;
typedef Projective<base_field, scalar_field, group_generator, weierstrass_b> proj;

template <class T>
int device_populate_random(T* d_elements, unsigned n) {
    T h_elements[n];
    for (unsigned i = 0; i < n; i++)
        h_elements[i] = T::rand_host();
    return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

template <class T>
int device_set(T* d_elements, T el, unsigned n) {
    T h_elements[n];
    for (unsigned i = 0; i < n; i++)
        h_elements[i] = el;
    return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

class PrimitivesTest : public ::testing::Test {
protected:
  static const unsigned n = 1 << 5;

  proj *points1{};
  proj *points2{};
  proj *zeroes{};
  proj *res1{};
  proj *res2{};

  PrimitivesTest() {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&points1, n * sizeof(proj)));
    assert(!hipMallocManaged(&points2, n * sizeof(proj)));
    assert(!hipMallocManaged(&zeroes, n * sizeof(proj)));
    assert(!hipMallocManaged(&res1, n * sizeof(proj)));
    assert(!hipMallocManaged(&res2, n * sizeof(proj)));
  }

  ~PrimitivesTest() override {
    hipFree(points1);
    hipFree(points2);
    hipFree(zeroes);
    hipFree(res1);
    hipFree(res2);
    hipDeviceReset();
  }

  void SetUp() override {
    ASSERT_EQ(device_populate_random<proj>(points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<proj>(points2, n), hipSuccess);
    ASSERT_EQ(device_set<proj>(zeroes, proj::zero(), n), hipSuccess);
    ASSERT_EQ(hipMemset(res1, 0, n * sizeof(proj)), hipSuccess);
    ASSERT_EQ(hipMemset(res2, 0, n * sizeof(proj)), hipSuccess);
  }
};

TEST_F(PrimitivesTest, RandomPointsAreOnCurve) {
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(proj::is_on_curve, points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionAndSubtractionCancel) {
  ASSERT_EQ(vec_add<proj>(points1, points2, res1, n), hipSuccess);
  ASSERT_EQ(vec_sub<proj>(res1, points2, res2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res2[i]);
}


int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

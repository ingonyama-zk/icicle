#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include "test_kernels.cuh"


template <class T>
int device_populate_random(T* d_elements, unsigned n) {
    T h_elements[n];
    for (unsigned i = 0; i < n; i++)
        h_elements[i] = T::rand_host();
    return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

template <class T>
int device_set(T* d_elements, T el, unsigned n) {
    T h_elements[n];
    for (unsigned i = 0; i < n; i++)
        h_elements[i] = el;
    return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

class PrimitivesTest : public ::testing::Test {
protected:
  static const unsigned n = 1 << 5;

  proj *points1{};
  proj *points2{};
  scalar_field *scalars1{};
  scalar_field *scalars2{};
  proj *zero_points{};
  scalar_field *zero_scalars{};
  scalar_field *one_scalars{};
  affine *aff_points{};
  proj *res_points1{};
  proj *res_points2{};
  scalar_field *res_scalars1{};
  scalar_field *res_scalars2{};

  PrimitivesTest() {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&points1, n * sizeof(proj)));
    assert(!hipMallocManaged(&points2, n * sizeof(proj)));
    assert(!hipMallocManaged(&scalars1, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&zero_points, n * sizeof(proj)));
    assert(!hipMallocManaged(&zero_scalars, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&aff_points, n * sizeof(affine)));
    assert(!hipMallocManaged(&res_points1, n * sizeof(proj)));
    assert(!hipMallocManaged(&res_points2, n * sizeof(proj)));
    assert(!hipMallocManaged(&res_scalars1, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&res_scalars2, n * sizeof(scalar_field)));
  }

  ~PrimitivesTest() override {
    hipFree(points1);
    hipFree(points2);
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_points);
    hipFree(zero_scalars);
    hipFree(one_scalars);
    hipFree(aff_points);
    hipFree(res_points1);
    hipFree(res_points2);
    hipFree(res_scalars1);
    hipFree(res_scalars2);
    hipDeviceReset();
  }

  void SetUp() override {
    ASSERT_EQ(device_populate_random<proj>(points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<proj>(points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_field>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_field>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set<proj>(zero_points, proj::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_field>(zero_scalars, scalar_field::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_field>(one_scalars, scalar_field::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(aff_points, 0, n * sizeof(affine)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points1, 0, n * sizeof(proj)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points2, 0, n * sizeof(proj)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars1, 0, n * sizeof(scalar_field)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars2, 0, n * sizeof(scalar_field)), hipSuccess);
  }
};

TEST_F(PrimitivesTest, FieldAdditionSubtractionCancel) {
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_scalars1, scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldZeroAddition) {
  ASSERT_EQ(vec_add(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldAdditionHostDeviceEq) {
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] + scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByOne) {
  ASSERT_EQ(vec_mul(scalars1, one_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByMinusOne) {
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, res_scalars1, res_scalars2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, res_scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], zero_scalars[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByZero) {
  ASSERT_EQ(vec_mul(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(zero_scalars[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationInverseCancel) {
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i] * res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationHostDeviceEq) {
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByTwoEqSum) {
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars2[i], scalars1[i] + scalars1[i]);
}

TEST_F(PrimitivesTest, FieldSqrHostDeviceEq) {
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationSqrEq) {
  ASSERT_EQ(vec_mul(scalars1, scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, ECRandomPointsAreOnCurve) {
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(proj::is_on_curve, points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionSubtractionCancel) {
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECPointZeroAddition) {
  ASSERT_EQ(vec_add(points1, zero_points, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionHostDeviceEq) {
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i] + points2[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationHostDeviceEq) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByOne) {
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByMinusOne) {
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByTwo) {
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationInverseCancel) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, res_points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverMultiplication) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, res_points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverAddition) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i] * points1[i], res_points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, ECProjectiveToAffine) {
  ASSERT_EQ(point_vec_to_affine(points1, aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], proj::from_affine(aff_points[i]));
}

TEST_F(PrimitivesTest, ECMixedPointAddition) {
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECMixedAdditionOfNegatedPointEqSubtraction) {
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], points1[i] + res_points2[i]);
}


int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

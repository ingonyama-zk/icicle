#include "test_kernels.cuh"
#include <boost/multiprecision/cpp_int.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>
namespace mp = boost::multiprecision;

template <class T>
int device_populate_random(T* d_elements, unsigned n)
{
  T h_elements[n];
  for (unsigned i = 0; i < n; i++)
    h_elements[i] = T::rand_host();
  return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

template <class T>
int device_set(T* d_elements, T el, unsigned n)
{
  T h_elements[n];
  for (unsigned i = 0; i < n; i++)
    h_elements[i] = el;
  return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

mp::int1024_t convert_to_boost_mp(uint32_t* a, uint32_t length)
{
  mp::int1024_t res = 0;
  for (uint32_t i = 0; i < length; i++) {
    res += (mp::int1024_t)(a[i]) << 32 * i;
  }
  return res;
}

class PrimitivesTest : public ::testing::Test
{
protected:
  static const unsigned n = 1 << 4;

  projective_t* points1{};
  projective_t* points2{};
  g2_projective_t* g2_points1{};
  g2_projective_t* g2_points2{};
  scalar_field_t* scalars1{};
  scalar_field_t* scalars2{};
  projective_t* zero_points{};
  g2_projective_t* g2_zero_points{};
  scalar_field_t* zero_scalars{};
  scalar_field_t* one_scalars{};
  affine_t* aff_points{};
  g2_affine_t* g2_aff_points{};
  projective_t* res_points1{};
  projective_t* res_points2{};
  g2_projective_t* g2_res_points1{};
  g2_projective_t* g2_res_points2{};
  scalar_field_t* res_scalars1{};
  scalar_field_t* res_scalars2{};
  scalar_field_t::Wide* res_scalars_wide{};
  scalar_field_t::Wide* res_scalars_wide_full{};

  PrimitivesTest()
  {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&points1, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&points2, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&g2_points1, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_points2, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&scalars1, n * sizeof(scalar_field_t)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(scalar_field_t)));
    assert(!hipMallocManaged(&zero_points, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&g2_zero_points, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&zero_scalars, n * sizeof(scalar_field_t)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(scalar_field_t)));
    assert(!hipMallocManaged(&aff_points, n * sizeof(affine_t)));
    assert(!hipMallocManaged(&g2_aff_points, n * sizeof(g2_affine_t)));
    assert(!hipMallocManaged(&res_points1, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&res_points2, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&g2_res_points1, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_res_points2, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&res_scalars1, n * sizeof(scalar_field_t)));
    assert(!hipMallocManaged(&res_scalars2, n * sizeof(scalar_field_t)));
    assert(!hipMallocManaged(&res_scalars_wide, n * sizeof(scalar_field_t::Wide)));
    assert(!hipMallocManaged(&res_scalars_wide_full, n * sizeof(scalar_field_t::Wide)));
  }

  ~PrimitivesTest() override
  {
    hipFree(points1);
    hipFree(points2);
    hipFree(g2_points1);
    hipFree(g2_points2);
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_points);
    hipFree(g2_zero_points);
    hipFree(zero_scalars);
    hipFree(one_scalars);
    hipFree(aff_points);
    hipFree(g2_aff_points);
    hipFree(res_points1);
    hipFree(res_points2);
    hipFree(g2_res_points1);
    hipFree(g2_res_points2);
    hipFree(res_scalars1);
    hipFree(res_scalars2);

    hipFree(res_scalars_wide);
    hipFree(res_scalars_wide_full);

    hipDeviceReset();
  }

  void SetUp() override
  {
    ASSERT_EQ(device_populate_random<projective_t>(points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<projective_t>(points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<g2_projective_t>(g2_points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<g2_projective_t>(g2_points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_field_t>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_field_t>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set<projective_t>(zero_points, projective_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<g2_projective_t>(g2_zero_points, g2_projective_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_field_t>(zero_scalars, scalar_field_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_field_t>(one_scalars, scalar_field_t::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(aff_points, 0, n * sizeof(affine_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_aff_points, 0, n * sizeof(g2_affine_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points1, 0, n * sizeof(projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points2, 0, n * sizeof(projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_res_points1, 0, n * sizeof(g2_projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_res_points2, 0, n * sizeof(g2_projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars1, 0, n * sizeof(scalar_field_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars2, 0, n * sizeof(scalar_field_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars_wide, 0, n * sizeof(scalar_field_t::Wide)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars_wide_full, 0, n * sizeof(scalar_field_t::Wide)), hipSuccess);
  }
};

TEST_F(PrimitivesTest, FieldAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_scalars1, scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldZeroAddition)
{
  ASSERT_EQ(vec_add(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] + scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(scalars1, one_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, res_scalars1, res_scalars2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, res_scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], zero_scalars[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByZero)
{
  ASSERT_EQ(vec_mul(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(zero_scalars[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i] * res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByTwoEqSum)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars2[i], scalars1[i] + scalars1[i]);
}

TEST_F(PrimitivesTest, FieldSqrHostDeviceEq)
{
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationSqrEq)
{
  ASSERT_EQ(vec_mul(scalars1, scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, ECRandomPointsAreOnCurve)
{
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(projective_t::is_on_curve, points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECPointZeroAddition)
{
  ASSERT_EQ(vec_add(points1, zero_points, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i] + points2[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByTwo)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, res_points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverMultiplication)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, res_points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverAddition)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i] * points1[i], res_points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, ECProjectiveToAffine)
{
  ASSERT_EQ(point_vec_to_affine(points1, aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], projective_t::from_affine(aff_points[i]));
}

TEST_F(PrimitivesTest, ECMixedPointAddition)
{
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECMixedAdditionOfNegatedPointEqSubtraction)
{
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, MP_LSB_MULT)
{
  // LSB multiply, check correctness of first TLC + 1 digits result.
  ASSERT_EQ(mp_lsb_mult(scalars1, scalars2, res_scalars_wide), hipSuccess);
  std::cout << "first GPU lsb mult output  = 0x";
  for (int i = 0; i < 2 * scalar_field_t::TLC; i++) {
    std::cout << std::hex << res_scalars_wide[0].limbs_storage.limbs[i];
  }
  std::cout << std::endl;

  ASSERT_EQ(mp_mult(scalars1, scalars2, res_scalars_wide_full), hipSuccess);
  std::cout << "first GPU full mult output = 0x";
  for (int i = 0; i < 2 * scalar_field_t::TLC; i++) {
    std::cout << std::hex << res_scalars_wide_full[0].limbs_storage.limbs[i];
  }
  std::cout << std::endl;
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < scalar_field_t::TLC + 1; i++) {
      ASSERT_EQ(res_scalars_wide_full[j].limbs_storage.limbs[i], res_scalars_wide[j].limbs_storage.limbs[i]);
    }
  }
}

TEST_F(PrimitivesTest, MP_MSB_MULT)
{
  // MSB multiply, take n msb bits of multiplication, assert that the error is up to 1.
  ASSERT_EQ(mp_msb_mult(scalars1, scalars2, res_scalars_wide), hipSuccess);
  std::cout << "first GPU msb mult output  = 0x";
  for (int i = 2 * scalar_field_t::TLC - 1; i >= 0; i--) {
    std::cout << std::hex << res_scalars_wide[0].limbs_storage.limbs[i] << " ";
  }
  std::cout << std::endl;

  ASSERT_EQ(mp_mult(scalars1, scalars2, res_scalars_wide_full), hipSuccess);
  std::cout << "first GPU full mult output = 0x";
  for (int i = 2 * scalar_field_t::TLC - 1; i >= 0; i--) {
    std::cout << std::hex << res_scalars_wide_full[0].limbs_storage.limbs[i] << " ";
  }

  std::cout << std::endl;

  for (int i = 0; i < 2 * scalar_field_t::TLC - 1; i++) {
    if (res_scalars_wide_full[0].limbs_storage.limbs[i] == res_scalars_wide[0].limbs_storage.limbs[i])
      std::cout << "matched word idx = " << i << std::endl;
  }
}

TEST_F(PrimitivesTest, INGO_MP_MULT)
{
  // MSB multiply, take n msb bits of multiplication, assert that the error is up to 1.
  ASSERT_EQ(ingo_mp_mult(scalars1, scalars2, res_scalars_wide), hipSuccess);
  std::cout << "INGO   = 0x";
  for (int i = 0; i < 2 * scalar_field_t::TLC; i++) {
    std::cout << std::hex << res_scalars_wide[0].limbs_storage.limbs[i] << " ";
  }
  std::cout << std::endl;

  ASSERT_EQ(mp_mult(scalars1, scalars2, res_scalars_wide_full), hipSuccess);
  std::cout << "ZKSYNC = 0x";
  for (int i = 0; i < 2 * scalar_field_t::TLC; i++) {
    std::cout << std::hex << res_scalars_wide_full[0].limbs_storage.limbs[i] << " ";
  }

  std::cout << std::endl;

  for (int i = 0; i < 2 * scalar_field_t::TLC - 1; i++) {
    if (res_scalars_wide_full[0].limbs_storage.limbs[i] == res_scalars_wide[0].limbs_storage.limbs[i])
      std::cout << "matched word idx = " << i << std::endl;
  }
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < 2 * scalar_field_t::TLC - 1; i++) {
      ASSERT_EQ(res_scalars_wide_full[j].limbs_storage.limbs[i], res_scalars_wide[j].limbs_storage.limbs[i]);
    }
  }
}

TEST_F(PrimitivesTest, INGO_MP_MSB_MULT)
{
  // MSB multiply, take n msb bits of multiplication, assert that the error is up to 1.
  ASSERT_EQ(ingo_mp_msb_mult(scalars1, scalars2, res_scalars_wide, n), hipSuccess);
  std::cout << "INGO MSB   = 0x";
  for (int i = 2 * scalar_field_t::TLC - 1; i >= 0; i--) {
    std::cout << std::hex << res_scalars_wide[0].limbs_storage.limbs[i] << " ";
  }
  std::cout << std::endl;

  ASSERT_EQ(mp_mult(scalars1, scalars2, res_scalars_wide_full), hipSuccess);
  std::cout << "ZKSYNC = 0x";
  for (int i = 2 * scalar_field_t::TLC - 1; i >= 0; i--) {
    std::cout << std::hex << res_scalars_wide_full[0].limbs_storage.limbs[i] << " ";
  }

  std::cout << std::endl;

  // for (int i=scalar_field::TLC; i < 2*scalar_field::TLC - 1; i++)
  // {
  //   ASSERT_EQ(in_bound, true);
  // }
  // for (int j=0; j<n; j++)
  // {
  //   for (int i=0; i < 2*scalar_field::TLC - 1; i++)
  //   {
  //     ASSERT_EQ(res_scalars_wide_full[j].limbs_storage.limbs[i], res_scalars_wide[j].limbs_storage.limbs[i]);
  //   }
  // }
  // mp testing
  mp::int1024_t scalar_1_mp = 0;
  mp::int1024_t scalar_2_mp = 0;
  mp::int1024_t res_mp = 0;
  mp::int1024_t res_gpu = 0;
  uint32_t num_limbs = scalar_field_t::TLC;

  for (int j = 0; j < n; j++) {
    uint32_t* scalar1_limbs = scalars1[j].limbs_storage.limbs;
    uint32_t* scalar2_limbs = scalars2[j].limbs_storage.limbs;
    scalar_1_mp = convert_to_boost_mp(scalar1_limbs, num_limbs);
    scalar_2_mp = convert_to_boost_mp(scalar2_limbs, num_limbs);
    res_mp = scalar_1_mp * scalar_2_mp;
    res_mp = res_mp >> (num_limbs * 32);
    res_gpu = convert_to_boost_mp(&(res_scalars_wide[j]).limbs_storage.limbs[num_limbs], num_limbs);
    std::cout << "res  mp = " << res_mp << std::endl;
    std::cout << "res gpu = " << res_gpu << std::endl;
    std::cout << "error = " << res_mp - res_gpu << std::endl;
    bool upper_bound = res_gpu <= res_mp;
    bool lower_bound = res_gpu > (res_mp - num_limbs);
    bool in_bound = upper_bound && lower_bound;

    ASSERT_EQ(in_bound, true);
  }
}

TEST_F(PrimitivesTest, INGO_MP_MOD_MULT)
{
  std::cout << " taking num limbs " << std::endl;
  uint32_t num_limbs = scalar_field_t::TLC;
  std::cout << " calling gpu... = " << std::endl;
  ASSERT_EQ(ingo_mp_mod_mult(scalars1, scalars2, res_scalars1, n), hipSuccess);
  std::cout << " gpu call done " << std::endl;
  // mp testing
  mp::int1024_t scalar_1_mp = 0;
  mp::int1024_t scalar_2_mp = 0;
  mp::int1024_t res_mp = 0;
  mp::int1024_t res_gpu = 0;
  mp::int1024_t p = convert_to_boost_mp(scalar_field_t::get_modulus().limbs, num_limbs);
  std::cout << " p = " << p << std::endl;

  for (int j = 0; j < n; j++) {
    uint32_t* scalar1_limbs = scalars1[j].limbs_storage.limbs;
    uint32_t* scalar2_limbs = scalars2[j].limbs_storage.limbs;
    scalar_1_mp = convert_to_boost_mp(scalar1_limbs, num_limbs);
    scalar_2_mp = convert_to_boost_mp(scalar2_limbs, num_limbs);
    // std::cout << " s1 = " << scalar_1_mp << std::endl;
    // std::cout << " s2 = " << scalar_2_mp << std::endl;
    res_mp = (scalar_1_mp * scalar_2_mp) % p;
    res_gpu = convert_to_boost_mp((res_scalars1[j]).limbs_storage.limbs, num_limbs);
    std::cout << "res  mp = " << res_mp << std::endl;
    std::cout << "res gpu = " << res_gpu << std::endl;
    std::cout << "error = " << res_mp - res_gpu << std::endl;
    ASSERT_EQ(res_gpu, res_mp);
  }
}

TEST_F(PrimitivesTest, G2ECRandomPointsAreOnCurve)
{
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(g2_projective_t::is_on_curve, g2_points1[i]);
}

TEST_F(PrimitivesTest, G2ECPointAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(g2_res_points1, g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECPointZeroAddition)
{
  ASSERT_EQ(vec_add(g2_points1, g2_zero_points, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECPointAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i] + g2_points2[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(g2_points1, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationByTwo)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_res_points1, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationIsDistributiveOverMultiplication)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, g2_res_points1, g2_res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationIsDistributiveOverAddition)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, g2_points1, g2_res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i] * g2_points1[i], g2_res_points1[i] + g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECProjectiveToAffine)
{
  ASSERT_EQ(point_vec_to_affine(g2_points1, g2_aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_projective_t::from_affine(g2_aff_points[i]));
}

TEST_F(PrimitivesTest, G2ECMixedPointAddition)
{
  ASSERT_EQ(point_vec_to_affine(g2_points2, g2_aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(g2_points1, g2_aff_points, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECMixedAdditionOfNegatedPointEqSubtraction)
{
  ASSERT_EQ(point_vec_to_affine(g2_points2, g2_aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(g2_points1, g2_aff_points, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_points1[i] + g2_res_points2[i]);
}

int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

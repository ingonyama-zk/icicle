#include <hip/hip_runtime.h>
#include "../curves/bls12_381/curve_config.cuh"
#include "../curves/bls12_377/curve_config.cuh"
#include "../curves/bn254/curve_config.cuh"
#include "projective.cuh"

extern "C" bool eq_bls12_381(BLS12_381::projective_t *point1, BLS12_381::projective_t *point2)
{
    return (*point1 == *point2) && 
    !((point1->x == BLS12_381::point_field_t::zero()) && (point1->y == BLS12_381::point_field_t::zero()) && (point1->z == BLS12_381::point_field_t::zero())) && 
    !((point2->x == BLS12_381::point_field_t::zero()) && (point2->y == BLS12_381::point_field_t::zero()) && (point2->z == BLS12_381::point_field_t::zero()));
}

extern "C" bool eq_bls12_377(BLS12_377::projective_t *point1, BLS12_377::projective_t *point2)
{
    return (*point1 == *point2) && 
    !((point1->x == BLS12_377::point_field_t::zero()) && (point1->y == BLS12_377::point_field_t::zero()) && (point1->z == BLS12_377::point_field_t::zero())) && 
    !((point2->x == BLS12_377::point_field_t::zero()) && (point2->y == BLS12_377::point_field_t::zero()) && (point2->z == BLS12_377::point_field_t::zero()));
}

extern "C" bool eq_bn254(BN254::projective_t *point1, BN254::projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BN254::point_field_t::zero()) && (point1->y == BN254::point_field_t::zero()) && (point1->z == BN254::point_field_t::zero())) && 
  !((point2->x == BN254::point_field_t::zero()) && (point2->y == BN254::point_field_t::zero()) && (point2->z == BN254::point_field_t::zero()));
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2_bls12_381(BLS12_381::g2_projective_t *point1, BLS12_381::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BLS12_381::g2_point_field_t::zero()) && (point1->y == BLS12_381::g2_point_field_t::zero()) && (point1->z == BLS12_381::g2_point_field_t::zero())) && 
  !((point2->x == BLS12_381::g2_point_field_t::zero()) && (point2->y == BLS12_381::g2_point_field_t::zero()) && (point2->z == BLS12_381::g2_point_field_t::zero()));
}

extern "C" bool eq_g2_bls12_377(BLS12_377::g2_projective_t *point1, BLS12_377::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BLS12_377::g2_point_field_t::zero()) && (point1->y == BLS12_377::g2_point_field_t::zero()) && (point1->z == BLS12_377::g2_point_field_t::zero())) && 
  !((point2->x == BLS12_377::g2_point_field_t::zero()) && (point2->y == BLS12_377::g2_point_field_t::zero()) && (point2->z == BLS12_377::g2_point_field_t::zero()));
}

extern "C" bool eq_g2_bn254(BN254::g2_projective_t *point1, BN254::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BN254::g2_point_field_t::zero()) && (point1->y == BN254::g2_point_field_t::zero()) && (point1->z == BN254::g2_point_field_t::zero())) && 
  !((point2->x == BN254::g2_point_field_t::zero()) && (point2->y == BN254::g2_point_field_t::zero()) && (point2->z == BN254::g2_point_field_t::zero()));
}
#endif
#include "curves/curve_config.cuh"
#include "projective.cuh"
#include <hip/hip_runtime.h>
#include "utils/utils.h"

#define projective_t  curve_config::projective_t // TODO: global to avoid lengthy texts
#define affine_t      curve_config::affine_t
#define point_field_t curve_config::point_field_t

extern "C" bool CONCAT_EXPAND(CURVE, Eq)(projective_t* point1, projective_t* point2)
{
  return (*point1 == *point2) &&
         !((point1->x == point_field_t::zero()) && (point1->y == point_field_t::zero()) &&
           (point1->z == point_field_t::zero())) &&
         !((point2->x == point_field_t::zero()) && (point2->y == point_field_t::zero()) &&
           (point2->z == point_field_t::zero()));
}

extern "C" void CONCAT_EXPAND(CURVE, ToAffine)(projective_t* point, affine_t* point_out)
{
  *point_out = projective_t::to_affine(*point);
}

extern "C" void CONCAT_EXPAND(CURVE, GenerateProjectivePoints)(projective_t* points, int size)
{
  projective_t::RandHostMany(points, size);
}

extern "C" void CONCAT_EXPAND(CURVE, GenerateAffinePoints)(affine_t* points, int size)
{
  projective_t::RandHostManyAffine(points, size);
}

#if defined(G2_DEFINED)

#define g2_projective_t  curve_config::g2_projective_t
#define g2_affine_t      curve_config::g2_affine_t
#define g2_point_field_t curve_config::g2_point_field_t

extern "C" bool CONCAT_EXPAND(CURVE, G2Eq)(g2_projective_t* point1, g2_projective_t* point2)
{
  return (*point1 == *point2) &&
         !((point1->x == g2_point_field_t::zero()) && (point1->y == g2_point_field_t::zero()) &&
           (point1->z == g2_point_field_t::zero())) &&
         !((point2->x == g2_point_field_t::zero()) && (point2->y == g2_point_field_t::zero()) &&
           (point2->z == g2_point_field_t::zero()));
}

extern "C" void CONCAT_EXPAND(CURVE, G2ToAffine)(g2_projective_t* point, g2_affine_t* point_out)
{
  *point_out = g2_projective_t::to_affine(*point);
}

extern "C" void CONCAT_EXPAND(CURVE, G2GenerateProjectivePoints)(g2_projective_t* points, int size)
{
  g2_projective_t::RandHostMany(points, size);
}

extern "C" void CONCAT_EXPAND(CURVE, G2GenerateAffinePoints)(g2_affine_t* points, int size)
{
  g2_projective_t::RandHostManyAffine(points, size);
}

#endif

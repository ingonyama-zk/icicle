#ifndef _BLS12_381_POSEIDON
#define _BLS12_381_POSEIDON
#include <hip/hip_runtime.h>
#include <stdexcept>
#include "../../appUtils/poseidon/poseidon.cu"
#include "curve_config.cuh"

template class Poseidon<BLS12_381::scalar_t>;

extern "C" int poseidon_multi_cuda_bls12_381(BLS12_381::scalar_t input[], BLS12_381::scalar_t* out,
                                             size_t number_of_blocks, int arity, size_t device_id = 0, hipStream_t stream = 0)
{
  try
  {
    // TODO: once we get bindings to pass a stream, we should make {stream} a required parameter and use it instead of 
    // creating a new stream
    if (stream == 0) {
        hipStreamCreate(&stream);
    }
    
    hipEvent_t start_event, end_event;
    hipEventCreate(&start_event);
    hipEventCreate(&end_event);
    hipEventRecord(start_event, stream);
    Poseidon<BLS12_381::scalar_t> poseidon(arity, stream);
    poseidon.hash_blocks(input, number_of_blocks, out, Poseidon<BLS12_381::scalar_t>::HashType::MerkleTree, stream);
    hipEventRecord(end_event, stream);
    hipEventSynchronize(end_event);

    #ifdef DEBUG
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start_event, end_event);
    printf("Time elapsed: %f", elapsedTime);
    #endif

    hipEventDestroy(start_event);
    hipEventDestroy(end_event);

    return hipSuccess;
  }
  catch (const std::runtime_error &ex)
  {
    printf("error %s", ex.what());
    return -1;
  }
}
#endif
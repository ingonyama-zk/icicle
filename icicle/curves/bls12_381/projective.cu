#include <hip/hip_runtime.h>
#include "curve_config.cuh"
#include "../../primitives/projective.cuh"

extern "C" bool eq_bls12_381(BLS12_381::projective_t *point1, BLS12_381::projective_t *point2)
{
    return (*point1 == *point2) && 
    !((point1->x == BLS12_381::point_field_t::zero()) && (point1->y == BLS12_381::point_field_t::zero()) && (point1->z == BLS12_381::point_field_t::zero())) && 
    !((point2->x == BLS12_381::point_field_t::zero()) && (point2->y == BLS12_381::point_field_t::zero()) && (point2->z == BLS12_381::point_field_t::zero()));
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2_bls12_381(BLS12_381::g2_projective_t *point1, BLS12_381::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BLS12_381::g2_point_field_t::zero()) && (point1->y == BLS12_381::g2_point_field_t::zero()) && (point1->z == BLS12_381::g2_point_field_t::zero())) && 
  !((point2->x == BLS12_381::g2_point_field_t::zero()) && (point2->y == BLS12_381::g2_point_field_t::zero()) && (point2->z == BLS12_381::g2_point_field_t::zero()));
}
#endif

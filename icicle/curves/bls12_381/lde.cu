#include "hip/hip_runtime.h"
#ifndef _BLS12_381_LDE
#define _BLS12_381_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" BLS12_381::scalar_t* build_domain_cuda_bls12_381(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, BLS12_381::scalar_t::omega_inv(logn), stream);
        } else {
            return fill_twiddle_factors_array(domain_size, BLS12_381::scalar_t::omega(logn), stream);
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_bls12_381(BLS12_381::scalar_t *arr, uint32_t n, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<BLS12_381::scalar_t,BLS12_381::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda_bls12_381(BLS12_381::projective_t *arr, uint32_t n, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<BLS12_381::projective_t,BLS12_381::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_bls12_381(BLS12_381::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<BLS12_381::scalar_t,BLS12_381::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda_bls12_381(BLS12_381::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<BLS12_381::projective_t,BLS12_381::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda_bls12_381(BLS12_381::scalar_t* d_out, BLS12_381::scalar_t *d_evaluations, BLS12_381::scalar_t *d_domain, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_bls12_381(BLS12_381::scalar_t* d_out, BLS12_381::scalar_t* d_evaluations, BLS12_381::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream); //TODO: we should avoid creating stream if default (hipStream_t stream = 0) is passed.
                                   //      but default is not working as expected as valgrind still reports errors
        auto result_code = interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, stream);
        hipStreamDestroy(stream); //TODO: hotfix for not freeing memory 
        return result_code;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda_bls12_381(BLS12_381::projective_t* d_out, BLS12_381::projective_t *d_evaluations, BLS12_381::scalar_t *d_domain, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda_bls12_381(BLS12_381::projective_t* d_out, BLS12_381::projective_t* d_evaluations, BLS12_381::scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        auto result_code = interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, stream);
        hipStreamDestroy(stream);
        return result_code; 
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda_bls12_381(BLS12_381::scalar_t* d_out, BLS12_381::scalar_t *d_coefficients, BLS12_381::scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_bls12_381(BLS12_381::scalar_t* d_out, BLS12_381::scalar_t* d_coefficients, BLS12_381::scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        auto result_code = evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, 0);
        hipStreamDestroy(stream);
        return result_code;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda_bls12_381(BLS12_381::projective_t* d_out, BLS12_381::projective_t *d_coefficients, BLS12_381::scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_bls12_381(BLS12_381::projective_t* d_out, BLS12_381::projective_t* d_coefficients, BLS12_381::scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BLS12_381::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        auto result_code = evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
        hipStreamDestroy(stream);
        return result_code;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_bls12_381(BLS12_381::scalar_t* d_out, BLS12_381::scalar_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, BLS12_381::scalar_t *coset_powers, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_bls12_381(BLS12_381::scalar_t* d_out, BLS12_381::scalar_t* d_coefficients, BLS12_381::scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, BLS12_381::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_bls12_381(BLS12_381::projective_t* d_out, BLS12_381::projective_t *d_coefficients, BLS12_381::scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, BLS12_381::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream); //TODO: don't create if default was passed, destroy what was created, same applies to all calls
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_bls12_381(BLS12_381::projective_t* d_out, BLS12_381::projective_t* d_coefficients, BLS12_381::scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, BLS12_381::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_inplace_batch_cuda_bls12_381(BLS12_381::scalar_t* d_inout, BLS12_381::scalar_t* d_twiddles,
                                           unsigned n, unsigned batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        ntt_inplace_batch_template(d_inout, d_twiddles, n, batch_size, inverse, stream, true);
        return hipSuccess; //TODO: we should implement this https://leimao.github.io/blog/Proper-CUDA-Error-Checking/
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda_bls12_381(BLS12_381::scalar_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_bls12_381(BLS12_381::scalar_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_bls12_381(BLS12_381::projective_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda_bls12_381(BLS12_381::projective_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
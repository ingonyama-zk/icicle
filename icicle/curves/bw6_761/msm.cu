#ifndef _BW6_761_MSM
#define _BW6_761_MSM
#include "../../appUtils/msm/msm.cu"
#include "curve_config.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>

extern "C" int msm_cuda_bw6_761(
  BW6_761::projective_t* out,
  BW6_761::affine_t points[],
  BW6_761::scalar_t scalars[],
  size_t count,
  unsigned large_bucket_factor,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    large_msm<BW6_761::scalar_t, BW6_761::projective_t, BW6_761::affine_t>(
      scalars, points, count, out, false, false, large_bucket_factor, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int msm_batch_cuda_bw6_761(
  BW6_761::projective_t* out,
  BW6_761::affine_t points[],
  BW6_761::scalar_t scalars[],
  size_t batch_size,
  size_t msm_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    batched_large_msm<BW6_761::scalar_t, BW6_761::projective_t, BW6_761::affine_t>(
      scalars, points, batch_size, msm_size, out, false, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

/**
 * Commit to a polynomial using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or
 * points.
 * @param d_out Ouptut point to write the result to.
 * @param d_scalars Scalars for the MSM. Must be on device.
 * @param d_points Points for the MSM. Must be on device.
 * @param count Length of `d_scalars` and `d_points` arrays (they should have equal length).
 */
extern "C" int commit_cuda_bw6_761(
  BW6_761::projective_t* d_out,
  BW6_761::scalar_t* d_scalars,
  BW6_761::affine_t* d_points,
  size_t count,
  unsigned large_bucket_factor,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    large_msm(d_scalars, d_points, count, d_out, true, false, large_bucket_factor, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

/**
 * Commit to a batch of polynomials using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or
 * points.
 * @param d_out Ouptut point to write the results to.
 * @param d_scalars Scalars for the MSMs of all polynomials. Must be on device.
 * @param d_points Points for the MSMs. Must be on device. It is assumed that this set of bases is used for each MSM.
 * @param count Length of `d_points` array, `d_scalar` has length `count` * `batch_size`.
 * @param batch_size Size of the batch.
 */
extern "C" int commit_batch_cuda_bw6_761(
  BW6_761::projective_t* d_out,
  BW6_761::scalar_t* d_scalars,
  BW6_761::affine_t* d_points,
  size_t count,
  size_t batch_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    batched_large_msm(d_scalars, d_points, batch_size, count, d_out, true, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

#if defined(G2_DEFINED)
extern "C" int msm_g2_cuda_bw6_761(
  BW6_761::g2_projective_t* out,
  BW6_761::g2_affine_t points[],
  BW6_761::scalar_t scalars[],
  size_t count,
  unsigned large_bucket_factor,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    large_msm<BW6_761::scalar_t, BW6_761::g2_projective_t, BW6_761::g2_affine_t>(
      scalars, points, count, out, false, false, large_bucket_factor, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int msm_batch_g2_cuda_bw6_761(
  BW6_761::g2_projective_t* out,
  BW6_761::g2_affine_t points[],
  BW6_761::scalar_t scalars[],
  size_t batch_size,
  size_t msm_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    batched_large_msm<BW6_761::scalar_t, BW6_761::g2_projective_t, BW6_761::g2_affine_t>(
      scalars, points, batch_size, msm_size, out, false, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

/**
 * Commit to a polynomial using the MSM in G2 group.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or
 * points.
 * @param d_out Ouptut G2 point to write the result to.
 * @param d_scalars Scalars for the MSM. Must be on device.
 * @param d_points G2 affine points for the MSM. Must be on device.
 * @param count Length of `d_scalars` and `d_points` arrays (they should have equal length).
 */
extern "C" int commit_g2_cuda_bw6_761(
  BW6_761::g2_projective_t* d_out,
  BW6_761::scalar_t* d_scalars,
  BW6_761::g2_affine_t* d_points,
  size_t count,
  unsigned large_bucket_factor,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  // TODO: use device_id when working with multiple devices
  (void)device_id;
  try {
    hipStreamCreate(&stream);
    large_msm(d_scalars, d_points, count, d_out, true, false, large_bucket_factor, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

/**
 * Commit to a batch of polynomials using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or
 * points.
 * @param d_out Ouptut G2 point to write the results to.
 * @param d_scalars Scalars for the MSMs of all polynomials. Must be on device.
 * @param d_points G2 affine points for the MSMs. Must be on device. It is assumed that this set of bases is used for
 * each MSM.
 * @param count Length of `d_points` array, `d_scalar` has length `count` * `batch_size`.
 * @param batch_size Size of the batch.
 */
extern "C" int commit_batch_g2_cuda_bw6_761(
  BW6_761::g2_projective_t* d_out,
  BW6_761::scalar_t* d_scalars,
  BW6_761::g2_affine_t* d_points,
  size_t count,
  size_t batch_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  // TODO: use device_id when working with multiple devices
  (void)device_id;
  try {
    hipStreamCreate(&stream);
    batched_large_msm(d_scalars, d_points, batch_size, count, d_out, true, stream);
    hipStreamSynchronize(stream);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}
#endif
#endif

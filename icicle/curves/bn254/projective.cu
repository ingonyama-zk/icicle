#include <hip/hip_runtime.h>
#include "curve_config.cuh"
#include "../../primitives/projective.cuh"

extern "C" bool eq_bn254(BN254::projective_t *point1, BN254::projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BN254::point_field_t::zero()) && (point1->y == BN254::point_field_t::zero()) && (point1->z == BN254::point_field_t::zero())) && 
  !((point2->x == BN254::point_field_t::zero()) && (point2->y == BN254::point_field_t::zero()) && (point2->z == BN254::point_field_t::zero()));
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2_bn254(BN254::g2_projective_t *point1, BN254::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BN254::g2_point_field_t::zero()) && (point1->y == BN254::g2_point_field_t::zero()) && (point1->z == BN254::g2_point_field_t::zero())) && 
  !((point2->x == BN254::g2_point_field_t::zero()) && (point2->y == BN254::g2_point_field_t::zero()) && (point2->z == BN254::g2_point_field_t::zero()));
}
#endif
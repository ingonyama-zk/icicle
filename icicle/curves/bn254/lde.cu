#include "hip/hip_runtime.h"
#ifndef _BN254_LDE
#define _BN254_LDE
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "../../utils/mont.cuh"
#include "curve_config.cuh"
#include <hip/hip_runtime.h>

extern "C" BN254::scalar_t* build_domain_cuda_bn254(
  uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    if (inverse) {
      return fill_twiddle_factors_array(domain_size, BN254::scalar_t::omega_inv(logn), stream);
    } else {
      return fill_twiddle_factors_array(domain_size, BN254::scalar_t::omega(logn), stream);
    }
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return nullptr;
  }
}

extern "C" int ntt_cuda_bn254(
  BN254::scalar_t* arr, uint32_t n, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return ntt_end2end_template<BN254::scalar_t, BN254::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());

    return -1;
  }
}

extern "C" int ecntt_cuda_bn254(
  BN254::projective_t* arr,
  uint32_t n,
  bool inverse,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return ntt_end2end_template<BN254::projective_t, BN254::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int ntt_batch_cuda_bn254(
  BN254::scalar_t* arr,
  uint32_t arr_size,
  uint32_t batch_size,
  bool inverse,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return ntt_end2end_batch_template<BN254::scalar_t, BN254::scalar_t>(
      arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int ecntt_batch_cuda_bn254(
  BN254::projective_t* arr,
  uint32_t arr_size,
  uint32_t batch_size,
  bool inverse,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return ntt_end2end_batch_template<BN254::projective_t, BN254::scalar_t>(
      arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int interpolate_scalars_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_evaluations,
  BN254::scalar_t* d_domain,
  unsigned n,
  unsigned device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int interpolate_scalars_batch_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_evaluations,
  BN254::scalar_t* d_domain,
  unsigned n,
  unsigned batch_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    hipStreamCreate(&stream);
    return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int interpolate_scalars_on_coset_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_evaluations,
  BN254::scalar_t* d_domain,
  unsigned n,
  BN254::scalar_t* coset_powers,
  unsigned device_id = 0,
  hipStream_t stream = 0)
{
  try {
    return interpolate(d_out, d_evaluations, d_domain, n, true, coset_powers, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int interpolate_scalars_batch_on_coset_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_evaluations,
  BN254::scalar_t* d_domain,
  unsigned n,
  unsigned batch_size,
  BN254::scalar_t* coset_powers,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, true, coset_powers, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int interpolate_points_cuda_bn254(
  BN254::projective_t* d_out,
  BN254::projective_t* d_evaluations,
  BN254::scalar_t* d_domain,
  unsigned n,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int interpolate_points_batch_cuda_bn254(
  BN254::projective_t* d_out,
  BN254::projective_t* d_evaluations,
  BN254::scalar_t* d_domain,
  unsigned n,
  unsigned batch_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    hipStreamCreate(&stream);
    return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_scalars_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  unsigned device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    hipStreamCreate(&stream);
    return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_scalars_batch_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  unsigned batch_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    hipStreamCreate(&stream);
    return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_points_cuda_bn254(
  BN254::projective_t* d_out,
  BN254::projective_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    hipStreamCreate(&stream);
    return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_points_batch_cuda_bn254(
  BN254::projective_t* d_out,
  BN254::projective_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  unsigned batch_size,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    BN254::scalar_t* _null = nullptr;
    hipStreamCreate(&stream);
    return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_scalars_on_coset_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  BN254::scalar_t* coset_powers,
  unsigned device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_bn254(
  BN254::scalar_t* d_out,
  BN254::scalar_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  unsigned batch_size,
  BN254::scalar_t* coset_powers,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_points_on_coset_cuda_bn254(
  BN254::projective_t* d_out,
  BN254::projective_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  BN254::scalar_t* coset_powers,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int evaluate_points_on_coset_batch_cuda_bn254(
  BN254::projective_t* d_out,
  BN254::projective_t* d_coefficients,
  BN254::scalar_t* d_domain,
  unsigned domain_size,
  unsigned n,
  unsigned batch_size,
  BN254::scalar_t* coset_powers,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int ntt_inplace_batch_cuda_bn254(
  BN254::scalar_t* d_inout,
  BN254::scalar_t* d_twiddles,
  unsigned n,
  unsigned batch_size,
  bool inverse,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    BN254::scalar_t* _null = nullptr;
    ntt_inplace_batch_template(d_inout, d_twiddles, n, batch_size, inverse, false, _null, stream, true);
    return hipSuccess; // TODO: we should implement this https://leimao.github.io/blog/Proper-CUDA-Error-Checking/
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int ntt_inplace_coset_batch_cuda_bn254(
  BN254::scalar_t* d_inout,
  BN254::scalar_t* d_twiddles,
  unsigned n,
  unsigned batch_size,
  bool inverse,
  bool is_coset,
  BN254::scalar_t* coset,
  size_t device_id = 0,
  hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    ntt_inplace_batch_template(d_inout, d_twiddles, n, batch_size, inverse, is_coset, coset, stream, true);
    return hipSuccess; // TODO: we should implement this https://leimao.github.io/blog/Proper-CUDA-Error-Checking/
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int sub_scalars_cuda_bn254(
  BN254::scalar_t* d_out, BN254::scalar_t* d_in1, BN254::scalar_t* d_in2, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return sub_polys(d_out, d_in1, d_in2, n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int add_scalars_cuda_bn254(
  BN254::scalar_t* d_out, BN254::scalar_t* d_in1, BN254::scalar_t* d_in2, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return add_polys(d_out, d_in1, d_in2, n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int to_montgomery_scalars_cuda_bn254(BN254::scalar_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return to_montgomery(d_inout, n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int from_montgomery_scalars_cuda_bn254(BN254::scalar_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return from_montgomery(d_inout, n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int to_montgomery_proj_points_cuda_bn254(BN254::projective_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return to_montgomery((BN254::point_field_t*)d_inout, 3 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int from_montgomery_proj_points_cuda_bn254(BN254::projective_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return from_montgomery((BN254::point_field_t*)d_inout, 3 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int to_montgomery_aff_points_cuda_bn254(BN254::affine_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return to_montgomery((BN254::point_field_t*)d_inout, 2 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int from_montgomery_aff_points_cuda_bn254(BN254::affine_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return from_montgomery((BN254::point_field_t*)d_inout, 2 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

#if defined(G2_DEFINED)
extern "C" int
to_montgomery_proj_points_g2_cuda_bn254(BN254::g2_projective_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return to_montgomery((BN254::point_field_t*)d_inout, 6 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int
from_montgomery_proj_points_g2_cuda_bn254(BN254::g2_projective_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return from_montgomery((BN254::point_field_t*)d_inout, 6 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int to_montgomery_aff_points_g2_cuda_bn254(BN254::g2_affine_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return to_montgomery((BN254::point_field_t*)d_inout, 4 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int
from_montgomery_aff_points_g2_cuda_bn254(BN254::g2_affine_t* d_inout, unsigned n, hipStream_t stream = 0)
{
  try {
    hipStreamCreate(&stream);
    return from_montgomery((BN254::point_field_t*)d_inout, 4 * n, stream);
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}
#endif

extern "C" int
reverse_order_scalars_cuda_bn254(BN254::scalar_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
  try {
    uint32_t logn = uint32_t(log(n) / log(2));
    hipStreamCreate(&stream);
    reverse_order(arr, n, logn, stream);
    hipStreamSynchronize(stream);
    return 0;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int reverse_order_scalars_batch_cuda_bn254(
  BN254::scalar_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
  try {
    uint32_t logn = uint32_t(log(n) / log(2));
    hipStreamCreate(&stream);
    reverse_order_batch(arr, n, logn, batch_size, stream);
    return 0;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int
reverse_order_points_cuda_bn254(BN254::projective_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
  try {
    uint32_t logn = uint32_t(log(n) / log(2));
    hipStreamCreate(&stream);
    reverse_order(arr, n, logn, stream);
    return 0;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int reverse_order_points_batch_cuda_bn254(
  BN254::projective_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
  try {
    uint32_t logn = uint32_t(log(n) / log(2));
    hipStreamCreate(&stream);
    reverse_order_batch(arr, n, logn, batch_size, stream);
    return 0;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}
#endif

#include "hip/hip_runtime.h"
#ifndef _BN254_LDE
#define _BN254_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" BN254::scalar_t* build_domain_cuda_bn254(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0)
{
    try
    {
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, BN254::scalar_t::omega_inv(logn));
        } else {
            return fill_twiddle_factors_array(domain_size, BN254::scalar_t::omega(logn));
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_bn254(BN254::scalar_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_template<BN254::scalar_t,BN254::scalar_t>(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda_bn254(BN254::projective_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_template<BN254::projective_t,BN254::scalar_t>(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_bn254(BN254::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch_template<BN254::scalar_t,BN254::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda_bn254(BN254::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch_template<BN254::projective_t,BN254::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t *d_evaluations, BN254::scalar_t *d_domain, unsigned n, unsigned device_id = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t* d_evaluations, BN254::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t *d_evaluations, BN254::scalar_t *d_domain, unsigned n, size_t device_id = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t* d_evaluations, BN254::scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t *d_coefficients, BN254::scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t *d_coefficients, BN254::scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t *d_coefficients, BN254::scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, BN254::scalar_t *coset_powers, unsigned device_id = 0)
{
    try
    {
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, BN254::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t *d_coefficients, BN254::scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, BN254::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, BN254::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda_bn254(BN254::scalar_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_bn254(BN254::scalar_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_bn254(BN254::projective_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda_bn254(BN254::projective_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
#include "hip/hip_runtime.h"
#ifndef _BN254_LDE
#define _BN254_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" BN254::scalar_t* build_domain_cuda_bn254(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, BN254::scalar_t::omega_inv(logn), stream);
        } else {
            return fill_twiddle_factors_array(domain_size, BN254::scalar_t::omega(logn), stream);
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_bn254(BN254::scalar_t *arr, uint32_t n, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<BN254::scalar_t,BN254::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda_bn254(BN254::projective_t *arr, uint32_t n, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<BN254::projective_t,BN254::scalar_t>(arr, n, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_bn254(BN254::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<BN254::scalar_t,BN254::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda_bn254(BN254::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<BN254::projective_t,BN254::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t *d_evaluations, BN254::scalar_t *d_domain, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t* d_evaluations, BN254::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t *d_evaluations, BN254::scalar_t *d_domain, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t* d_evaluations, BN254::scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t *d_coefficients, BN254::scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t *d_coefficients, BN254::scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        BN254::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t *d_coefficients, BN254::scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, BN254::scalar_t *coset_powers, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_bn254(BN254::scalar_t* d_out, BN254::scalar_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, BN254::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t *d_coefficients, BN254::scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, BN254::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_bn254(BN254::projective_t* d_out, BN254::projective_t* d_coefficients, BN254::scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, BN254::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_inplace_batch_cuda_bn254(BN254::scalar_t* d_inout, BN254::scalar_t* d_twiddles,
                                           unsigned n, unsigned batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        ntt_inplace_batch_template(d_inout, d_twiddles, n, batch_size, inverse, stream, true);
        return hipSuccess; //TODO: we should implement this https://leimao.github.io/blog/Proper-CUDA-Error-Checking/
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda_bn254(BN254::scalar_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_bn254(BN254::scalar_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_bn254(BN254::projective_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda_bn254(BN254::projective_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
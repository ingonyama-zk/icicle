#include "hip/hip_runtime.h"
#ifndef _CURVE_NAME_U_LDE
#define _CURVE_NAME_U_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" CURVE_NAME_U::scalar_t* build_domain_cuda_CURVE_NAME_L(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0)
{
    try
    {
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, CURVE_NAME_U::scalar_t::omega_inv(logn));
        } else {
            return fill_twiddle_factors_array(domain_size, CURVE_NAME_U::scalar_t::omega(logn));
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_template<CURVE_NAME_U::scalar_t,CURVE_NAME_U::scalar_t>(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_template<CURVE_NAME_U::projective_t,CURVE_NAME_U::scalar_t>(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch_template<CURVE_NAME_U::scalar_t,CURVE_NAME_U::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch_template<CURVE_NAME_U::projective_t,CURVE_NAME_U::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* d_out, CURVE_NAME_U::scalar_t *d_evaluations, CURVE_NAME_U::scalar_t *d_domain, unsigned n, unsigned device_id = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* d_out, CURVE_NAME_U::scalar_t* d_evaluations, CURVE_NAME_U::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::projective_t *d_evaluations, CURVE_NAME_U::scalar_t *d_domain, unsigned n, size_t device_id = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::projective_t* d_evaluations, CURVE_NAME_U::scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* d_out, CURVE_NAME_U::scalar_t *d_coefficients, CURVE_NAME_U::scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0)
{
    try
    {
        CURVE_NAME_U::scalar_t* _null = nullptr;
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* d_out, CURVE_NAME_U::scalar_t* d_coefficients, CURVE_NAME_U::scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        CURVE_NAME_U::scalar_t* _null = nullptr;
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::projective_t *d_coefficients, CURVE_NAME_U::scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0)
{
    try
    {
        CURVE_NAME_U::scalar_t* _null = nullptr;
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::projective_t* d_coefficients, CURVE_NAME_U::scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        CURVE_NAME_U::scalar_t* _null = nullptr;
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* d_out, CURVE_NAME_U::scalar_t *d_coefficients, CURVE_NAME_U::scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, CURVE_NAME_U::scalar_t *coset_powers, unsigned device_id = 0)
{
    try
    {
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* d_out, CURVE_NAME_U::scalar_t* d_coefficients, CURVE_NAME_U::scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, CURVE_NAME_U::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::projective_t *d_coefficients, CURVE_NAME_U::scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, CURVE_NAME_U::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::projective_t* d_coefficients, CURVE_NAME_U::scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, CURVE_NAME_U::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::scalar_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
#include "hip/hip_runtime.h"
#ifndef _${CURVE_NAME_U}_LDE
#define _${CURVE_NAME_U}_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" ${CURVE_NAME_U}::scalar_t* build_domain_cuda_${CURVE_NAME_L}(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, ${CURVE_NAME_U}::scalar_t::omega_inv(logn), stream);
        } else {
            return fill_twiddle_factors_array(domain_size, ${CURVE_NAME_U}::scalar_t::omega(logn), stream);
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t *arr, uint32_t n, bool inverse, Decimation decimation, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<${CURVE_NAME_U}::scalar_t,${CURVE_NAME_U}::scalar_t>(arr, n, inverse, decimation, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t *arr, uint32_t n, bool inverse, Decimation decimation, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_template<${CURVE_NAME_U}::projective_t,${CURVE_NAME_U}::scalar_t>(arr, n, inverse, decimation, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<${CURVE_NAME_U}::scalar_t,${CURVE_NAME_U}::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return ntt_end2end_batch_template<${CURVE_NAME_U}::projective_t,${CURVE_NAME_U}::scalar_t>(arr, arr_size, batch_size, inverse, stream); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t *d_evaluations, ${CURVE_NAME_U}::scalar_t *d_domain, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t* d_evaluations, ${CURVE_NAME_U}::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_on_coset_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t *d_evaluations, ${CURVE_NAME_U}::scalar_t *d_domain, unsigned n, ${CURVE_NAME_U}::scalar_t *coset_powers, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_on_coset_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t* d_evaluations, ${CURVE_NAME_U}::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, ${CURVE_NAME_U}::scalar_t* coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* d_out, ${CURVE_NAME_U}::projective_t *d_evaluations, ${CURVE_NAME_U}::scalar_t *d_domain, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* d_out, ${CURVE_NAME_U}::projective_t* d_evaluations, ${CURVE_NAME_U}::scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t *d_coefficients, ${CURVE_NAME_U}::scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t* d_coefficients, ${CURVE_NAME_U}::scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* d_out, ${CURVE_NAME_U}::projective_t *d_coefficients, ${CURVE_NAME_U}::scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* d_out, ${CURVE_NAME_U}::projective_t* d_coefficients, ${CURVE_NAME_U}::scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        ${CURVE_NAME_U}::scalar_t* _null = nullptr;
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t *d_coefficients, ${CURVE_NAME_U}::scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, ${CURVE_NAME_U}::scalar_t *coset_powers, unsigned device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* d_out, ${CURVE_NAME_U}::scalar_t* d_coefficients, ${CURVE_NAME_U}::scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, ${CURVE_NAME_U}::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* d_out, ${CURVE_NAME_U}::projective_t *d_coefficients, ${CURVE_NAME_U}::scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, ${CURVE_NAME_U}::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* d_out, ${CURVE_NAME_U}::projective_t* d_coefficients, ${CURVE_NAME_U}::scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, ${CURVE_NAME_U}::scalar_t *coset_powers, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        hipStreamSynchronize(stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::scalar_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* arr, int n, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order(arr, n, logn, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t* arr, int n, int batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        hipStreamCreate(&stream);
        reverse_order_batch(arr, n, logn, batch_size, stream);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
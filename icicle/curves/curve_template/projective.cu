#include <hip/hip_runtime.h>
#include "curve_config.cuh"
#include "../../primitives/projective.cuh"

extern "C" bool eq_${CURVE_NAME_L}(${CURVE_NAME_U}::projective_t *point1, ${CURVE_NAME_U}::projective_t *point2)
{
    return (*point1 == *point2) && 
    !((point1->x == ${CURVE_NAME_U}::point_field_t::zero()) && (point1->y == ${CURVE_NAME_U}::point_field_t::zero()) && (point1->z == ${CURVE_NAME_U}::point_field_t::zero())) && 
    !((point2->x == ${CURVE_NAME_U}::point_field_t::zero()) && (point2->y == ${CURVE_NAME_U}::point_field_t::zero()) && (point2->z == ${CURVE_NAME_U}::point_field_t::zero()));
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2_${CURVE_NAME_L}(${CURVE_NAME_U}::g2_projective_t *point1, ${CURVE_NAME_U}::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == ${CURVE_NAME_U}::g2_point_field_t::zero()) && (point1->y == ${CURVE_NAME_U}::g2_point_field_t::zero()) && (point1->z == ${CURVE_NAME_U}::g2_point_field_t::zero())) && 
  !((point2->x == ${CURVE_NAME_U}::g2_point_field_t::zero()) && (point2->y == ${CURVE_NAME_U}::g2_point_field_t::zero()) && (point2->z == ${CURVE_NAME_U}::g2_point_field_t::zero()));
}
#endif
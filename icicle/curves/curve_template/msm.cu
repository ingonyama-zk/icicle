#ifndef _CURVE_NAME_U_MSM
#define _CURVE_NAME_U_MSM
#include "../../appUtils/msm/msm.cu"
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "curve_config.cuh"


extern "C"
int msm_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t *out, CURVE_NAME_U::affine_t points[],
              CURVE_NAME_U::scalar_t scalars[], size_t count, size_t device_id = 0)
{
    try
    {
        if (count>256){
            large_msm<CURVE_NAME_U::scalar_t, CURVE_NAME_U::projective_t, CURVE_NAME_U::affine_t>(scalars, points, count, out, false);
        }
        else{
            short_msm<CURVE_NAME_U::scalar_t, CURVE_NAME_U::projective_t, CURVE_NAME_U::affine_t>(scalars, points, count, out, false);
        }

        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int msm_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* out, CURVE_NAME_U::affine_t points[],
                              CURVE_NAME_U::scalar_t scalars[], size_t batch_size, size_t msm_size, size_t device_id = 0)
{
  try
  {
    batched_large_msm<CURVE_NAME_U::scalar_t, CURVE_NAME_U::projective_t, CURVE_NAME_U::affine_t>(scalars, points, batch_size, msm_size, out, false);

    return hipSuccess;
  }
  catch (const std::runtime_error &ex)
  {
    printf("error %s", ex.what());
    return -1;
  }
}

/**
 * Commit to a polynomial using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
 * @param d_out Ouptut point to write the result to.
 * @param d_scalars Scalars for the MSM. Must be on device.
 * @param d_points Points for the MSM. Must be on device.
 * @param count Length of `d_scalars` and `d_points` arrays (they should have equal length).
 */
 extern "C"
 int commit_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::scalar_t* d_scalars, CURVE_NAME_U::affine_t* d_points, size_t count, size_t device_id = 0)
 {
     try
     {
         large_msm(d_scalars, d_points, count, d_out, true);
         return 0;
     }
     catch (const std::runtime_error &ex)
     {
         printf("error %s", ex.what());
         return -1;
     }
 }
 
 /**
  * Commit to a batch of polynomials using the MSM.
  * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
  * @param d_out Ouptut point to write the results to.
  * @param d_scalars Scalars for the MSMs of all polynomials. Must be on device.
  * @param d_points Points for the MSMs. Must be on device. It is assumed that this set of bases is used for each MSM.
  * @param count Length of `d_points` array, `d_scalar` has length `count` * `batch_size`.
  * @param batch_size Size of the batch.
  */
 extern "C"
 int commit_batch_cuda_CURVE_NAME_L(CURVE_NAME_U::projective_t* d_out, CURVE_NAME_U::scalar_t* d_scalars, CURVE_NAME_U::affine_t* d_points, size_t count, size_t batch_size, size_t device_id = 0)
 {
     try
     {
         batched_large_msm(d_scalars, d_points, batch_size, count, d_out, true);
         return 0;
     }
     catch (const std::runtime_error &ex)
     {
         printf("error %s", ex.what());
         return -1;
     }
 }

 #endif

#include "../../primitives/projective.cuh"
#include "curve_config.cuh"
#include <hip/hip_runtime.h>

extern "C" int random_projective_bls12_377(BLS12_377::projective_t* out) { 
  try {
    out[0] = BLS12_377::projective_t::rand_host();
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" BLS12_377::projective_t projective_zero_bls12_377() { return BLS12_377::projective_t::zero(); }

extern "C" bool projective_is_on_curve_bls12_377(BLS12_377::projective_t* point1)
{
  return BLS12_377::projective_t::is_on_curve(*point1);
}

extern "C" int projective_to_affine_bls12_377(BLS12_377::affine_t* out, BLS12_377::projective_t* point1)
{
  try {
    out[0] = BLS12_377::projective_t::to_affine(*point1);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int projective_from_affine_bls12_377(BLS12_377::projective_t* out, BLS12_377::affine_t* point1)
{
  try {
    out[0] = BLS12_377::projective_t::from_affine(*point1);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int random_scalar_bls12_377(BLS12_377::scalar_field_t* out) { 
  try {
    out[0] = BLS12_377::scalar_field_t::rand_host();
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" bool eq_bls12_377(BLS12_377::projective_t* point1, BLS12_377::projective_t* point2)
{
  return (*point1 == *point2) &&
         !((point1->x == BLS12_377::point_field_t::zero()) && (point1->y == BLS12_377::point_field_t::zero()) &&
           (point1->z == BLS12_377::point_field_t::zero())) &&
         !((point2->x == BLS12_377::point_field_t::zero()) && (point2->y == BLS12_377::point_field_t::zero()) &&
           (point2->z == BLS12_377::point_field_t::zero()));
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2_bls12_377(BLS12_377::g2_projective_t* point1, BLS12_377::g2_projective_t* point2)
{
  return (*point1 == *point2) &&
         !((point1->x == BLS12_377::g2_point_field_t::zero()) && (point1->y == BLS12_377::g2_point_field_t::zero()) &&
           (point1->z == BLS12_377::g2_point_field_t::zero())) &&
         !((point2->x == BLS12_377::g2_point_field_t::zero()) && (point2->y == BLS12_377::g2_point_field_t::zero()) &&
           (point2->z == BLS12_377::g2_point_field_t::zero()));
}

extern "C" int random_g2_projective_bls12_377(BLS12_377::g2_projective_t* out) 
{ 
  try {
    out[0] = BLS12_377::g2_projective_t::rand_host();
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int g2_projective_to_affine_bls12_377(BLS12_377::g2_affine_t* out, BLS12_377::g2_projective_t* point1)
{
  try {
    out[0] = BLS12_377::g2_projective_t::to_affine(*point1);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" int g2_projective_from_affine_bls12_377(BLS12_377::g2_projective_t* out, BLS12_377::g2_affine_t* point1)
{
  try {
    out[0] = BLS12_377::g2_projective_t::from_affine(*point1);
    return hipSuccess;
  } catch (const std::runtime_error& ex) {
    printf("error %s", ex.what());
    return -1;
  }
}

extern "C" bool g2_projective_is_on_curve_bls12_377(BLS12_377::g2_projective_t* point1)
{
  return BLS12_377::g2_projective_t::is_on_curve(*point1);
}

#endif

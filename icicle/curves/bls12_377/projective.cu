
#include <hip/hip_runtime.h>

#include "curve_config.cuh"

#include "../../primitives/projective.cuh"

extern "C" bool eq_bls12_377(BLS12_377::projective_t *point1, BLS12_377::projective_t *point2)
{
    return (*point1 == *point2) && 
    !((point1->x == BLS12_377::point_field_t::zero()) && (point1->y == BLS12_377::point_field_t::zero()) && (point1->z == BLS12_377::point_field_t::zero())) && 
    !((point2->x == BLS12_377::point_field_t::zero()) && (point2->y == BLS12_377::point_field_t::zero()) && (point2->z == BLS12_377::point_field_t::zero()));
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2_bls12_377(BLS12_377::g2_projective_t *point1, BLS12_377::g2_projective_t *point2)
{
  return (*point1 == *point2) && 
  !((point1->x == BLS12_377::g2_point_field_t::zero()) && (point1->y == BLS12_377::g2_point_field_t::zero()) && (point1->z == BLS12_377::g2_point_field_t::zero())) && 
  !((point2->x == BLS12_377::g2_point_field_t::zero()) && (point2->y == BLS12_377::g2_point_field_t::zero()) && (point2->z == BLS12_377::g2_point_field_t::zero()));
}
#endif

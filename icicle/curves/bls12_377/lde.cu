#include "hip/hip_runtime.h"
#ifndef _BLS12_377_LDE
#define _BLS12_377_LDE
#include <hip/hip_runtime.h>
#include "../../appUtils/ntt/lde.cu"
#include "../../appUtils/ntt/ntt.cuh"
#include "../../appUtils/vector_manipulation/ve_mod_mult.cuh"
#include "curve_config.cuh"

extern "C" BLS12_377::scalar_t* build_domain_cuda_bls12_377(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0)
{
    try
    {
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, BLS12_377::scalar_t::omega_inv(logn));
        } else {
            return fill_twiddle_factors_array(domain_size, BLS12_377::scalar_t::omega(logn));
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda_bls12_377(BLS12_377::scalar_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_template<BLS12_377::scalar_t,BLS12_377::scalar_t>(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda_bls12_377(BLS12_377::projective_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_template<BLS12_377::projective_t,BLS12_377::scalar_t>(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda_bls12_377(BLS12_377::scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch_template<BLS12_377::scalar_t,BLS12_377::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda_bls12_377(BLS12_377::projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch_template<BLS12_377::projective_t,BLS12_377::scalar_t>(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda_bls12_377(BLS12_377::scalar_t* d_out, BLS12_377::scalar_t *d_evaluations, BLS12_377::scalar_t *d_domain, unsigned n, unsigned device_id = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda_bls12_377(BLS12_377::scalar_t* d_out, BLS12_377::scalar_t* d_evaluations, BLS12_377::scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::projective_t *d_evaluations, BLS12_377::scalar_t *d_domain, unsigned n, size_t device_id = 0)
{
    try
    {
        return interpolate(d_out, d_evaluations, d_domain, n);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::projective_t* d_evaluations, BLS12_377::scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda_bls12_377(BLS12_377::scalar_t* d_out, BLS12_377::scalar_t *d_coefficients, BLS12_377::scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0)
{
    try
    {
        BLS12_377::scalar_t* _null = nullptr;
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda_bls12_377(BLS12_377::scalar_t* d_out, BLS12_377::scalar_t* d_coefficients, BLS12_377::scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        BLS12_377::scalar_t* _null = nullptr;
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::projective_t *d_coefficients, BLS12_377::scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0)
{
    try
    {
        BLS12_377::scalar_t* _null = nullptr;
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::projective_t* d_coefficients, BLS12_377::scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        BLS12_377::scalar_t* _null = nullptr;
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda_bls12_377(BLS12_377::scalar_t* d_out, BLS12_377::scalar_t *d_coefficients, BLS12_377::scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, BLS12_377::scalar_t *coset_powers, unsigned device_id = 0)
{
    try
    {
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda_bls12_377(BLS12_377::scalar_t* d_out, BLS12_377::scalar_t* d_coefficients, BLS12_377::scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, BLS12_377::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::projective_t *d_coefficients, BLS12_377::scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, BLS12_377::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::projective_t* d_coefficients, BLS12_377::scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, BLS12_377::scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda_bls12_377(BLS12_377::scalar_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda_bls12_377(BLS12_377::scalar_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda_bls12_377(BLS12_377::projective_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda_bls12_377(BLS12_377::projective_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
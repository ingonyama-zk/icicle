#ifndef _BLS12_377_MSM
#define _BLS12_377_MSM
#include "../../appUtils/msm/msm.cu"
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "curve_config.cuh"


extern "C"
int msm_cuda_bls12_377(BLS12_377::projective_t *out, BLS12_377::affine_t points[],
              BLS12_377::scalar_t scalars[], size_t count, unsigned large_bucket_factor, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        large_msm<BLS12_377::scalar_t, BLS12_377::projective_t, BLS12_377::affine_t>(scalars, points, count, out, false, false, large_bucket_factor, stream);
        hipStreamSynchronize(stream);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int msm_batch_cuda_bls12_377(BLS12_377::projective_t* out, BLS12_377::affine_t points[],
                              BLS12_377::scalar_t scalars[], size_t batch_size, size_t msm_size, size_t device_id = 0, hipStream_t stream = 0)
{
  try
  {
    hipStreamCreate(&stream);
    batched_large_msm<BLS12_377::scalar_t, BLS12_377::projective_t, BLS12_377::affine_t>(scalars, points, batch_size, msm_size, out, false, stream);
    hipStreamSynchronize(stream);

    return hipSuccess;
  }
  catch (const std::runtime_error &ex)
  {
    printf("error %s", ex.what());
    return -1;
  }
}

/**
 * Commit to a polynomial using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
 * @param d_out Ouptut point to write the result to.
 * @param d_scalars Scalars for the MSM. Must be on device.
 * @param d_points Points for the MSM. Must be on device.
 * @param count Length of `d_scalars` and `d_points` arrays (they should have equal length).
 */
 extern "C"
 int commit_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::scalar_t* d_scalars, BLS12_377::affine_t* d_points, size_t count, unsigned large_bucket_factor, size_t device_id = 0, hipStream_t stream = 0)
 {
     try
     {
         hipStreamCreate(&stream);
         large_msm(d_scalars, d_points, count, d_out, true, false, large_bucket_factor, stream);
         hipStreamSynchronize(stream);
         return hipSuccess;
     }
     catch (const std::runtime_error &ex)
     {
         printf("error %s", ex.what());
         return -1;
     }
 }
 
 /**
  * Commit to a batch of polynomials using the MSM.
  * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
  * @param d_out Ouptut point to write the results to.
  * @param d_scalars Scalars for the MSMs of all polynomials. Must be on device.
  * @param d_points Points for the MSMs. Must be on device. It is assumed that this set of bases is used for each MSM.
  * @param count Length of `d_points` array, `d_scalar` has length `count` * `batch_size`.
  * @param batch_size Size of the batch.
  */
 extern "C"
 int commit_batch_cuda_bls12_377(BLS12_377::projective_t* d_out, BLS12_377::scalar_t* d_scalars, BLS12_377::affine_t* d_points, size_t count, size_t batch_size, size_t device_id = 0, hipStream_t stream = 0)
 {
     try
     {
        hipStreamCreate(&stream);
         batched_large_msm(d_scalars, d_points, batch_size, count, d_out, true, stream);
         hipStreamSynchronize(stream);
         return hipSuccess;
     }
     catch (const std::runtime_error &ex)
     {
         printf("error %s", ex.what());
         return -1;
     }
 }

#if defined(G2_DEFINED)
extern "C"
int msm_g2_cuda_bls12_377(BLS12_377::g2_projective_t *out, BLS12_377::g2_affine_t points[],
              BLS12_377::scalar_t scalars[], size_t count, unsigned large_bucket_factor, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {   
        hipStreamCreate(&stream);
        large_msm<BLS12_377::scalar_t, BLS12_377::g2_projective_t, BLS12_377::g2_affine_t>(scalars, points, count, out, false, false, large_bucket_factor, stream);
        hipStreamSynchronize(stream);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int msm_batch_g2_cuda_bls12_377(BLS12_377::g2_projective_t* out, BLS12_377::g2_affine_t points[],
                              BLS12_377::scalar_t scalars[], size_t batch_size, size_t msm_size, size_t device_id = 0, hipStream_t stream = 0)
{
    try
    {
        hipStreamCreate(&stream);
        batched_large_msm<BLS12_377::scalar_t, BLS12_377::g2_projective_t, BLS12_377::g2_affine_t>(scalars, points, batch_size, msm_size, out, false, stream);
        hipStreamSynchronize(stream);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

/**
 * Commit to a polynomial using the MSM in G2 group.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
 * @param d_out Ouptut G2 point to write the result to.
 * @param d_scalars Scalars for the MSM. Must be on device.
 * @param d_points G2 affine points for the MSM. Must be on device.
 * @param count Length of `d_scalars` and `d_points` arrays (they should have equal length).
 */
extern "C"
int commit_g2_cuda_bls12_377(BLS12_377::g2_projective_t* d_out, BLS12_377::scalar_t* d_scalars, BLS12_377::g2_affine_t* d_points, size_t count, unsigned large_bucket_factor, size_t device_id = 0, hipStream_t stream = 0)
{
    // TODO: use device_id when working with multiple devices
    (void)device_id;
    try
    {
        hipStreamCreate(&stream);
        large_msm(d_scalars, d_points, count, d_out, true, false, large_bucket_factor, stream);
        hipStreamSynchronize(stream);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
 
 /**
  * Commit to a batch of polynomials using the MSM.
  * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
  * @param d_out Ouptut G2 point to write the results to.
  * @param d_scalars Scalars for the MSMs of all polynomials. Must be on device.
  * @param d_points G2 affine points for the MSMs. Must be on device. It is assumed that this set of bases is used for each MSM.
  * @param count Length of `d_points` array, `d_scalar` has length `count` * `batch_size`.
  * @param batch_size Size of the batch.
  */
extern "C"
int commit_batch_g2_cuda_bls12_377(BLS12_377::g2_projective_t* d_out, BLS12_377::scalar_t* d_scalars, BLS12_377::g2_affine_t* d_points, size_t count, size_t batch_size, size_t device_id = 0, hipStream_t stream = 0)
{
    // TODO: use device_id when working with multiple devices
    (void)device_id;
    try
    {
        hipStreamCreate(&stream);
        batched_large_msm(d_scalars, d_points, batch_size, count, d_out, true, stream);
        hipStreamSynchronize(stream);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}
#endif
#endif

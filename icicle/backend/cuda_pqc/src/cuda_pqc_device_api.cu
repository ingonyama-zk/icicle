#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include "icicle/device_api.h"
#include "icicle/errors.h"
#include "icicle/utils/log.h"
#include "icicle/runtime.h"
#include "hip/hip_runtime.h"

using namespace icicle;

class CudaPqcDeviceAPI : public DeviceAPI
{
public:
  eIcicleError set_device(const Device& device) override
  {
    hipError_t err = hipSetDevice(device.id);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::INVALID_DEVICE;
  }

  eIcicleError get_device_count(int& device_count) const override
  {
    hipError_t err = hipGetDeviceCount(&device_count);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::INVALID_DEVICE;
  }

  // Memory management
  eIcicleError allocate_memory(void** ptr, size_t size) const override
  {
    hipError_t err = hipMalloc(ptr, size);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError allocate_memory_async(void** ptr, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMallocAsync(ptr, size, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError free_memory(void* ptr) const override
  {
    hipError_t err = hipFree(ptr);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError free_memory_async(void* ptr, icicleStreamHandle stream) const override
  {
    hipError_t err = hipFreeAsync(ptr, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError get_available_memory(size_t& total /*OUT*/, size_t& free /*OUT*/) const override
  {
    hipError_t err = hipMemGetInfo(&free, &total);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::UNKNOWN_ERROR;
  }

  eIcicleError memset(void* ptr, int value, size_t size) const override
  {
    hipError_t err = hipMemset(ptr, value, size);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::UNKNOWN_ERROR;
  }

  eIcicleError memset_async(void* ptr, int value, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMemsetAsync(ptr, value, size, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::UNKNOWN_ERROR;
  }

  // Data transfer
  eIcicleError copy(void* dst, const void* src, size_t size, eCopyDirection direction) const override
  {
    hipMemcpyKind cuda_copy_kind = direction == eCopyDirection::HostToDevice   ? hipMemcpyHostToDevice
                                    : direction == eCopyDirection::DeviceToHost ? hipMemcpyDeviceToHost
                                                                                : hipMemcpyDeviceToDevice;
    hipError_t err = hipMemcpy(dst, src, size, cuda_copy_kind);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copy_async(
    void* dst, const void* src, size_t size, eCopyDirection direction, icicleStreamHandle stream) const override
  {
    hipMemcpyKind cuda_copy_kind = direction == eCopyDirection::HostToDevice   ? hipMemcpyHostToDevice
                                    : direction == eCopyDirection::DeviceToHost ? hipMemcpyDeviceToHost
                                                                                : hipMemcpyDeviceToDevice;
    hipError_t err = hipMemcpyAsync(dst, src, size, cuda_copy_kind, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  // Synchronization
  eIcicleError synchronize(icicleStreamHandle stream = nullptr) const override
  {
    hipError_t err =
      (stream == nullptr) ? hipDeviceSynchronize() : hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::SYNCHRONIZATION_FAILED;
  }

  // Stream management
  eIcicleError create_stream(icicleStreamHandle* stream) const override
  {
    hipStream_t cudaStream;
    hipError_t err = hipStreamCreate(&cudaStream);
    *stream = reinterpret_cast<icicleStreamHandle>(cudaStream);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_CREATION_FAILED;
  }

  eIcicleError destroy_stream(icicleStreamHandle stream) const override
  {
    hipError_t err = hipStreamDestroy(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_DESTRUCTION_FAILED;
  }

  eIcicleError get_device_properties(DeviceProperties& properties) const override
  {
    properties.using_host_memory = false;
    properties.num_memory_regions = 1;
    properties.supports_pinned_memory = false; // TODO support it for compatible devices
    return eIcicleError::SUCCESS;
  }
};

REGISTER_DEVICE_API("CUDA-PQC", CudaPqcDeviceAPI);

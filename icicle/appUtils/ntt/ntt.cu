#include "hip/hip_runtime.h"
#include "ntt.cuh"

#include <unordered_map>
#include <vector>

#include "curves/curve_config.cuh"
#include "utils/sharedmem.cuh"
#include "utils/utils_kernels.cuh"
#include "utils/utils.h"
#include "appUtils/ntt/ntt_impl.cuh"

namespace ntt {

  namespace {

    const uint32_t MAX_NUM_THREADS = 512;   // TODO: hotfix - should be 1024, currently limits shared memory size
    const uint32_t MAX_THREADS_BATCH = 512; // TODO: allows 100% occupancy for scalar NTT for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM_ELEMENT_SIZE = 32; // TODO: occupancy calculator, hardcoded for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM = MAX_SHARED_MEM_ELEMENT_SIZE * MAX_NUM_THREADS;

    template <typename E>
    __global__ void reverse_order_kernel(E* arr, E* arr_reversed, uint32_t n, uint32_t logn, uint32_t batch_size)
    {
      int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (threadId < n * batch_size) {
        int idx = threadId % n;
        int batch_idx = threadId / n;
        int idx_reversed = __brev(idx) >> (32 - logn);

        E val = arr[batch_idx * n + idx];
        if (arr == arr_reversed) { __syncthreads(); } // for in-place (when pointers arr==arr_reversed)
        arr_reversed[batch_idx * n + idx_reversed] = val;
      }
    }

    /**
     * Bit-reverses a batch of input arrays out-of-place inside GPU.
     * for example: on input array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr_in batch of arrays of some object of type T. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param batch_size the size of the batch.
     * @param arr_out buffer of the same size as `arr_in` on the GPU to write the bit-permuted array into.
     */
    template <typename E>
    void reverse_order_batch(E* arr_in, uint32_t n, uint32_t logn, uint32_t batch_size, hipStream_t stream, E* arr_out)
    {
      int number_of_threads = MAX_THREADS_BATCH;
      int number_of_blocks = (n * batch_size + number_of_threads - 1) / number_of_threads;
      reverse_order_kernel<<<number_of_blocks, number_of_threads, 0, stream>>>(arr_in, arr_out, n, logn, batch_size);
    }

    /**
     * Bit-reverses an input array out-of-place inside GPU.
     * for example: on array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr_in array of some object of type T of size which is a power of 2. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param arr_out buffer of the same size as `arr_in` on the GPU to write the bit-permuted array into.
     */
    template <typename E>
    void reverse_order(E* arr_in, uint32_t n, uint32_t logn, hipStream_t stream, E* arr_out)
    {
      reverse_order_batch(arr_in, n, logn, 1, stream, arr_out);
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared_rev(
      E* __restrict__ arr_in,
      int n,
      const S* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int ss,
      int logn,
      E* __restrict__ arr_out)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; ss < logn; ss++) {
            int s = logn - ss - 1;
            bool is_beginning = ss == 0;
            bool is_end = ss == (logn - 1);

            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;

            S tw = *(r_twiddles + (int)(j * n_twiddles_div));

            E u = is_beginning ? arr_in[offset + oij] : arr[oij];
            E v = is_beginning ? arr_in[offset + k] : arr[k];
            if (is_end) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = tw * (u - v);
            } else {
              arr[oij] = u + v;
              arr[k] = tw * (u - v);
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared(
      E* __restrict__ arr_in,
      int n,
      const S* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int s,
      int logn,
      E* __restrict__ arr_out)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; s < logn; s++) // TODO: this loop also can be unrolled
          {
            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;
            S tw = *(r_twiddles + (int)(j * n_twiddles_div));

            E u = s == 0 ? arr_in[offset + oij] : arr[oij];
            E v = s == 0 ? arr_in[offset + k] : arr[k];
            v = tw * v;
            if (s == (logn - 1)) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = u - v;
            } else {
              arr[oij] = u + v;
              arr[k] = u - v;
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tukey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     */
    template <typename E, typename S>
    __global__ void
    ntt_template_kernel(E* arr_in, int n, S* twiddles, int n_twiddles, int max_task, int s, bool rev, E* arr_out)
    {
      int task = blockIdx.x;
      int chunks = n / (blockDim.x * 2);

      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;
        uint32_t loop_limit = blockDim.x;

        if (l < loop_limit) {
          uint32_t ntw_i = task % chunks;

          uint32_t shift_s = 1 << s;
          uint32_t shift2_s = 1 << (s + 1);
          uint32_t n_twiddles_div = n_twiddles >> (s + 1);

          l = ntw_i * blockDim.x + l; // to l from chunks to full

          uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
          uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
          uint32_t k = i + j + shift_s;

          S tw = *(twiddles + (int)(j * n_twiddles_div));

          uint32_t offset = (task / chunks) * n;
          E u = arr_in[offset + i + j];
          E v = arr_in[offset + k];
          if (!rev) v = tw * v;
          arr_out[offset + i + j] = u + v;
          v = u - v;
          arr_out[offset + k] = rev ? tw * v : v;
        }
      }
    }

    /**
     * NTT/INTT inplace batch
     * Note: this function does not perform any bit-reverse permutations on its inputs or outputs.
     * @param d_input Input array
     * @param n Size of `d_input`
     * @param d_twiddles Twiddles
     * @param n_twiddles Size of `d_twiddles`
     * @param batch_size The size of the batch; the length of `d_inout` is `n` * `batch_size`.
     * @param inverse true for iNTT
     * @param coset should be array of length n or a nullptr if NTT is not computed on a coset
     * @param stream CUDA stream
     * @param is_async if false, perform sync of the supplied CUDA stream at the end of processing
     * @param d_output Output array
     */
    template <typename E, typename S>
    hipError_t ntt_inplace_batch_template(
      E* d_input,
      int n,
      S* d_twiddles,
      int n_twiddles,
      int batch_size,
      int logn,
      bool inverse,
      bool ct_buttterfly,
      S* arbitrary_coset,
      int coset_gen_index,
      hipStream_t stream,
      E* d_output)
    {
      CHK_INIT_IF_RETURN();

      bool is_shared_mem_enabled = sizeof(E) <= MAX_SHARED_MEM_ELEMENT_SIZE;
      const int log2_shmem_elems = is_shared_mem_enabled ? int(log(int(MAX_SHARED_MEM / sizeof(E))) / log(2)) : logn;
      int num_threads = max(min(min(n / 2, MAX_THREADS_BATCH), 1 << (log2_shmem_elems - 1)), 1);
      const int chunks = max(int((n / 2) / num_threads), 1);
      const int total_tasks = batch_size * chunks;
      int num_blocks = total_tasks;
      const int shared_mem = 2 * num_threads * sizeof(E); // TODO: calculator, as shared mem size may be more efficient
                                                          // less then max to allow more concurrent blocks on SM
      const int logn_shmem = is_shared_mem_enabled ? int(log(2 * num_threads) / log(2))
                                                   : 0; // TODO: shared memory support only for types <= 32 bytes
      int num_threads_coset = max(min(n / 2, MAX_NUM_THREADS), 1);
      int num_blocks_coset = (n * batch_size + num_threads_coset - 1) / num_threads_coset;

      if (inverse) {
        d_twiddles = d_twiddles + n_twiddles;
        n_twiddles = -n_twiddles;
      }

      bool is_on_coset = (coset_gen_index != 0) || arbitrary_coset;
      bool direct_coset = (!inverse && is_on_coset);
      if (direct_coset)
        utils_internal::BatchMulKernel<E, S><<<num_blocks_coset, num_threads_coset, 0, stream>>>(
          d_input, n, batch_size, arbitrary_coset ? arbitrary_coset : d_twiddles, arbitrary_coset ? 1 : coset_gen_index,
          n_twiddles, logn, ct_buttterfly, d_output);

      if (ct_buttterfly) {
        if (is_shared_mem_enabled)
          ntt_template_kernel_shared<<<num_blocks, num_threads, shared_mem, stream>>>(
            direct_coset ? d_output : d_input, 1 << logn_shmem, d_twiddles, n_twiddles, total_tasks, 0, logn_shmem,
            d_output);

        for (int s = logn_shmem; s < logn; s++) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<E, S><<<num_blocks, num_threads, 0, stream>>>(
            (direct_coset || (s > 0)) ? d_output : d_input, n, d_twiddles, n_twiddles, total_tasks, s, false, d_output);
        }
      } else {
        for (int s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<<<num_blocks, num_threads, 0, stream>>>(
            (direct_coset || (s < logn - 1)) ? d_output : d_input, n, d_twiddles, n_twiddles, total_tasks, s, true,
            d_output);
        }

        if (is_shared_mem_enabled)
          ntt_template_kernel_shared_rev<<<num_blocks, num_threads, shared_mem, stream>>>(
            (direct_coset || (logn > logn_shmem)) ? d_output : d_input, 1 << logn_shmem, d_twiddles, n_twiddles,
            total_tasks, 0, logn_shmem, d_output);
      }

      if (inverse) {
        if (is_on_coset)
          utils_internal::BatchMulKernel<E, S><<<num_blocks_coset, num_threads_coset, 0, stream>>>(
            d_output, n, batch_size, arbitrary_coset ? arbitrary_coset : d_twiddles,
            arbitrary_coset ? 1 : -coset_gen_index, -n_twiddles, logn, !ct_buttterfly, d_output);

        utils_internal::NormalizeKernel<E, S>
          <<<num_blocks_coset, num_threads_coset, 0, stream>>>(d_output, S::inv_log_size(logn), n * batch_size);
      }

      return CHK_LAST();
    }

  } // namespace

  /**
   * @struct Domain
   * Struct containing information about the domain on which (i)NTT is evaluated i.e. twiddle factors.
   * Twiddle factors are private, static and can only be set using [InitDomain](@ref InitDomain) function.
   * The internal representation of twiddles is prone to change in accordance with changing [NTT](@ref NTT) algorithm.
   * @tparam S The type of twiddle factors \f$ \{ \omega^i \} \f$. Must be a field.
   */
  template <typename S>
  class Domain
  {
    static inline int max_size = 0;
    static inline int max_log_size = 0;
    static inline S* twiddles = nullptr;
    static inline std::unordered_map<S, int> coset_index = {};

    static inline S* internal_twiddles = nullptr; // required by mixed-radix NTT
    static inline S* basic_twiddles = nullptr;    // required by mixed-radix NTT

  public:
    template <typename U>
    friend hipError_t InitDomain<U>(U primitive_root, device_context::DeviceContext& ctx);

    static hipError_t ReleaseDomain(device_context::DeviceContext& ctx);

    template <typename U, typename E>
    friend hipError_t NTT<U, E>(E* input, int size, NTTDir dir, NTTConfig<U>& config, E* output);
  };

  template <typename S>
  hipError_t InitDomain(S primitive_root, device_context::DeviceContext& ctx)
  {
    CHK_INIT_IF_RETURN();

    // only generate twiddles if they haven't been generated yet
    // please note that this is not thread-safe at all,
    // but it's a singleton that is supposed to be initialized once per program lifetime
    if (!Domain<S>::twiddles) {
      bool found_logn = false;
      S omega = primitive_root;
      unsigned omegas_count = S::get_omegas_count();
      for (int i = 0; i < omegas_count; i++) {
        omega = S::sqr(omega);
        if (!found_logn) {
          ++Domain<S>::max_log_size;
          found_logn = omega == S::one();
          if (found_logn) break;
        }
      }
      Domain<S>::max_size = (int)pow(2, Domain<S>::max_log_size);
      if (omega != S::one()) {
        throw IcicleError(
          IcicleError_t::InvalidArgument, "Primitive root provided to the InitDomain function is not in the subgroup");
      }

      // allocate and calculate twiddles on GPU
      // Note: radix-2 INTT needs ONE in last element (in addition to first element), therefore have n+1 elements
      // Managed allocation allows host to read the elements (logn) without copying all (n) TFs back to host
      CHK_IF_RETURN(hipMallocManaged(&Domain<S>::twiddles, (Domain<S>::max_size + 1) * sizeof(S)));
      CHK_IF_RETURN(generate_external_twiddles_generic(
        primitive_root, Domain<S>::twiddles, Domain<S>::internal_twiddles, Domain<S>::basic_twiddles,
        Domain<S>::max_log_size, ctx.stream));
      CHK_IF_RETURN(hipStreamSynchronize(ctx.stream));

      const bool is_map_only_powers_of_primitive_root = true;
      if (is_map_only_powers_of_primitive_root) {
        // populate the coset_index map. Note that only powers of the primitive-root are stored (1, PR, PR^2, PR^4, PR^8
        // etc.)
        Domain<S>::coset_index[S::one()] = 0;
        for (int i = 0; i < Domain<S>::max_log_size; ++i) {
          const int index = (int)pow(2, i);
          Domain<S>::coset_index[Domain<S>::twiddles[index]] = index;
        }
      } else {
        // populate all values
        for (int i = 0; i < Domain<S>::max_size; ++i) {
          Domain<S>::coset_index[Domain<S>::twiddles[i]] = i;
        }
      }
    }

    return CHK_LAST();
  }

  template <typename S>
  hipError_t Domain<S>::ReleaseDomain(device_context::DeviceContext& ctx)
  {
    CHK_INIT_IF_RETURN();

    max_size = 0;
    max_log_size = 0;
    hipFreeAsync(twiddles, ctx.stream);
    twiddles = nullptr;
    hipFreeAsync(internal_twiddles, ctx.stream);
    internal_twiddles = nullptr;
    hipFreeAsync(basic_twiddles, ctx.stream);
    basic_twiddles = nullptr;
    coset_index.clear();

    return CHK_LAST();
  }

  template <typename S, typename E>
  hipError_t NTT(E* input, int size, NTTDir dir, NTTConfig<S>& config, E* output)
  {
    CHK_INIT_IF_RETURN();
    if (size > Domain<S>::max_size) {
      std::cerr
        << "NTT size is too large for the domain. Consider generating your domain with a higher order root of unity"
        << '\n';
      throw -1;
    }

    hipStream_t& stream = config.ctx.stream;
    int batch_size = config.batch_size;
    int logn = int(log(size) / log(2));
    int input_size_bytes = size * batch_size * sizeof(E);
    bool are_inputs_on_device = config.are_inputs_on_device;
    bool are_outputs_on_device = config.are_outputs_on_device;

    E* d_input;
    if (are_inputs_on_device) {
      d_input = input;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&d_input, input_size_bytes, stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_input, input, input_size_bytes, hipMemcpyHostToDevice, stream));
    }
    E* d_output;
    if (are_outputs_on_device) {
      d_output = output;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&d_output, input_size_bytes, stream));
    }

    S* coset = nullptr;
    int coset_index = 0;
    try {
      coset_index = Domain<S>::coset_index.at(config.coset_gen);
    } catch (...) {
      // if coset index is not found in the subgroup, compute coset powers on CPU and move them to device
      std::vector<S> h_coset;
      h_coset.push_back(S::one());
      S coset_gen = (dir == NTTDir::kInverse) ? S::inverse(config.coset_gen) : config.coset_gen;
      for (int i = 1; i < size; i++) {
        h_coset.push_back(h_coset.at(i - 1) * coset_gen);
      }
      CHK_IF_RETURN(hipMallocAsync(&coset, size * sizeof(S), stream));
      CHK_IF_RETURN(hipMemcpyAsync(coset, &h_coset.front(), size * sizeof(S), hipMemcpyHostToDevice, stream));
      h_coset.clear();
    }

    const bool is_small_ntt = logn < 16;                  // cutoff point where mixed-radix is faster than radix-2
    const bool is_on_coset = (coset_index != 0) || coset; // coset not supported by mixed-radix algorithm yet
    const bool is_batch_ntt = batch_size > 1;             // batch not supported by mixed-radidx algorithm yet
    const bool is_NN = config.ordering == Ordering::kNN;  // TODO Yuval: relax this limitation
    const bool is_radix2_algorithm = config.is_force_radix2 || is_batch_ntt || is_small_ntt || is_on_coset || !is_NN;

    if (is_radix2_algorithm) {
      bool ct_butterfly = true;
      bool reverse_input = false;
      switch (config.ordering) {
      case Ordering::kNN:
        reverse_input = true;
        break;
      case Ordering::kNR:
        ct_butterfly = false;
        break;
      case Ordering::kRR:
        reverse_input = true;
        ct_butterfly = false;
        break;
      }

      if (reverse_input) reverse_order_batch(d_input, size, logn, batch_size, stream, d_output);

      CHK_IF_RETURN(ntt_inplace_batch_template(
        reverse_input ? d_output : d_input, size, Domain<S>::twiddles, Domain<S>::max_size, batch_size, logn,
        dir == NTTDir::kInverse, ct_butterfly, coset, coset_index, stream, d_output));

      if (coset) CHK_IF_RETURN(hipFreeAsync(coset, stream));
    } else { // mixed-radix algorithm
      CHK_IF_RETURN(ntt::mixed_radix_ntt(
        d_input, d_output, Domain<S>::twiddles, Domain<S>::internal_twiddles, Domain<S>::basic_twiddles, size,
        Domain<S>::max_log_size, dir == NTTDir::kInverse, config.ordering, stream));
    }

    if (!are_outputs_on_device)
      CHK_IF_RETURN(hipMemcpyAsync(output, d_output, input_size_bytes, hipMemcpyDeviceToHost, stream));

    if (!are_inputs_on_device) CHK_IF_RETURN(hipFreeAsync(d_input, stream));
    if (!are_outputs_on_device) CHK_IF_RETURN(hipFreeAsync(d_output, stream));
    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));

    return CHK_LAST();
  }

  template <typename S>
  NTTConfig<S> DefaultNTTConfig()
  {
    device_context::DeviceContext ctx = device_context::get_default_device_context();
    NTTConfig<S> config = {
      ctx,           // ctx
      S::one(),      // coset_gen
      1,             // batch_size
      Ordering::kNN, // ordering
      false,         // are_inputs_on_device
      false,         // are_outputs_on_device
      false,         // is_async
      false,         // is_force_radix2
    };
    return config;
  }

  /**
   * Extern "C" version of [InitDomain](@ref InitDomain) function with the following
   * value of template parameter (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   */
  extern "C" hipError_t
  CONCAT_EXPAND(CURVE, InitializeDomain)(curve_config::scalar_t primitive_root, device_context::DeviceContext& ctx)
  {
    return InitDomain(primitive_root, ctx);
  }

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, NTTCuda)(
    curve_config::scalar_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::scalar_t>(input, size, dir, config, output);
  }

#if defined(ECNTT_DEFINED)

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [projective representation](@ref projective_t) of the curve (i.e. EC NTT is computed);
   *  - `E` is the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, ECNTTCuda)(
    curve_config::projective_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::projective_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::projective_t>(input, size, dir, config, output);
  }

#endif

} // namespace ntt
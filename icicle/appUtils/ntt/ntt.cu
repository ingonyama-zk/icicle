#include "hip/hip_runtime.h"
#include "ntt.cuh"

#include "../../curves/curve_config.cuh"
#include "../../utils/sharedmem.cuh"
#include "../../utils/utils_kernels.cuh"

namespace ntt {

  namespace {

    const uint32_t MAX_NUM_THREADS = 512;
    const uint32_t MAX_THREADS_BATCH = 512;          // TODO: allows 100% occupancy for scalar NTT for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM_ELEMENT_SIZE = 32; // TODO: occupancy calculator, hardcoded for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM = MAX_SHARED_MEM_ELEMENT_SIZE * MAX_NUM_THREADS;

    /**
     * Computes the twiddle factors.
     * Outputs: d_twiddles[i] = omega^i.
     * @param d_twiddles input empty array.
     * @param n_twiddles number of twiddle factors.
     * @param omega multiplying factor.
     */
    template <typename S>
    __global__ void twiddle_factors_kernel(S* d_twiddles, int n_twiddles, S omega)
    {
      d_twiddles[0] = S::one();
      for (int i = 0; i < n_twiddles - 1; i++) {
        d_twiddles[i + 1] = omega * d_twiddles[i];
      }
    }

    template <typename E>
    __global__ void reverse_order_kernel(E* arr, E* arr_reversed, uint32_t n, uint32_t logn, uint32_t batch_size)
    {
      int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (threadId < n * batch_size) {
        int idx = threadId % n;
        int batch_idx = threadId / n;
        int idx_reversed = __brev(idx) >> (32 - logn);
        arr_reversed[batch_idx * n + idx_reversed] = arr[batch_idx * n + idx];
      }
    }

    /**
     * Bit-reverses a batch of input arrays in-place inside GPU.
     * for example: on input array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr batch of arrays of some object of type T. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param batch_size the size of the batch.
     */
    template <typename E>
    void reverse_order_batch(E* arr, uint32_t n, uint32_t logn, uint32_t batch_size, hipStream_t stream)
    {
      E* arr_reversed;
      hipMallocAsync(&arr_reversed, n * batch_size * sizeof(E), stream);
      int number_of_threads = MAX_THREADS_BATCH;
      int number_of_blocks = (n * batch_size + number_of_threads - 1) / number_of_threads;
      reverse_order_kernel<<<number_of_blocks, number_of_threads, 0, stream>>>(arr, arr_reversed, n, logn, batch_size);
      hipMemcpyAsync(arr, arr_reversed, n * batch_size * sizeof(E), hipMemcpyDefault, stream);
      hipFreeAsync(arr_reversed, stream);
    }

    /**
     * Bit-reverses an input array in-place inside GPU.
     * for example: on array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr array of some object of type T of size which is a power of 2. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     */
    template <typename E>
    void reverse_order(E* arr, uint32_t n, uint32_t logn, hipStream_t stream)
    {
      reverse_order_batch(arr, n, logn, 1, stream);
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared_rev(
      E* __restrict__ arr_g,
      uint32_t n,
      const S* __restrict__ r_twiddles,
      uint32_t n_twiddles,
      uint32_t max_task,
      uint32_t ss,
      uint32_t logn)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; ss < logn; ss++) {
            int s = logn - ss - 1;
            bool is_beginning = ss == 0;
            bool is_end = ss == (logn - 1);

            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;

            S tw = r_twiddles[j * n_twiddles_div];

            E u = is_beginning ? arr_g[offset + oij] : arr[oij];
            E v = is_beginning ? arr_g[offset + k] : arr[k];
            if (is_end) {
              arr_g[offset + oij] = u + v;
              arr_g[offset + k] = tw * (u - v);
            } else {
              arr[oij] = u + v;
              arr[k] = tw * (u - v);
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared(
      E* __restrict__ arr_g,
      uint32_t n,
      const S* __restrict__ r_twiddles,
      uint32_t n_twiddles,
      uint32_t max_task,
      uint32_t s,
      uint32_t logn)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; s < logn; s++) // TODO: this loop also can be unrolled
          {
            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;
            S tw = r_twiddles[j * n_twiddles_div];

            E u = s == 0 ? arr_g[offset + oij] : arr[oij];
            E v = s == 0 ? arr_g[offset + k] : arr[k];
            v = tw * v;
            if (s == (logn - 1)) {
              arr_g[offset + oij] = u + v;
              arr_g[offset + k] = u - v;
            } else {
              arr[oij] = u + v;
              arr[k] = u - v;
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tukey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     */
    template <typename E, typename S>
    __global__ void
    ntt_template_kernel(E* arr, uint32_t n, S* twiddles, uint32_t n_twiddles, uint32_t max_task, uint32_t s, bool rev)
    {
      int task = blockIdx.x;
      int chunks = n / (blockDim.x * 2);

      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;
        uint32_t loop_limit = blockDim.x;

        if (l < loop_limit) {
          uint32_t ntw_i = task % chunks;

          uint32_t shift_s = 1 << s;
          uint32_t shift2_s = 1 << (s + 1);
          uint32_t n_twiddles_div = n_twiddles >> (s + 1);

          l = ntw_i * blockDim.x + l; // to l from chunks to full

          uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
          uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
          uint32_t k = i + j + shift_s;

          S tw = twiddles[j * n_twiddles_div];

          uint32_t offset = (task / chunks) * n;
          E u = arr[offset + i + j];
          E v = arr[offset + k];
          if (!rev) v = tw * v;
          arr[offset + i + j] = u + v;
          v = u - v;
          arr[offset + k] = rev ? tw * v : v;
        }
      }
    }

    /**
     * NTT/INTT inplace batch
     * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
     * @param d_inout Array for inplace processing
     * @param d_twiddles
     * @param n Length of `d_twiddles` array
     * @param batch_size The size of the batch; the length of `d_inout` is `n` * `batch_size`.
     * @param inverse true for iNTT
     * @param is_coset true for multiplication by coset
     * @param coset should be array of lenght n - or in case of lesser than n, right-padded with zeroes
     * @param stream CUDA stream
     * @param is_sync_needed do perform sync of the supplied CUDA stream at the end of processing
     */
    template <typename E, typename S>
    void ntt_inplace_batch_template(
      E* d_inout,
      S* d_twiddles,
      unsigned n,
      unsigned batch_size,
      bool inverse,
      bool is_coset,
      S* coset,
      hipStream_t stream,
      bool is_sync_needed)
    {
      const int logn = int(log(n) / log(2));
      bool is_shared_mem_enabled = sizeof(E) <= MAX_SHARED_MEM_ELEMENT_SIZE;
      const int log2_shmem_elems = is_shared_mem_enabled ? int(log(int(MAX_SHARED_MEM / sizeof(E))) / log(2)) : logn;
      int num_threads = max(min(min(n / 2, MAX_THREADS_BATCH), 1 << (log2_shmem_elems - 1)), 1);
      const int chunks = max(int((n / 2) / num_threads), 1);
      const int total_tasks = batch_size * chunks;
      int num_blocks = total_tasks;
      const int shared_mem = 2 * num_threads * sizeof(E); // TODO: calculator, as shared mem size may be more efficient
                                                          // less then max to allow more concurrent blocks on SM
      const int logn_shmem = is_shared_mem_enabled ? int(log(2 * num_threads) / log(2))
                                                   : 0; // TODO: shared memory support only for types <= 32 bytes

      if (inverse) {
        if (is_shared_mem_enabled)
          ntt_template_kernel_shared<<<num_blocks, num_threads, shared_mem, stream>>>(
            d_inout, 1 << logn_shmem, d_twiddles, n, total_tasks, 0, logn_shmem);

        for (int s = logn_shmem; s < logn; s++) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<E, S>
            <<<num_blocks, num_threads, 0, stream>>>(d_inout, n, d_twiddles, n, total_tasks, s, false);
        }

        if (is_coset)
          utils_internal::BatchMulKernel<E, S><<<num_blocks, num_threads, 0, stream>>>(d_inout, coset, n, batch_size);

        num_threads = max(min(n / 2, MAX_NUM_THREADS), 1);
        num_blocks = (n * batch_size + num_threads - 1) / num_threads;
        utils_internal::NormalizeKernel<E, S>
          <<<num_blocks, num_threads, 0, stream>>>(d_inout, S::inv_log_size(logn), n * batch_size);
      } else {
        if (is_coset)
          utils_internal::BatchMulKernel<E, S><<<num_blocks, num_threads, 0, stream>>>(d_inout, coset, n, batch_size);

        for (int s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<<<num_blocks, num_threads, 0, stream>>>(d_inout, n, d_twiddles, n, total_tasks, s, true);
        }

        if (is_shared_mem_enabled)
          ntt_template_kernel_shared_rev<<<num_blocks, num_threads, shared_mem, stream>>>(
            d_inout, 1 << logn_shmem, d_twiddles, n, total_tasks, 0, logn_shmem);
      }

      if (!is_sync_needed) return;

      hipStreamSynchronize(stream);
    }

  } // namespace

  template <typename S>
  hipError_t GenerateTwiddleFactors(S* d_twiddles, int n_twiddles, S omega, device_context::DeviceContext ctx)
  {
    twiddle_factors_kernel<S><<<1, 1, 0, ctx.stream>>>(d_twiddles, n_twiddles, omega);
    hipStreamSynchronize(ctx.stream);
    return hipSuccess;
  }

  template <typename E, typename S>
  hipError_t NTT(NTTConfig<E, S>* config)
  {
    CHECK_LAST_CUDA_ERROR();

    hipStream_t stream = config->ctx.stream;
    int size = config->size;
    int batch_size = config->batch_size;
    bool is_inverse = config->is_inverse;
    int n_twiddles = size;
    int logn = int(log(size) / log(2));
    int input_size_bytes = size * batch_size * sizeof(E);
    bool is_input_on_device = config->are_inputs_on_device;
    bool is_output_on_device = config->is_output_on_device;
    bool is_forward_twiddle_empty = config->twiddles == nullptr;
    bool is_inverse_twiddle_empty = config->inv_twiddles == nullptr;
    bool is_generating_twiddles = (is_forward_twiddle_empty && is_inverse_twiddle_empty) ||
                                  (is_forward_twiddle_empty && !is_inverse) || (is_inverse_twiddle_empty && is_inverse);

    S* d_twiddles;
    if (is_generating_twiddles) {
      hipMallocAsync(&d_twiddles, n_twiddles * sizeof(S), stream);
      S omega = is_inverse ? S::omega_inv(logn) : S::omega(logn);
      GenerateTwiddleFactors(d_twiddles, n_twiddles, omega, config->ctx);
    } else {
      d_twiddles = is_inverse ? config->inv_twiddles : config->twiddles;
    }

    E* d_inout;
    if (is_input_on_device) {
      d_inout = config->inout;
    } else {
      hipMallocAsync(&d_inout, input_size_bytes, stream);
      hipMemcpyAsync(d_inout, config->inout, input_size_bytes, hipMemcpyHostToDevice, stream);
    }

    bool reverse_input;
    bool reverse_output;
    switch (config->ordering) {
    case Ordering::kNN:
      reverse_input = is_inverse;
      reverse_output = !is_inverse;
      break;
    case Ordering::kNR:
      reverse_input = is_inverse;
      reverse_output = is_inverse;
      break;
    case Ordering::kRN:
      reverse_input = !is_inverse;
      reverse_output = !is_inverse;
      break;
    case Ordering::kRR:
      reverse_input = !is_inverse;
      reverse_output = is_inverse;
      break;
    }
    CHECK_LAST_CUDA_ERROR();

    if (reverse_input) reverse_order_batch(d_inout, size, logn, config->batch_size, stream);
    CHECK_LAST_CUDA_ERROR();

    ntt_inplace_batch_template(
      d_inout, d_twiddles, size, batch_size, is_inverse, config->is_coset, config->coset_gen, stream, false);
    CHECK_LAST_CUDA_ERROR();

    if (reverse_output) reverse_order_batch(d_inout, size, logn, batch_size, stream);
    CHECK_LAST_CUDA_ERROR();

    if (is_output_on_device) {
      // free(config->inout); // TODO: ? or callback?+
      config->inout = d_inout;
    } else {
      if (is_input_on_device) {
        E* h_output = (E*)malloc(input_size_bytes); // TODO: caller responsible for memory management
        hipMemcpyAsync(h_output, d_inout, input_size_bytes, hipMemcpyDeviceToHost, stream);
        config->inout = h_output;
        CHECK_LAST_CUDA_ERROR();
      } else {
        hipMemcpyAsync(config->inout, d_inout, input_size_bytes, hipMemcpyDeviceToHost, stream);
        CHECK_LAST_CUDA_ERROR();
      }
      hipFreeAsync(d_inout, stream); // TODO: make it optional? so can be reused
    }
    CHECK_LAST_CUDA_ERROR();

    if (is_generating_twiddles && !config->is_preserving_twiddles) { hipFreeAsync(d_twiddles, stream); }

    if (config->is_preserving_twiddles) {
      if (is_inverse)
        config->inv_twiddles = d_twiddles;
      else {
        config->twiddles = d_twiddles;
      }
    }

    hipStreamSynchronize(stream);

    CHECK_LAST_CUDA_ERROR();

    return hipSuccess;
  }

  /**
   * Extern version of [ntt](@ref ntt) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t NTTCuda(NTTConfig<curve_config::scalar_t, curve_config::scalar_t>* config)
  {
    return NTT<curve_config::scalar_t, curve_config::scalar_t>(config);
  }

  /**
   * Extern version of [ntt](@ref ntt) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  template <typename E, typename S>
  hipError_t NTTDefaultContext(NTTConfig<E, S>* config)
  {
    // TODO: if empty - create default
    hipMemPool_t mempool;
    hipDeviceGetDefaultMemPool(&mempool, config->ctx.device_id);

    device_context::DeviceContext context = {
      config->ctx.device_id,
      0, // default stream
      mempool};

    config->ctx = context;

    return NTT<E, S>(config);
  }

  /**
   * Extern version of [ntt](@ref ntt) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t NTTDefaultContextCuda(NTTConfig<curve_config::scalar_t, curve_config::scalar_t>* config)
  {
    return NTTDefaultContext(config);
  }

#if defined(ECNTT_DEFINED)

  /**
   * Extern version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [projective representation](@ref projective_t) of the curve (i.e. EC NTT is computed);
   *  - `E` is the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t ECNTTCuda(NTTConfig<curve_config::projective_t, curve_config::scalar_t>* config)
  {
    return NTT<curve_config::projective_t, curve_config::scalar_t>(config);
  }

#endif

} // namespace ntt

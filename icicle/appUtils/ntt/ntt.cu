#include "hip/hip_runtime.h"
#include "ntt.cuh"

#include <unordered_map>
#include <vector>

#include "../../curves/curve_config.cuh"
#include "../../utils/sharedmem.cuh"
#include "../../utils/utils_kernels.cuh"

namespace ntt {

  namespace {

    const uint32_t MAX_NUM_THREADS = 512;   // TODO: hotfix - should be 1024, currently limits shared memory size
    const uint32_t MAX_THREADS_BATCH = 512; // TODO: allows 100% occupancy for scalar NTT for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM_ELEMENT_SIZE = 32; // TODO: occupancy calculator, hardcoded for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM = MAX_SHARED_MEM_ELEMENT_SIZE * MAX_NUM_THREADS;

    template <typename E>
    __global__ void reverse_order_kernel(E* arr, E* arr_reversed, uint32_t n, uint32_t logn, uint32_t batch_size)
    {
      int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (threadId < n * batch_size) {
        int idx = threadId % n;
        int batch_idx = threadId / n;
        int idx_reversed = __brev(idx) >> (32 - logn);
        arr_reversed[batch_idx * n + idx_reversed] = arr[batch_idx * n + idx];
      }
    }

    /**
     * Bit-reverses a batch of input arrays out-of-place inside GPU.
     * for example: on input array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr_in batch of arrays of some object of type T. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param batch_size the size of the batch.
     * @param arr_out buffer of the same size as `arr_in` on the GPU to write the bit-permuted array into.
     */
    template <typename E>
    void reverse_order_batch(E* arr_in, uint32_t n, uint32_t logn, uint32_t batch_size, hipStream_t stream, E* arr_out)
    {
      int number_of_threads = MAX_THREADS_BATCH;
      int number_of_blocks = (n * batch_size + number_of_threads - 1) / number_of_threads;
      reverse_order_kernel<<<number_of_blocks, number_of_threads, 0, stream>>>(arr_in, arr_out, n, logn, batch_size);
    }

    /**
     * Bit-reverses an input array out-of-place inside GPU.
     * for example: on array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr_in array of some object of type T of size which is a power of 2. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param arr_out buffer of the same size as `arr_in` on the GPU to write the bit-permuted array into.
     */
    template <typename E>
    void reverse_order(E* arr_in, uint32_t n, uint32_t logn, hipStream_t stream, E* arr_out)
    {
      reverse_order_batch(arr_in, n, logn, 1, stream, arr_out);
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared_rev(
      E* __restrict__ arr_in,
      int n,
      const S* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int ss,
      int logn,
      E* __restrict__ arr_out)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; ss < logn; ss++) {
            int s = logn - ss - 1;
            bool is_beginning = ss == 0;
            bool is_end = ss == (logn - 1);

            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;

            S tw = *(r_twiddles + (int)(j * n_twiddles_div));

            E u = is_beginning ? arr_in[offset + oij] : arr[oij];
            E v = is_beginning ? arr_in[offset + k] : arr[k];
            if (is_end) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = tw * (u - v);
            } else {
              arr[oij] = u + v;
              arr[k] = tw * (u - v);
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared(
      E* __restrict__ arr_in,
      int n,
      const S* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int s,
      int logn,
      E* __restrict__ arr_out)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; s < logn; s++) // TODO: this loop also can be unrolled
          {
            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;
            S tw = *(r_twiddles + (int)(j * n_twiddles_div));

            E u = s == 0 ? arr_in[offset + oij] : arr[oij];
            E v = s == 0 ? arr_in[offset + k] : arr[k];
            v = tw * v;
            if (s == (logn - 1)) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = u - v;
            } else {
              arr[oij] = u + v;
              arr[k] = u - v;
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tukey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     */
    template <typename E, typename S>
    __global__ void
    ntt_template_kernel(E* arr_in, int n, S* twiddles, int n_twiddles, int max_task, int s, bool rev, E* arr_out)
    {
      int task = blockIdx.x;
      int chunks = n / (blockDim.x * 2);

      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;
        uint32_t loop_limit = blockDim.x;

        if (l < loop_limit) {
          uint32_t ntw_i = task % chunks;

          uint32_t shift_s = 1 << s;
          uint32_t shift2_s = 1 << (s + 1);
          uint32_t n_twiddles_div = n_twiddles >> (s + 1);

          l = ntw_i * blockDim.x + l; // to l from chunks to full

          uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
          uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
          uint32_t k = i + j + shift_s;

          S tw = *(twiddles + (int)(j * n_twiddles_div));

          uint32_t offset = (task / chunks) * n;
          E u = arr_in[offset + i + j];
          E v = arr_in[offset + k];
          if (!rev) v = tw * v;
          arr_out[offset + i + j] = u + v;
          v = u - v;
          arr_out[offset + k] = rev ? tw * v : v;
        }
      }
    }

    /**
     * NTT/INTT inplace batch
     * Note: this function does not perform any bit-reverse permutations on its inputs or outputs.
     * @param d_input Input array
     * @param n Size of `d_input`
     * @param d_twiddles Twiddles
     * @param n_twiddles Size of `d_twiddles`
     * @param batch_size The size of the batch; the length of `d_inout` is `n` * `batch_size`.
     * @param inverse true for iNTT
     * @param coset should be array of lenght n or a nullptr if NTT is not computed on a coset
     * @param stream CUDA stream
     * @param is_async if false, perform sync of the supplied CUDA stream at the end of processing
     * @param d_output Output array
     */
    template <typename E, typename S>
    hipError_t ntt_inplace_batch_template(
      E* d_input,
      int n,
      S* d_twiddles,
      int n_twiddles,
      int batch_size,
      int logn,
      bool inverse,
      bool ct_buttterfly,
      S* arbitrary_coset,
      int coset_gen_index,
      hipStream_t stream,
      E* d_output)
    {
      CHK_INIT_IF_RETURN();

      bool is_shared_mem_enabled = sizeof(E) <= MAX_SHARED_MEM_ELEMENT_SIZE;
      const int log2_shmem_elems = is_shared_mem_enabled ? int(log(int(MAX_SHARED_MEM / sizeof(E))) / log(2)) : logn;
      int num_threads = max(min(min(n / 2, MAX_THREADS_BATCH), 1 << (log2_shmem_elems - 1)), 1);
      const int chunks = max(int((n / 2) / num_threads), 1);
      const int total_tasks = batch_size * chunks;
      int num_blocks = total_tasks;
      const int shared_mem = 2 * num_threads * sizeof(E); // TODO: calculator, as shared mem size may be more efficient
                                                          // less then max to allow more concurrent blocks on SM
      const int logn_shmem = is_shared_mem_enabled ? int(log(2 * num_threads) / log(2))
                                                   : 0; // TODO: shared memory support only for types <= 32 bytes
      int num_threads_coset = max(min(n / 2, MAX_NUM_THREADS), 1);
      int num_blocks_coset = (n * batch_size + num_threads_coset - 1) / num_threads_coset;

      if (inverse) {
        d_twiddles = d_twiddles + n_twiddles;
        n_twiddles = -n_twiddles;
      }

      bool is_on_coset = (coset_gen_index != 0) || arbitrary_coset;
      bool direct_coset = (!inverse && is_on_coset);
      if (direct_coset)
        utils_internal::BatchMulKernel<E, S><<<num_blocks_coset, num_threads_coset, 0, stream>>>(
          d_input, n, batch_size, arbitrary_coset ? arbitrary_coset : d_twiddles, arbitrary_coset ? 1 : coset_gen_index,
          n_twiddles, logn, ct_buttterfly, d_output);

      if (ct_buttterfly) {
        if (is_shared_mem_enabled)
          ntt_template_kernel_shared<<<num_blocks, num_threads, shared_mem, stream>>>(
            direct_coset ? d_output : d_input, 1 << logn_shmem, d_twiddles, n_twiddles, total_tasks, 0, logn_shmem,
            d_output);

        for (int s = logn_shmem; s < logn; s++) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<E, S><<<num_blocks, num_threads, 0, stream>>>(
            (direct_coset || (s > 0)) ? d_output : d_input, n, d_twiddles, n_twiddles, total_tasks, s, false, d_output);
        }
      } else {
        for (int s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<<<num_blocks, num_threads, 0, stream>>>(
            (direct_coset || (s < logn - 1)) ? d_output : d_input, n, d_twiddles, n_twiddles, total_tasks, s, true,
            d_output);
        }

        if (is_shared_mem_enabled)
          ntt_template_kernel_shared_rev<<<num_blocks, num_threads, shared_mem, stream>>>(
            (direct_coset || (logn > logn_shmem)) ? d_output : d_input, 1 << logn_shmem, d_twiddles, n_twiddles,
            total_tasks, 0, logn_shmem, d_output);
      }

      if (inverse) {
        if (is_on_coset)
          utils_internal::BatchMulKernel<E, S><<<num_blocks_coset, num_threads_coset, 0, stream>>>(
            d_output, n, batch_size, arbitrary_coset ? arbitrary_coset : d_twiddles,
            arbitrary_coset ? 1 : -coset_gen_index, -n_twiddles, logn, !ct_buttterfly, d_output);

        utils_internal::NormalizeKernel<E, S>
          <<<num_blocks_coset, num_threads_coset, 0, stream>>>(d_output, S::inv_log_size(logn), n * batch_size);
      }

      return CHK_LAST();
    }

  } // namespace

  /**
   * @struct Domain
   * Struct containing information about the domain on which (i)NTT is evaluated i.e. twiddle factors.
   * Twiddle factors are private, static and can only be set using [InitDomain](@ref InitDomain) function.
   * The internal representation of twiddles is prone to change in accordance with changing [NTT](@ref NTT) algorithm.
   * @tparam S The type of twiddle factors \f$ \{ \omega^i \} \f$. Must be a field.
   */
  template <typename S>
  class Domain
  {
    static int max_size;
    static S* twiddles;
    static std::unordered_map<S, int> coset_index;

  public:
    template <typename U>
    friend hipError_t InitDomain<U>(U primitive_root, device_context::DeviceContext& ctx);

    template <typename U, typename E>
    friend hipError_t NTT<U, E>(E* input, int size, NTTDir dir, NTTConfig<U>& config, E* output);
  };

  template <typename S>
  int Domain<S>::max_size = 0;
  template <typename S>
  S* Domain<S>::twiddles = nullptr;
  template <typename S>
  std::unordered_map<S, int> Domain<S>::coset_index = {};

  template <typename S>
  hipError_t InitDomain(S primitive_root, device_context::DeviceContext& ctx)
  {
    CHK_INIT_IF_RETURN();

    // only generate twiddles if they haven't been generated yet
    // please note that this is not thread-safe at all,
    // but it's a singleton that is supposed to be initialized once per program lifetime
    if (!Domain<S>::twiddles) {
      S omega = primitive_root;
      for (int i = 0; i < S::TWO_ADICITY; i++)
        omega = S::sqr(omega);
      if (omega != S::one()) {
        std::cerr << "Primitive root provided to the InitDomain function is not in the subgroup" << '\n';
        throw -1;
      }

      std::vector<S> h_twiddles;
      h_twiddles.push_back(S::one());
      int n = 1;
      do {
        Domain<S>::coset_index[h_twiddles.at(n - 1)] = n - 1;
        h_twiddles.push_back(h_twiddles.at(n - 1) * primitive_root);
      } while (h_twiddles.at(n++) != S::one());

      CHK_IF_RETURN(hipMallocAsync(&Domain<S>::twiddles, n * sizeof(S), ctx.stream));
      CHK_IF_RETURN(
        hipMemcpyAsync(Domain<S>::twiddles, &h_twiddles.front(), n * sizeof(S), hipMemcpyHostToDevice, ctx.stream));

      Domain<S>::max_size = n - 1;
      CHK_IF_RETURN(hipStreamSynchronize(ctx.stream));
    }

    return CHK_LAST();
  }

  template <typename S, typename E>
  hipError_t NTT(E* input, int size, NTTDir dir, NTTConfig<S>& config, E* output)
  {
    CHK_INIT_IF_RETURN();

    hipStream_t& stream = config.ctx.stream;
    int batch_size = config.batch_size;
    int logn = int(log(size) / log(2));
    int input_size_bytes = size * batch_size * sizeof(E);
    bool are_inputs_on_device = config.are_inputs_on_device; // TODO: unify name to is_
    bool are_outputs_on_device = config.are_outputs_on_device;

    S* coset = nullptr;
    int coset_index = 0;
    try {
      coset_index = Domain<S>::coset_index.at(config.coset_gen);
    } catch (...) {
      // if coset index is not found in the subgroup, compute coset powers on CPU and move them to device
      std::vector<S> h_coset;
      h_coset.push_back(S::one());
      S coset_gen = (dir == NTTDir::kInverse) ? S::inverse(config.coset_gen) : config.coset_gen;
      for (int i = 1; i < size; i++) {
        h_coset.push_back(h_coset.at(i - 1) * coset_gen);
      }
      CHK_IF_RETURN(hipMallocAsync(&coset, size * sizeof(S), stream));
      CHK_IF_RETURN(hipMemcpyAsync(coset, &h_coset.front(), size * sizeof(S), hipMemcpyHostToDevice, stream));
      h_coset.clear();
    }

    E* d_input;
    if (are_inputs_on_device) {
      d_input = input;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&d_input, input_size_bytes, stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_input, input, input_size_bytes, hipMemcpyHostToDevice, stream));
    }
    E* d_output;
    if (are_outputs_on_device) {
      d_output = output;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&d_output, input_size_bytes, stream));
    }

    bool ct_butterfly = true;
    bool reverse_input = false;
    switch (config.ordering) {
    case Ordering::kNN:
      reverse_input = true;
      break;
    case Ordering::kNR:
      ct_butterfly = false;
      break;
    case Ordering::kRR:
      reverse_input = true;
      ct_butterfly = false;
      break;
    }

    if (reverse_input) reverse_order_batch(d_input, size, logn, batch_size, stream, d_output);

    CHK_IF_RETURN(ntt_inplace_batch_template(
      reverse_input ? d_output : d_input, size, Domain<S>::twiddles, Domain<S>::max_size, batch_size, logn,
      dir == NTTDir::kInverse, ct_butterfly, coset, coset_index, stream, d_output));

    if (!are_outputs_on_device)
      CHK_IF_RETURN(hipMemcpyAsync(output, d_output, input_size_bytes, hipMemcpyDeviceToHost, stream));

    if (coset) CHK_IF_RETURN(hipFreeAsync(coset, stream));
    if (!are_inputs_on_device) CHK_IF_RETURN(hipFreeAsync(d_input, stream));
    if (!are_outputs_on_device) CHK_IF_RETURN(hipFreeAsync(d_output, stream));
    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));

    return CHK_LAST();
  }

  template <typename S>
  NTTConfig<S> DefaultNTTConfig()
  {
    device_context::DeviceContext ctx = device_context::get_default_device_context();
    NTTConfig<S> config = {
      ctx,           // ctx
      S::one(),      // coset_gen
      1,             // batch_size
      Ordering::kNN, // ordering
      false,         // are_inputs_on_device
      false,         // are_outputs_on_device
      false,         // is_async
    };
    return config;
  }

  /**
   * Extern "C" version of [DefaultNTTConfig](@ref DefaultNTTConfig) function with the following
   * value of template parameter (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   * @return Default [NTTConfig](@ref NTTConfig).
   */
  extern "C" NTTConfig<curve_config::scalar_t> GetDefaultNTTConfig()
  {
    return DefaultNTTConfig<curve_config::scalar_t>();
  }

  /**
   * Extern "C" version of [InitDomain](@ref InitDomain) function with the following
   * value of template parameter (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   */
  extern "C" hipError_t InitializeDomain(curve_config::scalar_t primitive_root, device_context::DeviceContext& ctx)
  {
    return InitDomain(primitive_root, ctx);
  }

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t NTTCuda(
    curve_config::scalar_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::scalar_t>(input, size, dir, config, output);
  }

#if defined(ECNTT_DEFINED)

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [projective representation](@ref projective_t) of the curve (i.e. EC NTT is computed);
   *  - `E` is the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t ECNTTCuda(
    curve_config::projective_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::projective_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::projective_t>(input, size, dir, config, output);
  }

#endif

} // namespace ntt
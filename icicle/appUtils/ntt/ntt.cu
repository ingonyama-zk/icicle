#include "hip/hip_runtime.h"
#include "ntt.cuh"

#include <unordered_map>
#include <vector>

#include "curves/curve_config.cuh"
#include "utils/sharedmem.cuh"
#include "utils/utils_kernels.cuh"
#include "utils/utils.h"
#include "appUtils/ntt/ntt_impl.cuh"

#include <mutex>

namespace ntt {

  namespace {

    const uint32_t MAX_NUM_THREADS = 512;   // TODO: hotfix - should be 1024, currently limits shared memory size
    const uint32_t MAX_THREADS_BATCH = 512; // TODO: allows 100% occupancy for scalar NTT for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM_ELEMENT_SIZE = 32; // TODO: occupancy calculator, hardcoded for sm_86..sm_89
    const uint32_t MAX_SHARED_MEM = MAX_SHARED_MEM_ELEMENT_SIZE * MAX_NUM_THREADS;

    template <typename E>
    __global__ void reverse_order_kernel(E* arr, E* arr_reversed, uint32_t n, uint32_t logn, uint32_t batch_size)
    {
      int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (threadId < n * batch_size) {
        int idx = threadId % n;
        int batch_idx = threadId / n;
        int idx_reversed = __brev(idx) >> (32 - logn);

        E val = arr[batch_idx * n + idx];
        if (arr == arr_reversed) { __syncthreads(); } // for in-place (when pointers arr==arr_reversed)
        arr_reversed[batch_idx * n + idx_reversed] = val;
      }
    }

    /**
     * Bit-reverses a batch of input arrays out-of-place inside GPU.
     * for example: on input array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr_in batch of arrays of some object of type T. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param batch_size the size of the batch.
     * @param arr_out buffer of the same size as `arr_in` on the GPU to write the bit-permuted array into.
     */
    template <typename E>
    void reverse_order_batch(E* arr_in, uint32_t n, uint32_t logn, uint32_t batch_size, hipStream_t stream, E* arr_out)
    {
      int number_of_threads = MAX_THREADS_BATCH;
      int number_of_blocks = (n * batch_size + number_of_threads - 1) / number_of_threads;
      reverse_order_kernel<<<number_of_blocks, number_of_threads, 0, stream>>>(arr_in, arr_out, n, logn, batch_size);
    }

    /**
     * Bit-reverses an input array out-of-place inside GPU.
     * for example: on array ([a[0],a[1],a[2],a[3]], 4, 2) it returns
     * [a[0],a[3],a[2],a[1]] (elements at indices 3 and 1 swhich places).
     * @param arr_in array of some object of type T of size which is a power of 2. Should be on GPU.
     * @param n length of `arr`.
     * @param logn log(n).
     * @param arr_out buffer of the same size as `arr_in` on the GPU to write the bit-permuted array into.
     */
    template <typename E>
    void reverse_order(E* arr_in, uint32_t n, uint32_t logn, hipStream_t stream, E* arr_out)
    {
      reverse_order_batch(arr_in, n, logn, 1, stream, arr_out);
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared_rev(
      E* __restrict__ arr_in,
      int n,
      const S* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int ss,
      int logn,
      E* __restrict__ arr_out)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; ss < logn; ss++) {
            int s = logn - ss - 1;
            bool is_beginning = ss == 0;
            bool is_end = ss == (logn - 1);

            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;

            S tw = *(r_twiddles + (int)(j * n_twiddles_div));

            E u = is_beginning ? arr_in[offset + oij] : arr[oij];
            E v = is_beginning ? arr_in[offset + k] : arr[k];
            if (is_end) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = tw * (u - v);
            } else {
              arr[oij] = u + v;
              arr[k] = tw * (u - v);
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tuckey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr_in input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     * @param arr_out buffer for the output.
     */
    template <typename E, typename S>
    __global__ void ntt_template_kernel_shared(
      E* __restrict__ arr_in,
      int n,
      const S* __restrict__ r_twiddles,
      int n_twiddles,
      int max_task,
      int s,
      int logn,
      E* __restrict__ arr_out)
    {
      SharedMemory<E> smem;
      E* arr = smem.getPointer();

      uint32_t task = blockIdx.x;
      uint32_t loop_limit = blockDim.x;
      uint32_t chunks = n / (loop_limit * 2);
      uint32_t offset = (task / chunks) * n;
      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;

        if (l < loop_limit) {
#pragma unroll
          for (; s < logn; s++) // TODO: this loop also can be unrolled
          {
            uint32_t ntw_i = task % chunks;

            uint32_t n_twiddles_div = n_twiddles >> (s + 1);

            uint32_t shift_s = 1 << s;
            uint32_t shift2_s = 1 << (s + 1);

            l = ntw_i * loop_limit + l; // to l from chunks to full

            uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
            uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
            uint32_t oij = i + j;
            uint32_t k = oij + shift_s;
            S tw = *(r_twiddles + (int)(j * n_twiddles_div));

            E u = s == 0 ? arr_in[offset + oij] : arr[oij];
            E v = s == 0 ? arr_in[offset + k] : arr[k];
            v = tw * v;
            if (s == (logn - 1)) {
              arr_out[offset + oij] = u + v;
              arr_out[offset + k] = u - v;
            } else {
              arr[oij] = u + v;
              arr[k] = u - v;
            }

            __syncthreads();
          }
        }
      }
    }

    /**
     * Cooley-Tukey NTT.
     * NOTE! this function assumes that d_twiddles are located in the device memory.
     * @param arr input array of type E (elements).
     * @param n length of d_arr.
     * @param twiddles twiddle factors of type S (scalars) array allocated on the device memory (must be a power of 2).
     * @param n_twiddles length of twiddles, should be negative for intt.
     * @param max_task max count of parallel tasks.
     * @param s log2(n) loop index.
     */
    template <typename E, typename S>
    __global__ void
    ntt_template_kernel(E* arr_in, int n, S* twiddles, int n_twiddles, int max_task, int s, bool rev, E* arr_out)
    {
      int task = blockIdx.x;
      int chunks = n / (blockDim.x * 2);

      if (task < max_task) {
        // flattened loop allows parallel processing
        uint32_t l = threadIdx.x;
        uint32_t loop_limit = blockDim.x;

        if (l < loop_limit) {
          uint32_t ntw_i = task % chunks;

          uint32_t shift_s = 1 << s;
          uint32_t shift2_s = 1 << (s + 1);
          uint32_t n_twiddles_div = n_twiddles >> (s + 1);

          l = ntw_i * blockDim.x + l; // to l from chunks to full

          uint32_t j = l & (shift_s - 1);               // Equivalent to: l % (1 << s)
          uint32_t i = ((l >> s) * shift2_s) & (n - 1); // (..) % n (assuming n is power of 2)
          uint32_t k = i + j + shift_s;

          S tw = *(twiddles + (int)(j * n_twiddles_div));

          uint32_t offset = (task / chunks) * n;
          E u = arr_in[offset + i + j];
          E v = arr_in[offset + k];
          if (!rev) v = tw * v;
          arr_out[offset + i + j] = u + v;
          v = u - v;
          arr_out[offset + k] = rev ? tw * v : v;
        }
      }
    }

    /**
     * NTT/INTT inplace batch
     * Note: this function does not perform any bit-reverse permutations on its inputs or outputs.
     * @param d_input Input array
     * @param n Size of `d_input`
     * @param d_twiddles Twiddles
     * @param n_twiddles Size of `d_twiddles`
     * @param batch_size The size of the batch; the length of `d_inout` is `n` * `batch_size`.
     * @param inverse true for iNTT
     * @param coset should be array of length n or a nullptr if NTT is not computed on a coset
     * @param stream CUDA stream
     * @param is_async if false, perform sync of the supplied CUDA stream at the end of processing
     * @param d_output Output array
     */
    template <typename E, typename S>
    hipError_t ntt_inplace_batch_template(
      E* d_input,
      int n,
      S* d_twiddles,
      int n_twiddles,
      int batch_size,
      int logn,
      bool inverse,
      bool dit,
      S* arbitrary_coset,
      int coset_gen_index,
      hipStream_t stream,
      E* d_output)
    {
      CHK_INIT_IF_RETURN();

      bool is_shared_mem_enabled = sizeof(E) <= MAX_SHARED_MEM_ELEMENT_SIZE;
      const int log2_shmem_elems = is_shared_mem_enabled ? int(log(int(MAX_SHARED_MEM / sizeof(E))) / log(2)) : logn;
      int num_threads = max(min(min(n / 2, MAX_THREADS_BATCH), 1 << (log2_shmem_elems - 1)), 1);
      const int chunks = max(int((n / 2) / num_threads), 1);
      const int total_tasks = batch_size * chunks;
      int num_blocks = total_tasks;
      const int shared_mem = 2 * num_threads * sizeof(E); // TODO: calculator, as shared mem size may be more efficient
                                                          // less then max to allow more concurrent blocks on SM
      const int logn_shmem = is_shared_mem_enabled ? int(log(2 * num_threads) / log(2))
                                                   : 0; // TODO: shared memory support only for types <= 32 bytes
      int num_threads_coset = max(min(n / 2, MAX_NUM_THREADS), 1);
      int num_blocks_coset = (n * batch_size + num_threads_coset - 1) / num_threads_coset;

      if (inverse) {
        d_twiddles = d_twiddles + n_twiddles;
        n_twiddles = -n_twiddles;
      }

      bool is_on_coset = (coset_gen_index != 0) || arbitrary_coset;
      bool direct_coset = (!inverse && is_on_coset);
      if (direct_coset)
        utils_internal::BatchMulKernel<E, S><<<num_blocks_coset, num_threads_coset, 0, stream>>>(
          d_input, n, batch_size, arbitrary_coset ? arbitrary_coset : d_twiddles, arbitrary_coset ? 1 : coset_gen_index,
          n_twiddles, logn, dit, d_output);

      if (dit) {
        if (is_shared_mem_enabled)
          ntt_template_kernel_shared<<<num_blocks, num_threads, shared_mem, stream>>>(
            direct_coset ? d_output : d_input, 1 << logn_shmem, d_twiddles, n_twiddles, total_tasks, 0, logn_shmem,
            d_output);

        for (int s = logn_shmem; s < logn; s++) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<E, S><<<num_blocks, num_threads, 0, stream>>>(
            (direct_coset || (s > 0)) ? d_output : d_input, n, d_twiddles, n_twiddles, total_tasks, s, false, d_output);
        }
      } else {
        for (int s = logn - 1; s >= logn_shmem; s--) // TODO: this loop also can be unrolled
        {
          ntt_template_kernel<<<num_blocks, num_threads, 0, stream>>>(
            (direct_coset || (s < logn - 1)) ? d_output : d_input, n, d_twiddles, n_twiddles, total_tasks, s, true,
            d_output);
        }

        if (is_shared_mem_enabled)
          ntt_template_kernel_shared_rev<<<num_blocks, num_threads, shared_mem, stream>>>(
            (direct_coset || (logn > logn_shmem)) ? d_output : d_input, 1 << logn_shmem, d_twiddles, n_twiddles,
            total_tasks, 0, logn_shmem, d_output);
      }

      if (inverse) {
        if (is_on_coset)
          utils_internal::BatchMulKernel<E, S><<<num_blocks_coset, num_threads_coset, 0, stream>>>(
            d_output, n, batch_size, arbitrary_coset ? arbitrary_coset : d_twiddles,
            arbitrary_coset ? 1 : -coset_gen_index, -n_twiddles, logn, !dit, d_output);

        utils_internal::NormalizeKernel<E, S>
          <<<num_blocks_coset, num_threads_coset, 0, stream>>>(d_output, S::inv_log_size(logn), n * batch_size);
      }

      return CHK_LAST();
    }

  } // namespace

  /**
   * @struct Domain
   * Struct containing information about the domain on which (i)NTT is evaluated i.e. twiddle factors.
   * Twiddle factors are private, static and can only be set using [InitDomain](@ref InitDomain) function.
   * The internal representation of twiddles is prone to change in accordance with changing [NTT](@ref NTT) algorithm.
   * @tparam S The type of twiddle factors \f$ \{ \omega^i \} \f$. Must be a field.
   */
  template <typename S>
  class Domain
  {
    // Mutex for protecting access to the domain/device container array
    static inline std::mutex device_domain_mutex;
    // The domain-per-device container - assumption is InitDomain is called once per device per program.

    int max_size = 0;
    int max_log_size = 0;
    S* twiddles = nullptr;
    bool initialized = false; // protection for multi-threaded case
    std::unordered_map<S, int> coset_index = {};

    S* internal_twiddles = nullptr; // required by mixed-radix NTT
    S* basic_twiddles = nullptr;    // required by mixed-radix NTT

    // mixed-radix NTT supports a fast-twiddle option at the cost of additional 4N memory (where N is max NTT size)
    S* fast_external_twiddles = nullptr;     // required by mixed-radix NTT (fast-twiddles mode)
    S* fast_internal_twiddles = nullptr;     // required by mixed-radix NTT (fast-twiddles mode)
    S* fast_basic_twiddles = nullptr;        // required by mixed-radix NTT (fast-twiddles mode)
    S* fast_external_twiddles_inv = nullptr; // required by mixed-radix NTT (fast-twiddles mode)
    S* fast_internal_twiddles_inv = nullptr; // required by mixed-radix NTT (fast-twiddles mode)
    S* fast_basic_twiddles_inv = nullptr;    // required by mixed-radix NTT (fast-twiddles mode)

  public:
    template <typename U>
    friend hipError_t InitDomain<U>(U primitive_root, device_context::DeviceContext& ctx, bool fast_tw);

    hipError_t ReleaseDomain(device_context::DeviceContext& ctx);

    template <typename U, typename E>
    friend hipError_t NTT<U, E>(E* input, int size, NTTDir dir, NTTConfig<U>& config, E* output);
  };

  template <typename S>
  static inline Domain<S> domains_for_devices[device_context::MAX_DEVICES] = {};

  template <typename S>
  hipError_t InitDomain(S primitive_root, device_context::DeviceContext& ctx, bool fast_twiddles_mode)
  {
    CHK_INIT_IF_RETURN();

    Domain<S>& domain = domains_for_devices<S>[ctx.device_id];

    // only generate twiddles if they haven't been generated yet
    // please note that this offers just basic thread-safety,
    // it's assumed a singleton (non-enforced) that is supposed
    // to be initialized once per device per program lifetime
    if (!domain.initialized) {
      // Mutex is automatically released when lock goes out of scope, even in case of exceptions
      std::lock_guard<std::mutex> lock(Domain<S>::device_domain_mutex);
      // double check locking
      if (domain.initialized) return CHK_LAST(); // another thread is already initializing the domain

      bool found_logn = false;
      S omega = primitive_root;
      unsigned omegas_count = S::get_omegas_count();
      for (int i = 0; i < omegas_count; i++) {
        omega = S::sqr(omega);
        if (!found_logn) {
          ++domain.max_log_size;
          found_logn = omega == S::one();
          if (found_logn) break;
        }
      }

      domain.max_size = (int)pow(2, domain.max_log_size);
      if (omega != S::one()) {
        THROW_ICICLE_ERR(
          IcicleError_t::InvalidArgument, "Primitive root provided to the InitDomain function is not in the subgroup");
      }

      // allocate and calculate twiddles on GPU
      // Note: radix-2 INTT needs ONE in last element (in addition to first element), therefore have n+1 elements
      // Managed allocation allows host to read the elements (logn) without copying all (n) TFs back to host
      CHK_IF_RETURN(hipMallocManaged(&domain.twiddles, (domain.max_size + 1) * sizeof(S)));
      CHK_IF_RETURN(generate_external_twiddles_generic(
        primitive_root, domain.twiddles, domain.internal_twiddles, domain.basic_twiddles, domain.max_log_size,
        ctx.stream));

      if (fast_twiddles_mode) {
        // generating fast-twiddles (note that this cost 4N additional memory)
        CHK_IF_RETURN(hipMallocAsync(&domain.fast_external_twiddles, domain.max_size * sizeof(S) * 2, ctx.stream));
        CHK_IF_RETURN(hipMallocAsync(&domain.fast_external_twiddles_inv, domain.max_size * sizeof(S) * 2, ctx.stream));

        // fast-twiddles forward NTT
        CHK_IF_RETURN(generate_external_twiddles_fast_twiddles_mode(
          primitive_root, domain.fast_external_twiddles, domain.fast_internal_twiddles, domain.fast_basic_twiddles,
          domain.max_log_size, ctx.stream));

        // fast-twiddles inverse NTT
        S primitive_root_inv;
        CHK_IF_RETURN(hipMemcpyAsync(
          &primitive_root_inv, &domain.twiddles[domain.max_size - 1], sizeof(S), hipMemcpyDeviceToHost, ctx.stream));
        CHK_IF_RETURN(generate_external_twiddles_fast_twiddles_mode(
          primitive_root_inv, domain.fast_external_twiddles_inv, domain.fast_internal_twiddles_inv,
          domain.fast_basic_twiddles_inv, domain.max_log_size, ctx.stream));
      }
      CHK_IF_RETURN(hipStreamSynchronize(ctx.stream));

      const bool is_map_only_powers_of_primitive_root = true;
      if (is_map_only_powers_of_primitive_root) {
        // populate the coset_index map. Note that only powers of the primitive-root are stored (1, PR, PR^2, PR^4, PR^8
        // etc.)
        domain.coset_index[S::one()] = 0;
        for (int i = 0; i < domain.max_log_size; ++i) {
          const int index = (int)pow(2, i);
          domain.coset_index[domain.twiddles[index]] = index;
        }
      } else {
        // populate all values
        for (int i = 0; i < domain.max_size; ++i) {
          domain.coset_index[domain.twiddles[i]] = i;
        }
      }
      domain.initialized = true;
    }

    return CHK_LAST();
  }

  template <typename S>
  hipError_t Domain<S>::ReleaseDomain(device_context::DeviceContext& ctx)
  {
    CHK_INIT_IF_RETURN();

    max_size = 0;
    max_log_size = 0;
    hipFreeAsync(twiddles, ctx.stream);
    twiddles = nullptr;
    hipFreeAsync(internal_twiddles, ctx.stream);
    internal_twiddles = nullptr;
    hipFreeAsync(basic_twiddles, ctx.stream);
    basic_twiddles = nullptr;
    coset_index.clear();

    hipFreeAsync(fast_external_twiddles, ctx.stream);
    fast_external_twiddles = nullptr;
    hipFreeAsync(fast_internal_twiddles, ctx.stream);
    fast_internal_twiddles = nullptr;
    hipFreeAsync(fast_basic_twiddles, ctx.stream);
    fast_basic_twiddles = nullptr;
    hipFreeAsync(fast_external_twiddles_inv, ctx.stream);
    fast_external_twiddles_inv = nullptr;
    hipFreeAsync(fast_internal_twiddles_inv, ctx.stream);
    fast_internal_twiddles_inv = nullptr;
    hipFreeAsync(fast_basic_twiddles_inv, ctx.stream);
    fast_basic_twiddles_inv = nullptr;

    return CHK_LAST();
  }

  template <typename S>
  static bool is_choose_radix2_algorithm(int logn, int batch_size, const NTTConfig<S>& config)
  {
    const bool is_mixed_radix_alg_supported = (logn > 3 && logn != 7);
    const bool is_user_selected_radix2_alg = config.ntt_algorithm == NttAlgorithm::Radix2;
    const bool is_force_radix2 = !is_mixed_radix_alg_supported || is_user_selected_radix2_alg;
    if (is_force_radix2) return true;

    const bool is_user_selected_mixed_radix_alg = config.ntt_algorithm == NttAlgorithm::MixedRadix;
    if (is_user_selected_mixed_radix_alg) return false;

    // Heuristic to automatically select an algorithm
    // Note that generally the decision depends on {logn, batch, ordering, inverse, coset, in-place, coeff-field} and
    // the specific GPU.
    // the following heuristic is a simplification based on measurements. Users can try both and select the algorithm
    // based on the specific case via the 'NTTConfig.ntt_algorithm' field

    if (logn >= 16) return false; // mixed-radix is typically faster in those cases
    if (logn <= 11) return true;  //  radix-2 is typically faster for batch<=256 in those cases
    const int log_batch = (int)log2(batch_size);
    return (logn + log_batch <= 18); // almost the cutoff point where both are equal
  }

  template <typename S, typename E>
  hipError_t radix2_ntt(
    E* d_input,
    E* d_output,
    S* twiddles,
    int ntt_size,
    int max_size,
    int batch_size,
    bool is_inverse,
    Ordering ordering,
    S* arbitrary_coset,
    int coset_gen_index,
    hipStream_t hip_stream)
  {
    CHK_INIT_IF_RETURN();

    const int logn = int(log2(ntt_size));

    bool dit = true;
    bool reverse_input = false;
    switch (ordering) {
    case Ordering::kNN:
      reverse_input = true;
      break;
    case Ordering::kNR:
    case Ordering::kNM:
      dit = false;
      break;
    case Ordering::kRR:
      reverse_input = true;
      dit = false;
      break;
    case Ordering::kRN:
    case Ordering::kMN:
      dit = true;
      reverse_input = false;
    }

    if (reverse_input) reverse_order_batch(d_input, ntt_size, logn, batch_size, hip_stream, d_output);

    CHK_IF_RETURN(ntt_inplace_batch_template(
      reverse_input ? d_output : d_input, ntt_size, twiddles, max_size, batch_size, logn, is_inverse, dit,
      arbitrary_coset, coset_gen_index, hip_stream, d_output));

    return CHK_LAST();
  }

  template <typename S, typename E>
  hipError_t NTT(E* input, int size, NTTDir dir, NTTConfig<S>& config, E* output)
  {
    CHK_INIT_IF_RETURN();

    Domain<S>& domain = domains_for_devices<S>[config.ctx.device_id];

    if (size > domain.max_size) {
      std::ostringstream oss;
      oss << "NTT size=" << size
          << " is too large for the domain. Consider generating your domain with a higher order root of unity.\n";
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, oss.str().c_str());
    }

    int logn = int(log2(size));
    const bool is_size_power_of_two = size == (1 << logn);
    if (!is_size_power_of_two) {
      std::ostringstream oss;
      oss << "NTT size=" << size << " is not supported since it is not a power of two.\n";
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, oss.str().c_str());
    }

    hipStream_t& stream = config.ctx.stream;
    size_t batch_size = config.batch_size;
    size_t input_size_bytes = (size_t)size * batch_size * sizeof(E);
    bool are_inputs_on_device = config.are_inputs_on_device;
    bool are_outputs_on_device = config.are_outputs_on_device;

    E* d_input;
    if (are_inputs_on_device) {
      d_input = input;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&d_input, input_size_bytes, stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_input, input, input_size_bytes, hipMemcpyHostToDevice, stream));
    }
    E* d_output;
    if (are_outputs_on_device) {
      d_output = output;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&d_output, input_size_bytes, stream));
    }

    S* coset = nullptr;
    int coset_index = 0;
    try {
      coset_index = domain.coset_index.at(config.coset_gen);
    } catch (...) {
      // if coset index is not found in the subgroup, compute coset powers on CPU and move them to device
      std::vector<S> h_coset;
      h_coset.push_back(S::one());
      S coset_gen = (dir == NTTDir::kInverse) ? S::inverse(config.coset_gen) : config.coset_gen;
      for (int i = 1; i < size; i++) {
        h_coset.push_back(h_coset.at(i - 1) * coset_gen);
      }
      CHK_IF_RETURN(hipMallocAsync(&coset, size * sizeof(S), stream));
      CHK_IF_RETURN(hipMemcpyAsync(coset, &h_coset.front(), size * sizeof(S), hipMemcpyHostToDevice, stream));
      h_coset.clear();
    }

    const bool is_radix2_algorithm = is_choose_radix2_algorithm(logn, batch_size, config);
    const bool is_inverse = dir == NTTDir::kInverse;

    if (is_radix2_algorithm) {
      CHK_IF_RETURN(ntt::radix2_ntt(
        d_input, d_output, domain.twiddles, size, domain.max_size, batch_size, is_inverse, config.ordering, coset,
        coset_index, stream));
    } else {
      const bool is_on_coset = (coset_index != 0) || coset;
      const bool is_fast_twiddles_enabled = (domain.fast_external_twiddles != nullptr) && !is_on_coset;
      S* twiddles = is_fast_twiddles_enabled
                      ? (is_inverse ? domain.fast_external_twiddles_inv : domain.fast_external_twiddles)
                      : domain.twiddles;
      S* internal_twiddles = is_fast_twiddles_enabled
                               ? (is_inverse ? domain.fast_internal_twiddles_inv : domain.fast_internal_twiddles)
                               : domain.internal_twiddles;
      S* basic_twiddles = is_fast_twiddles_enabled
                            ? (is_inverse ? domain.fast_basic_twiddles_inv : domain.fast_basic_twiddles)
                            : domain.basic_twiddles;

      CHK_IF_RETURN(ntt::mixed_radix_ntt(
        d_input, d_output, twiddles, internal_twiddles, basic_twiddles, size, domain.max_log_size, batch_size,
        is_inverse, is_fast_twiddles_enabled, config.ordering, coset, coset_index, stream));
    }

    if (!are_outputs_on_device)
      CHK_IF_RETURN(hipMemcpyAsync(output, d_output, input_size_bytes, hipMemcpyDeviceToHost, stream));

    if (coset) CHK_IF_RETURN(hipFreeAsync(coset, stream));
    if (!are_inputs_on_device) CHK_IF_RETURN(hipFreeAsync(d_input, stream));
    if (!are_outputs_on_device) CHK_IF_RETURN(hipFreeAsync(d_output, stream));
    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));

    return CHK_LAST();
  }

  template <typename S>
  NTTConfig<S> DefaultNTTConfig()
  {
    device_context::DeviceContext ctx = device_context::get_default_device_context();
    NTTConfig<S> config = {
      ctx,                // ctx
      S::one(),           // coset_gen
      1,                  // batch_size
      Ordering::kNN,      // ordering
      false,              // are_inputs_on_device
      false,              // are_outputs_on_device
      false,              // is_async
      NttAlgorithm::Auto, // ntt_algorithm
    };
    return config;
  }

  /**
   * Extern "C" version of [InitDomain](@ref InitDomain) function with the following
   * value of template parameter (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, InitializeDomain)(
    curve_config::scalar_t* primitive_root, device_context::DeviceContext& ctx, bool fast_twiddles_mode)
  {
    return InitDomain(*primitive_root, ctx, fast_twiddles_mode);
  }

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, NTTCuda)(
    curve_config::scalar_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::scalar_t>(input, size, dir, config, output);
  }

#if defined(ECNTT_DEFINED)

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [projective representation](@ref projective_t) of the curve (i.e. EC NTT is computed);
   *  - `E` is the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(CURVE, ECNTTCuda)(
    curve_config::projective_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::projective_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::projective_t>(input, size, dir, config, output);
  }

#endif

} // namespace ntt
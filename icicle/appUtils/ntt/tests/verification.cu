
#define CURVE_ID BLS12_381

#include "primitives/field.cuh"
#include "primitives/projective.cuh"
#include <chrono>
#include <iostream>
#include <vector>

#include "curves/curve_config.cuh"
#include "ntt/ntt.cu"
#include "ntt/ntt_impl.cuh"
#include <memory>

#ifdef ECNTT_DEFINED
typedef curve_config::scalar_t test_scalar;
typedef curve_config::projective_t test_data;
#else
typedef curve_config::scalar_t test_scalar;
typedef curve_config::scalar_t test_data;
#endif

#include "kernel_ntt.cu"

void random_samples(test_data* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i < 1000 ? test_data::rand_host() : res[i - 1000];
}

void incremental_values(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++) {
    res[i] = i ? res[i - 1] + test_scalar::one() : test_scalar::zero();
  }
}

int main(int argc, char** argv)
{
  hipEvent_t icicle_start, icicle_stop, new_start, new_stop;
  float icicle_time, new_time;

  int NTT_LOG_SIZE = (argc > 1) ? atoi(argv[1]) : 19;
  int NTT_SIZE = 1 << NTT_LOG_SIZE;
  bool INPLACE = (argc > 2) ? atoi(argv[2]) : false;
  int INV = (argc > 3) ? atoi(argv[3]) : true;
  int BATCH_SIZE = (argc > 4) ? atoi(argv[4]) : 1;
  int COSET_IDX = (argc > 5) ? atoi(argv[5]) : 0;
  const ntt::Ordering ordering = (argc > 6) ? ntt::Ordering(atoi(argv[6])) : ntt::Ordering::kNN;
  bool FAST_TW = (argc > 7) ? atoi(argv[7]) : true;

  // Note: NM, MN are not expected to be equal when comparing mixed-radix and radix-2 NTTs
  const char* ordering_str = ordering == ntt::Ordering::kNN   ? "NN"
                             : ordering == ntt::Ordering::kNR ? "NR"
                             : ordering == ntt::Ordering::kRN ? "RN"
                             : ordering == ntt::Ordering::kRR ? "RR"
                             : ordering == ntt::Ordering::kNM ? "NM"
                                                              : "MN";

  printf(
    "running ntt 2^%d, inplace=%d, inverse=%d, batch_size=%d, coset-idx=%d, ordering=%s, fast_tw=%d\n", NTT_LOG_SIZE,
    INPLACE, INV, BATCH_SIZE, COSET_IDX, ordering_str, FAST_TW);

  CHK_IF_RETURN(hipFree(nullptr)); // init GPU context (warmup)

  // init domain
  auto ntt_config = ntt::DefaultNTTConfig<test_scalar>();
  ntt_config.ordering = ordering;
  ntt_config.are_inputs_on_device = true;
  ntt_config.are_outputs_on_device = true;
  ntt_config.batch_size = BATCH_SIZE;

  CHK_IF_RETURN(hipEventCreate(&icicle_start));
  CHK_IF_RETURN(hipEventCreate(&icicle_stop));
  CHK_IF_RETURN(hipEventCreate(&new_start));
  CHK_IF_RETURN(hipEventCreate(&new_stop));

  auto start = std::chrono::high_resolution_clock::now();
  const test_scalar basic_root = test_scalar::omega(NTT_LOG_SIZE);
  ntt::InitDomain(basic_root, ntt_config.ctx, FAST_TW);
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
  std::cout << "initDomain took: " << duration / 1000 << " MS" << std::endl;

  // cpu allocation
  auto CpuScalars = std::make_unique<test_data[]>(NTT_SIZE * BATCH_SIZE);
  auto CpuOutputOld = std::make_unique<test_data[]>(NTT_SIZE * BATCH_SIZE);
  auto CpuOutputNew = std::make_unique<test_data[]>(NTT_SIZE * BATCH_SIZE);

  // gpu allocation
  test_data *GpuScalars, *GpuOutputOld, *GpuOutputNew;
  CHK_IF_RETURN(hipMalloc(&GpuScalars, sizeof(test_data) * NTT_SIZE * BATCH_SIZE));
  CHK_IF_RETURN(hipMalloc(&GpuOutputOld, sizeof(test_data) * NTT_SIZE * BATCH_SIZE));
  CHK_IF_RETURN(hipMalloc(&GpuOutputNew, sizeof(test_data) * NTT_SIZE * BATCH_SIZE));

  // init inputs
  // incremental_values(CpuScalars.get(), NTT_SIZE * BATCH_SIZE);
  random_samples(CpuScalars.get(), NTT_SIZE * BATCH_SIZE);
  CHK_IF_RETURN(
    hipMemcpy(GpuScalars, CpuScalars.get(), NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyHostToDevice));

  // inplace
  if (INPLACE) {
    CHK_IF_RETURN(
      hipMemcpy(GpuOutputNew, GpuScalars, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToDevice));
  }

  for (int coset_idx = 0; coset_idx < COSET_IDX; ++coset_idx) {
    ntt_config.coset_gen = ntt_config.coset_gen * basic_root;
  }

  auto benchmark = [&](bool is_print, int iterations) -> hipError_t {
    // NEW
    CHK_IF_RETURN(hipEventRecord(new_start, ntt_config.ctx.stream));
    ntt_config.ntt_algorithm = ntt::NttAlgorithm::MixedRadix;
    for (size_t i = 0; i < iterations; i++) {
      CHK_IF_RETURN(ntt::NTT(
        INPLACE ? GpuOutputNew : GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config,
        GpuOutputNew));
    }
    CHK_IF_RETURN(hipEventRecord(new_stop, ntt_config.ctx.stream));
    CHK_IF_RETURN(hipStreamSynchronize(ntt_config.ctx.stream));
    CHK_IF_RETURN(hipEventElapsedTime(&new_time, new_start, new_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    // OLD
    CHK_IF_RETURN(hipEventRecord(icicle_start, ntt_config.ctx.stream));
    ntt_config.ntt_algorithm = ntt::NttAlgorithm::Radix2;
    for (size_t i = 0; i < iterations; i++) {
      CHK_IF_RETURN(
        ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld));
    }
    CHK_IF_RETURN(hipEventRecord(icicle_stop, ntt_config.ctx.stream));
    CHK_IF_RETURN(hipStreamSynchronize(ntt_config.ctx.stream));
    CHK_IF_RETURN(hipEventElapsedTime(&icicle_time, icicle_start, icicle_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    if (is_print) {
      printf("Old Runtime=%0.3f MS\n", icicle_time / iterations);
      printf("New Runtime=%0.3f MS\n", new_time / iterations);
    }

    return CHK_LAST();
  };

  CHK_IF_RETURN(benchmark(false /*=print*/, 1)); // warmup
  int count = INPLACE ? 1 : 10;
  if (INPLACE) {
    CHK_IF_RETURN(
      hipMemcpy(GpuOutputNew, GpuScalars, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToDevice));
  }
  CHK_IF_RETURN(benchmark(true /*=print*/, count));

  // verify
  CHK_IF_RETURN(
    hipMemcpy(CpuOutputNew.get(), GpuOutputNew, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToHost));
  CHK_IF_RETURN(
    hipMemcpy(CpuOutputOld.get(), GpuOutputOld, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToHost));

  bool success = true;
  for (int i = 0; i < NTT_SIZE * BATCH_SIZE; i++) {
    if (CpuOutputNew[i] != CpuOutputOld[i]) {
      success = false;
      // std::cout << i << " ref " << CpuOutputOld[i] << " != " << CpuOutputNew[i] << std::endl;
      break;
    } else {
      // std::cout << i << " ref " << CpuOutputOld[i] << " == " << CpuOutputNew[i] << std::endl;
      // break;
    }
  }
  const char* success_str = success ? "SUCCESS!" : "FAIL!";
  printf("%s\n", success_str);

  CHK_IF_RETURN(hipFree(GpuScalars));
  CHK_IF_RETURN(hipFree(GpuOutputOld));
  CHK_IF_RETURN(hipFree(GpuOutputNew));

  return CHK_LAST();
}
#include "hip/hip_runtime.h"
#ifndef T_NTT
#define T_NTT
#pragma once

#include <stdio.h>
#include <stdint.h>
#include "curves/curve_config.cuh"

struct stage_metadata {
  uint32_t th_stride;
  uint32_t ntt_block_size;
  uint32_t ntt_block_id;
  uint32_t ntt_inp_id;
};

#define STAGE_SIZES_DATA                                                                                               \
  {                                                                                                                    \
    {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 0, 0, 0, 0}, {5, 0, 0, 0, 0},              \
      {6, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 4, 0, 0, 0}, {5, 4, 0, 0, 0}, {5, 5, 0, 0, 0}, {6, 5, 0, 0, 0},            \
      {6, 6, 0, 0, 0}, {4, 5, 4, 0, 0}, {4, 6, 4, 0, 0}, {5, 5, 5, 0, 0}, {6, 4, 6, 0, 0}, {6, 5, 6, 0, 0},            \
      {6, 6, 6, 0, 0}, {6, 5, 4, 4, 0}, {5, 5, 5, 5, 0}, {6, 5, 5, 5, 0}, {6, 5, 5, 6, 0}, {6, 6, 6, 5, 0},            \
      {6, 6, 6, 6, 0}, {5, 5, 5, 5, 5}, {6, 5, 4, 5, 6}, {6, 5, 5, 5, 6}, {6, 5, 6, 5, 6}, {6, 6, 5, 6, 6},            \
      {6, 6, 6, 6, 6},                                                                                                 \
  }
uint32_t constexpr STAGE_SIZES_HOST[31][5] = STAGE_SIZES_DATA;
__device__ constexpr uint32_t STAGE_SIZES_DEVICE[31][5] = STAGE_SIZES_DATA;

// construction for fast-twiddles
uint32_t constexpr STAGE_PREV_SIZES[31] = {0,  0,  0,  0,  0,  0,  0,  0,  4,  5,  5,  6,  6,  9,  9, 10,
                                           11, 11, 12, 15, 15, 16, 16, 18, 18, 20, 21, 21, 22, 23, 24};

#define STAGE_SIZES_DATA_FAST_TW                                                                                       \
  {                                                                                                                    \
    {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 0, 0, 0, 0}, {5, 0, 0, 0, 0},              \
      {6, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 4, 0, 0, 0}, {5, 4, 0, 0, 0}, {5, 5, 0, 0, 0}, {6, 5, 0, 0, 0},            \
      {6, 6, 0, 0, 0}, {5, 4, 4, 0, 0}, {5, 4, 5, 0, 0}, {5, 5, 5, 0, 0}, {6, 5, 5, 0, 0}, {6, 5, 6, 0, 0},            \
      {6, 6, 6, 0, 0}, {5, 5, 5, 4, 0}, {5, 5, 5, 5, 0}, {6, 5, 5, 5, 0}, {6, 5, 5, 6, 0}, {6, 6, 6, 5, 0},            \
      {6, 6, 6, 6, 0}, {5, 5, 5, 5, 5}, {6, 5, 5, 5, 5}, {6, 5, 5, 5, 6}, {6, 5, 5, 6, 6}, {6, 6, 6, 5, 6},            \
      {6, 6, 6, 6, 6},                                                                                                 \
  }
uint32_t constexpr STAGE_SIZES_HOST_FT[31][5] = STAGE_SIZES_DATA_FAST_TW;
__device__ uint32_t constexpr STAGE_SIZES_DEVICE_FT[31][5] = STAGE_SIZES_DATA_FAST_TW;

template <typename E, typename S>
class NTTEngine
{
public:
  E X[8];
  S WB[3];
  S WI[7];
  S WE[8];

  __device__ __forceinline__ void loadBasicTwiddles(S* basic_twiddles)
  {
#pragma unroll
    for (int i = 0; i < 3; i++) {
      WB[i] = basic_twiddles[i];
    }
  }

  __device__ __forceinline__ void loadBasicTwiddlesGeneric(S* basic_twiddles, bool inv)
  {
#pragma unroll
    for (int i = 0; i < 3; i++) {
      WB[i] = basic_twiddles[inv ? i + 3 : i];
    }
  }

  __device__ __forceinline__ void loadInternalTwiddles64(S* data, bool stride)
  {
#pragma unroll
    for (int i = 0; i < 7; i++) {
      WI[i] = data[((stride ? (threadIdx.x >> 3) : (threadIdx.x)) & 0x7) * (i + 1)];
    }
  }

  __device__ __forceinline__ void loadInternalTwiddles32(S* data, bool stride)
  {
#pragma unroll
    for (int i = 0; i < 7; i++) {
      WI[i] = data[2 * ((stride ? (threadIdx.x >> 4) : (threadIdx.x)) & 0x3) * (i + 1)];
    }
  }

  __device__ __forceinline__ void loadInternalTwiddles16(S* data, bool stride)
  {
#pragma unroll
    for (int i = 0; i < 7; i++) {
      WI[i] = data[4 * ((stride ? (threadIdx.x >> 5) : (threadIdx.x)) & 0x1) * (i + 1)];
    }
  }

  __device__ __forceinline__ void loadInternalTwiddlesGeneric64(S* data, bool stride, bool inv)
  {
#pragma unroll
    for (int i = 0; i < 7; i++) {
      uint32_t exp = ((stride ? (threadIdx.x >> 3) : (threadIdx.x)) & 0x7) * (i + 1);
      WI[i] = data[(inv && exp) ? 64 - exp : exp]; // if exp = 0 we also take exp and not 64-exp
    }
  }

  __device__ __forceinline__ void loadInternalTwiddlesGeneric32(S* data, bool stride, bool inv)
  {
#pragma unroll
    for (int i = 0; i < 7; i++) {
      uint32_t exp = 2 * ((stride ? (threadIdx.x >> 4) : (threadIdx.x)) & 0x3) * (i + 1);
      WI[i] = data[(inv && exp) ? 64 - exp : exp];
    }
  }

  __device__ __forceinline__ void loadInternalTwiddlesGeneric16(S* data, bool stride, bool inv)
  {
#pragma unroll
    for (int i = 0; i < 7; i++) {
      uint32_t exp = 4 * ((stride ? (threadIdx.x >> 5) : (threadIdx.x)) & 0x1) * (i + 1);
      WI[i] = data[(inv && exp) ? 64 - exp : exp];
    }
  }

  __device__ __forceinline__ void
  loadExternalTwiddles64(S* data, uint32_t tw_order, uint32_t tw_log_order, bool strided, stage_metadata s_meta)
  {
    data += tw_order * s_meta.ntt_inp_id + (s_meta.ntt_block_id & (tw_order - 1));

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      WE[i] = data[8 * i * tw_order + (1 << tw_log_order + 6) - 1];
    }
  }

  __device__ __forceinline__ void
  loadExternalTwiddles32(S* data, uint32_t tw_order, uint32_t tw_log_order, bool strided, stage_metadata s_meta)
  {
    data += tw_order * s_meta.ntt_inp_id * 2 + (s_meta.ntt_block_id & (tw_order - 1));

#pragma unroll
    for (uint32_t j = 0; j < 2; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 4; i++) {
        WE[4 * j + i] = data[(8 * i + j) * tw_order + (1 << tw_log_order + 5) - 1];
      }
    }
  }

  __device__ __forceinline__ void
  loadExternalTwiddles16(S* data, uint32_t tw_order, uint32_t tw_log_order, bool strided, stage_metadata s_meta)
  {
    data += tw_order * s_meta.ntt_inp_id * 4 + (s_meta.ntt_block_id & (tw_order - 1));

#pragma unroll
    for (uint32_t j = 0; j < 4; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 2; i++) {
        WE[2 * j + i] = data[(8 * i + j) * tw_order + (1 << tw_log_order + 4) - 1];
      }
    }
  }

  __device__ __forceinline__ void loadExternalTwiddlesGeneric64(
    S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta, uint32_t tw_log_size, bool inv)
  {
#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      uint32_t exp = (s_meta.ntt_inp_id + 8 * i) * (s_meta.ntt_block_id & (tw_order - 1))
                     << (tw_log_size - tw_log_order - 6);
      WE[i] = data[(inv && exp) ? ((1 << tw_log_size) - exp) : exp];
    }
  }

  __device__ __forceinline__ void loadExternalTwiddlesGeneric32(
    S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta, uint32_t tw_log_size, bool inv)
  {
#pragma unroll
    for (uint32_t j = 0; j < 2; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 4; i++) {
        uint32_t exp = (s_meta.ntt_inp_id * 2 + 8 * i + j) * (s_meta.ntt_block_id & (tw_order - 1))
                       << (tw_log_size - tw_log_order - 5);
        WE[4 * j + i] = data[(inv && exp) ? ((1 << tw_log_size) - exp) : exp];
      }
    }
  }

  __device__ __forceinline__ void loadExternalTwiddlesGeneric16(
    S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta, uint32_t tw_log_size, bool inv)
  {
#pragma unroll
    for (uint32_t j = 0; j < 4; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 2; i++) {
        uint32_t exp = (s_meta.ntt_inp_id * 4 + 8 * i + j) * (s_meta.ntt_block_id & (tw_order - 1))
                       << (tw_log_size - tw_log_order - 4);
        WE[2 * j + i] = data[(inv && exp) ? ((1 << tw_log_size) - exp) : exp];
      }
    }
  }

  __device__ __forceinline__ void loadGlobalData(
    E* data, uint32_t data_stride, uint32_t log_data_stride, uint32_t log_size, bool strided, stage_metadata s_meta)
  {
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride * s_meta.ntt_block_size;
    } else {
      data += s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      X[i] = data[s_meta.th_stride * i * data_stride];
    }
  }

  __device__ __forceinline__ void storeGlobalData(
    E* data, uint32_t data_stride, uint32_t log_data_stride, uint32_t log_size, bool strided, stage_metadata s_meta)
  {
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride * s_meta.ntt_block_size;
    } else {
      data += s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      data[s_meta.th_stride * i * data_stride] = X[i];
    }
  }

  __device__ __forceinline__ void loadGlobalData32(
    E* data, uint32_t data_stride, uint32_t log_data_stride, uint32_t log_size, bool strided, stage_metadata s_meta)
  {
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride * s_meta.ntt_inp_id * 2 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride * s_meta.ntt_block_size;
    } else {
      data += s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 2;
    }

#pragma unroll
    for (uint32_t j = 0; j < 2; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 4; i++) {
        X[4 * j + i] = data[(8 * i + j) * data_stride];
      }
    }
  }

  __device__ __forceinline__ void storeGlobalData32(
    E* data, uint32_t data_stride, uint32_t log_data_stride, uint32_t log_size, bool strided, stage_metadata s_meta)
  {
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride * s_meta.ntt_inp_id * 2 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride * s_meta.ntt_block_size;
    } else {
      data += s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 2;
    }

#pragma unroll
    for (uint32_t j = 0; j < 2; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 4; i++) {
        data[(8 * i + j) * data_stride] = X[4 * j + i];
      }
    }
  }

  __device__ __forceinline__ void loadGlobalData16(
    E* data, uint32_t data_stride, uint32_t log_data_stride, uint32_t log_size, bool strided, stage_metadata s_meta)
  {
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride * s_meta.ntt_inp_id * 4 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride * s_meta.ntt_block_size;
    } else {
      data += s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 4;
    }

#pragma unroll
    for (uint32_t j = 0; j < 4; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 2; i++) {
        X[2 * j + i] = data[(8 * i + j) * data_stride];
      }
    }
  }

  __device__ __forceinline__ void storeGlobalData16(
    E* data, uint32_t data_stride, uint32_t log_data_stride, uint32_t log_size, bool strided, stage_metadata s_meta)
  {
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride * s_meta.ntt_inp_id * 4 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride * s_meta.ntt_block_size;
    } else {
      data += s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 4;
    }

#pragma unroll
    for (uint32_t j = 0; j < 4; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 2; i++) {
        data[(8 * i + j) * data_stride] = X[2 * j + i];
      }
    }
  }

  __device__ __forceinline__ void ntt4_2()
  {
#pragma unroll
    for (int i = 0; i < 2; i++) {
      ntt4(X[4 * i], X[4 * i + 1], X[4 * i + 2], X[4 * i + 3]);
    }
  }

  __device__ __forceinline__ void ntt2_4()
  {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      ntt2(X[2 * i], X[2 * i + 1]);
    }
  }

  __device__ __forceinline__ void ntt2(E& X0, E& X1)
  {
    E T;

    T = X0 + X1;
    X1 = X0 - X1;
    X0 = T;
  }

  __device__ __forceinline__ void ntt4(E& X0, E& X1, E& X2, E& X3)
  {
    E T;

    T = X0 + X2;
    X2 = X0 - X2;
    X0 = X1 + X3;
    X1 = X1 - X3; // T has X0, X0 has X1, X2 has X2, X1 has X3

    X1 = X1 * WB[0];

    X3 = X2 - X1;
    X1 = X2 + X1;
    X2 = T - X0;
    X0 = T + X0;
  }

  // rbo version
  __device__ __forceinline__ void ntt4rbo(E& X0, E& X1, E& X2, E& X3)
  {
    E T;

    T = X0 - X1;
    X0 = X0 + X1;
    X1 = X2 + X3;
    X3 = X2 - X3; // T has X0, X0 has X1, X2 has X2, X1 has X3

    X3 = X3 * WB[0];

    X2 = X0 - X1;
    X0 = X0 + X1;
    X1 = T + X3;
    X3 = T - X3;
  }

  __device__ __forceinline__ void ntt8(E& X0, E& X1, E& X2, E& X3, E& X4, E& X5, E& X6, E& X7)
  {
    E T;

    // out of 56,623,104 possible mappings, we have:
    T = X3 - X7;
    X7 = X3 + X7;
    X3 = X1 - X5;
    X5 = X1 + X5;
    X1 = X2 + X6;
    X2 = X2 - X6;
    X6 = X0 + X4;
    X0 = X0 - X4;

    T = T * WB[1];
    X2 = X2 * WB[1];

    X4 = X6 + X1;
    X6 = X6 - X1;
    X1 = X3 + T;
    X3 = X3 - T;
    T = X5 + X7;
    X5 = X5 - X7;
    X7 = X0 + X2;
    X0 = X0 - X2;

    X1 = X1 * WB[0];
    X5 = X5 * WB[1];
    X3 = X3 * WB[2];

    X2 = X6 + X5;
    X6 = X6 - X5;
    X5 = X7 - X1;
    X1 = X7 + X1;
    X7 = X0 - X3;
    X3 = X0 + X3;
    X0 = X4 + T;
    X4 = X4 - T;
  }

  __device__ __forceinline__ void ntt8win()
  {
    E T;

    T = X[3] - X[7];
    X[7] = X[3] + X[7];
    X[3] = X[1] - X[5];
    X[5] = X[1] + X[5];
    X[1] = X[2] + X[6];
    X[2] = X[2] - X[6];
    X[6] = X[0] + X[4];
    X[0] = X[0] - X[4];

    X[2] = X[2] * WB[0];

    X[4] = X[6] + X[1];
    X[6] = X[6] - X[1];
    X[1] = X[3] + T;
    X[3] = X[3] - T;
    T = X[5] + X[7];
    X[5] = X[5] - X[7];
    X[7] = X[0] + X[2];
    X[0] = X[0] - X[2];

    X[1] = X[1] * WB[1];
    X[5] = X[5] * WB[0];
    X[3] = X[3] * WB[2];

    X[2] = X[6] + X[5];
    X[6] = X[6] - X[5];

    X[5] = X[1] + X[3];
    X[3] = X[1] - X[3];

    X[1] = X[7] + X[5];
    X[5] = X[7] - X[5];
    X[7] = X[0] - X[3];
    X[3] = X[0] + X[3];
    X[0] = X[4] + T;
    X[4] = X[4] - T;
  }

  __device__ __forceinline__ void SharedData64Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x7 : threadIdx.x >> 3;
    uint32_t column_id = stride ? threadIdx.x >> 3 : threadIdx.x & 0x7;

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 64 + i * 8 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 64 + i * 8 + column_id];
      }
    }
  }

  __device__ __forceinline__ void SharedData64Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x7 : threadIdx.x >> 3;
    uint32_t row_id = stride ? threadIdx.x >> 3 : threadIdx.x & 0x7;

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 64 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 64 + row_id * 8 + i];
      }
    }
  }

  __device__ __forceinline__ void SharedData32Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t column_id = stride ? threadIdx.x >> 4 : threadIdx.x & 0x3;

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 32 + i * 4 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 32 + i * 4 + column_id];
      }
    }
  }

  __device__ __forceinline__ void SharedData32Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t row_id = stride ? threadIdx.x >> 4 : threadIdx.x & 0x3;

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 32 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 32 + row_id * 8 + i];
      }
    }
  }

  __device__ __forceinline__ void SharedData32Columns4_2(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t column_id = (stride ? threadIdx.x >> 4 : threadIdx.x & 0x3) * 2;

#pragma unroll
    for (uint32_t j = 0; j < 2; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 4; i++) {
        if (store) {
          shmem[ntt_id * 32 + i * 8 + column_id + j] = X[4 * j + i];
        } else {
          X[4 * j + i] = shmem[ntt_id * 32 + i * 8 + column_id + j];
        }
      }
    }
  }

  __device__ __forceinline__ void SharedData32Rows4_2(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t row_id = (stride ? threadIdx.x >> 4 : threadIdx.x & 0x3) * 2;

#pragma unroll
    for (uint32_t j = 0; j < 2; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 4; i++) {
        if (store) {
          shmem[ntt_id * 32 + row_id * 4 + 4 * j + i] = X[4 * j + i];
        } else {
          X[4 * j + i] = shmem[ntt_id * 32 + row_id * 4 + 4 * j + i];
        }
      }
    }
  }

  __device__ __forceinline__ void SharedData16Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t column_id = stride ? threadIdx.x >> 5 : threadIdx.x & 0x1;

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 16 + i * 2 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 16 + i * 2 + column_id];
      }
    }
  }

  __device__ __forceinline__ void SharedData16Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t row_id = stride ? threadIdx.x >> 5 : threadIdx.x & 0x1;

#pragma unroll
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 16 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 16 + row_id * 8 + i];
      }
    }
  }

  __device__ __forceinline__ void SharedData16Columns2_4(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t column_id = (stride ? threadIdx.x >> 5 : threadIdx.x & 0x1) * 4;

#pragma unroll
    for (uint32_t j = 0; j < 4; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + i * 8 + column_id + j] = X[2 * j + i];
        } else {
          X[2 * j + i] = shmem[ntt_id * 16 + i * 8 + column_id + j];
        }
      }
    }
  }

  __device__ __forceinline__ void SharedData16Rows2_4(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t row_id = (stride ? threadIdx.x >> 5 : threadIdx.x & 0x1) * 4;

#pragma unroll
    for (uint32_t j = 0; j < 4; j++) {
#pragma unroll
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + row_id * 2 + 2 * j + i] = X[2 * j + i];
        } else {
          X[2 * j + i] = shmem[ntt_id * 16 + row_id * 2 + 2 * j + i];
        }
      }
    }
  }

  __device__ __forceinline__ void twiddlesInternal()
  {
#pragma unroll
    for (int i = 1; i < 8; i++) {
      X[i] = X[i] * WI[i - 1];
    }
  }

  __device__ __forceinline__ void twiddlesExternal()
  {
#pragma unroll
    for (int i = 0; i < 8; i++) {
      X[i] = X[i] * WE[i];
    }
  }
};

#endif
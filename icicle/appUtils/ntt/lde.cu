#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ntt.cuh"
#include "../vector_manipulation/ve_mod_mult.cuh"
#include "lde.cuh"


/**
 * Interpolate a batch of polynomials from their evaluations on the same subgroup.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
 * @param d_out The variable to write coefficients of the resulting polynomials into (the coefficients are in bit-reversed order if the evaluations weren't bit-reversed and vice-versa).
 * @param d_evaluations Input array of evaluations of all polynomials of type E (elements).
 * @param d_domain Domain on which the polynomials are evaluated. Must be a subgroup.
 * @param n Length of `d_domain` array, also equal to the number of evaluations of each polynomial.
 * @param batch_size The size of the batch; the length of `d_evaluations` is `n` * `batch_size`.
 */
template <typename E, typename S> int interpolate_batch(E * d_out, E * d_evaluations, S * d_domain, unsigned n, unsigned batch_size) {
  uint32_t logn = uint32_t(log(n) / log(2));
  hipMemcpy(d_out, d_evaluations, sizeof(E) * n * batch_size, hipMemcpyDeviceToDevice);
  
  int NUM_THREADS = min(n / 2, MAX_THREADS_BATCH);
  int NUM_BLOCKS = batch_size * max(int((n / 2) / NUM_THREADS), 1);
  for (uint32_t s = 0; s < logn; s++) //TODO: this loop also can be unrolled
  {
    ntt_template_kernel <E, S> <<<NUM_BLOCKS, NUM_THREADS>>>(d_out, n, d_domain, n, NUM_BLOCKS, s, false);
  }

  NUM_BLOCKS = (n * batch_size + NUM_THREADS - 1) / NUM_THREADS;
  template_normalize_kernel <E, S> <<<NUM_BLOCKS, NUM_THREADS>>> (d_out, n * batch_size, scalar_t::inv_log_size(logn));
  return 0;
}

/**
 * Interpolate a polynomial from its evaluations on a subgroup.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
 * @param d_out The variable to write coefficients of the resulting polynomial into (the coefficients are in bit-reversed order if the evaluations weren't bit-reversed and vice-versa).
 * @param d_evaluations Input array of evaluations that have type E (elements).
 * @param d_domain Domain on which the polynomial is evaluated. Must be a subgroup.
 * @param n Length of `d_evaluations` and the size `d_domain` arrays (they should have equal length).
 */
template <typename E, typename S> int interpolate(E * d_out, E * d_evaluations, S * d_domain, unsigned n) {
  return interpolate_batch <E, S> (d_out, d_evaluations, d_domain, n, 1);
}

template < typename E > __global__ void fill_array(E * arr, E val, uint32_t n) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < n) {
    arr[tid] = val;
  }
}

/**
 * Evaluate a batch of polynomials on the same coset.
 * @param d_out The evaluations of the polynomials on coset `u` * `d_domain`.
 * @param d_coefficients Input array of coefficients of all polynomials of type E (elements) to be evaluated in-place on a coset.
 * @param d_domain Domain on which the polynomials are evaluated (see `coset` flag). Must be a subgroup.
 * @param domain_size Length of `d_domain` array, on which the polynomial is computed.
 * @param n The number of coefficients, which might be different from `domain_size`.
 * @param batch_size The size of the batch; the length of `d_coefficients` is `n` * `batch_size`.
 * @param coset The flag that indicates whether to evaluate on a coset. If false, evaluate on a subgroup `d_domain`.
 * @param coset_powers If `coset` is true, a list of powers `[1, u, u^2, ..., u^{n-1}]` where `u` is the generator of the coset.
 */
template <typename E, typename S>
int evaluate_batch(E * d_out, E * d_coefficients, S * d_domain, unsigned domain_size, unsigned n, unsigned batch_size, bool coset, S * coset_powers) {
  uint32_t logn = uint32_t(log(domain_size) / log(2));
  if (domain_size > n) {
    // allocate and initialize an array of stream handles to parallelize data copying across batches
    hipStream_t *memcpy_streams = (hipStream_t *) malloc(batch_size * sizeof(hipStream_t));
    for (int i = 0; i < batch_size; i++)
    {
      hipStreamCreate(&(memcpy_streams[i]));

      hipMemcpyAsync(&d_out[i * domain_size], &d_coefficients[i * n], n * sizeof(E), hipMemcpyDeviceToDevice, memcpy_streams[i]);
      int NUM_THREADS = MAX_THREADS_BATCH;
      int NUM_BLOCKS = (domain_size - n + NUM_THREADS - 1) / NUM_THREADS;
      fill_array <E> <<<NUM_BLOCKS, NUM_THREADS, 0, memcpy_streams[i]>>> (&d_out[i * domain_size + n], E::zero(), domain_size - n);

      hipStreamSynchronize(memcpy_streams[i]);
      hipStreamDestroy(memcpy_streams[i]);
    }
  } else
    hipMemcpy(d_out, d_coefficients, sizeof(E) * domain_size * batch_size, hipMemcpyDeviceToDevice);

  if (coset)
    batch_vector_mult(coset_powers, d_out, domain_size, batch_size);

  int NUM_THREADS = min(domain_size / 2, MAX_THREADS_BATCH);
  int chunks = max(int((domain_size / 2) / NUM_THREADS), 1);
  int NUM_BLOCKS = batch_size * chunks;
  for (uint32_t s = 0; s < logn; s++) //TODO: this loop also can be unrolled
  {
    ntt_template_kernel <E, S> <<<NUM_BLOCKS, NUM_THREADS>>>(d_out, domain_size, d_domain, domain_size, batch_size * chunks, logn - s - 1, true);
  }
  return 0;
}

/**
 * Evaluate a polynomial on a coset.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs, so the order of outputs is bit-reversed.
 * @param d_out The evaluations of the polynomial on coset `u` * `d_domain`.
 * @param d_coefficients Input array of coefficients of a polynomial of type E (elements).
 * @param d_domain Domain on which the polynomial is evaluated (see `coset` flag). Must be a subgroup.
 * @param domain_size Length of `d_domain` array, on which the polynomial is computed.
 * @param n The number of coefficients, which might be different from `domain_size`.
 * @param coset The flag that indicates whether to evaluate on a coset. If false, evaluate on a subgroup `d_domain`.
 * @param coset_powers If `coset` is true, a list of powers `[1, u, u^2, ..., u^{n-1}]` where `u` is the generator of the coset.
 */
template <typename E, typename S> 
int evaluate(E * d_out, E * d_coefficients, S * d_domain, unsigned domain_size, unsigned n, bool coset, S * coset_powers) {
  return evaluate_batch <E, S> (d_out, d_coefficients, d_domain, domain_size, n, 1, coset, coset_powers);
}

int interpolate_scalars(scalar_t* d_out, scalar_t* d_evaluations, scalar_t* d_domain, unsigned n) {
  return interpolate(d_out, d_evaluations, d_domain, n);
}

int interpolate_scalars_batch(scalar_t* d_out, scalar_t* d_evaluations, scalar_t* d_domain, unsigned n, unsigned batch_size) {
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
}

int interpolate_points(projective_t* d_out, projective_t* d_evaluations, scalar_t* d_domain, unsigned n) {
  return interpolate(d_out, d_evaluations, d_domain, n);
}

int interpolate_points_batch(projective_t* d_out, projective_t* d_evaluations, scalar_t* d_domain, unsigned n, unsigned batch_size) {
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size);
}

int evaluate_scalars(scalar_t* d_out, scalar_t* d_coefficients, scalar_t* d_domain, unsigned domain_size, unsigned n) {
  scalar_t* _null = nullptr;
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
}

int evaluate_scalars_batch(scalar_t* d_out, scalar_t* d_coefficients, scalar_t* d_domain, unsigned domain_size, unsigned n, unsigned batch_size) {
  scalar_t* _null = nullptr;
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
}

int evaluate_points(projective_t* d_out, projective_t* d_coefficients, scalar_t* d_domain, unsigned domain_size, unsigned n) {
  scalar_t* _null = nullptr;
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null);
}

int evaluate_points_batch(projective_t* d_out, projective_t* d_coefficients, scalar_t* d_domain, 
                          unsigned domain_size, unsigned n, unsigned batch_size) {
  scalar_t* _null = nullptr;
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null);
}

int evaluate_scalars_on_coset(scalar_t* d_out, scalar_t* d_coefficients, scalar_t* d_domain, 
                              unsigned domain_size, unsigned n, scalar_t* coset_powers) {
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
}

int evaluate_scalars_on_coset_batch(scalar_t* d_out, scalar_t* d_coefficients, scalar_t* d_domain, unsigned domain_size, 
                                    unsigned n, unsigned batch_size, scalar_t* coset_powers) {
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
}

int evaluate_points_on_coset(projective_t* d_out, projective_t* d_coefficients, scalar_t* d_domain, 
                             unsigned domain_size, unsigned n, scalar_t* coset_powers) {
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers);
}

int evaluate_points_on_coset_batch(projective_t* d_out, projective_t* d_coefficients, scalar_t* d_domain, unsigned domain_size,
                                   unsigned n, unsigned batch_size, scalar_t* coset_powers) {
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers);
}

extern "C" scalar_t* build_domain_cuda(uint32_t domain_size, uint32_t logn, bool inverse, size_t device_id = 0)
{
    try
    {
        if (inverse) {
            return fill_twiddle_factors_array(domain_size, scalar_t::omega_inv(logn));
        } else {
            return fill_twiddle_factors_array(domain_size, scalar_t::omega(logn));
        }
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return nullptr;
    }
}

extern "C" int ntt_cuda(scalar_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        
        return -1;        
    }
}

extern "C" int ecntt_cuda(projective_t *arr, uint32_t n, bool inverse, size_t device_id = 0)
{
    try
    {
        return ecntt_end2end(arr, n, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ntt_batch_cuda(scalar_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ntt_end2end_batch(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int ecntt_batch_cuda(projective_t *arr, uint32_t arr_size, uint32_t batch_size, bool inverse, size_t device_id = 0)
{
    try
    {
        return ecntt_end2end_batch(arr, arr_size, batch_size, inverse); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_cuda(scalar_t* d_out, scalar_t *d_evaluations, scalar_t *d_domain, unsigned n, unsigned device_id = 0)
{
    try
    {
        return interpolate_scalars(d_out, d_evaluations, d_domain, n); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_scalars_batch_cuda(scalar_t* d_out, scalar_t* d_evaluations, scalar_t* d_domain, unsigned n,
                                              unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_scalars_batch(d_out, d_evaluations, d_domain, n, batch_size); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_cuda(projective_t* d_out, projective_t *d_evaluations, scalar_t *d_domain, unsigned n, size_t device_id = 0)
{
    try
    {
        return interpolate_points(d_out, d_evaluations, d_domain, n); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int interpolate_points_batch_cuda(projective_t* d_out, projective_t* d_evaluations, scalar_t* d_domain,
                                             unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return interpolate_points_batch(d_out, d_evaluations, d_domain, n, batch_size); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_cuda(scalar_t* d_out, scalar_t *d_coefficients, scalar_t *d_domain, 
                                     unsigned domain_size, unsigned n, unsigned device_id = 0)
{
    try
    {
        return evaluate_scalars(d_out, d_coefficients, d_domain, domain_size, n); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_batch_cuda(scalar_t* d_out, scalar_t* d_coefficients, scalar_t* d_domain, unsigned domain_size,
                                           unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return evaluate_scalars_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_cuda(projective_t* d_out, projective_t *d_coefficients, scalar_t *d_domain, 
                                    unsigned domain_size, unsigned n, size_t device_id = 0)
{
    try
    {
        return evaluate_points(d_out, d_coefficients, d_domain, domain_size, n); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_batch_cuda(projective_t* d_out, projective_t* d_coefficients, scalar_t* d_domain, unsigned domain_size,
                                          unsigned n, unsigned batch_size, size_t device_id = 0)
{
    try
    {
        return evaluate_points_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_cuda(scalar_t* d_out, scalar_t *d_coefficients, scalar_t *d_domain, unsigned domain_size,
                                              unsigned n, scalar_t *coset_powers, unsigned device_id = 0)
{
    try
    {
        return evaluate_scalars_on_coset(d_out, d_coefficients, d_domain, domain_size, n, coset_powers); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_scalars_on_coset_batch_cuda(scalar_t* d_out, scalar_t* d_coefficients, scalar_t* d_domain, unsigned domain_size, 
                                                    unsigned n, unsigned batch_size, scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_scalars_on_coset_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, coset_powers); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_cuda(projective_t* d_out, projective_t *d_coefficients, scalar_t *d_domain, unsigned domain_size,
                                             unsigned n, scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_points_on_coset(d_out, d_coefficients, d_domain, domain_size, n, coset_powers); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int evaluate_points_on_coset_batch_cuda(projective_t* d_out, projective_t* d_coefficients, scalar_t* d_domain, unsigned domain_size, 
                                                   unsigned n, unsigned batch_size, scalar_t *coset_powers, size_t device_id = 0)
{
    try
    {
        return evaluate_points_on_coset_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, coset_powers); // TODO: pass device_id
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_cuda(scalar_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_scalars_batch_cuda(scalar_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_cuda(projective_t* arr, int n, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order(arr, n, logn);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

extern "C" int reverse_order_points_batch_cuda(projective_t* arr, int n, int batch_size, size_t device_id = 0)
{
    try
    {
        uint32_t logn = uint32_t(log(n) / log(2));
        reverse_order_batch(arr, n, logn, batch_size);
        return 0;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

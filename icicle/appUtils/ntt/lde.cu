#include "hip/hip_runtime.h"
#ifndef LDE
#define LDE
#include <hip/hip_runtime.h>
#include "ntt.cuh"
#include "lde.cuh"
#include "../vector_manipulation/ve_mod_mult.cuh"

template < typename E, bool SUB > __global__ void add_sub_array(E* res, E* in1, E* in2, uint32_t n) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < n) {
      res[tid] = SUB ? in1[tid] - in2[tid] : in1[tid] + in2[tid];
    }
  }
  
  template <typename E>
  int sub_polys(E* d_out, E* d_in1, E* d_in2, unsigned n, hipStream_t stream) {
    uint32_t NUM_THREADS = MAX_THREADS_BATCH;
    uint32_t NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;
  
    add_sub_array <E, true> <<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(d_out, d_in1, d_in2, n);
  
    return 0;
  }
  
  template <typename E>
  int add_polys(E* d_out, E* d_in1, E* d_in2, unsigned n, hipStream_t stream) {
    uint32_t NUM_THREADS = MAX_THREADS_BATCH;
    uint32_t NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;
  
    add_sub_array <E, false> <<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(d_out, d_in1, d_in2, n);
  
    return 0;
  }
  
/**
 * Interpolate a batch of polynomials from their evaluations on the same subgroup.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
 * @param d_out The variable to write coefficients of the resulting polynomials into (the coefficients are in bit-reversed order if the evaluations weren't bit-reversed and vice-versa).
 * @param d_evaluations Input array of evaluations of all polynomials of type E (elements).
 * @param d_domain Domain on which the polynomials are evaluated. Must be a subgroup.
 * @param n Length of `d_domain` array, also equal to the number of evaluations of each polynomial.
 * @param batch_size The size of the batch; the length of `d_evaluations` is `n` * `batch_size`.
 */
template <typename E, typename S> int interpolate_batch(E * d_out, E * d_evaluations, S * d_domain, unsigned n, unsigned batch_size, bool coset, S * coset_powers, hipStream_t stream) {
  hipMemcpyAsync(d_out, d_evaluations, sizeof(E) * n * batch_size, hipMemcpyDeviceToDevice, stream);
  ntt_inplace_batch_template(d_out, d_domain, n, batch_size, true, coset, coset_powers, stream, true);
  return 0;
}

/**
 * Interpolate a polynomial from its evaluations on a subgroup.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs.
 * @param d_out The variable to write coefficients of the resulting polynomial into (the coefficients are in bit-reversed order if the evaluations weren't bit-reversed and vice-versa).
 * @param d_evaluations Input array of evaluations that have type E (elements).
 * @param d_domain Domain on which the polynomial is evaluated. Must be a subgroup.
 * @param n Length of `d_evaluations` and the size `d_domain` arrays (they should have equal length).
 */
template <typename E, typename S> int interpolate(E * d_out, E * d_evaluations, S * d_domain, unsigned n, bool coset, S * coset_powers, hipStream_t stream) {
  return interpolate_batch <E, S> (d_out, d_evaluations, d_domain, n, 1, coset, coset_powers, stream);
}

template < typename E > __global__ void fill_array(E * arr, E val, uint32_t n) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < n) {
    arr[tid] = val;
  }
}

/**
 * Evaluate a batch of polynomials on the same coset.
 * @param d_out The evaluations of the polynomials on coset `u` * `d_domain`.
 * @param d_coefficients Input array of coefficients of all polynomials of type E (elements) to be evaluated in-place on a coset.
 * @param d_domain Domain on which the polynomials are evaluated (see `coset` flag). Must be a subgroup.
 * @param domain_size Length of `d_domain` array, on which the polynomial is computed.
 * @param n The number of coefficients, which might be different from `domain_size`.
 * @param batch_size The size of the batch; the length of `d_coefficients` is `n` * `batch_size`.
 * @param coset The flag that indicates whether to evaluate on a coset. If false, evaluate on a subgroup `d_domain`.
 * @param coset_powers If `coset` is true, a list of powers `[1, u, u^2, ..., u^{n-1}]` where `u` is the generator of the coset.
 */
template <typename E, typename S>
int evaluate_batch(E * d_out, E * d_coefficients, S * d_domain, unsigned domain_size, unsigned n, unsigned batch_size, bool coset, S * coset_powers, hipStream_t stream) {
  uint32_t logn = uint32_t(log(domain_size) / log(2));
  if (domain_size > n) {
    // allocate and initialize an array of stream handles to parallelize data copying across batches
    hipStream_t *memcpy_streams = (hipStream_t *) malloc(batch_size * sizeof(hipStream_t));
    for (unsigned i = 0; i < batch_size; i++)
    {
      hipStreamCreate(&(memcpy_streams[i]));

      hipMemcpyAsync(&d_out[i * domain_size], &d_coefficients[i * n], n * sizeof(E), hipMemcpyDeviceToDevice, memcpy_streams[i]);
      uint32_t NUM_THREADS = MAX_THREADS_BATCH;
      uint32_t NUM_BLOCKS = (domain_size - n + NUM_THREADS - 1) / NUM_THREADS;
      fill_array <E> <<<NUM_BLOCKS, NUM_THREADS, 0, memcpy_streams[i]>>> (&d_out[i * domain_size + n], E::zero(), domain_size - n);

      hipStreamSynchronize(memcpy_streams[i]);
      hipStreamDestroy(memcpy_streams[i]);
    }
  } else
    hipMemcpyAsync(d_out, d_coefficients, sizeof(E) * domain_size * batch_size, hipMemcpyDeviceToDevice, stream);

  if (coset)
    batch_vector_mult(coset_powers, d_out, domain_size, batch_size, stream);
  
  S* _null = nullptr;
  ntt_inplace_batch_template(d_out, d_domain, domain_size, batch_size, false, false, _null, stream, true);
  return 0;
}

/**
 * Evaluate a polynomial on a coset.
 * Note: this function does not preform any bit-reverse permutations on its inputs or outputs, so the order of outputs is bit-reversed.
 * @param d_out The evaluations of the polynomial on coset `u` * `d_domain`.
 * @param d_coefficients Input array of coefficients of a polynomial of type E (elements).
 * @param d_domain Domain on which the polynomial is evaluated (see `coset` flag). Must be a subgroup.
 * @param domain_size Length of `d_domain` array, on which the polynomial is computed.
 * @param n The number of coefficients, which might be different from `domain_size`.
 * @param coset The flag that indicates whether to evaluate on a coset. If false, evaluate on a subgroup `d_domain`.
 * @param coset_powers If `coset` is true, a list of powers `[1, u, u^2, ..., u^{n-1}]` where `u` is the generator of the coset.
 */
template <typename E, typename S> 
int evaluate(E * d_out, E * d_coefficients, S * d_domain, unsigned domain_size, unsigned n, bool coset, S * coset_powers, hipStream_t stream) {
  return evaluate_batch <E, S> (d_out, d_coefficients, d_domain, domain_size, n, 1, coset, coset_powers, stream);
}

template <typename S> 
int interpolate_scalars(S* d_out, S* d_evaluations, S* d_domain, unsigned n, hipStream_t stream) {
  S* _null = nullptr;
  return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
}

template <typename S> 
int interpolate_scalars_batch(S* d_out, S* d_evaluations, S* d_domain, unsigned n, unsigned batch_size, hipStream_t stream) {
  S* _null = nullptr;
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
}

template <typename E, typename S> 
int interpolate_points(E* d_out, E* d_evaluations, S* d_domain, unsigned n, hipStream_t stream) {
  S* _null = nullptr;
  return interpolate(d_out, d_evaluations, d_domain, n, false, _null, stream);
}

template <typename E, typename S> 
int interpolate_points_batch(E* d_out, E* d_evaluations, S* d_domain, unsigned n, unsigned batch_size, hipStream_t stream) {
  S* _null = nullptr;
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, false, _null, stream);
}

template <typename S> 
int evaluate_scalars(S* d_out, S* d_coefficients, S* d_domain, unsigned domain_size, unsigned n, hipStream_t stream) {
  S* _null = nullptr;
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
}

template <typename S> 
int evaluate_scalars_batch(S* d_out, S* d_coefficients, S* d_domain, unsigned domain_size, unsigned n, unsigned batch_size, hipStream_t stream) {
  S* _null = nullptr;
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
}

template <typename E, typename S> 
int evaluate_points(E* d_out, E* d_coefficients, S* d_domain, unsigned domain_size, unsigned n, hipStream_t stream) {
  S* _null = nullptr;
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, false, _null, stream);
}

template <typename E, typename S> 
int evaluate_points_batch(E* d_out, E* d_coefficients, S* d_domain, 
                          unsigned domain_size, unsigned n, unsigned batch_size, hipStream_t stream) {
  S* _null = nullptr;
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, false, _null, stream);
}

template <typename S> 
int interpolate_scalars_on_coset(S* d_out, S* d_evaluations, S* d_domain,
                                 unsigned n, S* coset_powers, hipStream_t stream) {
  return interpolate(d_out, d_evaluations, d_domain, n, true, coset_powers, stream);
}

template <typename S> 
int interpolate_scalars_on_coset_batch(S* d_out, S* d_evaluations, S* d_domain,
                                       unsigned n, unsigned batch_size, S* coset_powers, hipStream_t stream) {
  return interpolate_batch(d_out, d_evaluations, d_domain, n, batch_size, true, coset_powers, stream);
}

template <typename S> 
int evaluate_scalars_on_coset(S* d_out, S* d_coefficients, S* d_domain, 
                              unsigned domain_size, unsigned n, S* coset_powers, hipStream_t stream) {
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
}

template <typename E, typename S> 
int evaluate_scalars_on_coset_batch(S* d_out, S* d_coefficients, S* d_domain, unsigned domain_size, 
                                    unsigned n, unsigned batch_size, S* coset_powers, hipStream_t stream) {
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
}

template <typename E, typename S> 
int evaluate_points_on_coset(E* d_out, E* d_coefficients, S* d_domain, 
                             unsigned domain_size, unsigned n, S* coset_powers, hipStream_t stream) {
  return evaluate(d_out, d_coefficients, d_domain, domain_size, n, true, coset_powers, stream);
}

template <typename E, typename S> 
int evaluate_points_on_coset_batch(E* d_out, E* d_coefficients, S* d_domain, unsigned domain_size,
                                   unsigned n, unsigned batch_size, S* coset_powers, hipStream_t stream) {
  return evaluate_batch(d_out, d_coefficients, d_domain, domain_size, n, batch_size, true, coset_powers, stream);
}
#endif
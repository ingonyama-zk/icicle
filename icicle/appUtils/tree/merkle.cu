#include "hip/hip_runtime.h"
#include "merkle.cuh"

namespace merkle {
  /// Flattens the tree digests and sum them up to get
  /// the memory needed to contain all the digests
  template <typename S>
  size_t get_digests_len(uint32_t height, uint32_t arity)
  {
    size_t digests_len = 0;
    size_t row_length = 1;
    for (int i = 1; i < height; i++) {
      digests_len += row_length;
      row_length *= arity;
    }

    return digests_len;
  }

  /// Constructs merkle subtree without parallelization
  /// The digests are aligned sequentially per row
  /// Example:
  ///
  /// Big tree:
  ///
  ///        1
  ///       / \
  ///      2   3
  ///     / \ / \
  ///    4  5 6  7
  ///
  /// Subtree 1    Subtree 2
  ///    2            3
  ///   / \          / \
  ///  4   5        6   7
  ///
  /// Digests array for subtree 1:
  /// [4 5 . . 2 . .]
  /// |   |    |
  /// -----    V
  ///   |    Segment (offset = 4, subtree_idx = 0)
  ///   v
  /// Segment (offset = 0, subtree_idx = 0)
  ///
  /// Digests array for subtree 2:
  /// [. . 6 7 . 3 .]
  ///     |   |
  ///     -----
  ///       |
  ///       v
  ///    Segment (offset = 0, subtree_idx = 1)
  ///
  /// Total digests array:
  /// [4 5 6 7 2 3 .]
  template <typename S, int T>
  hipError_t build_merkle_subtree(
    S* state,
    S* digests,
    size_t subtree_idx,
    size_t subtree_height,
    S* big_tree_digests,
    size_t start_segment_size,
    size_t start_segment_offset,
    int keep_rows,
    const PoseidonConstants<S>& poseidon,
    hipStream_t& stream)
  {
    int arity = T - 1;

    PoseidonConfig config = default_poseidon_config<S>(T);
    config.are_inputs_on_device = true;
    config.are_outputs_on_device = true;
    config.input_is_a_state = true;
    config.loop_state = true;
    config.ctx.stream = stream;

    size_t leaves_size = pow(arity, subtree_height - 1);
    uint32_t number_of_blocks = leaves_size / arity;
    size_t segment_size = start_segment_size;
    size_t segment_offset = start_segment_offset;

    while (number_of_blocks > 0) {
      hipError_t poseidon_res = poseidon_hash<S, T>(state, digests, number_of_blocks, poseidon, config);
      CHK_IF_RETURN(poseidon_res);

      if (!keep_rows || subtree_height <= keep_rows + 1) {
        S* digests_with_offset = big_tree_digests + segment_offset + subtree_idx * number_of_blocks;
        CHK_IF_RETURN(
          hipMemcpyAsync(digests_with_offset, digests, number_of_blocks * sizeof(S), hipMemcpyDeviceToHost, stream));
        segment_offset += segment_size;
      }

      segment_size /= arity;
      subtree_height--;
      number_of_blocks /= arity;
      config.aligned = true;
    }

    return CHK_LAST();
  }

  template <typename S, int T>
  hipError_t build_merkle_tree(
    const S* leaves,
    S* digests,
    uint32_t height,
    const poseidon::PoseidonConstants<S>& poseidon,
    const TreeBuilderConfig& config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = config.ctx.stream;

    int arity = T - 1;
    uint32_t number_of_leaves = pow(arity, (height - 1));

    // This will determine how much splitting do we need to do
    // `number_of_streams` subtrees should fit in the device
    // This means each subtree should fit in `STREAM_CHUNK_SIZE` memory
    uint32_t number_of_subtrees = 1;
    uint32_t subtree_height = height;
    uint32_t subtree_leaves_size = pow(arity, height - 1);
    uint32_t subtree_state_size = subtree_leaves_size / arity * T;
    uint32_t subtree_digests_size = get_digests_len<S>(subtree_height, arity);
    size_t subtree_memory_required = sizeof(S) * (subtree_state_size + subtree_digests_size);
    while (subtree_memory_required > STREAM_CHUNK_SIZE) {
      number_of_subtrees *= arity;
      subtree_height--;
      subtree_leaves_size /= arity;
      subtree_state_size = subtree_leaves_size / arity * T;
      subtree_digests_size = subtree_state_size / arity;
      subtree_memory_required = sizeof(S) * (subtree_state_size + subtree_digests_size);
    }
    int cap_height = height - subtree_height + 1;
    size_t caps_len = pow(arity, cap_height - 1);

    size_t available_memory, _total_memory;
    CHK_IF_RETURN(hipMemGetInfo(&available_memory, &_total_memory));
    available_memory -= GIGA / 8; // Leave 128 MB

    // We can effectively parallelize memory copy with streams
    // as long as they don't operate on more than `STREAM_CHUNK_SIZE` bytes
    const size_t number_of_streams = std::min((uint32_t)(available_memory / STREAM_CHUNK_SIZE), number_of_subtrees);
    hipStream_t* streams = static_cast<hipStream_t*>(malloc(sizeof(hipStream_t) * number_of_streams));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamCreate(&streams[i]));
    }

#if !defined(__CUDA_ARCH__) && defined(MERKLE_DEBUG)
    std::cout << "Available memory = " << available_memory / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Number of streams = " << number_of_streams << std::endl;
    std::cout << "Number of subtrees = " << number_of_subtrees << std::endl;
    std::cout << "Height of a subtree = " << subtree_height << std::endl;
    std::cout << "Cutoff height = " << height - subtree_height + 1 << std::endl;
    std::cout << "Number of leaves in a subtree = " << subtree_leaves_size << std::endl;
    std::cout << "State of a subtree = " << subtree_state_size << std::endl;
    std::cout << "Digest elements for a subtree = " << get_digests_len<S>(subtree_height, arity) << std::endl;
    std::cout << "Size of 1 subtree states = " << subtree_state_size * sizeof(S) / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Size of 1 subtree digests = " << subtree_digests_size * sizeof(S) / 1024 / 1024 << " MB" << std::endl;
#endif

    // Allocate memory for the leaves and digests
    // These are shared by streams in a pool
    S *states_ptr, *digests_ptr;
    CHK_IF_RETURN(hipMallocAsync(&states_ptr, subtree_state_size * number_of_streams * sizeof(S), stream))
    CHK_IF_RETURN(hipMallocAsync(&digests_ptr, subtree_digests_size * number_of_streams * sizeof(S), stream))
    // Wait for these allocations to finish
    CHK_IF_RETURN(hipStreamSynchronize(stream));

    bool caps_mode = config.keep_rows && config.keep_rows < cap_height;
    S* caps;
    if (caps_mode) { caps = static_cast<S*>(malloc(caps_len * sizeof(S))); }

    for (size_t subtree_idx = 0; subtree_idx < number_of_subtrees; subtree_idx++) {
      size_t stream_idx = subtree_idx % number_of_streams;
      hipStream_t subtree_stream = streams[stream_idx];

      const S* subtree_leaves = leaves + subtree_idx * subtree_leaves_size;
      S* subtree_state = states_ptr + stream_idx * subtree_state_size;
      S* subtree_digests = digests_ptr + stream_idx * subtree_digests_size;

      // Copy the first level from RAM / device to device
      // The pitch property of hipMemcpy2D resolves shape differences
      CHK_IF_RETURN(hipMemcpy2DAsync(
        subtree_state, T * sizeof(S),      // Device pointer and device pitch
        subtree_leaves, arity * sizeof(S), // Host pointer and pitch
        arity * sizeof(S),                 // Size of the source matrix (Arity)
        subtree_leaves_size / arity,       // Size of the source matrix (Number of blocks)
        config.are_inputs_on_device ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice, subtree_stream));

      int subtree_keep_rows = 0;
      if (config.keep_rows) {
        int diff = config.keep_rows - cap_height + 1;
        subtree_keep_rows = diff <= 0 ? 1 : diff;
      }
      size_t start_segment_size = number_of_leaves / arity;
      hipError_t subtree_result = build_merkle_subtree<S, T>(
        subtree_state,              // state
        subtree_digests,            // digests
        subtree_idx,                // subtree_idx
        subtree_height,             // subtree_height
        caps_mode ? caps : digests, // big_tree_digests
        start_segment_size,         // start_segment_size
        0,                          // start_segment_offset
        subtree_keep_rows,          // keep_rows
        poseidon,                   // hash
        subtree_stream              // stream
      );
      CHK_IF_RETURN(subtree_result);
    }

    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
    }

    // Finish the top-level tree if any
    if (cap_height > 1) {
      size_t start_segment_size = caps_len / arity;
      size_t start_segment_offset = 0;
      if (!caps_mode) {
        size_t layer_size = pow(arity, config.keep_rows - 1);
        for (int i = 0; i < config.keep_rows - cap_height + 1; i++) {
          start_segment_offset += layer_size;
          layer_size /= arity;
        }
      }
      CHK_IF_RETURN(hipMemcpy2DAsync(
        states_ptr, T * sizeof(S), caps_mode ? caps : (digests + start_segment_offset - caps_len), arity * sizeof(S),
        arity * sizeof(S),
        caps_len / arity,                 // Size of the source
        hipMemcpyHostToDevice, stream)); // Direction and stream

      hipError_t top_tree_result = build_merkle_subtree<S, T>(
        states_ptr,           // state
        digests_ptr,          // digests
        0,                    // subtree_idx
        cap_height,           // subtree_height
        digests,              // big_tree_digests
        start_segment_size,   // start_segment_size
        start_segment_offset, // start_segment_offset
        config.keep_rows,     // keep_rows
        poseidon,             // hash
        stream                // stream
      );
      CHK_IF_RETURN(top_tree_result);
      if (caps_mode) { free(caps); }
    }

    CHK_IF_RETURN(hipFreeAsync(states_ptr, stream));
    CHK_IF_RETURN(hipFreeAsync(digests_ptr, stream));
    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
      CHK_IF_RETURN(hipStreamDestroy(streams[i]));
    }
    free(streams);
    return CHK_LAST();
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BuildPoseidonMerkleTree)(
    const curve_config::scalar_t* leaves,
    curve_config::scalar_t* digests,
    uint32_t height,
    int arity,
    PoseidonConstants<curve_config::scalar_t>& constants,
    TreeBuilderConfig& config)
  {
    switch (arity) {
    case 2:
      return build_merkle_tree<curve_config::scalar_t, 3>(leaves, digests, height, constants, config);
    case 4:
      return build_merkle_tree<curve_config::scalar_t, 5>(leaves, digests, height, constants, config);
    case 8:
      return build_merkle_tree<curve_config::scalar_t, 9>(leaves, digests, height, constants, config);
    case 11:
      return build_merkle_tree<curve_config::scalar_t, 12>(leaves, digests, height, constants, config);
    default:
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "BuildPoseidonMerkleTree: #arity must be one of [2, 4, 8, 11]");
    }
    return CHK_LAST();
  }
} // namespace merkle
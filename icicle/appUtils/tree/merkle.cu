#include "hip/hip_runtime.h"
#include "../poseidon/poseidon.cuh"
#include <iostream>
#include <math.h>

using namespace poseidon;

static constexpr size_t GIGA = 1024 * 1024 * 1024;

/// Bytes per stream
static constexpr size_t STREAM_CHUNK_SIZE = 1024 * 1024 * 1024;

/// Flattens the tree digests and sum them up to get
/// the memory needed to contain all the digests
size_t get_digests_len(uint32_t height, uint32_t arity)
{
  size_t digests_len = 0;
  size_t row_length = 1;
  for (int i = 1; i < height; i++) {
    digests_len += row_length;
    row_length *= arity;
  }

  return digests_len;
}

/// Construct merkle subtree without parallelization
template <typename S>
void __build_merkle_subtree(
  S* state,
  S* digests,
  size_t subtree_idx,
  size_t leaves_size,
  S* big_tree_digests,
  size_t start_segment_size,
  Poseidon<S>& poseidon,
  hipStream_t& stream)
{
  // We would need to align the digests of a big tree correctly
  // The digests are aligned sequentially per row
  // Example:
  //
  // Big tree:
  //
  //        1
  //       / \
    //      2   3
  //     / \ / \
    //    4  5 6  7
  //
  // Subtree 1    Subtree 2
  //    2            3
  //   / \          / \
    //  4   5        6   7
  //
  // Digests array for subtree 1:
  // [4 5 . . 2 .]
  // |   |    |
  // -----    V
  //   |    Segment (offset = 4, subtree_idx = 0)
  //   v
  // Segment (offset = 0, subtree_idx = 0)
  //
  // Digests array for subtree 2:
  // [. . 6 7 . 3]
  //     |   |
  //     -----
  //       |
  //       v
  //    Segment (offset = 0, subtree_idx = 1)
  //
  // Total digests array:
  // [4 5 6 7 2 3]

  uint32_t number_of_blocks = leaves_size / poseidon.arity;
  size_t segment_size = start_segment_size;
  size_t segment_offset = 0;

  bool first_iteration = true;
  while (number_of_blocks > 0) {
    poseidon.poseidon_hash(
      state, number_of_blocks, digests, Poseidon<S>::HashType::MerkleTree, stream, !first_iteration, true);

    S* digests_with_offset = big_tree_digests + segment_offset + subtree_idx * number_of_blocks;
    hipMemcpyAsync(digests_with_offset, digests, number_of_blocks * sizeof(S), hipMemcpyDeviceToHost, stream);

    number_of_blocks /= poseidon.arity;
    segment_offset += segment_size;
    segment_size /= poseidon.arity;
    first_iteration = false;
  }
}

/// Constructs the merkle tree
///
///=====================================================
/// # Arguments
/// * `leaves`  - a host pointer to the leaves array. Expected to have arity ^ (height - 1) elements
/// * `digests` - a host pointer to write digests to. Expected to have `sum(arity ^ (i)) for i in [0..height-1]`
/// elements
/// * `height`  - the height of a tree
/// * `poseidon` - an instance of the poseidon hasher
/// * `stream` - a cuda stream for top-level operations
///
/// # Algorithm
/// The function will split large trees into many subtrees of size that will fit `STREAM_CHUNK_SIZE`.
/// The subtrees will be constructed in streams pool. Each stream will handle a subtree
/// After all subtrees are constructed - the function will combine the resulting sub-digests into the final top-tree
///======================================================
template <typename S>
void build_merkle_tree(const S* leaves, S* digests, uint32_t height, Poseidon<S>& poseidon, hipStream_t stream)
{
  uint32_t number_of_leaves = pow(poseidon.arity, (height - 1));

  // This will determine how much splitting do we need to do
  // `number_of_streams` subtrees should fit in the device
  // This means each subtree should fit in `STREAM_CHUNK_SIZE` memory
  uint32_t number_of_subtrees = 1;
  uint32_t subtree_height = height;
  uint32_t subtree_leaves_size = pow(poseidon.arity, height - 1);
  uint32_t subtree_state_size = subtree_leaves_size / poseidon.arity * poseidon.t;
  uint32_t subtree_digests_size = subtree_state_size / poseidon.arity;
  size_t subtree_memory_required = sizeof(S) * (subtree_state_size + subtree_digests_size);
  while (subtree_memory_required > STREAM_CHUNK_SIZE) {
    number_of_subtrees *= poseidon.arity;
    subtree_height--;
    subtree_leaves_size /= poseidon.arity;
    subtree_state_size = subtree_leaves_size / poseidon.arity * poseidon.t;
    subtree_digests_size = subtree_state_size / poseidon.arity;
    subtree_memory_required = sizeof(S) * (subtree_state_size + subtree_digests_size);
  }

  size_t available_memory, _total_memory;
  hipMemGetInfo(&available_memory, &_total_memory);
  available_memory -= GIGA / 8; // Leave 128 MB

  // We can effectively parallelize memory copy with streams
  // as long as they don't operate on more than `STREAM_CHUNK_SIZE` bytes
  const size_t number_of_streams = std::min((uint32_t)(available_memory / STREAM_CHUNK_SIZE), number_of_subtrees);
  hipStream_t* streams = static_cast<hipStream_t*>(malloc(sizeof(hipStream_t) * number_of_streams));
  for (size_t i = 0; i < number_of_streams; i++) {
    hipStreamCreate(&streams[i]);
  }

#if !defined(__CUDA_ARCH__) && defined(MERKLE_DEBUG)
  std::cout << "Available memory = " << available_memory / 1024 / 1024 << " MB" << std::endl;
  std::cout << "Number of streams = " << number_of_streams << std::endl;
  std::cout << "Number of subtrees = " << number_of_subtrees << std::endl;
  std::cout << "Height of a subtree = " << subtree_height << std::endl;
  std::cout << "Cutoff height = " << height - subtree_height + 1 << std::endl;
  std::cout << "Number of leaves in a subtree = " << subtree_leaves_size << std::endl;
  std::cout << "State of a subtree = " << subtree_state_size << std::endl;
  std::cout << "Digest elements for a subtree = " << get_digests_len(subtree_height, poseidon.arity) << std::endl;
  std::cout << "Size of 1 subtree states = " << subtree_state_size * sizeof(S) / 1024 / 1024 << " MB" << std::endl;
  std::cout << "Size of 1 subtree digests = " << subtree_digests_size * sizeof(S) / 1024 / 1024 << " MB" << std::endl;
#endif

  // Allocate memory for the leaves and digests
  // These are shared by streams in a pool
  S *states_ptr, *digests_ptr;
  if (hipMallocAsync(&states_ptr, subtree_state_size * number_of_streams * sizeof(S), stream) != hipSuccess) {
    throw std::runtime_error("Failed memory allocation of states on the device");
  }
  if (hipMallocAsync(&digests_ptr, subtree_digests_size * number_of_streams * sizeof(S), stream) != hipSuccess) {
    throw std::runtime_error("Failed memory allocation of digests on the device");
  }
  // We should wait for these allocations to finish in order to proceed
  hipStreamSynchronize(stream);

  for (size_t subtree_idx = 0; subtree_idx < number_of_subtrees; subtree_idx++) {
#if !defined(__CUDA_ARCH__) && defined(MERKLE_DEBUG)
    std::cout << "Processing subtree #" << subtree_idx << std::endl;
#endif
    size_t stream_idx = subtree_idx % number_of_streams;
    hipStream_t subtree_stream = streams[stream_idx];

    const S* subtree_leaves = leaves + subtree_idx * subtree_leaves_size;
    S* subtree_state = states_ptr + stream_idx * subtree_state_size;
    S* subtree_digests = digests_ptr + stream_idx * subtree_digests_size;

    // We need to copy the first level from RAM to device
    // The pitch property of hipMemcpy2D will allow us to deal with shape differences
    hipMemcpy2DAsync(
      subtree_state, poseidon.t * sizeof(S),      // Device pointer and device pitch
      subtree_leaves, poseidon.arity * sizeof(S), // Host pointer and pitch
      poseidon.arity * sizeof(S),                 // Size of the source matrix (Arity)
      subtree_leaves_size / poseidon.arity,       // Size of the source matrix (Number of blocks)
      hipMemcpyHostToDevice, subtree_stream);    // Direction and stream

    __build_merkle_subtree<S>(
      subtree_state, subtree_digests, subtree_idx, subtree_leaves_size, digests, number_of_leaves / poseidon.arity,
      poseidon, subtree_stream);
  }

  hipFreeAsync(states_ptr, stream);
  hipFreeAsync(digests_ptr, stream);
  for (size_t i = 0; i < number_of_streams; i++) {
    hipStreamDestroy(streams[i]);
  }
  free(streams);
}
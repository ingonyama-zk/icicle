#include "hip/hip_runtime.h"
#define DEBUG
#define MERKLE_DEBUG
#include "../../curves/bls12_381/curve_config.cuh"
#include "../../curves/bls12_381/merkle.cu"

#ifndef __CUDA_ARCH__
#include <iostream>
#include <chrono>
#include <fstream>
#include <math.h>

void dump(BLS12_381::scalar_t * buf, size_t size, std::string filepath) {
  std::ofstream outfile(filepath, std::ios::binary);
  for (int i = 0; i < size; i++) {
    BLS12_381::scalar_t el = buf[i];
    outfile.write((char *)&el, sizeof(BLS12_381::scalar_t));
  }
  outfile.close();
}

int main(int argc, char* argv[]) {
  using FpMilliseconds = 
    std::chrono::duration<float, std::chrono::milliseconds::period>;
  using FpMicroseconds = 
    std::chrono::duration<float, std::chrono::microseconds::period>;
  
  const int arity = 2;
  const int t = arity + 1;

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start_event, end_event;
  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
  hipEventRecord(start_event, stream);
  auto start_time1 = std::chrono::high_resolution_clock::now();
  Poseidon<BLS12_381::scalar_t> poseidon(arity, stream);

  auto end_time1 = std::chrono::high_resolution_clock::now();
  auto elapsed_time1 = std::chrono::duration_cast<std::chrono::microseconds>(end_time1 - start_time1);
  printf("Elapsed time poseidon: %.0f us\n", FpMicroseconds(elapsed_time1).count());

  uint32_t tree_height = 30;
  uint32_t number_of_leaves = pow(arity, (tree_height - 1));
  uint32_t cutoff_height = 7;
  size_t cutoff_leaves_len = pow(arity, (cutoff_height - 1));
  std::cout << "Cutoff leaves = " << cutoff_leaves_len << std::endl;
  size_t cutoff_digests_len = get_digests_len(cutoff_height, arity);
  std::cout << "Cutoff digests = " << cutoff_digests_len << std::endl;

  auto start_time2 = std::chrono::high_resolution_clock::now();
  BLS12_381::scalar_t input = BLS12_381::scalar_t::zero();
  BLS12_381::scalar_t * leaves = static_cast< BLS12_381::scalar_t * >(malloc(number_of_leaves * sizeof(BLS12_381::scalar_t)));
  // memset(leaves, 1, number_of_leaves * sizeof(BLS12_381::scalar_t));
  for (uint32_t i = 0; i < number_of_leaves; i++) {
    leaves[i] = input;
    input = input + BLS12_381::scalar_t::one();
  }

  std::cout << "INPUT:" << std::endl;
  for (uint32_t i = 0; i < 10; i++) {
    std::cout << leaves[number_of_leaves - 1 - i] << std::endl;
  }

  auto end_time2 = std::chrono::high_resolution_clock::now();
  auto elapsed_time2 = std::chrono::duration_cast<std::chrono::microseconds>(end_time2 - start_time2);
  printf("Leaves allocation: %.0f us\n", FpMicroseconds(elapsed_time2).count());

  size_t leaves_mem = number_of_leaves * sizeof(BLS12_381::scalar_t);
  std::cout << "Memory for leaves = " << leaves_mem / 1024 / 1024 << " MB; " << leaves_mem / 1024 / 1024 / 1024 << " GB" << std::endl;
  std::cout << "Number of leaves = " << number_of_leaves << std::endl;

  auto start_time = std::chrono::high_resolution_clock::now();
  auto digests_len = get_digests_len(tree_height, arity);
  BLS12_381::scalar_t * digests = static_cast< BLS12_381::scalar_t * >(malloc(digests_len * sizeof(BLS12_381::scalar_t)));
  // memset(digests, 0, digests_len * sizeof(BLS12_381::scalar_t));
  size_t digests_mem = digests_len * sizeof(BLS12_381::scalar_t);
  std::cout << "Memory for digests = " << digests_mem / 1024 / 1024 << " MB; " << digests_mem / 1024 / 1024 / 1024 << " GB" << std::endl;
  std::cout << "Number of digest elements = " << digests_len << std::endl;

  std::cout << "Total RAM consumption = " << (digests_mem + leaves_mem) / 1024 / 1024 << " MB; " << (digests_mem + leaves_mem) / 1024 / 1024 / 1024 << " GB" << std::endl;
  build_merkle_tree<BLS12_381::scalar_t>(leaves, digests, tree_height, poseidon, stream);
  auto end_time = std::chrono::high_resolution_clock::now();
  auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
  auto tree_time = FpMicroseconds(elapsed_time).count();
  printf("Elapsed time in merkle tree building: %.0f us, %.0f ms, %.2f s\n", tree_time, tree_time / 1000, tree_time / 1000 / 1000);
  hipEventRecord(end_event, stream);
  hipEventSynchronize(end_event);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start_event, end_event);
  printf("Elapsed time: %8.3f ms\n", elapsedTime);

  // for (int i = 0; i < digests_len; i++) {
  //   std::cout << i << ": " << digests[i] << std::endl;
  // }
  // std::cout << std::endl;

  for (int i = 0; i < 10; i++) {
    std::cout << digests[digests_len - cutoff_digests_len - 1 - i] << std::endl;
  }

  hipEventDestroy(start_event);
  hipEventDestroy(end_event);
  free(leaves);
}

#endif
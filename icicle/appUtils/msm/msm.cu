#include "hip/hip_runtime.h"
#include "msm.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdexcept>
#include <vector>

#include "../../curves/curve_config.cuh"
#include "../../primitives/affine.cuh"
#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include "../../utils/error_handler.cuh"
#include "../../utils/mont.cuh"

namespace msm {

  namespace {

#define MAX_TH 256

    // #define SIGNED_DIG //WIP
    // #define BIG_TRIANGLE
    // #define SSM_SUM  //WIP

    template <typename S>
    int get_optimal_c(int bitsize)
    {
      return max((int)ceil(log2(bitsize)) - 4, 1);
    }

    template <typename P>
    __global__ void single_stage_multi_reduction_kernel(
      P* v,
      P* v_r,
      unsigned block_size,
      unsigned write_stride,
      unsigned write_phase,
      unsigned padding,
      unsigned num_of_threads)
    {
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid >= num_of_threads) { return; }

      int jump = block_size / 2;
      int tid_p = padding ? (tid / (2 * padding)) * padding + tid % padding : tid;
      int block_id = tid_p / jump;
      int block_tid = tid_p % jump;
      unsigned read_ind = block_size * block_id + block_tid;
      unsigned write_ind = tid;
      unsigned v_r_key = write_stride
                           ? ((write_ind / write_stride) * 2 + write_phase) * write_stride + write_ind % write_stride
                           : write_ind;
      P v_r_value = padding ? (tid % (2 * padding) < padding) ? v[read_ind] + v[read_ind + jump] : P::zero()
                            : v[read_ind] + v[read_ind + jump];

      v_r[v_r_key] = v_r_value;
    }

    // this kernel performs single scalar multiplication
    // each thread multilies a single scalar and point
    template <typename P, typename S>
    __global__ void ssm_kernel(S* scalars, P* points, P* results, unsigned N)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid < N) results[tid] = scalars[tid] * points[tid];
    }

    // this kernel sums all the elements in a given vector using multiple threads
    template <typename P>
    __global__ void sum_reduction_kernel(P* v, P* v_r)
    {
      unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

      // Start at 1/2 block stride and divide by two each iteration
      for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
        // Each thread does work unless it is further than the stride
        if (threadIdx.x < s) { v[tid] = v[tid] + v[tid + s]; }
        __syncthreads();
      }

      // Let the thread 0 for this block write the final result
      if (threadIdx.x == 0) { v_r[blockIdx.x] = v[tid]; }
    }

    // this kernel initializes the buckets with zero points
    // each thread initializes a different bucket
    template <typename P>
    __global__ void initialize_buckets_kernel(P* buckets, unsigned N)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid < N) buckets[tid] = P::zero(); // zero point
    }

    // this kernel splits the scalars into digits of size c
    // each thread splits a single scalar into nof_bms digits
    template <typename S>
    __global__ void split_scalars_kernel(
      unsigned* buckets_indices,
      unsigned* point_indices,
      S* scalars,
      unsigned total_size,
      unsigned points_size,
      unsigned msm_size,
      unsigned nof_bms,
      unsigned bm_bitsize,
      unsigned c)
    {
      // constexpr unsigned sign_mask = 0x80000000;
      // constexpr unsigned trash_bucket = 0x80000000;
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      unsigned bucket_index;
      // unsigned bucket_index2;
      unsigned current_index;
      unsigned msm_index = tid / msm_size;
      // unsigned borrow = 0;
      if (tid < total_size) {
        S scalar = scalars[tid];
        for (unsigned bm = 0; bm < nof_bms; bm++) {
          bucket_index = scalar.get_scalar_digit(bm, c);
#ifdef SIGNED_DIG
          bucket_index += borrow;
          borrow = 0;
          unsigned sign = 0;
          if (bucket_index > (1 << (c - 1))) {
            bucket_index = (1 << c) - bucket_index;
            borrow = 1;
            sign = sign_mask;
          }
#endif
          current_index = bm * total_size + tid;
#ifdef SIGNED_DIG
          point_indices[current_index] = sign | tid; // the point index is saved for later
#else
          buckets_indices[current_index] =
            (msm_index << (c + bm_bitsize)) | (bm << c) |
            bucket_index; // the bucket module number and the msm number are appended at the msbs
          if (scalar == S::zero() || bucket_index == 0) buckets_indices[current_index] = 0; // will be skipped
          point_indices[current_index] = tid % points_size; // the point index is saved for later
#endif
        }
      }
    }

    template <typename S>
    __global__ void
    find_cutoff_kernel(unsigned* v, unsigned size, unsigned cutoff, unsigned run_length, unsigned* result)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      const unsigned nof_threads = (size + run_length - 1) / run_length;
      if (tid >= nof_threads) { return; }
      const unsigned start_index = tid * run_length;
      for (int i = start_index; i < min(start_index + run_length, size - 1); i++) {
        if (v[i] > cutoff && v[i + 1] <= cutoff) {
          result[0] = i + 1;
          return;
        }
      }
      if (tid == 0 && v[size - 1] > cutoff) { result[0] = size; }
    }

    template <typename S>
    __global__ void
    find_max_size(unsigned* bucket_sizes, unsigned* single_bucket_indices, unsigned c, unsigned* largest_bucket_size)
    {
      for (int i = 0;; i++) {
        if (single_bucket_indices[i] & ((1 << c) - 1)) {
          largest_bucket_size[0] = bucket_sizes[i];
          largest_bucket_size[1] = i;
          break;
        }
      }
    }

    // this kernel adds up the points in each bucket
    //  __global__ void accumulate_buckets_kernel(P *__restrict__ buckets, unsigned *__restrict__ bucket_offsets,
    //   unsigned *__restrict__ bucket_sizes, unsigned *__restrict__ single_bucket_indices, unsigned *__restrict__
    //   point_indices, A *__restrict__ points, unsigned nof_buckets, unsigned batch_size, unsigned msm_idx_shift){
    template <typename P, typename A>
    __global__ void accumulate_buckets_kernel(
      P* __restrict__ buckets,
      unsigned* __restrict__ bucket_offsets,
      unsigned* __restrict__ bucket_sizes,
      unsigned* __restrict__ single_bucket_indices,
      const unsigned* __restrict__ point_indices,
      A* __restrict__ points,
      const unsigned nof_buckets,
      const unsigned nof_buckets_to_compute,
      const unsigned msm_idx_shift,
      const unsigned c)
    {
      // constexpr unsigned sign_mask = 0x80000000;
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid >= nof_buckets_to_compute) return;
      if ((single_bucket_indices[tid] & ((1 << c) - 1)) == 0) {
        return; // skip zero buckets
      }
#ifdef SIGNED_DIG // todo - fix
      const unsigned msm_index = single_bucket_indices[tid] >> msm_idx_shift;
      const unsigned bm_index = (single_bucket_indices[tid] & ((1 << msm_idx_shift) - 1)) >> c;
      const unsigned bucket_index =
        msm_index * nof_buckets + bm_index * ((1 << (c - 1)) + 1) + (single_bucket_indices[tid] & ((1 << c) - 1));
#else
      unsigned msm_index = single_bucket_indices[tid] >> msm_idx_shift;
      unsigned bucket_index = msm_index * nof_buckets + (single_bucket_indices[tid] & ((1 << msm_idx_shift) - 1));
#endif
      const unsigned bucket_offset = bucket_offsets[tid];
      const unsigned bucket_size = bucket_sizes[tid];

      P bucket; // get rid of init buckets? no.. because what about buckets with no points
      for (unsigned i = 0; i < bucket_size;
           i++) { // add the relevant points starting from the relevant offset up to the bucket size
        unsigned point_ind = point_indices[bucket_offset + i];
#ifdef SIGNED_DIG
        unsigned sign = point_ind & sign_mask;
        point_ind &= ~sign_mask;
        A point = points[point_ind];
        if (sign) point = A::neg(point);
#else
        A point = points[point_ind];
#endif
        bucket = i ? bucket + point : P::from_affine(point);
      }
      buckets[bucket_index] = bucket;
    }

    template <typename P, typename A>
    __global__ void accumulate_large_buckets_kernel(
      P* __restrict__ buckets,
      unsigned* __restrict__ bucket_offsets,
      unsigned* __restrict__ bucket_sizes,
      unsigned* __restrict__ single_bucket_indices,
      const unsigned* __restrict__ point_indices,
      A* __restrict__ points,
      const unsigned nof_buckets,
      const unsigned nof_buckets_to_compute,
      const unsigned msm_idx_shift,
      const unsigned c,
      const unsigned threads_per_bucket,
      const unsigned max_run_length)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      unsigned large_bucket_index = tid / threads_per_bucket;
      unsigned bucket_segment_index = tid % threads_per_bucket;
      if (tid >= nof_buckets_to_compute * threads_per_bucket) { return; }
      if ((single_bucket_indices[large_bucket_index] & ((1 << c) - 1)) == 0) { // dont need
        return;                                                                // skip zero buckets
      }
      unsigned write_bucket_index = bucket_segment_index * nof_buckets_to_compute + large_bucket_index;
      const unsigned bucket_offset = bucket_offsets[large_bucket_index] + bucket_segment_index * max_run_length;
      const unsigned bucket_size = bucket_sizes[large_bucket_index] > bucket_segment_index * max_run_length
                                     ? bucket_sizes[large_bucket_index] - bucket_segment_index * max_run_length
                                     : 0;
      P bucket;
      unsigned run_length = min(bucket_size, max_run_length);
      for (unsigned i = 0; i < run_length;
           i++) { // add the relevant points starting from the relevant offset up to the bucket size
        unsigned point_ind = point_indices[bucket_offset + i];
        A point = points[point_ind];
        bucket = i ? bucket + point : P::from_affine(point); // init empty buckets
      }
      buckets[write_bucket_index] = run_length ? bucket : P::zero();
    }

    template <typename P>
    __global__ void
    distribute_large_buckets_kernel(P* large_buckets, P* buckets, unsigned* single_bucket_indices, unsigned size)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid >= size) { return; }
      buckets[single_bucket_indices[tid]] = large_buckets[tid];
    }

    // this kernel sums the entire bucket module
    // each thread deals with a single bucket module
    template <typename P>
    __global__ void big_triangle_sum_kernel(P* buckets, P* final_sums, unsigned nof_bms, unsigned c)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid >= nof_bms) return;
#ifdef SIGNED_DIG
      unsigned buckets_in_bm = (1 << c) + 1;
#else
      unsigned buckets_in_bm = (1 << c);
#endif
      P line_sum = buckets[(tid + 1) * buckets_in_bm - 1];
      final_sums[tid] = line_sum;
      for (unsigned i = buckets_in_bm - 2; i > 0; i--) {
        line_sum = line_sum + buckets[tid * buckets_in_bm + i]; // using the running sum method
        final_sums[tid] = final_sums[tid] + line_sum;
      }
    }

    // this kernel uses single scalar multiplication to multiply each bucket by its index
    // each thread deals with a single bucket
    template <typename P, typename S>
    __global__ void ssm_buckets_kernel(P* buckets, unsigned* single_bucket_indices, unsigned nof_buckets, unsigned c)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid > nof_buckets) return;
      unsigned bucket_index = single_bucket_indices[tid];
      S scalar_bucket_multiplier;
      scalar_bucket_multiplier = {
        bucket_index & ((1 << c) - 1), 0, 0, 0, 0, 0, 0, 0}; // the index without the bucket module index
      buckets[bucket_index] = scalar_bucket_multiplier * buckets[bucket_index];
    }

    template <typename P>
    __global__ void last_pass_kernel(P* final_buckets, P* final_sums, unsigned num_sums)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid > num_sums) return;
      final_sums[tid] = final_buckets[2 * tid + 1];
    }

    // this kernel computes the final result using the double and add algorithm
    // it is done by a single thread
    template <typename P, typename S>
    __global__ void
    final_accumulation_kernel(P* final_sums, P* final_results, unsigned nof_msms, unsigned nof_bms, unsigned c)
    {
      unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
      if (tid > nof_msms) return;
      P final_result = P::zero();
      for (unsigned i = nof_bms; i > 1; i--) {
        final_result = final_result + final_sums[i - 1 + tid * nof_bms]; // add
        for (unsigned j = 0; j < c; j++)                                 // double
        {
          final_result = final_result + final_result;
        }
      }
      final_results[tid] = final_result + final_sums[tid * nof_bms];
    }

    // this function computes msm using the bucket method
    template <typename S, typename P, typename A>
    hipError_t bucket_method_msm(
      int bitsize,
      int c,
      S* scalars,
      A* points,
      int size,
      P* final_result,
      bool are_scalars_on_device,
      bool are_scalars_montgomery_form,
      bool are_points_on_device,
      bool are_points_montgomery_form,
      bool is_result_on_device,
      bool is_big_triangle,
      int large_bucket_factor,
      bool is_async,
      hipStream_t stream)
    {
      CHK_INIT_IF_RETURN();

      S* d_scalars;
      A* d_points;
      if (!are_scalars_on_device) {
        // copy scalars to gpu
        CHK_IF_RETURN(hipMallocAsync(&d_scalars, sizeof(S) * size, stream));
        CHK_IF_RETURN(hipMemcpyAsync(d_scalars, scalars, sizeof(S) * size, hipMemcpyHostToDevice, stream));
      } else {
        d_scalars = scalars;
      }
      hipStream_t stream_points;
      if (!are_points_on_device || are_points_montgomery_form) CHK_IF_RETURN(hipStreamCreate(&stream_points));
      if (!are_points_on_device) {
        // copy points to gpu
        CHK_IF_RETURN(hipMallocAsync(&d_points, sizeof(A) * size, stream_points));
        CHK_IF_RETURN(hipMemcpyAsync(d_points, points, sizeof(A) * size, hipMemcpyHostToDevice, stream_points));
      } else {
        d_points = points;
      }
      if (are_scalars_montgomery_form) CHK_IF_RETURN(mont::FromMontgomery(d_scalars, size, stream));
      if (are_points_montgomery_form) CHK_IF_RETURN(mont::FromMontgomery(d_points, size, stream_points));
      hipEvent_t event_points_uploaded;
      if (!are_points_on_device || are_points_montgomery_form) {
        CHK_IF_RETURN(hipEventCreateWithFlags(&event_points_uploaded, hipEventDisableTiming));
        CHK_IF_RETURN(hipEventRecord(event_points_uploaded, stream_points));
      }

      P* buckets;
      // compute number of bucket modules and number of buckets in each module
      unsigned nof_bms = (bitsize + c - 1) / c;
      unsigned bm_bitsize = (unsigned)ceil(log2(nof_bms));
#ifdef SIGNED_DIG
      unsigned nof_buckets = nof_bms * ((1 << (c - 1)) + 1); // signed digits
#else
      unsigned nof_buckets = nof_bms << c;
#endif
      CHK_IF_RETURN(hipMallocAsync(&buckets, sizeof(P) * nof_buckets, stream));

      // launch the bucket initialization kernel with maximum threads
      unsigned NUM_THREADS = 1 << 10;
      unsigned NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
      initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, nof_buckets);

      unsigned* bucket_indices;
      unsigned* point_indices;
      CHK_IF_RETURN(hipMallocAsync(&bucket_indices, sizeof(unsigned) * size * (nof_bms + 1), stream));
      CHK_IF_RETURN(hipMallocAsync(&point_indices, sizeof(unsigned) * size * (nof_bms + 1), stream));

      // split scalars into digits
      NUM_THREADS = 1 << 10;
      NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
      split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
        bucket_indices + size, point_indices + size, d_scalars, size, size, size, nof_bms, bm_bitsize, c);
      //+size - leaving the first bm free for the out of place sort later

      // sort indices - the indices are sorted from smallest to largest in order to group together the points that
      // belong to each bucket
      unsigned* sort_indices_temp_storage{};
      size_t sort_indices_temp_storage_bytes;
      // The second to last parameter is the default value supplied explicitly to allow passing the stream
      // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for
      // more info
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairs(
        sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + size, bucket_indices,
        point_indices + size, point_indices, size, 0, sizeof(unsigned) * 8, stream));
      CHK_IF_RETURN(hipMallocAsync(&sort_indices_temp_storage, sort_indices_temp_storage_bytes, stream));
      for (unsigned i = 0; i < nof_bms; i++) {
        unsigned offset_out = i * size;
        unsigned offset_in = offset_out + size;
        // The second to last parameter is the default value supplied explicitly to allow passing the stream
        // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for
        // more info
        CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairs(
          sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + offset_in,
          bucket_indices + offset_out, point_indices + offset_in, point_indices + offset_out, size, 0,
          sizeof(unsigned) * 8, stream));
      }
      CHK_IF_RETURN(hipFreeAsync(sort_indices_temp_storage, stream));

      // find bucket_sizes
      unsigned* single_bucket_indices;
      unsigned* bucket_sizes;
      unsigned* nof_buckets_to_compute;
      CHK_IF_RETURN(hipMallocAsync(&single_bucket_indices, sizeof(unsigned) * nof_buckets, stream));
      CHK_IF_RETURN(hipMallocAsync(&bucket_sizes, sizeof(unsigned) * nof_buckets, stream));
      CHK_IF_RETURN(hipMallocAsync(&nof_buckets_to_compute, sizeof(unsigned), stream));
      unsigned* encode_temp_storage{};
      size_t encode_temp_storage_bytes = 0;
      CHK_IF_RETURN(hipcub::DeviceRunLengthEncode::Encode(
        encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
        nof_buckets_to_compute, nof_bms * size, stream));
      CHK_IF_RETURN(hipMallocAsync(&encode_temp_storage, encode_temp_storage_bytes, stream));
      CHK_IF_RETURN(hipcub::DeviceRunLengthEncode::Encode(
        encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
        nof_buckets_to_compute, nof_bms * size, stream));
      CHK_IF_RETURN(hipFreeAsync(encode_temp_storage, stream));

      // get offsets - where does each new bucket begin
      unsigned* bucket_offsets;
      CHK_IF_RETURN(hipMallocAsync(&bucket_offsets, sizeof(unsigned) * nof_buckets, stream));
      unsigned* offsets_temp_storage{};
      size_t offsets_temp_storage_bytes = 0;
      CHK_IF_RETURN(hipcub::DeviceScan::ExclusiveSum(
        offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets, stream));
      CHK_IF_RETURN(hipMallocAsync(&offsets_temp_storage, offsets_temp_storage_bytes, stream));
      CHK_IF_RETURN(hipcub::DeviceScan::ExclusiveSum(
        offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets, stream));
      CHK_IF_RETURN(hipFreeAsync(offsets_temp_storage, stream));

      // sort by bucket sizes
      unsigned h_nof_buckets_to_compute;
      CHK_IF_RETURN(hipMemcpyAsync(
        &h_nof_buckets_to_compute, nof_buckets_to_compute, sizeof(unsigned), hipMemcpyDeviceToHost, stream));

      // if all points are 0 just return point 0
      if (h_nof_buckets_to_compute == 0) {
        if (!is_result_on_device)
          final_result[0] = P::zero();
        else {
          P* h_final_result = (P*)malloc(sizeof(P));
          h_final_result[0] = P::zero();
          CHK_IF_RETURN(hipMemcpyAsync(final_result, h_final_result, sizeof(P), hipMemcpyHostToDevice, stream));
        }

        return CHK_LAST();
      }

      unsigned* sorted_bucket_sizes;
      CHK_IF_RETURN(hipMallocAsync(&sorted_bucket_sizes, sizeof(unsigned) * h_nof_buckets_to_compute, stream));
      unsigned* sorted_bucket_offsets;
      CHK_IF_RETURN(hipMallocAsync(&sorted_bucket_offsets, sizeof(unsigned) * h_nof_buckets_to_compute, stream));
      unsigned* sort_offsets_temp_storage{};
      size_t sort_offsets_temp_storage_bytes = 0;
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairsDescending(
        sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, bucket_sizes, sorted_bucket_sizes, bucket_offsets,
        sorted_bucket_offsets, h_nof_buckets_to_compute, 0, sizeof(unsigned) * 8, stream));
      CHK_IF_RETURN(hipMallocAsync(&sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, stream));
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairsDescending(
        sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, bucket_sizes, sorted_bucket_sizes, bucket_offsets,
        sorted_bucket_offsets, h_nof_buckets_to_compute, 0, sizeof(unsigned) * 8, stream));
      CHK_IF_RETURN(hipFreeAsync(sort_offsets_temp_storage, stream));

      unsigned* sorted_single_bucket_indices;
      CHK_IF_RETURN(
        hipMallocAsync(&sorted_single_bucket_indices, sizeof(unsigned) * h_nof_buckets_to_compute, stream));
      unsigned* sort_single_temp_storage{};
      size_t sort_single_temp_storage_bytes = 0;
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairsDescending(
        sort_single_temp_storage, sort_single_temp_storage_bytes, bucket_sizes, sorted_bucket_sizes,
        single_bucket_indices, sorted_single_bucket_indices, h_nof_buckets_to_compute, 0, sizeof(unsigned) * 8,
        stream));
      CHK_IF_RETURN(hipMallocAsync(&sort_single_temp_storage, sort_single_temp_storage_bytes, stream));
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairsDescending(
        sort_single_temp_storage, sort_single_temp_storage_bytes, bucket_sizes, sorted_bucket_sizes,
        single_bucket_indices, sorted_single_bucket_indices, h_nof_buckets_to_compute, 0, sizeof(unsigned) * 8,
        stream));
      CHK_IF_RETURN(hipFreeAsync(sort_single_temp_storage, stream));

      // find large buckets
      unsigned avarage_size = size / (1 << c);
      unsigned bucket_th = large_bucket_factor * avarage_size;
      unsigned* nof_large_buckets;
      CHK_IF_RETURN(hipMallocAsync(&nof_large_buckets, sizeof(unsigned), stream));
      CHK_IF_RETURN(hipMemset(nof_large_buckets, 0, sizeof(unsigned)));

      unsigned TOTAL_THREADS = 129000; // todo - device dependant
      unsigned cutoff_run_length = max(2, h_nof_buckets_to_compute / TOTAL_THREADS);
      unsigned cutoff_nof_runs = (h_nof_buckets_to_compute + cutoff_run_length - 1) / cutoff_run_length;
      NUM_THREADS = min(1 << 5, cutoff_nof_runs);
      NUM_BLOCKS = (cutoff_nof_runs + NUM_THREADS - 1) / NUM_THREADS;
      find_cutoff_kernel<S><<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
        sorted_bucket_sizes, h_nof_buckets_to_compute, bucket_th, cutoff_run_length, nof_large_buckets);

      unsigned h_nof_large_buckets;
      CHK_IF_RETURN(
        hipMemcpyAsync(&h_nof_large_buckets, nof_large_buckets, sizeof(unsigned), hipMemcpyDeviceToHost, stream));

      unsigned* max_res;
      CHK_IF_RETURN(hipMallocAsync(&max_res, sizeof(unsigned) * 2, stream));
      find_max_size<S><<<1, 1, 0, stream>>>(sorted_bucket_sizes, sorted_single_bucket_indices, c, max_res);

      unsigned h_max_res[2];
      CHK_IF_RETURN(hipMemcpyAsync(h_max_res, max_res, sizeof(unsigned) * 2, hipMemcpyDeviceToHost, stream));
      unsigned h_largest_bucket_size = h_max_res[0];
      unsigned h_nof_zero_large_buckets = h_max_res[1];
      unsigned large_buckets_to_compute =
        h_nof_large_buckets > h_nof_zero_large_buckets ? h_nof_large_buckets - h_nof_zero_large_buckets : 0;

      if (!are_points_on_device || are_points_montgomery_form) {
        // by this point, points need to be already uploaded and un-Montgomeried
        CHK_IF_RETURN(hipStreamWaitEvent(stream, event_points_uploaded));
        CHK_IF_RETURN(hipStreamDestroy(stream_points));
      }

      hipStream_t stream_large_buckets;
      hipEvent_t event_large_buckets_accumulated;
      P* large_buckets;
      if (large_buckets_to_compute > 0 && bucket_th > 0) {
        CHK_IF_RETURN(hipStreamCreate(&stream_large_buckets));
        CHK_IF_RETURN(hipEventCreateWithFlags(&event_large_buckets_accumulated, hipEventDisableTiming));

        unsigned threads_per_bucket =
          1 << (unsigned)ceil(log2((h_largest_bucket_size + bucket_th - 1) / bucket_th)); // global param
        unsigned max_bucket_size_run_length = (h_largest_bucket_size + threads_per_bucket - 1) / threads_per_bucket;
        unsigned total_large_buckets_size = large_buckets_to_compute * threads_per_bucket;
        CHK_IF_RETURN(hipMallocAsync(&large_buckets, sizeof(P) * total_large_buckets_size, stream));

        NUM_THREADS = min(1 << 8, total_large_buckets_size);
        NUM_BLOCKS = (total_large_buckets_size + NUM_THREADS - 1) / NUM_THREADS;
        accumulate_large_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream_large_buckets>>>(
          large_buckets, sorted_bucket_offsets + h_nof_zero_large_buckets,
          sorted_bucket_sizes + h_nof_zero_large_buckets, sorted_single_bucket_indices + h_nof_zero_large_buckets,
          point_indices, d_points, nof_buckets, large_buckets_to_compute, c + bm_bitsize, c, threads_per_bucket,
          max_bucket_size_run_length);

        // reduce
        for (int s = total_large_buckets_size >> 1; s > large_buckets_to_compute - 1; s >>= 1) {
          NUM_THREADS = min(MAX_TH, s);
          NUM_BLOCKS = (s + NUM_THREADS - 1) / NUM_THREADS;
          single_stage_multi_reduction_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream_large_buckets>>>(
            large_buckets, large_buckets, s * 2, 0, 0, 0, s);
        }

        // distribute
        NUM_THREADS = min(MAX_TH, large_buckets_to_compute);
        NUM_BLOCKS = (large_buckets_to_compute + NUM_THREADS - 1) / NUM_THREADS;
        distribute_large_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream_large_buckets>>>(
          large_buckets, buckets, sorted_single_bucket_indices + h_nof_zero_large_buckets, large_buckets_to_compute);

        CHK_IF_RETURN(hipEventRecord(event_large_buckets_accumulated, stream_large_buckets));
        CHK_IF_RETURN(hipStreamDestroy(stream_large_buckets));
      } else {
        h_nof_large_buckets = 0;
      }

      // launch the accumulation kernel with maximum threads
      if (h_nof_buckets_to_compute > h_nof_large_buckets) {
        NUM_THREADS = 1 << 8;
        NUM_BLOCKS = (h_nof_buckets_to_compute - h_nof_large_buckets + NUM_THREADS - 1) / NUM_THREADS;
        accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
          buckets, sorted_bucket_offsets + h_nof_large_buckets, sorted_bucket_sizes + h_nof_large_buckets,
          sorted_single_bucket_indices + h_nof_large_buckets, point_indices, d_points, nof_buckets,
          h_nof_buckets_to_compute - h_nof_large_buckets, c + bm_bitsize, c);
      }

      if (large_buckets_to_compute > 0 && bucket_th > 0)
        // all the large buckets need to be accumulated before the final summation
        CHK_IF_RETURN(hipStreamWaitEvent(stream, event_large_buckets_accumulated));

#ifdef SSM_SUM
      // sum each bucket
      NUM_THREADS = 1 << 10;
      NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
      ssm_buckets_kernel<fake_point, fake_scalar>
        <<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, single_bucket_indices, nof_buckets, c);

      // sum each bucket module
      P* final_results;
      CHK_IF_RETURN(hipMallocAsync(&final_results, sizeof(P) * nof_bms, stream));
      NUM_THREADS = 1 << c;
      NUM_BLOCKS = nof_bms;
      sum_reduction_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, final_results);
#endif

      P* d_final_result;
      if (!is_result_on_device) CHK_IF_RETURN(hipMallocAsync(&d_final_result, sizeof(P), stream));

      P* final_results;
      if (is_big_triangle) {
        CHK_IF_RETURN(hipMallocAsync(&final_results, sizeof(P) * nof_bms, stream));
        // launch the bucket module sum kernel - a thread for each bucket module
        NUM_THREADS = nof_bms;
        NUM_BLOCKS = 1;
#ifdef SIGNED_DIG
        big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
          buckets, final_results, nof_bms, c - 1); // sighed digits
#else
        big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, final_results, nof_bms, c);
#endif
      } else {
        unsigned source_bits_count = c;
        // bool odd_source_c = source_bits_count % 2;
        unsigned source_windows_count = nof_bms;
        unsigned source_buckets_count = nof_buckets;
        P* source_buckets = buckets;
        buckets = nullptr;
        P* target_buckets;
        P* temp_buckets1;
        P* temp_buckets2;
        for (unsigned i = 0;; i++) {
          const unsigned target_bits_count = (source_bits_count + 1) >> 1;                 // c/2=8
          const unsigned target_windows_count = source_windows_count << 1;                 // nof bms*2 = 32
          const unsigned target_buckets_count = target_windows_count << target_bits_count; // bms*2^c = 32*2^8
          CHK_IF_RETURN(
            hipMallocAsync(&target_buckets, sizeof(P) * target_buckets_count, stream)); // 32*2^8*2^7 buckets
          CHK_IF_RETURN(
            hipMallocAsync(&temp_buckets1, sizeof(P) * source_buckets_count / 2, stream)); // 32*2^8*2^7 buckets
          CHK_IF_RETURN(
            hipMallocAsync(&temp_buckets2, sizeof(P) * source_buckets_count / 2, stream)); // 32*2^8*2^7 buckets

          if (source_bits_count > 0) {
            for (unsigned j = 0; j < target_bits_count; j++) {
              // unsigned last_j = target_bits_count - 1;
              unsigned nof_threads = (source_buckets_count >> (1 + j));
              NUM_THREADS = min(MAX_TH, nof_threads);
              NUM_BLOCKS = (nof_threads + NUM_THREADS - 1) / NUM_THREADS;
              single_stage_multi_reduction_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
                j == 0 ? source_buckets : temp_buckets1, j == target_bits_count - 1 ? target_buckets : temp_buckets1,
                1 << (source_bits_count - j), j == target_bits_count - 1 ? 1 << target_bits_count : 0, 0, 0,
                nof_threads);

              NUM_THREADS = min(MAX_TH, nof_threads);
              NUM_BLOCKS = (nof_threads + NUM_THREADS - 1) / NUM_THREADS;
              single_stage_multi_reduction_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
                j == 0 ? source_buckets : temp_buckets2, j == target_bits_count - 1 ? target_buckets : temp_buckets2,
                1 << (target_bits_count - j), j == target_bits_count - 1 ? 1 << target_bits_count : 0, 1, 0,
                nof_threads);
            }
          }
          if (target_bits_count == 1) {
            nof_bms = bitsize;
            CHK_IF_RETURN(hipMallocAsync(&final_results, sizeof(P) * nof_bms, stream));
            NUM_THREADS = 32;
            NUM_BLOCKS = (nof_bms + NUM_THREADS - 1) / NUM_THREADS;
            last_pass_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(target_buckets, final_results, nof_bms);
            c = 1;
            CHK_IF_RETURN(hipFreeAsync(source_buckets, stream));
            CHK_IF_RETURN(hipFreeAsync(target_buckets, stream));
            CHK_IF_RETURN(hipFreeAsync(temp_buckets1, stream));
            CHK_IF_RETURN(hipFreeAsync(temp_buckets2, stream));
            break;
          }
          CHK_IF_RETURN(hipFreeAsync(source_buckets, stream));
          CHK_IF_RETURN(hipFreeAsync(temp_buckets1, stream));
          CHK_IF_RETURN(hipFreeAsync(temp_buckets2, stream));
          source_buckets = target_buckets;
          target_buckets = nullptr;
          temp_buckets1 = nullptr;
          temp_buckets2 = nullptr;
          source_bits_count = target_bits_count;
          // odd_source_c = source_bits_count % 2;
          source_windows_count = target_windows_count;
          source_buckets_count = target_buckets_count;
        }
      }

      // launch the double and add kernel, a single thread
      final_accumulation_kernel<P, S>
        <<<1, 1, 0, stream>>>(final_results, is_result_on_device ? final_result : d_final_result, 1, nof_bms, c);
      CHK_IF_RETURN(hipFreeAsync(final_results, stream));

      if (!is_result_on_device)
        CHK_IF_RETURN(hipMemcpyAsync(final_result, d_final_result, sizeof(P), hipMemcpyDeviceToHost, stream));

      // free memory
      if (!are_scalars_on_device) CHK_IF_RETURN(hipFreeAsync(d_scalars, stream));
      if (!are_points_on_device) CHK_IF_RETURN(hipFreeAsync(d_points, stream));
      if (!is_result_on_device) CHK_IF_RETURN(hipFreeAsync(d_final_result, stream));
      CHK_IF_RETURN(hipFreeAsync(buckets, stream));
#ifndef PHASE1_TEST
      CHK_IF_RETURN(hipFreeAsync(bucket_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(point_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(single_bucket_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(bucket_sizes, stream));
      CHK_IF_RETURN(hipFreeAsync(nof_buckets_to_compute, stream));
      CHK_IF_RETURN(hipFreeAsync(bucket_offsets, stream));
#endif
      CHK_IF_RETURN(hipFreeAsync(sorted_bucket_sizes, stream));
      CHK_IF_RETURN(hipFreeAsync(sorted_bucket_offsets, stream));
      CHK_IF_RETURN(hipFreeAsync(sorted_single_bucket_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(nof_large_buckets, stream));
      CHK_IF_RETURN(hipFreeAsync(max_res, stream));
      if (large_buckets_to_compute > 0 && bucket_th > 0) CHK_IF_RETURN(hipFreeAsync(large_buckets, stream));

      if (!is_async) CHK_IF_RETURN(hipStreamSynchronize(stream));

      return CHK_LAST();
    }

    // this function computes multiple msms using the bucket method
    template <typename S, typename P, typename A>
    hipError_t batched_bucket_method_msm(
      unsigned bitsize,
      unsigned c,
      S* scalars,
      A* points,
      unsigned batch_size,
      unsigned msm_size,
      unsigned points_size,
      P* final_results,
      bool are_scalars_on_device,
      bool are_scalars_montgomery_form,
      bool are_points_on_device,
      bool are_points_montgomery_form,
      bool are_results_on_device,
      bool is_async,
      hipStream_t stream)
    {
      CHK_INIT_IF_RETURN();

      unsigned total_size = batch_size * msm_size;
      S* d_scalars;
      A* d_points;
      if (!are_scalars_on_device) {
        // copy scalars to gpu
        CHK_IF_RETURN(hipMallocAsync(&d_scalars, sizeof(S) * total_size, stream));
        CHK_IF_RETURN(hipMemcpyAsync(d_scalars, scalars, sizeof(S) * total_size, hipMemcpyHostToDevice, stream));
      } else {
        d_scalars = scalars;
      }
      hipStream_t stream_points;
      if (!are_points_on_device || are_points_montgomery_form) hipStreamCreate(&stream_points);
      if (!are_points_on_device) {
        // copy points to gpu
        hipMallocAsync(&d_points, sizeof(A) * points_size, stream_points);
        hipMemcpyAsync(d_points, points, sizeof(A) * points_size, hipMemcpyHostToDevice, stream_points);
      } else {
        d_points = points;
      }
      if (are_scalars_montgomery_form) mont::FromMontgomery(d_scalars, total_size, stream);
      if (are_points_montgomery_form) mont::FromMontgomery(d_points, points_size, stream_points);
      hipEvent_t event_points_uploaded;
      if (!are_points_on_device || are_points_montgomery_form) {
        hipEventCreateWithFlags(&event_points_uploaded, hipEventDisableTiming);
        hipEventRecord(event_points_uploaded, stream_points);
      }

      P* buckets;
      // compute number of bucket modules and number of buckets in each module
      unsigned nof_bms = (bitsize + c - 1) / c;
      unsigned bm_bitsize = (unsigned)ceil(log2(nof_bms));
      unsigned nof_buckets = (nof_bms << c);
      unsigned total_nof_buckets = nof_buckets * batch_size;
      CHK_IF_RETURN(hipMallocAsync(&buckets, sizeof(P) * total_nof_buckets, stream));

      // lanch the bucket initialization kernel with maximum threads
      unsigned NUM_THREADS = 1 << 10;
      unsigned NUM_BLOCKS = (total_nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
      initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, total_nof_buckets);

      unsigned* bucket_indices;
      unsigned* point_indices;
      CHK_IF_RETURN(hipMallocAsync(&bucket_indices, sizeof(unsigned) * total_size * nof_bms, stream));
      CHK_IF_RETURN(hipMallocAsync(&point_indices, sizeof(unsigned) * total_size * nof_bms, stream));

      // split scalars into digits
      NUM_THREADS = 1 << 10;
      NUM_BLOCKS = (total_size + NUM_THREADS - 1) / NUM_THREADS;
      split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
        bucket_indices, point_indices, d_scalars, total_size, points_size, msm_size, nof_bms, bm_bitsize, c);

      // sort indices - the indices are sorted from smallest to largest in order to group together the points that
      // belong to each bucket
      unsigned* sorted_bucket_indices;
      unsigned* sorted_point_indices;
      CHK_IF_RETURN(hipMallocAsync(&sorted_bucket_indices, sizeof(unsigned) * (total_size * nof_bms), stream));
      CHK_IF_RETURN(hipMallocAsync(&sorted_point_indices, sizeof(unsigned) * (total_size * nof_bms), stream));

      unsigned* sort_indices_temp_storage{};
      size_t sort_indices_temp_storage_bytes;
      // The second to last parameter is the default value supplied explicitly to allow passing the stream
      // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for
      // more info
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairs(
        sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices, sorted_bucket_indices,
        point_indices, sorted_point_indices, total_size * nof_bms, 0, sizeof(unsigned) * 8, stream));
      CHK_IF_RETURN(hipMallocAsync(&sort_indices_temp_storage, sort_indices_temp_storage_bytes, stream));
      // The second to last parameter is the default value supplied explicitly to allow passing the stream
      // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for
      // more info
      CHK_IF_RETURN(hipcub::DeviceRadixSort::SortPairs(
        sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices, sorted_bucket_indices,
        point_indices, sorted_point_indices, total_size * nof_bms, 0, sizeof(unsigned) * 8, stream));
      CHK_IF_RETURN(hipFreeAsync(sort_indices_temp_storage, stream));

      // find bucket_sizes
      unsigned* single_bucket_indices;
      unsigned* bucket_sizes;
      unsigned* total_nof_buckets_to_compute;
      CHK_IF_RETURN(hipMallocAsync(&single_bucket_indices, sizeof(unsigned) * total_nof_buckets, stream));
      CHK_IF_RETURN(hipMallocAsync(&bucket_sizes, sizeof(unsigned) * total_nof_buckets, stream));
      CHK_IF_RETURN(hipMallocAsync(&total_nof_buckets_to_compute, sizeof(unsigned), stream));
      unsigned* encode_temp_storage{};
      size_t encode_temp_storage_bytes = 0;
      CHK_IF_RETURN(hipcub::DeviceRunLengthEncode::Encode(
        encode_temp_storage, encode_temp_storage_bytes, sorted_bucket_indices, single_bucket_indices, bucket_sizes,
        total_nof_buckets_to_compute, nof_bms * total_size, stream));
      CHK_IF_RETURN(hipMallocAsync(&encode_temp_storage, encode_temp_storage_bytes, stream));
      CHK_IF_RETURN(hipcub::DeviceRunLengthEncode::Encode(
        encode_temp_storage, encode_temp_storage_bytes, sorted_bucket_indices, single_bucket_indices, bucket_sizes,
        total_nof_buckets_to_compute, nof_bms * total_size, stream));
      CHK_IF_RETURN(hipFreeAsync(encode_temp_storage, stream));

      // get offsets - where does each new bucket begin
      unsigned* bucket_offsets;
      CHK_IF_RETURN(hipMallocAsync(&bucket_offsets, sizeof(unsigned) * total_nof_buckets, stream));
      unsigned* offsets_temp_storage{};
      size_t offsets_temp_storage_bytes = 0;
      CHK_IF_RETURN(hipcub::DeviceScan::ExclusiveSum(
        offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, total_nof_buckets, stream));
      CHK_IF_RETURN(hipMallocAsync(&offsets_temp_storage, offsets_temp_storage_bytes, stream));
      CHK_IF_RETURN(hipcub::DeviceScan::ExclusiveSum(
        offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, total_nof_buckets, stream));
      CHK_IF_RETURN(hipFreeAsync(offsets_temp_storage, stream));

      unsigned h_nof_buckets_to_compute;
      CHK_IF_RETURN(hipMemcpyAsync(
        &h_nof_buckets_to_compute, total_nof_buckets_to_compute, sizeof(unsigned), hipMemcpyDeviceToHost, stream));

      if (!are_points_on_device || are_points_montgomery_form) {
        // by this point, points need to be already uploaded and un-Montgomeried
        hipStreamWaitEvent(stream, event_points_uploaded);
        hipStreamDestroy(stream_points);
      }

      // launch the accumulation kernel with maximum threads
      NUM_THREADS = 1 << 8;
      NUM_BLOCKS = (total_nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
      accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
        buckets, bucket_offsets, bucket_sizes, single_bucket_indices, sorted_point_indices, d_points, nof_buckets,
        h_nof_buckets_to_compute, c + bm_bitsize, c);

      // #ifdef SSM_SUM
      //   //sum each bucket
      //   NUM_THREADS = 1 << 10;
      //   NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
      //   ssm_buckets_kernel<P, S><<<NUM_BLOCKS, NUM_THREADS>>>(buckets, single_bucket_indices, nof_buckets, c);

      //   //sum each bucket module
      //   P* final_results;
      //   hipMalloc(&final_results, sizeof(P) * nof_bms);
      //   NUM_THREADS = 1<<c;
      //   NUM_BLOCKS = nof_bms;
      //   sum_reduction_kernel<<<NUM_BLOCKS,NUM_THREADS>>>(buckets, final_results);
      // #endif

      // #ifdef BIG_TRIANGLE
      P* bm_sums;
      CHK_IF_RETURN(hipMallocAsync(&bm_sums, sizeof(P) * nof_bms * batch_size, stream));
      // launch the bucket module sum kernel - a thread for each bucket module
      NUM_THREADS = 1 << 8;
      NUM_BLOCKS = (nof_bms * batch_size + NUM_THREADS - 1) / NUM_THREADS;
      big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, bm_sums, nof_bms * batch_size, c);
      // #endif

      P* d_final_results;
      if (!are_results_on_device) CHK_IF_RETURN(hipMallocAsync(&d_final_results, sizeof(P) * batch_size, stream));

      // launch the double and add kernel, a single thread for each msm
      NUM_THREADS = 1 << 8;
      NUM_BLOCKS = (batch_size + NUM_THREADS - 1) / NUM_THREADS;
      final_accumulation_kernel<P, S><<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(
        bm_sums, are_results_on_device ? final_results : d_final_results, batch_size, nof_bms, c);

      // copy final result to host
      if (!are_results_on_device)
        CHK_IF_RETURN(
          hipMemcpyAsync(final_results, d_final_results, sizeof(P) * batch_size, hipMemcpyDeviceToHost, stream));

      // free memory
      if (!are_scalars_on_device) CHK_IF_RETURN(hipFreeAsync(d_scalars, stream));
      if (!are_points_on_device) CHK_IF_RETURN(hipFreeAsync(d_points, stream));
      if (!are_results_on_device) CHK_IF_RETURN(hipFreeAsync(d_final_results, stream));
      CHK_IF_RETURN(hipFreeAsync(buckets, stream));
      CHK_IF_RETURN(hipFreeAsync(bucket_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(point_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(sorted_bucket_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(sorted_point_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(single_bucket_indices, stream));
      CHK_IF_RETURN(hipFreeAsync(bucket_sizes, stream));
      CHK_IF_RETURN(hipFreeAsync(total_nof_buckets_to_compute, stream));
      CHK_IF_RETURN(hipFreeAsync(bucket_offsets, stream));
      CHK_IF_RETURN(hipFreeAsync(bm_sums, stream));

      if (is_async) return CHK_LAST();
      return CHK_STICKY(hipStreamSynchronize(stream));
    }

  } // namespace

  extern "C" MSMConfig DefaultMSMConfig()
  {
    device_context::DeviceContext ctx = device_context::get_default_device_context();
    MSMConfig config = {
      ctx,   // ctx
      0,     // points_size
      1,     // precompute_factor
      0,     // c
      0,     // bitsize
      10,    // large_bucket_factor
      1,     // batch_size
      false, // are_scalars_on_device
      false, // are_scalars_montgomery_form
      false, // are_points_on_device
      false, // are_points_montgomery_form
      false, // are_results_on_device
      false, // is_big_triangle
      false, // is_async
    };
    return config;
  }

  template <typename S, typename A, typename P>
  hipError_t MSM(S* scalars, A* points, int msm_size, MSMConfig& config, P* results)
  {
    int bitsize = (config.bitsize == 0) ? S::NBITS : config.bitsize;
    hipStream_t& stream = config.ctx.stream;
    // TODO: DmytroTym/HadarIngonyama - unify the implementation of the bucket method and the batched bucket method in
    // one function
    if (config.batch_size == 1)
      return CHK_STICKY(bucket_method_msm(
        bitsize, 16, scalars, points, msm_size, results, config.are_scalars_on_device,
        config.are_scalars_montgomery_form, config.are_points_on_device, config.are_points_montgomery_form,
        config.are_results_on_device, config.is_big_triangle, config.large_bucket_factor, config.is_async, stream));
    else
      return CHK_STICKY(batched_bucket_method_msm(
        bitsize, (config.c == 0) ? get_optimal_c<S>(msm_size) : config.c, scalars, points, config.batch_size, msm_size,
        (config.points_size == 0) ? msm_size : config.points_size, results, config.are_scalars_on_device,
        config.are_scalars_montgomery_form, config.are_points_on_device, config.are_points_montgomery_form,
        config.are_results_on_device, config.is_async, stream));
  }

  /**
   * Extern "C" version of [MSM](@ref MSM) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   *  - `A` is the [affine representation](@ref affine_t) of curve points;
   *  - `P` is the [projective representation](@ref projective_t) of curve points.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t MSMCuda(
    curve_config::scalar_t* scalars,
    curve_config::affine_t* points,
    int msm_size,
    MSMConfig& config,
    curve_config::projective_t* out)
  {
    return MSM<curve_config::scalar_t, curve_config::affine_t, curve_config::projective_t>(
      scalars, points, msm_size, config, out);
  }

#if defined(G2_DEFINED)

  /**
   * Extern "C" version of [MSM](@ref MSM) function with the following values of template parameters
   * (where the curve is given by `-DCURVE` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   *  - `A` is the [affine representation](@ref g2_affine_t) of G2 curve points;
   *  - `P` is the [projective representation](@ref g2_projective_t) of G2 curve points.
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t G2MSMCuda(
    curve_config::scalar_t* scalars,
    curve_config::g2_affine_t* points,
    int msm_size,
    MSMConfig& config,
    curve_config::g2_projective_t* out)
  {
    return MSM<curve_config::scalar_t, curve_config::g2_affine_t, curve_config::g2_projective_t>(
      scalars, points, msm_size, config, out);
  }

#endif

} // namespace msm
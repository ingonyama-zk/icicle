#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>
#include "../../utils/cuda_utils.cuh"
#include "../../primitives/projective.cuh"
#include "../../primitives/field.cuh"
#include "../../curves/curve_config.cuh"
#include "msm.cuh"


#define BIG_TRIANGLE
// #define SSM_SUM

//this kernel performs single scalar multiplication
//each thread multilies a single scalar and point
template <typename P, typename S>
__global__ void ssm_kernel(S *scalars, P *points, P *results, unsigned N) {

  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) results[tid] = scalars[tid]*points[tid];

}

//this kernel sums all the elements in a given vector using multiple threads
template <typename P>
__global__ void sum_reduction_kernel(P *v, P* v_r) {

	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Start at 1/2 block stride and divide by two each iteration
	for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			v[tid] = v[tid] + v[tid + s];
		}
    __syncthreads();
	}

	// Let the thread 0 for this block write the final result
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = v[tid];
	}
}

//this kernel initializes the buckets with zero points
//each thread initializes a different bucket
template <typename P>
__global__ void initialize_buckets_kernel(P *buckets, unsigned N) {
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) buckets[tid] = P::zero(); //zero point

}

//this kernel splits the scalars into digits of size c
//each thread splits a single scalar into nof_bms digits
template <typename S>
__global__ void split_scalars_kernel(unsigned *buckets_indices, unsigned *point_indices, S *scalars, unsigned problem_size, unsigned nof_bms, unsigned c){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned bucket_index;
  unsigned current_index;
  if (tid < problem_size){
    S scalar = scalars[tid];

    for (unsigned bm = 0; bm < nof_bms; bm++)
    {
      bucket_index = scalar.get_scalar_digit(bm, c);
      current_index = bm * problem_size + tid;
      buckets_indices[current_index] = (bm<<c) | bucket_index;  //the bucket module number is appended at the msbs
      point_indices[current_index] = tid; //the point index is saved for later
    }
  }
}

//this kernel adds up the points in each bucket
template <typename P, typename A>
__global__ void accumulate_buckets_kernel(P *__restrict__ buckets, unsigned *__restrict__ bucket_offsets,
               unsigned *__restrict__ bucket_sizes, unsigned *__restrict__ single_bucket_indices, unsigned *__restrict__ point_indices, A *__restrict__ points, unsigned nof_buckets){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned bucket_index = single_bucket_indices[tid];
  unsigned bucket_size = bucket_sizes[tid];
  if (tid>=nof_buckets || bucket_size == 0){ //if the bucket is empty we don't need to continue
    return;
  }
  unsigned bucket_offset = bucket_offsets[tid];
  for (unsigned i = 0; i < bucket_sizes[tid]; i++)  //add the relevant points starting from the relevant offset up to the bucket size
  {
    buckets[bucket_index] = buckets[bucket_index] + points[point_indices[bucket_offset+i]];
  }
}

//this kernel sums the entire bucket module
//each thread deals with a single bucket module
template <typename P>
__global__ void big_triangle_sum_kernel(P* buckets, P* final_sums, unsigned nof_bms, unsigned c){

  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>nof_bms) return;
  P line_sum = buckets[(tid+1)*(1<<c)-1];
  final_sums[tid] = line_sum;
  for (unsigned i = (1<<c)-2; i >0; i--)
  {
    line_sum = line_sum + buckets[tid*(1<<c) + i];  //using the running sum method
    final_sums[tid] = final_sums[tid] + line_sum;
  }
}

//this kernel uses single scalar multiplication to multiply each bucket by its index
//each thread deals with a single bucket
template <typename P, typename S>
__global__ void ssm_buckets_kernel(P* buckets, unsigned* single_bucket_indices, unsigned nof_buckets, unsigned c){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>nof_buckets) return;
  unsigned bucket_index = single_bucket_indices[tid];
  S scalar_bucket_multiplier;
  scalar_bucket_multiplier = {bucket_index&((1<<c)-1), 0, 0, 0, 0, 0, 0, 0}; //the index without the bucket module index
  buckets[bucket_index] = scalar_bucket_multiplier*buckets[bucket_index];

}

//this kernel computes the final result using the double and add algorithm
//it is done by a single thread
template <typename P, typename S>
__global__ void final_accumulation_kernel(P* final_sums, P* final_result, unsigned nof_bms, unsigned c){
  
  *final_result = P().zero();
  S digit_base = {unsigned(1<<c)};
  for (unsigned i = nof_bms; i >0; i--)
  {
    *final_result = digit_base*(*final_result) + final_sums[i-1];
  }
  

}

//this function computes msm using the bucket method
template <typename S, typename P, typename A>
void bucket_method_msm(unsigned bitsize, unsigned c, S *h_scalars, A *h_points, unsigned size, P*h_final_result){
  
  //copy scalars and point to gpu
  S *scalars;
  A *points;

  hipMalloc(&scalars, sizeof(S) * size);
  hipMalloc(&points, sizeof(A) * size);
  hipMemcpy(scalars, h_scalars, sizeof(S) * size, hipMemcpyHostToDevice);
  hipMemcpy(points, h_points, sizeof(A) * size, hipMemcpyHostToDevice);

  P *buckets;
  //compute number of bucket modules and number of buckets in each module
  unsigned nof_bms = bitsize/c;
  if (bitsize%c){
    nof_bms++;
  }
  unsigned nof_buckets = nof_bms<<c;
  hipMalloc(&buckets, sizeof(P) * nof_buckets); 

  //lanch the bucket initialization kernel with maximum threads
  unsigned NUM_THREADS = 1 << 10;
  unsigned NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, nof_buckets);

  unsigned *bucket_indices;
  unsigned *point_indices;
  hipMalloc(&bucket_indices, sizeof(unsigned) * size * (nof_bms+1));
  hipMalloc(&point_indices, sizeof(unsigned) * size * (nof_bms+1));

  //split scalars into digits
  NUM_THREADS = 1 << 10;
  NUM_BLOCKS = (size * (nof_bms+1) + NUM_THREADS - 1) / NUM_THREADS;
  split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(bucket_indices + size, point_indices + size, scalars, size, nof_bms, c);

  //sort indices - the indices are sorted from smallest to largest in order to group together the points that belong to each bucket
  unsigned *sort_indices_temp_storage{};
  size_t sort_indices_temp_storage_bytes;
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + size, bucket_indices,
                                 point_indices + size, point_indices, size);
  hipMalloc(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);
  for (unsigned i = 0; i < nof_bms; i++) {
    unsigned offset_out = i * size;
    unsigned offset_in = offset_out + size;
    hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + offset_in,
                                  bucket_indices + offset_out, point_indices + offset_in, point_indices + offset_out, size);
  }
  hipFree(sort_indices_temp_storage);

  //find bucket_sizes
  unsigned *single_bucket_indices;
  unsigned *bucket_sizes;
  unsigned *nof_buckets_to_compute;
  hipMalloc(&single_bucket_indices, sizeof(unsigned)*nof_buckets);
  hipMalloc(&bucket_sizes, sizeof(unsigned)*nof_buckets);
  hipMalloc(&nof_buckets_to_compute, sizeof(unsigned));
  unsigned *encode_temp_storage{};
  size_t encode_temp_storage_bytes = 0;
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                        nof_buckets_to_compute, nof_bms*size);
  hipMalloc(&encode_temp_storage, encode_temp_storage_bytes);
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                        nof_buckets_to_compute, nof_bms*size);
  hipFree(encode_temp_storage);

  //get offsets - where does each new bucket begin
  unsigned* bucket_offsets;
  hipMalloc(&bucket_offsets, sizeof(unsigned)*nof_buckets);
  unsigned* offsets_temp_storage{};
  size_t offsets_temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets);
  hipMalloc(&offsets_temp_storage, offsets_temp_storage_bytes);
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets);
  hipFree(offsets_temp_storage);

  //launch the accumulation kernel with maximum threads
  NUM_THREADS = 1 << 8;
  NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, points, nof_buckets);

  #ifdef SSM_SUM
    //sum each bucket
    NUM_THREADS = 1 << 10;
    NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
    ssm_buckets_kernel<fake_point, fake_scalar><<<NUM_BLOCKS, NUM_THREADS>>>(buckets, single_bucket_indices, nof_buckets, c);
   
    //sum each bucket module
    P* final_results;
    hipMalloc(&final_results, sizeof(P) * nof_bms);
    NUM_THREADS = 1<<c;
    NUM_BLOCKS = nof_bms;
    sum_reduction_kernel<<<NUM_BLOCKS,NUM_THREADS>>>(buckets, final_results);
  #endif

  #ifdef BIG_TRIANGLE
    P* final_results;
    hipMalloc(&final_results, sizeof(P) * nof_bms);
    //launch the bucket module sum kernel - a thread for each bucket module
    NUM_THREADS = nof_bms;
    NUM_BLOCKS = 1;
    big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, final_results, nof_buckets, c);
    
  #endif

  P* final_result;
  hipMalloc(&final_result, sizeof(P));
  //launch the double and add kernel, a single thread
  final_accumulation_kernel<P, S><<<1,1>>>(final_results, final_result, nof_bms, c);
  
  //copy final result to host
  hipDeviceSynchronize();
  hipMemcpy(h_final_result, final_result, sizeof(P), hipMemcpyDeviceToHost);

  //free memory
  hipFree(buckets);
  hipFree(points);
  hipFree(scalars);
  hipFree(bucket_indices);
  hipFree(point_indices);
  hipFree(single_bucket_indices);
  hipFree(bucket_sizes);
  hipFree(nof_buckets_to_compute);
  hipFree(bucket_offsets);
  hipFree(final_results);
  hipFree(final_result);

}

//this kernel converts affine points to projective points
//each thread deals with a single point
template <typename P, typename A>
__global__ void to_proj_kernel(A* affine_points, P* proj_points, unsigned N){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) proj_points[tid] = P::from_affine(affine_points[tid]);
}

//the function computes msm using ssm
template <typename S, typename P, typename A>
void short_msm(S *h_scalars, A *h_points, unsigned size, P* h_final_result){ //works up to 2^8
  
  S *scalars;
  A *a_points;
  P *p_points;
  P *results;

  hipMalloc(&scalars, sizeof(S) * size);
  hipMalloc(&a_points, sizeof(A) * size);
  hipMalloc(&p_points, sizeof(P) * size);
  hipMalloc(&results, sizeof(P) * size);

  //copy inputs to device
  hipMemcpy(scalars, h_scalars, sizeof(S) * size, hipMemcpyHostToDevice);
  hipMemcpy(a_points, h_points, sizeof(A) * size, hipMemcpyHostToDevice);

  //convert to projective representation and multiply each point by its scalar using single scalar multiplication
  unsigned NUM_THREADS = size;
  to_proj_kernel<<<1,NUM_THREADS>>>(a_points, p_points, size);
  ssm_kernel<<<1,NUM_THREADS>>>(scalars, p_points, results, size);

  P *final_result;
  hipMalloc(&final_result, sizeof(P));

  //assuming msm size is a power of 2
  //sum all the ssm results
  NUM_THREADS = size;
  sum_reduction_kernel<<<1,NUM_THREADS>>>(results, final_result);

  //copy result to host
  hipDeviceSynchronize();
  hipMemcpy(h_final_result, final_result, sizeof(P), hipMemcpyDeviceToHost);

  //free memory
  hipFree(scalars);
  hipFree(a_points);
  hipFree(p_points);
  hipFree(results);
  hipFree(final_result);

}

//the function computes msm on the host using the naive method
template <typename A, typename S, typename P>
void reference_msm(S* scalars, A* a_points, unsigned size){
  
  P points[size];
  for (unsigned i = 0; i < size ; i++)
  {
    points[i] = P::from_affine(a_points[i]);
  }
  

  P res = P::zero();
  
  for (unsigned i = 0; i < size; i++)
  {
    res = res + scalars[i]*points[i];
  }

  std::cout<<"reference results"<<std::endl;
  std::cout<<P::to_affine(res)<<std::endl;
  
}

//this function is used to compute msms of size larger than 1024
template <typename S, typename P, typename A>
void large_msm(S* scalars, A* points, unsigned size, P* result){
  unsigned c = 10;
  unsigned bitsize = 255;
  bucket_method_msm(bitsize, c, scalars, points, size, result);
}

extern "C"
int msm_cuda(projective_t *out, affine_t points[],
              scalar_t scalars[], size_t count, size_t device_id = 0)
{
    try
    {
        if (count>256){
            large_msm<scalar_t, projective_t, affine_t>(scalars, points, count, out);
        }
        else{
            short_msm<scalar_t, projective_t, affine_t>(scalars, points, count, out);
        }

        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}


int main()
{
  // fake_point p1;
  // fake_point p2;
  // p1.val = 8;
  // p2.val = 9;
  // std::cout<<(p1+p2)<<std::endl;

  // unsigned N = 4;
  unsigned N = 1<<26;

  // fake_scalar scalars[N];
  // fake_point points[N];

  scalar_t *scalars = new scalar_t[N];
  affine_t *points = new affine_t[N];
  // std::vector<scalar_t> scalars;
  // std::vector<affine_t> points;
  // scalars.reserve(N);
  // points.reserve(N);

  // srand(time(NULL));
  // for (unsigned i = 0; i < N; i++)
  // {
  //   // scalars[i].val = rand()%(1<<10);
  //   scalars[i] = {unsigned(rand()%(1<<10))};
  //   // std::cout<<scalars[i].val<<std::endl;
  //   // points[i].val = rand()%(1<<10);
  //   points[i] = {{unsigned(rand()%(1<<10)), 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
  //             {unsigned(rand()%(1<<10)), 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0}};
  //   // std::cout<<points[i].val<<std::endl;
  // }

  scalars[0] = {3827484040, 2874625294, 4134484179, 2522098891, 1684039639, 4190761864, 1674792009, 1733172596};
  scalars[1] = {2859778174, 247198543, 1069683537, 986951671, 18230349, 1865405355, 3834758898, 1605705230};
  scalars[2] = {870702623, 2140804196, 1118323047, 4097847923, 733285949, 2517599604, 2748585063, 1465198310};
  scalars[3] = {2997668077, 4130616472, 4255624276, 3713720096, 813455961, 1818410993, 1796699074, 1289452986};

  points[0].x = {927572523, 3123811016, 1178179586, 448270957, 3269025417, 873655910, 946685814, 846160237, 2311665546, 894701547, 1123227996, 414748152};
  points[0].y = {2100670837, 1657590303, 4206131811, 3111559769, 3261363570, 430821050, 2016803245, 2664358421, 3132350727, 189414955, 1844185218, 11036570};
  points[1].x = {606938594, 3862011666, 3396180143, 765820065, 3281167117, 634141057, 210831039, 670764991, 3442481388, 2417967610, 1382165347, 243748907};
  points[1].y = {2486871565, 3199940895, 3186416593, 2451721591, 4108712975, 2604984942, 1165376591, 854454192, 1479545654, 1006124383, 1570319433, 22366661};
  points[2].x = {183039612, 256454025, 4250922080, 2485970688, 3679755773, 1397028634, 1298805238, 3413182507, 2291846949, 1280816489, 1119750210, 122833203};
  points[2].y = {3025851512, 1147574033, 1323495323, 569405769, 382481561, 1330634004, 3879950484, 1158208050, 2740575984, 2745897444, 3101936482, 405605297};
  points[3].x = {4006417784, 3580973450, 2524244405, 3414509667, 4142213295, 3876406748, 4116037682, 877187559, 3606672288, 3459819278, 3198860768, 30571621};
  points[3].y = {182896763, 2741166359, 626891178, 1601768019, 1967793394, 706302600, 2612369182, 2051460370, 2918333441, 1902350841, 475238909, 239719017};

/*correct result:
1557917178, 269077943, 1116505460, 728110787, 4176849812, 3140203189, 2756051319, 197704154, 1838744007, 2201658078, 1505047534, 239949230, 
2029063365, 2557489072, 3905272471, 2418563649, 2077595491, 357415053, 3188715161, 1890916285, 354886608, 410171932, 1437862573, 206970588, 
4160033405, 2697065480, 1940009895, 2097886176, 4019146882, 2931880476, 3425684730, 2783686325, 1918054479, 1505257125, 3268347217, 269536830, */

  for (unsigned i = 1; i < N/4; i++)
  {
    scalars[4*i+0] = scalars[0];
    scalars[4*i+1] = scalars[1];
    scalars[4*i+2] = scalars[2];
    scalars[4*i+3] = scalars[3];
    points[4*i+0] = points[0];
    points[4*i+1] = points[1];
    points[4*i+2] = points[2];
    points[4*i+3] = points[3];
  }
  

  // std::cout<<"scalars"<<std::endl;
  // for (unsigned j = 0; j<N ; j++){

  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << scalars[j].limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";

  // }

  // std::cout<<"points"<<std::endl;
  // for (unsigned j = 0; j<N ; j++){
  // for (unsigned i = 0; i < 12; i++) {
  //   std::cout << points[j].x.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";
  // for (unsigned i = 0; i < 12; i++) {
  //   std::cout << points[j].y.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";

  // // return 0;
  // // for (unsigned i = 0; i < 8; i++) {
  // //   std::cout << points[j].z.limbs_storage.limbs[i] << ", ";
  // // }
  // // std::cout << "\n";
  // }
  
  // projective_t test_p = projective_t::zero();

  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << test_p.x.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";
  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << test_p.y.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";
  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << test_p.z.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";

  // projective_t test_mul = scalars[0]*points[0];

  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << test_mul.x.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";
  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << test_mul.y.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";
  // for (unsigned i = 0; i < 8; i++) {
  //   std::cout << test_mul.z.limbs_storage.limbs[i] << ", ";
  // }
  // std::cout << "\n";

  // scalars[0].val = 456;
  // scalars[1].val = 51;
  // scalars[2].val = 984;
  // scalars[3].val = 15;

  // points[0].val = 0;
  // points[1].val = 1;
  // points[2].val = 2;
  // points[3].val = 3;
  // cuda_ctx my_ctx = cuda_ctx(0);
  // large_msm<fake_point, fake_scalar>(my_ctx, points, scalars, N);

  // bucket_method_msm<scalar_t,projective_t, affine_t>(255, 10, scalars, points, N);
  // projective_t pr=short_msm<scalar_t,projective_t,affine_t>(scalars, points, N);

  projective_t *short_res = (projective_t*)malloc(sizeof(projective_t));
  projective_t *large_res = (projective_t*)malloc(sizeof(projective_t));

  // projective_t short_res[1];
  // projective_t large_res[1];

  // short_msm<scalar_t, projective_t, affine_t>(scalars, points, N, short_res);
  large_msm<scalar_t, projective_t, affine_t>(scalars, points, N, large_res);

  // std::cout<<"final result short"<<std::endl;
  // std::cout<<*short_res<<std::endl;
  std::cout<<"final result large"<<std::endl;
  // std::cout<<projective_t::to_affine(*large_res)<<std::endl;
  std::cout<<*large_res<<std::endl;

  // reference_msm<affine_t, scalar_t, projective_t>(scalars, points, N);
  // std::cout<<"final result short"<<std::endl;
  // std::cout<<pr<<std::endl;

  return 0;
}
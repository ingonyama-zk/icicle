#include <hip/hip_runtime.h>
#include "commit.cuh"
#include "msm.cuh"


/**
 * Commit to a polynomial using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
 * @param d_out Ouptut point to write the result to.
 * @param d_scalars Scalars for the MSM. Must be on device.
 * @param d_points Points for the MSM. Must be on device.
 * @param count Length of `d_scalars` and `d_points` arrays (they should have equal length).
 */
extern "C"
int commit_cuda(projective_t *d_out, scalar_t* d_scalars, affine_t* d_points, size_t count, size_t device_id = 0)
{
    try
    {
        // TODO: set c depending on `count` instead of just 10
        bucket_method_msm(scalar_t::NBITS, 10, d_scalars, d_points, count, d_out, true);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

/**
 * Commit to a batch of polynomials using the MSM.
 * Note: this function just calls the MSM, it doesn't convert between evaluation and coefficient form of scalars or points.
 * @param d_out Ouptut point to write the results to.
 * @param d_scalars Scalars for the MSMs of all polynomials. Must be on device.
 * @param d_points Points for the MSMs. Must be on device. It is assumed that this set of bases is used for each MSM.
 * @param count Length of `d_points` array, `d_scalar` has length `count` * `batch_size`.
 * @param batch_size Size of the batch.
 */
extern "C"
int commit_batch_cuda(projective_t *d_out, scalar_t* d_scalars, affine_t* d_points, 
                      size_t count, size_t batch_size, size_t device_id = 0)
{
    try
    {
        // TODO: set c depending on `count` instead of just 10
        batched_bucket_method_msm(scalar_t::NBITS, 10, d_scalars, d_points, batch_size, count, d_out, true);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        printf("error %s", ex.what());
        return -1;
    }
}

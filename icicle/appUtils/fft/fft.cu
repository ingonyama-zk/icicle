#include <iostream>
#include <iomanip>
#include <chrono>
#include <nvml.h>
#include <vector>

#include "curves/curve_config.cuh"
#include "utils/device_context.cuh"
#include "utils/vec_ops.cu"

namespace fft {
  template <typename S>
  hipError_t fft(
    S* input, S* output, S* ws, int n, bool invert)
  {
    CHK_INIT_IF_RETURN();

    std::cout << STREAM_CHUNK_SIZE << std::endl;

    return CHK_LAST();
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, FftEvaluate)(
    curve_config::scalar_t* inout,
    curve_config::scalar_t* ws,
    int n)
  {

    return fft<curve_config::scalar_t>(inout, ws, n);
  }
}

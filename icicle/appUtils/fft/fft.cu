#include "hip/hip_runtime.h"
#include "fft.cuh"

namespace fft {
  __device__ uint32_t device_reverse_bits(uint32_t x) {
      x = ((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1);
      x = ((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2);
      x = ((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4);
      x = ((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8);
      return (x >> 16) | (x << 16);
  }

  template <typename S>
  __global__ void swap_bits(S* b, uint32_t n, uint32_t log_n) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid * 2; i < tid * 2 + 2; i++) {
      uint32_t rev = device_reverse_bits(i);
      rev = rev >> (32 - log_n);

      if (i < rev) {
        S tmp = b[i];
        b[i] = b[rev];
        b[rev] = tmp;
      }
    }
  }

  template <typename S>
  __global__ void fft_kernel(S* b, uint32_t n, uint32_t power, uint32_t ws_index, S* ws) {
      uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

      uint32_t len = 1 << power;
      uint32_t len2 = len >> 1; // len2 = len / 2
      uint32_t q = tid >> (power - 1);
      uint32_t i = q * len;
      uint32_t j = tid - q * len2;

      S w;
      w = ws[ws_index + j];

      S u = b[i + j];
      S v = b[i + j + len / 2] * w;
      b[i + j] = u + v;
      b[i + j + len / 2] = u - v;
  }

  template <typename S>
  hipError_t fft(
    S* inout, S* ws, int n, bool invert)
  {
    CHK_INIT_IF_RETURN();

    S* device_inout;
    S* device_ws;
    // allocate device array
    hipMalloc((void**)&device_inout, n * sizeof(S));
    hipMalloc((void**)&device_ws, n * sizeof(S));

    // copy from host to device
    auto err = hipMemcpy(device_inout, inout, n * sizeof(S), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      std::cerr << "Failed to copy data from host to device - " << hipGetErrorString(err) << std::endl;
      return err;
    }
    err = hipMemcpy(device_ws, ws, n * sizeof(S), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      std::cerr << "Failed to copy data from host to device - " << hipGetErrorString(err) << std::endl;
      return err;
    }

    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);

    // Set the grid and block dimensions
    int worker_count = n >> 1;
    int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
    int num_blocks = (worker_count + num_threads - 1) / num_threads;

    const int log_n = log2(n);
    // Swap bits
    swap_bits<<< num_blocks, num_threads  >>> (device_inout, n, log_n);

    // main loop
    int ws_index = 0;
    for (int pow = 1; ; pow++) {
      int len = 1 << pow;
      if (len > n) {
        break;
      }

      fft_kernel<<< num_blocks, num_threads  >>> (device_inout, n, pow, ws_index, device_ws);

      ws_index += len >> 1;
    }

    // copy back to host
    err = hipMemcpy(inout, device_inout, n * sizeof(S), hipMemcpyDeviceToHost);

    hipFree(device_inout);
    hipFree(device_ws);

    return CHK_LAST();
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, FftEvaluate)(
    curve_config::scalar_t* inout,
    curve_config::scalar_t* ws,
    int n)
  {
    return fft<curve_config::scalar_t>(inout, ws, n, false);
  }
}

#include "utils/utils.h"
#include "fields/field_config.cuh"

namespace ntt {
  // Explicit instantiation for scalar type
  template hipError_t generate_external_twiddles_generic(
    const curve_config::scalar_t& basic_root,
    curve_config::scalar_t* external_twiddles,
    curve_config::scalar_t*& internal_twiddles,
    curve_config::scalar_t*& basic_twiddles,
    uint32_t log_size,
    hipStream_t& stream);

  template hipError_t generate_external_twiddles_fast_twiddles_mode(
    const curve_config::scalar_t& basic_root,
    curve_config::scalar_t* external_twiddles,
    curve_config::scalar_t*& internal_twiddles,
    curve_config::scalar_t*& basic_twiddles,
    uint32_t log_size,
    hipStream_t& stream);

  template hipError_t mixed_radix_ntt<curve_config::scalar_t, curve_config::scalar_t>(
    curve_config::scalar_t* d_input,
    curve_config::scalar_t* d_output,
    curve_config::scalar_t* external_twiddles,
    curve_config::scalar_t* internal_twiddles,
    curve_config::scalar_t* basic_twiddles,
    int ntt_size,
    int max_logn,
    int batch_size,
    bool columns_batch,
    bool is_inverse,
    bool fast_tw,
    Ordering ordering,
    curve_config::scalar_t* arbitrary_coset,
    int coset_gen_index,
    hipStream_t hip_stream);

  /**
   * Extern "C" version of [InitDomain](@ref InitDomain) function with the following
   * value of template parameter (where the curve is given by `-DFIELD` env variable during build):
   *  - `S` is the [scalar field](@ref scalar_t) of the curve;
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, InitializeDomain)(
    curve_config::scalar_t* primitive_root, device_context::DeviceContext& ctx, bool fast_twiddles_mode)
  {
    return InitDomain(*primitive_root, ctx, fast_twiddles_mode);
  }

  /**
   * Extern "C" version of [NTT](@ref NTT) function with the following values of template parameters
   * (where the curve is given by `-DFIELD` env variable during build):
   *  - `S` and `E` are both the [scalar field](@ref scalar_t) of the curve;
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, NTTCuda)(
    curve_config::scalar_t* input,
    int size,
    NTTDir dir,
    NTTConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* output)
  {
    return NTT<curve_config::scalar_t, curve_config::scalar_t>(input, size, dir, config, output);
  }
}
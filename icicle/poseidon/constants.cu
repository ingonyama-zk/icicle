#include "poseidon.cuh"

/// These are pre-calculated constants for different curves
#if CURVE_ID == BN254
#include "appUtils/poseidon/constants/bn254_poseidon.h"
using namespace poseidon_constants_bn254;
#elif CURVE_ID == BLS12_381
#include "appUtils/poseidon/constants/bls12_381_poseidon.h"
using namespace poseidon_constants_bls12_381;
#elif CURVE_ID == BLS12_377
#include "appUtils/poseidon/constants/bls12_377_poseidon.h"
using namespace poseidon_constants_bls12_377;
#elif CURVE_ID == BW6_761
#include "appUtils/poseidon/constants/bw6_761_poseidon.h"
using namespace poseidon_constants_bw6_761;
#elif CURVE_ID == GRUMPKIN
#include "appUtils/poseidon/constants/grumpkin_poseidon.h"
using namespace poseidon_constants_grumpkin;
#endif

namespace poseidon {
  template <typename S>
  hipError_t create_optimized_poseidon_constants(
    int arity,
    int full_rounds_half,
    int partial_rounds,
    const S* constants,
    device_context::DeviceContext& ctx,
    PoseidonConstants<S>* poseidon_constants)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = ctx.stream;
    int width = arity + 1;
    int round_constants_len = width * full_rounds_half * 2 + partial_rounds;
    int mds_matrix_len = width * width;
    int sparse_matrices_len = (width * 2 - 1) * partial_rounds;
    int constants_len = round_constants_len + mds_matrix_len * 2 + sparse_matrices_len;

    // Malloc memory for copying constants
    S* d_constants;
    CHK_IF_RETURN(hipMallocAsync(&d_constants, sizeof(S) * constants_len, stream));

    // Copy constants
    CHK_IF_RETURN(hipMemcpyAsync(d_constants, constants, sizeof(S) * constants_len, hipMemcpyHostToDevice, stream));

    S* round_constants = d_constants;
    S* mds_matrix = round_constants + round_constants_len;
    S* non_sparse_matrix = mds_matrix + mds_matrix_len;
    S* sparse_matrices = non_sparse_matrix + mds_matrix_len;

    // Pick the domain_tag accordinaly
    // For now, we only support Merkle tree mode
    uint32_t tree_domain_tag_value = 1;
    tree_domain_tag_value = (tree_domain_tag_value << (width - 1)) - tree_domain_tag_value;
    S domain_tag = S::from(tree_domain_tag_value);

    // Make sure all the constants have been copied
    CHK_IF_RETURN(hipStreamSynchronize(stream));
    *poseidon_constants = {arity,      partial_rounds,    full_rounds_half, round_constants,
                           mds_matrix, non_sparse_matrix, sparse_matrices,  domain_tag};

    return CHK_LAST();
  }

  template <typename S>
  hipError_t init_optimized_poseidon_constants(
    int arity, device_context::DeviceContext& ctx, PoseidonConstants<S>* poseidon_constants)
  {
    CHK_INIT_IF_RETURN();
    int full_rounds_half = FULL_ROUNDS_DEFAULT;
    int partial_rounds;
    unsigned char* constants;
    switch (arity) {
    case 2:
      constants = poseidon_constants_2;
      partial_rounds = partial_rounds_2;
      break;
    case 4:
      constants = poseidon_constants_4;
      partial_rounds = partial_rounds_4;
      break;
    case 8:
      constants = poseidon_constants_8;
      partial_rounds = partial_rounds_8;
      break;
    case 11:
      constants = poseidon_constants_11;
      partial_rounds = partial_rounds_11;
      break;
    default:
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument, "init_optimized_poseidon_constants: #arity must be one of [2, 4, 8, 11]");
    }
    S* h_constants = reinterpret_cast<S*>(constants);

    create_optimized_poseidon_constants(arity, full_rounds_half, partial_rounds, h_constants, ctx, poseidon_constants);

    return CHK_LAST();
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, CreateOptimizedPoseidonConstants)(
    int arity,
    int full_rounds_half,
    int partial_rounds,
    const curve_config::scalar_t* constants,
    device_context::DeviceContext& ctx,
    PoseidonConstants<curve_config::scalar_t>* poseidon_constants)
  {
    return create_optimized_poseidon_constants<curve_config::scalar_t>(
      arity, full_rounds_half, partial_rounds, constants, ctx, poseidon_constants);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, InitOptimizedPoseidonConstants)(
    int arity, device_context::DeviceContext& ctx, PoseidonConstants<curve_config::scalar_t>* constants)
  {
    return init_optimized_poseidon_constants<curve_config::scalar_t>(arity, ctx, constants);
  }
} // namespace poseidon
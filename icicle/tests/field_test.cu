#include "utils/test_functions.cuh"
#include "fields/field_config.cuh"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

using namespace field_config;

template <class T>
class FieldTest : public ::testing::Test
{
protected:
  static const unsigned n = 1 << 4;

  T* scalars1{};
  T* scalars2{};
  T* zero_scalars{};
  T* one_scalars{};
  T* res_scalars1{};
  T* res_scalars2{};

  FieldTest()
  {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&scalars1, n * sizeof(T)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(T)));
    assert(!hipMallocManaged(&zero_scalars, n * sizeof(T)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(T)));
    assert(!hipMallocManaged(&res_scalars1, n * sizeof(T)));
    assert(!hipMallocManaged(&res_scalars2, n * sizeof(T)));
  }

  ~FieldTest() override
  {
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_scalars);
    hipFree(one_scalars);
    hipFree(res_scalars1);
    hipFree(res_scalars2);

    hipDeviceReset();
  }

  void SetUp() override
  {
    ASSERT_EQ(device_populate_random<T>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<T>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set(zero_scalars, T::zero(), n), hipSuccess);
    ASSERT_EQ(device_set(one_scalars, T::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars1, 0, n * sizeof(T)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars2, 0, n * sizeof(T)), hipSuccess);
  }
};

#ifdef EXT_FIELD
typedef testing::Types<scalar_t, extension_t> FTImplementations;
#else
typedef testing::Types<scalar_t> FTImplementations;
#endif

TYPED_TEST_SUITE(FieldTest, FTImplementations);

TYPED_TEST(FieldTest, FieldAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(this->scalars1, this->scalars2, this->res_scalars1, this->n), hipSuccess);
  ASSERT_EQ(vec_sub(this->res_scalars1, this->scalars2, this->res_scalars2, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i], this->res_scalars2[i]);
}

TYPED_TEST(FieldTest, FieldZeroAddition)
{
  ASSERT_EQ(vec_add(this->scalars1, this->zero_scalars, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i], this->res_scalars1[i]);
}

TYPED_TEST(FieldTest, FieldAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(this->scalars1, this->scalars2, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i] + this->scalars2[i], this->res_scalars1[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(this->scalars1, this->one_scalars, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i], this->res_scalars1[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(this->one_scalars, this->res_scalars1, this->n), hipSuccess);
  ASSERT_EQ(vec_mul(this->scalars1, this->res_scalars1, this->res_scalars2, this->n), hipSuccess);
  ASSERT_EQ(vec_add(this->scalars1, this->res_scalars2, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->res_scalars1[i], this->zero_scalars[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationByZero)
{
  ASSERT_EQ(vec_mul(this->scalars1, this->zero_scalars, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->zero_scalars[i], this->res_scalars1[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(this->scalars1, this->scalars2, this->res_scalars1, this->n), hipSuccess);
  ASSERT_EQ(field_vec_inv(this->scalars2, this->res_scalars2, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i], this->res_scalars1[i] * this->res_scalars2[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(this->scalars1, this->scalars2, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i] * this->scalars2[i], this->res_scalars1[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationByTwoEqSum)
{
  ASSERT_EQ(vec_add(this->one_scalars, this->one_scalars, this->res_scalars1, this->n), hipSuccess);
  ASSERT_EQ(vec_mul(this->res_scalars1, this->scalars1, this->res_scalars2, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->res_scalars2[i], this->scalars1[i] + this->scalars1[i]);
}

TYPED_TEST(FieldTest, FieldSqrHostDeviceEq)
{
  ASSERT_EQ(field_vec_sqr(this->scalars1, this->res_scalars1, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->scalars1[i] * this->scalars1[i], this->res_scalars1[i]);
}

TYPED_TEST(FieldTest, FieldMultiplicationSqrEq)
{
  ASSERT_EQ(vec_mul(this->scalars1, this->scalars1, this->res_scalars1, this->n), hipSuccess);
  ASSERT_EQ(field_vec_sqr(this->scalars1, this->res_scalars2, this->n), hipSuccess);
  for (unsigned i = 0; i < this->n; i++)
    ASSERT_EQ(this->res_scalars1[i], this->res_scalars2[i]);
}

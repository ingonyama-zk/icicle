#include "utils/test_functions.cuh"
#include "fields/field_config.cuh"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

using namespace field_config;

class FieldTest : public ::testing::Test
{
protected:
  static const unsigned n = 1 << 4;

  scalar_t* scalars1{};
  scalar_t* scalars2{};
  scalar_t* zero_scalars{};
  scalar_t* one_scalars{};
  scalar_t* res_scalars1{};
  scalar_t* res_scalars2{};

  FieldTest()
  {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&scalars1, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&zero_scalars, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&res_scalars1, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&res_scalars2, n * sizeof(scalar_t)));
  }

  ~FieldTest() override
  {
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_scalars);
    hipFree(one_scalars);
    hipFree(res_scalars1);
    hipFree(res_scalars2);

    hipDeviceReset();
  }

  void SetUp() override
  {
    ASSERT_EQ(device_populate_random<scalar_t>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_t>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set(zero_scalars, scalar_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set(one_scalars, scalar_t::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars1, 0, n * sizeof(scalar_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars2, 0, n * sizeof(scalar_t)), hipSuccess);
  }
};

TEST_F(FieldTest, FieldAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_scalars1, scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars2[i]);
}

TEST_F(FieldTest, FieldZeroAddition)
{
  ASSERT_EQ(vec_add(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(FieldTest, FieldAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] + scalars2[i], res_scalars1[i]);
}

TEST_F(FieldTest, FieldMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(scalars1, one_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(FieldTest, FieldMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, res_scalars1, res_scalars2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, res_scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], zero_scalars[i]);
}

TEST_F(FieldTest, FieldMultiplicationByZero)
{
  ASSERT_EQ(vec_mul(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(zero_scalars[i], res_scalars1[i]);
}

TEST_F(FieldTest, FieldMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i] * res_scalars2[i]);
}

TEST_F(FieldTest, FieldMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars2[i], res_scalars1[i]);
}

TEST_F(FieldTest, FieldMultiplicationByTwoEqSum)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars2[i], scalars1[i] + scalars1[i]);
}

TEST_F(FieldTest, FieldSqrHostDeviceEq)
{
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars1[i], res_scalars1[i]);
}

TEST_F(FieldTest, FieldMultiplicationSqrEq)
{
  ASSERT_EQ(vec_mul(scalars1, scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], res_scalars2[i]);
}

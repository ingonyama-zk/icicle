#include "hip/hip_runtime.h"
#include "../utils/error_handler.cuh" // Include your error handling header file
#include <gtest/gtest.h>

__global__ void a_kernel_with_conditional_sticky_error(bool is_failing)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx == 0) {
    assert(hipGetLastError() == hipSuccess);
    assert(hipGetLastError() == hipSuccess);
    // Deliberately assert false
    assert(!is_failing);
  }
}

// Test Fixture for CUDA tests
class CudaErrorTest : public ::testing::Test
{
protected:
  void SetUp() override
  {
    // Perform any setup needed before each test
    hipError_t err = hipGetLastError(); // Clear any existing errors
  }

  void TearDown() override
  {
    // Clean up after each test if necessary
  }
};

// Test Case for Non-Sticky Error
TEST_F(CudaErrorTest, NonStickyErrorTest)
{
  hipGetLastError();

  // Deliberately cause a non-sticky CUDA error
  hipError_t err = hipMalloc(nullptr, 0); // This should cause hipErrorInvalidValue
  hipError_t err2;

  // Check if the macro correctly reports the error without throwing an exception
  EXPECT_EQ(err, hipErrorInvalidValue);
  EXPECT_NO_THROW({ err2 = CHK_LAST(); });
  EXPECT_EQ(err2, err);

  // Optionally, clear the error if needed
  EXPECT_EQ(hipGetLastError(), hipSuccess);
}

// Test Case for Sticky Error
TEST_F(CudaErrorTest, StickyErrorTest)
{
  EXPECT_EQ(hipGetLastError(), hipSuccess);

  // Deliberately cause a sticky CUDA error
  a_kernel_with_conditional_sticky_error<<<1, 1>>>(true);

  EXPECT_EQ(hipGetLastError(), hipSuccess); // no error until synchronization

  // Launch without error
  a_kernel_with_conditional_sticky_error<<<1, 1>>>(false);

  EXPECT_EQ(hipGetLastError(), hipSuccess);

  hipError_t sync_error = hipDeviceSynchronize(); // only hipDeviceSynchronize() can help
                                                    // determine sticky error reliably,
                                                    // returning same error as failed kernel

  EXPECT_NE(sync_error, hipSuccess);
  EXPECT_EQ(sync_error, hipErrorAssert);

  EXPECT_EQ(hipGetLastError(), hipErrorAssert); // reports error after hipDeviceSynchronize
  EXPECT_EQ(hipGetLastError(), hipSuccess);     // resets error, despite it's sticky

  // Check if the macro correctly throws an exception for a sticky error
  EXPECT_THROW({ CHK_STICKY(hipDeviceSynchronize()); }, IcicleError);
}

// Test Case for Sticky Error
TEST_F(CudaErrorTest, StickyErrorTestNotThrowing)
{
  EXPECT_EQ(hipGetLastError(), hipSuccess);

  // Deliberately cause a sticky CUDA error
  a_kernel_with_conditional_sticky_error<<<1, 1>>>(true);

  EXPECT_EQ(hipDeviceSynchronize(), hipErrorAssert);

  // Check if the macro correctly throws an exception for a sticky error
  hipError_t err = CHK_LAST_STICKY_NO_THROW();
  EXPECT_EQ(err, hipErrorAssert);
}
#include "utils/test_functions.cuh"
#include "curves/curve_config.cuh"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

using namespace curve_config;

class CurveTest : public ::testing::Test
{
protected:
  static const unsigned n = 1 << 4;

  projective_t* points1{};
  projective_t* points2{};
  scalar_t* scalars1{};
  scalar_t* scalars2{};
  projective_t* zero_points{};
  scalar_t* one_scalars{};
  affine_t* aff_points{};
  projective_t* res_points1{};
  projective_t* res_points2{};
  scalar_t* res_scalars{};

#ifdef G2
  g2_projective_t* g2_points1{};
  g2_projective_t* g2_points2{};
  g2_projective_t* g2_zero_points{};
  g2_affine_t* g2_aff_points{};
  g2_projective_t* g2_res_points1{};
  g2_projective_t* g2_res_points2{};
#endif

  CurveTest()
  {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&points1, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&points2, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&scalars1, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&zero_points, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&aff_points, n * sizeof(affine_t)));
    assert(!hipMallocManaged(&res_points1, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&res_points2, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&res_scalars, n * sizeof(scalar_t)));

#ifdef G2
    assert(!hipMallocManaged(&g2_points1, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_points2, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_zero_points, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_aff_points, n * sizeof(g2_affine_t)));
    assert(!hipMallocManaged(&g2_res_points1, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_res_points2, n * sizeof(g2_projective_t)));
#endif
  }

  ~CurveTest() override
  {
    hipFree(points1);
    hipFree(points2);
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_points);
    hipFree(one_scalars);
    hipFree(aff_points);
    hipFree(res_points1);
    hipFree(res_points2);
    hipFree(res_scalars);

#ifdef G2
    hipFree(g2_points1);
    hipFree(g2_points2);
    hipFree(g2_zero_points);
    hipFree(g2_aff_points);
    hipFree(g2_res_points1);
    hipFree(g2_res_points2);
#endif

    hipDeviceReset();
  }

  void SetUp() override
  {
    ASSERT_EQ(device_populate_random<projective_t>(points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<projective_t>(points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_t>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_t>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set<projective_t>(zero_points, projective_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_t>(one_scalars, scalar_t::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(aff_points, 0, n * sizeof(affine_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points1, 0, n * sizeof(projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points2, 0, n * sizeof(projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars, 0, n * sizeof(scalar_t)), hipSuccess);

#ifdef G2
    ASSERT_EQ(device_populate_random<g2_projective_t>(g2_points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<g2_projective_t>(g2_points2, n), hipSuccess);
    ASSERT_EQ(device_set<g2_projective_t>(g2_zero_points, g2_projective_t::zero(), n), hipSuccess);
    ASSERT_EQ(hipMemset(g2_aff_points, 0, n * sizeof(g2_affine_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_res_points1, 0, n * sizeof(g2_projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_res_points2, 0, n * sizeof(g2_projective_t)), hipSuccess);
#endif
  }
};

TEST_F(CurveTest, ECRandomPointsAreOnCurve)
{
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(projective_t::is_on_curve, points1[i]);
}

TEST_F(CurveTest, ECPointAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(CurveTest, ECPointZeroAddition)
{
  ASSERT_EQ(vec_add(points1, zero_points, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(CurveTest, ECPointAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i] + points2[i], res_points1[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * points1[i], res_points1[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationByTwo)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * points1[i], res_points1[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, res_points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationIsDistributiveOverMultiplication)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, res_points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(CurveTest, ECScalarMultiplicationIsDistributiveOverAddition)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars[i] * points1[i], res_points1[i] + res_points2[i]);
}

TEST_F(CurveTest, ECProjectiveToAffine)
{
  ASSERT_EQ(point_vec_to_affine(points1, aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], projective_t::from_affine(aff_points[i]));
}

TEST_F(CurveTest, ECMixedPointAddition)
{
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(CurveTest, ECMixedAdditionOfNegatedPointEqSubtraction)
{
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], points1[i] + res_points2[i]);
}

#ifdef G2
TEST_F(CurveTest, G2ECRandomPointsAreOnCurve)
{
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(g2_projective_t::is_on_curve, g2_points1[i]);
}

TEST_F(CurveTest, G2ECPointAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(g2_res_points1, g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points2[i]);
}

TEST_F(CurveTest, G2ECPointZeroAddition)
{
  ASSERT_EQ(vec_add(g2_points1, g2_zero_points, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points1[i]);
}

TEST_F(CurveTest, G2ECPointAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i] + g2_points2[i], g2_res_points1[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * g2_points1[i], g2_res_points1[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points1[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(g2_points1, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationByTwo)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * g2_points1[i], g2_res_points1[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, g2_res_points1, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points2[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationIsDistributiveOverMultiplication)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, g2_res_points1, g2_res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(CurveTest, G2ECScalarMultiplicationIsDistributiveOverAddition)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, g2_points1, g2_res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars[i] * g2_points1[i], g2_res_points1[i] + g2_res_points2[i]);
}

TEST_F(CurveTest, G2ECProjectiveToAffine)
{
  ASSERT_EQ(point_vec_to_affine(g2_points1, g2_aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_projective_t::from_affine(g2_aff_points[i]));
}

TEST_F(CurveTest, G2ECMixedPointAddition)
{
  ASSERT_EQ(point_vec_to_affine(g2_points2, g2_aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(g2_points1, g2_aff_points, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(CurveTest, G2ECMixedAdditionOfNegatedPointEqSubtraction)
{
  ASSERT_EQ(point_vec_to_affine(g2_points2, g2_aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(g2_points1, g2_aff_points, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_points1[i] + g2_res_points2[i]);
}
#endif
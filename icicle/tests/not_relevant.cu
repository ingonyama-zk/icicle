#include "hip/hip_runtime.h"
#include <vector>
#include <cupqc.hpp>
#include <cassert>
#include <cstdio>
#include <string>
#include <hip/hip_runtime.h>   
using namespace cupqc;
    
#define DEBUG_KEY_GEN false // Disable debugging for key generation
    
using MLKEM768Key = decltype(ML_KEM_768()
                            + Function<function::Keygen>()
                            + Block()
                            + BlockDim<32>());  // Optional operator with default config
    
using MLKEM768Encaps = decltype(ML_KEM_768()
                               + Function<function::Encaps>()
                               + Block()
                               + BlockDim<32>());  // Optional operator with default config

using MLKEM768Decaps = decltype(ML_KEM_768()
                               + Function<function::Decaps>()
                               + Block()
                               + BlockDim<32>());  // Optional operator with default config

__global__ void keygen_kernel(uint8_t* public_keys, uint8_t* secret_keys, uint8_t* workspace, uint8_t* randombytes)
{
    __shared__ uint8_t smem_ptr[MLKEM768Key::shared_memory_size];
    int block = blockIdx.x;
    auto public_key = public_keys + block * MLKEM768Key::public_key_size;
    auto secret_key = secret_keys + block * MLKEM768Key::secret_key_size;
    auto entropy    = randombytes + block * MLKEM768Key::entropy_size;
    auto work       = workspace   + block * MLKEM768Key::workspace_size;
    
    MLKEM768Key().execute(public_key, secret_key, entropy, work, smem_ptr);
}

__global__ void encaps_kernel(uint8_t* ciphertexts, uint8_t* shared_secrets, const uint8_t* public_keys, uint8_t* workspace, uint8_t* randombytes)
{   
    __shared__ uint8_t smem_ptr[MLKEM768Encaps::shared_memory_size];
    int block = blockIdx.x;
    auto shared_secret = shared_secrets + block * MLKEM768Encaps::shared_secret_size;
    auto ciphertext    = ciphertexts + block * MLKEM768Encaps::ciphertext_size;
    auto public_key    = public_keys + block * MLKEM768Encaps::public_key_size;
    auto entropy       = randombytes + block * MLKEM768Encaps::entropy_size;
    auto work          = workspace   + block * MLKEM768Encaps::workspace_size;

    MLKEM768Encaps().execute(ciphertext, shared_secret, public_key, entropy, work, smem_ptr);
}
        
__global__ void decaps_kernel(uint8_t* shared_secrets, const uint8_t* ciphertexts, const uint8_t* secret_keys, uint8_t* workspace)
{       
    __shared__ uint8_t smem_ptr[MLKEM768Decaps::shared_memory_size];
    int block = blockIdx.x;
    auto shared_secret = shared_secrets + block * MLKEM768Decaps::shared_secret_size;
    auto ciphertext    = ciphertexts + block * MLKEM768Decaps::ciphertext_size;
    auto secret_key    = secret_keys + block * MLKEM768Decaps::secret_key_size;
    auto work          = workspace   + block * MLKEM768Decaps::workspace_size;

    MLKEM768Decaps().execute(shared_secret, ciphertext, secret_key, work, smem_ptr);
}

void benchmark(const std::string& operation_name, const hipEvent_t& start, const hipEvent_t& stop, unsigned int batch) {
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    double seconds = milliseconds / 1000.0;
    double throughput = batch / seconds;
    printf("%s Throughput: ~%.2f ops/sec\n", operation_name.c_str(), throughput);
}

void ml_kem_keygen(std::vector<uint8_t>& public_keys, std::vector<uint8_t>& secret_keys, const unsigned int batch)
{
    auto length_public_key = MLKEM768Key::public_key_size;
    auto length_secret_key = MLKEM768Key::secret_key_size;

    auto workspace   = make_workspace<MLKEM768Key>(batch);
    auto randombytes = get_entropy<MLKEM768Key>(batch);

    uint8_t* d_public_key = nullptr;
    uint8_t* d_secret_key = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_public_key), length_public_key * batch);
    hipMalloc(reinterpret_cast<void**>(&d_secret_key), length_secret_key * batch);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    keygen_kernel<<<batch, MLKEM768Key::BlockDim>>>(d_public_key, d_secret_key, workspace, randombytes);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(public_keys.data(), d_public_key, length_public_key * batch, hipMemcpyDeviceToHost);
    hipMemcpy(secret_keys.data(), d_secret_key, length_secret_key * batch, hipMemcpyDeviceToHost);

    benchmark("Key Generation", start, stop, batch);

    hipFree(d_public_key);
    hipFree(d_secret_key);
    destroy_workspace(workspace);
    release_entropy(randombytes);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void ml_kem_encaps(std::vector<uint8_t>& ciphertexts, std::vector<uint8_t>& shared_secrets,
                   const std::vector<uint8_t>& public_keys, const unsigned int batch)
{
    auto length_ciphertext   = MLKEM768Encaps::ciphertext_size;
    auto length_sharedsecret = MLKEM768Encaps::shared_secret_size;
    auto length_public_key   = MLKEM768Encaps::public_key_size;

    auto workspace   = make_workspace<MLKEM768Encaps>(batch);
    auto randombytes = get_entropy<MLKEM768Encaps>(batch);

    uint8_t* d_ciphertext   = nullptr;
    uint8_t* d_sharedsecret = nullptr;
    uint8_t* d_public_key   = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_ciphertext), length_ciphertext * batch);
    hipMalloc(reinterpret_cast<void**>(&d_sharedsecret), length_sharedsecret * batch);
    hipMalloc(reinterpret_cast<void**>(&d_public_key), length_public_key * batch);

    hipMemcpy(d_public_key, public_keys.data(), length_public_key * batch, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    encaps_kernel<<<batch, MLKEM768Encaps::BlockDim>>>(d_ciphertext, d_sharedsecret, d_public_key, workspace, randombytes);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(ciphertexts.data(), d_ciphertext, length_ciphertext * batch, hipMemcpyDeviceToHost);
    hipMemcpy(shared_secrets.data(), d_sharedsecret, length_sharedsecret * batch, hipMemcpyDeviceToHost);

    benchmark("Encapsulation", start, stop, batch);

    hipFree(d_ciphertext);
    hipFree(d_sharedsecret);
    hipFree(d_public_key);
    destroy_workspace(workspace);
    release_entropy(randombytes);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void ml_kem_decaps(std::vector<uint8_t>& shared_secrets, const std::vector<uint8_t>& ciphertexts,
                   const std::vector<uint8_t>& secret_keys, const unsigned int batch)
{
    auto length_ciphertext   = MLKEM768Decaps::ciphertext_size;
    auto length_sharedsecret = MLKEM768Decaps::shared_secret_size;
    auto length_secret_key   = MLKEM768Decaps::secret_key_size;

    auto workspace   = make_workspace<MLKEM768Decaps>(batch);

    uint8_t* d_ciphertext   = nullptr;
    uint8_t* d_sharedsecret = nullptr;
    uint8_t* d_secret_key   = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_ciphertext), length_ciphertext * batch);
    hipMalloc(reinterpret_cast<void**>(&d_sharedsecret), length_sharedsecret * batch);
    hipMalloc(reinterpret_cast<void**>(&d_secret_key), length_secret_key * batch);

    hipMemcpy(d_ciphertext, ciphertexts.data(), length_ciphertext * batch, hipMemcpyHostToDevice);
    hipMemcpy(d_secret_key, secret_keys.data(), length_secret_key * batch, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    decaps_kernel<<<batch, MLKEM768Decaps::BlockDim>>>(d_sharedsecret, d_ciphertext, d_secret_key, workspace);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(shared_secrets.data(), d_sharedsecret, length_sharedsecret * batch, hipMemcpyDeviceToHost);

    benchmark("Decapsulation", start, stop, batch);

    hipFree(d_ciphertext);
    hipFree(d_sharedsecret);
    hipFree(d_secret_key);
    destroy_workspace(workspace);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[]) {
    for (unsigned int batch = 1 << 9; batch <= (1 << 22); batch <<= 1) {
        printf("\nBenchmarking with batch size: %u\n", batch);

        std::vector<uint8_t> public_keys(MLKEM768Key::public_key_size * batch);
        std::vector<uint8_t> secret_keys(MLKEM768Key::secret_key_size * batch);
        std::vector<uint8_t> ciphertexts(MLKEM768Encaps::ciphertext_size * batch);
        std::vector<uint8_t> encaps_shared_secrets(MLKEM768Encaps::shared_secret_size * batch);
        std::vector<uint8_t> decaps_shared_secrets(MLKEM768Decaps::shared_secret_size * batch);

        ml_kem_keygen(public_keys, secret_keys, batch);

        ml_kem_encaps(ciphertexts, encaps_shared_secrets, public_keys, batch);

        ml_kem_decaps(decaps_shared_secrets, ciphertexts, secret_keys, batch);

        printf("Benchmarking completed successfully for batch size %u\n", batch);
    }
}
#include "../primitives/test_kernels.cuh"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

template <class T>
int device_populate_random(T* d_elements, unsigned n)
{
  T h_elements[n];
  for (unsigned i = 0; i < n; i++)
    h_elements[i] = T::rand_host();
  return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

template <class T>
int device_set(T* d_elements, T el, unsigned n)
{
  T h_elements[n];
  for (unsigned i = 0; i < n; i++)
    h_elements[i] = el;
  return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

class PrimitivesTest : public ::testing::Test
{
protected:
  static const unsigned n = 1 << 4;

  projective_t* points1{};
  projective_t* points2{};
  g2_projective_t* g2_points1{};
  g2_projective_t* g2_points2{};
  scalar_t* scalars1{};
  scalar_t* scalars2{};
  projective_t* zero_points{};
  g2_projective_t* g2_zero_points{};
  scalar_t* zero_scalars{};
  scalar_t* one_scalars{};
  affine_t* aff_points{};
  g2_affine_t* g2_aff_points{};
  projective_t* res_points1{};
  projective_t* res_points2{};
  g2_projective_t* g2_res_points1{};
  g2_projective_t* g2_res_points2{};
  scalar_t* res_scalars1{};
  scalar_t* res_scalars2{};
  scalar_t::Wide* res_scalars_wide{};
  scalar_t::Wide* res_scalars_wide_full{};

  PrimitivesTest()
  {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&points1, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&points2, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&g2_points1, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_points2, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&scalars1, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&zero_points, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&g2_zero_points, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&zero_scalars, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&aff_points, n * sizeof(affine_t)));
    assert(!hipMallocManaged(&g2_aff_points, n * sizeof(g2_affine_t)));
    assert(!hipMallocManaged(&res_points1, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&res_points2, n * sizeof(projective_t)));
    assert(!hipMallocManaged(&g2_res_points1, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&g2_res_points2, n * sizeof(g2_projective_t)));
    assert(!hipMallocManaged(&res_scalars1, n * sizeof(scalar_t)));
    assert(!hipMallocManaged(&res_scalars2, n * sizeof(scalar_t)));
  }

  ~PrimitivesTest() override
  {
    hipFree(points1);
    hipFree(points2);
    hipFree(g2_points1);
    hipFree(g2_points2);
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_points);
    hipFree(g2_zero_points);
    hipFree(zero_scalars);
    hipFree(one_scalars);
    hipFree(aff_points);
    hipFree(g2_aff_points);
    hipFree(res_points1);
    hipFree(res_points2);
    hipFree(g2_res_points1);
    hipFree(g2_res_points2);
    hipFree(res_scalars1);
    hipFree(res_scalars2);

    hipDeviceReset();
  }

  void SetUp() override
  {
    ASSERT_EQ(device_populate_random<projective_t>(points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<projective_t>(points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<g2_projective_t>(g2_points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<g2_projective_t>(g2_points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_t>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_t>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set<projective_t>(zero_points, projective_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<g2_projective_t>(g2_zero_points, g2_projective_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_t>(zero_scalars, scalar_t::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_t>(one_scalars, scalar_t::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(aff_points, 0, n * sizeof(affine_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_aff_points, 0, n * sizeof(g2_affine_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points1, 0, n * sizeof(projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points2, 0, n * sizeof(projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_res_points1, 0, n * sizeof(g2_projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(g2_res_points2, 0, n * sizeof(g2_projective_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars1, 0, n * sizeof(scalar_t)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars2, 0, n * sizeof(scalar_t)), hipSuccess);
  }
};

TEST_F(PrimitivesTest, FieldAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_scalars1, scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldZeroAddition)
{
  ASSERT_EQ(vec_add(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] + scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(scalars1, one_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, res_scalars1, res_scalars2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, res_scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], zero_scalars[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByZero)
{
  ASSERT_EQ(vec_mul(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(zero_scalars[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i] * res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByTwoEqSum)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars2[i], scalars1[i] + scalars1[i]);
}

TEST_F(PrimitivesTest, FieldSqrHostDeviceEq)
{
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationSqrEq)
{
  ASSERT_EQ(vec_mul(scalars1, scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, ECRandomPointsAreOnCurve)
{
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(projective_t::is_on_curve, points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECPointZeroAddition)
{
  ASSERT_EQ(vec_add(points1, zero_points, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i] + points2[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByTwo)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, res_points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverMultiplication)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, res_points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverAddition)
{
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i] * points1[i], res_points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, ECProjectiveToAffine)
{
  ASSERT_EQ(point_vec_to_affine(points1, aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], projective_t::from_affine(aff_points[i]));
}

TEST_F(PrimitivesTest, ECMixedPointAddition)
{
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECMixedAdditionOfNegatedPointEqSubtraction)
{
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECRandomPointsAreOnCurve)
{
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(g2_projective_t::is_on_curve, g2_points1[i]);
}

TEST_F(PrimitivesTest, G2ECPointAdditionSubtractionCancel)
{
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(g2_res_points1, g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECPointZeroAddition)
{
  ASSERT_EQ(vec_add(g2_points1, g2_zero_points, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECPointAdditionHostDeviceEq)
{
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i] + g2_points2[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationHostDeviceEq)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationByOne)
{
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationByMinusOne)
{
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(g2_points1, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationByTwo)
{
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * g2_points1[i], g2_res_points1[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationInverseCancel)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_res_points1, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationIsDistributiveOverMultiplication)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, g2_res_points1, g2_res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECScalarMultiplicationIsDistributiveOverAddition)
{
  ASSERT_EQ(vec_mul(scalars1, g2_points1, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, g2_points1, g2_res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i] * g2_points1[i], g2_res_points1[i] + g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECProjectiveToAffine)
{
  ASSERT_EQ(point_vec_to_affine(g2_points1, g2_aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_points1[i], g2_projective_t::from_affine(g2_aff_points[i]));
}

TEST_F(PrimitivesTest, G2ECMixedPointAddition)
{
  ASSERT_EQ(point_vec_to_affine(g2_points2, g2_aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(g2_points1, g2_aff_points, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(g2_points1, g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_res_points2[i]);
}

TEST_F(PrimitivesTest, G2ECMixedAdditionOfNegatedPointEqSubtraction)
{
  ASSERT_EQ(point_vec_to_affine(g2_points2, g2_aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(g2_points1, g2_aff_points, g2_res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(g2_points2, g2_res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(g2_res_points1[i], g2_points1[i] + g2_res_points2[i]);
}
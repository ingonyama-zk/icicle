#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

#include "error_handler_test.cu"
// #include "primitives_test.cu"
#include "device_error_test.cu"

int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

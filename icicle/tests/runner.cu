#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>

// include list of test files
// Ensure the device_error_test.cu is last to prevent aborting mid-test run
#include "field_test.cu"
#ifdef CURVE_ID
#include "curve_test.cu"
#endif
#include "error_handler_test.cu"
#include "device_error_test.cu"

int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  printf("running gtests...\n");
  return RUN_ALL_TESTS();
}
#include "hip/hip_runtime.h"

#include <gtest/gtest.h>
#include <iostream>
#include <memory>
#include <vector>
#include <list>

#include "curves/curve_config.cuh"
typedef curve_config::scalar_t test_type;

#include "polynomials/polynomials.h"
#include "appUtils/ntt/ntt.cuh"

using FpMicroseconds = std::chrono::duration<float, std::chrono::microseconds::period>;
#define START_TIMER(timer) auto timer##_start = std::chrono::high_resolution_clock::now();
#define END_TIMER(timer, msg, enable)                                                                                  \
  if (enable)                                                                                                          \
    printf(                                                                                                            \
      "%s: %.3f ms\n", msg, FpMicroseconds(std::chrono::high_resolution_clock::now() - timer##_start).count() / 1000);

using namespace polynomials;

typedef Polynomial<test_type> Polynomial_t;

class PolynomialTest : public ::testing::Test
{
public:
  static inline const int MAX_NTT_LOG_SIZE = 24;
  static inline const bool MEASURE = true;

  // SetUpTestSuite/TearDownTestSuite are called once for the entire test suite
  static void SetUpTestSuite()
  {
    // init NTT domain
    auto ntt_config = ntt::DefaultNTTConfig<test_type>();
    const test_type basic_root = test_type::omega(MAX_NTT_LOG_SIZE);
    ntt::InitDomain(basic_root, ntt_config.ctx);
    // initializing polynoimals factory for CUDA backend
    Polynomial_t::initialize(std::make_unique<CUDAPolynomialFactory<>>());
  }

  static void TearDownTestSuite() {}

  void SetUp() override
  {
    // code that executes before each test
  }

  void TearDown() override
  {
    // code that executes before each test
  }

  static Polynomial_t randomize_polynomial(uint32_t size, bool random = true)
  {
    auto coeff = std::make_unique<test_type[]>(size);
    if (random) {
      random_samples(coeff.get(), size);
    } else {
      incremental_values(coeff.get(), size);
    }
    return Polynomial_t::from_coefficients(coeff.get(), size);
  }

  static void random_samples(test_type* res, uint32_t count)
  {
    for (int i = 0; i < count; i++)
      res[i] = test_type::rand_host();
  }

  static void incremental_values(test_type* res, uint32_t count)
  {
    for (int i = 0; i < count; i++) {
      res[i] = i ? res[i - 1] + test_type::one() : test_type::one();
    }
  }

  static void assert_equal(Polynomial_t& lhs, Polynomial_t& rhs)
  {
    const int deg_lhs = lhs.degree();
    const int deg_rhs = rhs.degree();
    ASSERT_EQ(deg_lhs, deg_rhs);

    auto lhs_coeffs = std::make_unique<test_type[]>(deg_lhs);
    auto rhs_coeffs = std::make_unique<test_type[]>(deg_rhs);
    lhs.get_coefficients_on_host(lhs_coeffs.get(), 1, deg_lhs - 1);
    rhs.get_coefficients_on_host(rhs_coeffs.get(), 1, deg_rhs - 1);

    ASSERT_EQ(0, memcmp(lhs_coeffs.get(), rhs_coeffs.get(), deg_lhs * sizeof(test_type)));
  }

  static Polynomial_t vanishing_polynomial(int degree)
  {
    test_type coeffs_v[degree + 1] = {0};
    coeffs_v[0] = minus_one; // -1
    coeffs_v[degree] = one;  // +x^n
    auto v = Polynomial_t::from_coefficients(coeffs_v, degree + 1);
    return v;
  }

  const static inline auto zero = test_type::zero();
  const static inline auto one = test_type::one();
  const static inline auto two = test_type::from(2);
  const static inline auto three = test_type::from(3);
  const static inline auto four = test_type::from(4);
  const static inline auto five = test_type::from(5);
  const static inline auto minus_one = zero - one;
};

TEST_F(PolynomialTest, evaluation)
{
  const test_type coeffs[3] = {one, two, three};
  auto f = Polynomial_t::from_coefficients(coeffs, 3);
  test_type x = test_type::rand_host();

  auto f_x = f(x); // evaluation

  auto expected_f_x = one + two * x + three * x * x;

  EXPECT_EQ(f_x, expected_f_x);
}

TEST_F(PolynomialTest, fromEvaluations)
{
  const int size = 100;
  const int log_size = (int)ceil(log2(size));
  const int nof_evals = 1 << log_size;
  auto f = randomize_polynomial(size);

  // evaluate f on roots of unity
  test_type omega = test_type::omega(log_size);
  test_type evals[nof_evals] = {0};
  test_type x = one;
  for (int i = 0; i < nof_evals; ++i) {
    evals[i] = f(x);
    x = x * omega;
  }

  // construct g from f's evaluations
  auto g = Polynomial_t::from_rou_evaluations(evals, nof_evals);

  // make sure they are equal, that is f-g=0
  auto h = f - g;
  EXPECT_EQ(h.degree(), -1); // degree -1 is the zero polynomial
}

TEST_F(PolynomialTest, fromEvaluationsNotPowerOfTwo)
{
  const int size = 100;
  const int log_size = (int)ceil(log2(size));
  const int nof_evals = size;
  auto f = randomize_polynomial(size);

  // evaluate f on roots of unity
  test_type omega = test_type::omega(log_size);
  test_type evals[nof_evals] = {0};
  test_type x = one;
  for (int i = 0; i < nof_evals; ++i) {
    evals[i] = f(x);
    x = x * omega;
  }

  // construct g from f's evaluations
  auto g = Polynomial_t::from_rou_evaluations(evals, nof_evals);

  test_type r = test_type::rand_host();

  // since NTT works on a power of two (therefore the extra elements are arbitrary), f!=g but they should evaluate to
  // the same values on the roots of unity due to construction.
  x = one;
  for (int i = 0; i < nof_evals; ++i) {
    EXPECT_EQ(f(x), g(x));
    x = x * omega;
  }
}

TEST_F(PolynomialTest, addition)
{
  const int size_0 = 12, size_1 = 17;
  auto f = randomize_polynomial(size_0);
  auto g = randomize_polynomial(size_1);

  test_type x = test_type::rand_host();
  auto f_x = f(x);
  auto g_x = g(x);
  auto fx_plus_gx = f_x + g_x;

  auto s = f + g;
  auto s_x = s(x);

  EXPECT_EQ(fx_plus_gx, s_x);
}

TEST_F(PolynomialTest, addition_inplace)
{
  const int size_0 = 2, size_1 = 2;
  auto f = randomize_polynomial(size_0);
  auto g = randomize_polynomial(size_1);

  test_type x = test_type::rand_host();
  auto f_x = f(x);
  auto g_x = g(x);
  auto fx_plus_gx = f_x + g_x;

  f += g;
  auto s_x = f(x);

  EXPECT_EQ(fx_plus_gx, s_x);
}

TEST_F(PolynomialTest, cAPI)
{
  const int size = 3;
  auto coeff = std::make_unique<test_type[]>(size);
  random_samples(coeff.get(), size);

  auto f = polynomial_create_from_coefficients(coeff.get(), size);
  auto g = polynomial_create_from_coefficients(coeff.get(), size);
  auto s = polynomial_add(f, g);

  test_type x = test_type::rand_host();
  // TODO Yuval: use C-API for evaluate too
  auto f_x = f->evaluate(x);
  auto g_x = g->evaluate(x);
  auto fx_plus_gx = f_x + g_x;
  auto s_x = s->evaluate(x);
  EXPECT_EQ(fx_plus_gx, s_x);

  polynomial_delete(f);
  polynomial_delete(g);
  polynomial_delete(s);
}

TEST_F(PolynomialTest, multiplication)
{
  const int size_0 = 1 << 15, size_1 = 1 << 12;
  auto f = randomize_polynomial(size_0);
  auto g = randomize_polynomial(size_1);

  test_type x = test_type::rand_host();
  auto f_x = f(x);
  auto g_x = g(x);
  auto fx_mul_gx = f_x * g_x;

  START_TIMER(poly_mult_start);
  auto m = f * g;
  END_TIMER(poly_mult_start, "Polynomial multiplication took", MEASURE);

  auto m_x = m(x);

  EXPECT_EQ(fx_mul_gx, m_x);
}

TEST_F(PolynomialTest, multiplicationScalar)
{
  const int size = 17;
  auto f = randomize_polynomial(size);

  auto g = two * f;
  auto h = f * three;

  test_type x = test_type::rand_host();
  auto f_x = f(x);
  auto g_x = g(x);
  auto h_x = h(x);

  EXPECT_EQ(g_x, f_x * two);
  EXPECT_EQ(h_x, f_x * three);

  EXPECT_EQ(g.degree(), f.degree());
  EXPECT_EQ(h.degree(), f.degree());
}

TEST_F(PolynomialTest, monomials)
{
  const test_type coeffs[3] = {one, zero, two}; // 1+2x^2
  auto f = Polynomial_t::from_coefficients(coeffs, 3);
  const auto x = three;
  const auto expected_f_x = one + two * x * x;
  auto f_x = f(x);

  EXPECT_EQ(f_x, expected_f_x);

  f.add_monomial_inplace(three, 1); // add 3x
  const auto expected_addmonmon_f_x = f_x + three * x;
  const auto addmonom_f_x = f(x);

  EXPECT_EQ(addmonom_f_x, expected_addmonmon_f_x);

  f.sub_monomial_inplace(one); // subtract 1. equivalent to 'f-1'
  const auto expected_submonom_f_x = addmonom_f_x - one;
  const auto submonom_f_x = f(x);

  EXPECT_EQ(submonom_f_x, expected_submonom_f_x);
}

TEST_F(PolynomialTest, ReadCoeffsToHost)
{
  const test_type coeffs_f[3] = {zero, one, two}; // x+2x^2
  auto f = Polynomial_t::from_coefficients(coeffs_f, 3);
  const test_type coeffs_g[3] = {one, one, one}; // 1+x+x^2
  auto g = Polynomial_t::from_coefficients(coeffs_g, 3);

  auto h = f + g; // 1+2x+3x^3
  const auto h0 = h.get_coefficient_on_host(0);
  const auto h1 = h.get_coefficient_on_host(1);
  const auto h2 = h.get_coefficient_on_host(2);
  EXPECT_EQ(h0, one);
  EXPECT_EQ(h1, two);
  EXPECT_EQ(h2, three);

  int64_t nof_coeffs = h.get_coefficients_on_host(nullptr); // query #coeffs
  EXPECT_GE(nof_coeffs, 3);                                 // can be larger due to padding to powers of two
  test_type h_coeffs[3] = {0};
  nof_coeffs = h.get_coefficients_on_host(h_coeffs, 0, 2); // read the coefficients
  EXPECT_EQ(nof_coeffs, 3);                                // expecting 3 due to specified indices

  test_type expected_h_coeffs[nof_coeffs] = {one, two, three};
  for (int i = 0; i < nof_coeffs; ++i) {
    EXPECT_EQ(expected_h_coeffs[i], h_coeffs[i]);
  }
}

TEST_F(PolynomialTest, divisionSimple)
{
  const test_type coeffs_a[4] = {five, zero, four, three}; // 3x^3+4x^2+5
  const test_type coeffs_b[3] = {minus_one, zero, one};    // x^2-1
  auto a = Polynomial_t::from_coefficients(coeffs_a, 4);
  auto b = Polynomial_t::from_coefficients(coeffs_b, 3);

  auto [q, r] = a.divide(b);
  test_type q_coeffs[2] = {0}; // 3x+4
  test_type r_coeffs[2] = {0}; // 3x+9
  const auto q_nof_coeffs = q.get_coefficients_on_host(q_coeffs, 0, 1);
  const auto r_nof_coeffs = r.get_coefficients_on_host(r_coeffs, 0, 1);

  ASSERT_EQ(q_nof_coeffs, 2);
  ASSERT_EQ(r_nof_coeffs, 2);
  ASSERT_EQ(q_coeffs[0], test_type::from(4));
  ASSERT_EQ(q_coeffs[1], test_type::from(3));
  ASSERT_EQ(r_coeffs[0], test_type::from(9));
  ASSERT_EQ(r_coeffs[1], test_type::from(3));
}

TEST_F(PolynomialTest, divisionLarge)
{
  const int size_0 = 1 << 12, size_1 = 1 << 2;
  auto a = randomize_polynomial(size_0);
  auto b = randomize_polynomial(size_1);

  START_TIMER(poly_mult_start);
  auto [q, r] = a.divide(b);
  END_TIMER(poly_mult_start, "Polynomial division took", MEASURE);

  test_type x = test_type::rand_host();
  auto a_x = a(x);
  auto b_x = b(x);
  auto q_x = q(x);
  auto r_x = r(x);

  // a(x) = b(x)*q(x)+r(x)
  EXPECT_EQ(a_x, b_x * q_x + r_x);
}

TEST_F(PolynomialTest, divideByVanishingPolynomial)
{
  const test_type coeffs_v[5] = {minus_one, zero, zero, zero, one}; // x^4-1 vanishes on 4th roots of unity
  auto v = Polynomial_t::from_coefficients(coeffs_v, 5);
  auto h = randomize_polynomial(1 << 11, false);
  auto hv = h * v;

  START_TIMER(poly_div_long);
  auto [h_div, R] = hv.divide(v);
  END_TIMER(poly_div_long, "Polynomial division by vanishing (long division) took", MEASURE);
  assert_equal(h_div, h);

  START_TIMER(poly_div_vanishing);
  auto h_div_by_vanishing = hv.divide_by_vanishing_polynomial(4);
  END_TIMER(poly_div_vanishing, "Polynomial division by vanishing (fast) took", MEASURE);
  assert_equal(h_div_by_vanishing, h);
}

// TODO Yuval: move to examples ??
TEST_F(PolynomialTest, QAP)
{
  // this examples is randomizing N private numbers and proving that I know N numbers such that their product is equal
  // to 'out'.
  //
  // Circuit:
  //
  // in0  in1
  //  \   /
  //   \ /  in2
  //   (X)  /
  //   t0\ /  in3
  //     (X)  /
  //     t1\ /
  //       (X)
  //        .
  //        .
  //        .
  //        |
  //       out
  //
  // simple construction: t0=in0*in1, t1=t0*in2, t2=t1*in3 and so on to simplify the example

  // (1) randomize N numbers and construct the witness as [1,out,...N inputs..., ... intermediate values...]
  const int nof_inputs = 5;
  const int nof_outputs = 1;
  const int nof_intermediates = nof_inputs - 2; // same as #multiplication gates minus last one (which is the output)
  const int witness_size =
    1 + nof_outputs + nof_inputs + nof_intermediates; // witness = [1, out, inputs..., intermediates...]

  const int input_offset = 1 + nof_outputs;
  const int intermediate_offset = input_offset + nof_inputs;

  std::vector<test_type> witness(witness_size, test_type::zero());
  witness[0] = test_type::one();
  random_samples(witness.data() + input_offset, nof_inputs); // randomize inputs
  // compute intermediate values (based on the circuit above)
  for (int i = 0; i < nof_intermediates; ++i) {
    const auto& left_input = i == 0 ? witness[input_offset] : witness[intermediate_offset + i - 1];
    const auto& right_input = witness[input_offset + i + 1];
    witness[intermediate_offset + i] = left_input * right_input;
  }
  // compute output as last_input * last_intermediate
  witness[1] = witness[input_offset + nof_inputs - 1] * witness[intermediate_offset + nof_intermediates - 1];

  // (2) construt matrices A,B,C (based on the circuit)
  const int nof_constraints = nof_inputs - 1;
  // allocating such that columns are consecutive in memory for more efficient polynomial construction from consecutive
  // evaluations
  const int nof_cols = witness_size;
  const int nof_rows = nof_constraints;
  std::vector<test_type> L(nof_cols * nof_rows, test_type::zero());
  std::vector<test_type> R(nof_cols * nof_rows, test_type::zero());
  std::vector<test_type> O(nof_cols * nof_rows, test_type::zero());

  test_type* L_data = L.data();
  test_type* R_data = R.data();
  test_type* O_data = O.data();

  // filling the R1CS matrices (where cols are consecutive, not rows)
  for (int row = 0; row < nof_rows; ++row) {
    const int L_col = row == 0 ? input_offset : intermediate_offset + row - 1;
    *(L_data + L_col * nof_rows + row) = test_type::one();

    const int R_col = input_offset + row + 1;
    *(R_data + R_col * nof_rows + row) = test_type::one();

    const int O_col = row == nof_rows - 1 ? 1 : intermediate_offset + row;
    *(O_data + O_col * nof_rows + row) = test_type::one();
  }

  // (3) interpolate the columns of L,R,O to build the polynomials
  std::vector<Polynomial_t> L_QAP, R_QAP, O_QAP;
  L_QAP.reserve(nof_cols);
  R_QAP.reserve(nof_cols);
  O_QAP.reserve(nof_cols);
  for (int col = 0; col < nof_cols; ++col) { // #polynomials is equal to witness_size
    L_QAP.push_back(std::move(Polynomial_t::from_rou_evaluations(L_data + col * nof_rows, nof_rows)));
    R_QAP.push_back(std::move(Polynomial_t::from_rou_evaluations(R_data + col * nof_rows, nof_rows)));
    O_QAP.push_back(std::move(Polynomial_t::from_rou_evaluations(O_data + col * nof_rows, nof_rows)));
  }

  // (4) using the witness, compute L(x),R(x),O(x)
  Polynomial_t& Lx = L_QAP[0]; // TODO Yuval: probably better copy
  Polynomial_t& Rx = R_QAP[0]; // TODO Yuval: probably better copy
  Polynomial_t& Ox = O_QAP[0]; // TODO Yuval: probably better copy
  std::cout << "Lx.degree()=" << Lx.degree() << std::endl;
  for (int col = 1; col < nof_cols; ++col) {
    Lx += witness[col] * L_QAP[col];
    Rx += witness[col] * R_QAP[col];
    Ox += witness[col] * O_QAP[col];
    std::cout << "Lx[col].degree()=" << L_QAP[col].degree() << std::endl;
  }

  //  (4b) sanity check: verify that it divides with no remainder
  {
    auto v = vanishing_polynomial(nof_constraints - 1 /*=degree*/);
    std::cout << "Lx.degree()=" << Lx.degree() << std::endl;
    std::cout << "Rx.degree()=" << Rx.degree() << std::endl;
    std::cout << "Ox.degree()=" << Ox.degree() << std::endl;
    std::cout << "LxRx.degree()=" << (Lx * Rx).degree() << std::endl;
    std::cout << "LxRx-Ox.degree()=" << (Lx * Rx - Ox).degree() << std::endl;
    std::cout << "v.degree()=" << v.degree() << std::endl;
    auto [q, r] = (Lx * Rx - Ox).divide(v);
    Polynomial_t h = (Lx * Rx - Ox).divide_by_vanishing_polynomial(nof_constraints);
    std::cout << "q.degree()=" << q.degree() << std::endl;
    std::cout << "r.degree()=" << r.degree() << std::endl;
    std::cout << "h.degree()=" << h.degree() << std::endl;
    EXPECT_EQ(r.degree(), -1);              // zero polynomial
    EXPECT_EQ(q.degree(), nof_constraints); // L(X)R(x)-O(x) is degree 2N so expecting N after division
  }

  // (5) compute h(x) as '(L(x)R(x)-O(x)) / t(x)'
  Polynomial_t h = (Lx * Rx - Ox).divide_by_vanishing_polynomial(nof_constraints);

  // (6) compute A,B,C via MSMs
}

int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
#include <iostream>
#include <cstring>
#include "icicle/device_api.h"
#include "icicle/errors.h"

#include "hip/hip_runtime.h"

using namespace icicle;

class CUDADeviceAPI : public DeviceAPI
{
public:
  eIcicleError setDevice(const Device& device) const
  {
    hipError_t err = hipSetDevice(device.id);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::INVALID_DEVICE;
  }

  // Memory management
  eIcicleError allocateMemory(const Device& device, void** ptr, size_t size) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMalloc(ptr, size);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError allocateMemoryAsync(const Device& device, void** ptr, size_t size, IcicleStreamHandle stream) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMallocAsync(ptr, size, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError freeMemory(const Device& device, void* ptr) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipFree(ptr);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError freeMemoryAsync(const Device& device, void* ptr, IcicleStreamHandle stream) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipFreeAsync(ptr, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError getAvailableMemory(const Device& device, size_t& total /*OUT*/, size_t& free /*OUT*/) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMemGetInfo(&free, &total);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::UNKNOWN_ERROR;
  }

  // Data transfer
  eIcicleError copyToHost(const Device& device, void* dst, const void* src, size_t size) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError
  copyToHostAsync(const Device& device, void* dst, const void* src, size_t size, IcicleStreamHandle stream) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copyToDevice(const Device& device, void* dst, const void* src, size_t size) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError
  copyToDeviceAsync(const Device& device, void* dst, const void* src, size_t size, IcicleStreamHandle stream) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  // Synchronization
  eIcicleError synchronize(const Device& device, IcicleStreamHandle stream = nullptr) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err =
      (stream == nullptr) ? hipDeviceSynchronize() : hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::SYNCHRONIZATION_FAILED;
  }

  // Stream management
  eIcicleError createStream(const Device& device, IcicleStreamHandle* stream) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipStream_t cudaStream;
    hipError_t err = hipStreamCreate(&cudaStream);
    *stream = reinterpret_cast<IcicleStreamHandle>(cudaStream);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_CREATION_FAILED;
  }

  eIcicleError destroyStream(const Device& device, IcicleStreamHandle stream) override
  {
    if (eIcicleError err = setDevice(device); err != eIcicleError::SUCCESS) return err;
    hipError_t err = hipStreamDestroy(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_DESTRUCTION_FAILED;
  }
};

REGISTER_DEVICE_API("CUDA", CUDADeviceAPI);
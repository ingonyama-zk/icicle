#include <iostream>
#include <cstring>
#include "icicle/device_api.h"
#include "icicle/errors.h"

#include "hip/hip_runtime.h"

using namespace icicle;

class CUDADeviceAPI : public DeviceAPI
{
public:
    IcicleError setDevice(const Device& device) const {
        hipError_t err = hipSetDevice(device.id);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::INVALID_DEVICE;
    }

    // Memory management
    IcicleError allocateMemory(const Device& device, void** ptr, size_t size) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMalloc(ptr, size);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::ALLOCATION_FAILED;
    }

    IcicleError allocateMemoryAsync(const Device& device, void** ptr, size_t size, IcicleStreamHandle stream) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMallocAsync(ptr, size, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::ALLOCATION_FAILED;
    }

    IcicleError freeMemory(const Device& device, void* ptr) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipFree(ptr);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::DEALLOCATION_FAILED;
    }

    IcicleError freeMemoryAsync(const Device& device, void* ptr, IcicleStreamHandle stream) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipFreeAsync(ptr, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::DEALLOCATION_FAILED;
    }

    IcicleError getAvailableMemory(const Device& device, size_t& total /*OUT*/, size_t& free /*OUT*/) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMemGetInfo(&free, &total);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::UNKNOWN_ERROR;
    }

    // Data transfer
    IcicleError copyToHost(const Device& device, void* dst, const void* src, size_t size) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    IcicleError copyToHostAsync(const Device& device, void* dst, const void* src, size_t size, IcicleStreamHandle stream) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    IcicleError copyToDevice(const Device& device, void* dst, const void* src, size_t size) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    IcicleError copyToDeviceAsync(const Device& device, void* dst, const void* src, size_t size, IcicleStreamHandle stream) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    // Synchronization
    IcicleError synchronize(const Device& device, IcicleStreamHandle stream = nullptr) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = (stream == nullptr) ? hipDeviceSynchronize() : hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::SYNCHRONIZATION_FAILED;
    }

    // Stream management
    IcicleError createStream(const Device& device, IcicleStreamHandle* stream) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipStream_t cudaStream;
        hipError_t err = hipStreamCreate(&cudaStream);
        *stream = reinterpret_cast<IcicleStreamHandle>(cudaStream);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::STREAM_CREATION_FAILED;
    }

    IcicleError destroyStream(const Device& device, IcicleStreamHandle stream) override
    {
        if(IcicleError err = setDevice(device); err != IcicleError::SUCCESS) return err;
        hipError_t err = hipStreamDestroy(reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::STREAM_DESTRUCTION_FAILED;
    }
};

REGISTER_DEVICE_API("CUDA", CUDADeviceAPI);
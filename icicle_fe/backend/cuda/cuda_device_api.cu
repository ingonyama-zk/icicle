#include <iostream>
#include <cstring>
#include "icicle/device_api.h"
#include "icicle/errors.h"

#include "hip/hip_runtime.h"

using namespace icicle;

class CUDADeviceAPI : public DeviceAPI
{
public:
    // Memory management
    IcicleError allocateMemory(const Device& device, void** ptr, size_t size) override
    {
        hipError_t err = hipMalloc(ptr, size);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::ALLOCATION_FAILED;
    }

    IcicleError allocateMemoryAsync(const Device& device, void** ptr, size_t size, IcicleStreamHandle stream) override
    {
        hipError_t err = hipMallocAsync(ptr, size, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::ALLOCATION_FAILED;
    }

    IcicleError freeMemory(const Device& device, void* ptr) override
    {
        hipError_t err = hipFree(ptr);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::DEALLOCATION_FAILED;
    }

    IcicleError freeMemoryAsync(const Device& device, void* ptr, IcicleStreamHandle stream) override
    {
        hipError_t err = hipFreeAsync(ptr, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::DEALLOCATION_FAILED;
    }

    IcicleError getAvailableMemory(const Device& device, size_t& total /*OUT*/, size_t& free /*OUT*/) override
    {
        size_t freeMem = 0, totalMem = 0;
        hipError_t err = hipMemGetInfo(&freeMem, &totalMem);
        if (err != hipSuccess) {
            return IcicleError::UNKNOWN_ERROR;
        }

        free = freeMem;
        total = totalMem;
        return IcicleError::SUCCESS;
    }

    // Data transfer
    IcicleError copyToHost(const Device& device, void* dst, const void* src, size_t size) override
    {
        hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    IcicleError copyToHostAsync(const Device& device, void* dst, const void* src, size_t size, IcicleStreamHandle stream) override
    {
        hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    IcicleError copyToDevice(const Device& device, void* dst, const void* src, size_t size) override
    {
        hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    IcicleError copyToDeviceAsync(const Device& device, void* dst, const void* src, size_t size, IcicleStreamHandle stream) override
    {
        hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::COPY_FAILED;
    }

    // Synchronization
    IcicleError synchronize(const Device& device, IcicleStreamHandle stream = nullptr) override
    {
        hipError_t err = (stream == nullptr) ? hipDeviceSynchronize() : hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::SYNCHRONIZATION_FAILED;
    }

    // Stream management
    IcicleError createStream(const Device& device, IcicleStreamHandle* stream) override
    {
        hipStream_t cudaStream;
        hipError_t err = hipStreamCreate(&cudaStream);
        *stream = reinterpret_cast<IcicleStreamHandle>(cudaStream);
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::STREAM_CREATION_FAILED;
    }

    IcicleError destroyStream(const Device& device, IcicleStreamHandle stream) override
    {
        hipError_t err = hipStreamDestroy(reinterpret_cast<hipStream_t>(stream));
        return (err == hipSuccess) ? IcicleError::SUCCESS : IcicleError::STREAM_DESTRUCTION_FAILED;
    }
};

REGISTER_DEVICE_API("CUDA", CUDADeviceAPI);
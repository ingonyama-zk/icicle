#include <iostream>
#include <cstring>
#include "icicle/device_api.h"
#include "icicle/errors.h"

#include "hip/hip_runtime.h"

using namespace icicle;

class CUDADeviceAPI : public DeviceAPI
{
public:
  eIcicleError setDevice(const Device& device) override
  {
    hipError_t err = hipSetDevice(device.id);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::INVALID_DEVICE;
  }

  // Memory management
  eIcicleError allocateMemory(void** ptr, size_t size) const override
  {
    hipError_t err = hipMalloc(ptr, size);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError allocateMemoryAsync(void** ptr, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMallocAsync(ptr, size, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError freeMemory(void* ptr) const override
  {
    hipError_t err = hipFree(ptr);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError freeMemoryAsync(void* ptr, icicleStreamHandle stream) const override
  {
    hipError_t err = hipFreeAsync(ptr, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError getAvailableMemory(size_t& total /*OUT*/, size_t& free /*OUT*/) const override
  {
    hipError_t err = hipMemGetInfo(&free, &total);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::UNKNOWN_ERROR;
  }

  // Data transfer
  eIcicleError copyToHost(void* dst, const void* src, size_t size) const override
  {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copyToHostAsync(void* dst, const void* src, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copyToDevice(void* dst, const void* src, size_t size) const override
  {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copyToDeviceAsync(void* dst, const void* src, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  // Synchronization
  eIcicleError synchronize(icicleStreamHandle stream = nullptr) const override
  {
    hipError_t err =
      (stream == nullptr) ? hipDeviceSynchronize() : hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::SYNCHRONIZATION_FAILED;
  }

  // Stream management
  eIcicleError createStream(icicleStreamHandle* stream) const override
  {
    hipStream_t cudaStream;
    hipError_t err = hipStreamCreate(&cudaStream);
    *stream = reinterpret_cast<icicleStreamHandle>(cudaStream);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_CREATION_FAILED;
  }

  eIcicleError destroyStream(icicleStreamHandle stream) const override
  {
    hipError_t err = hipStreamDestroy(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_DESTRUCTION_FAILED;
  }
};

REGISTER_DEVICE_API("CUDA", CUDADeviceAPI);
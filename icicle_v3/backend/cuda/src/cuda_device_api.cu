#include <iostream>
#include <cstring>
#include "icicle/device_api.h"
#include "icicle/errors.h"

#include "hip/hip_runtime.h"

using namespace icicle;

class CudaDeviceAPI : public DeviceAPI
{
public:
  eIcicleError set_device(const Device& device) override
  {
    hipError_t err = hipSetDevice(device.id);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::INVALID_DEVICE;
  }

  // Memory management
  eIcicleError allocate_memory(void** ptr, size_t size) const override
  {
    hipError_t err = hipMalloc(ptr, size);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError allocate_memory_async(void** ptr, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMallocAsync(ptr, size, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::ALLOCATION_FAILED;
  }

  eIcicleError free_memory(void* ptr) const override
  {
    hipError_t err = hipFree(ptr);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError free_memory_async(void* ptr, icicleStreamHandle stream) const override
  {
    hipError_t err = hipFreeAsync(ptr, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::DEALLOCATION_FAILED;
  }

  eIcicleError get_available_memory(size_t& total /*OUT*/, size_t& free /*OUT*/) const override
  {
    hipError_t err = hipMemGetInfo(&free, &total);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::UNKNOWN_ERROR;
  }

  // Data transfer
  eIcicleError copy_to_host(void* dst, const void* src, size_t size) const override
  {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copy_to_host_async(void* dst, const void* src, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copy_to_device(void* dst, const void* src, size_t size) const override
  {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  eIcicleError copy_to_device_async(void* dst, const void* src, size_t size, icicleStreamHandle stream) const override
  {
    hipError_t err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::COPY_FAILED;
  }

  // Synchronization
  eIcicleError synchronize(icicleStreamHandle stream = nullptr) const override
  {
    hipError_t err =
      (stream == nullptr) ? hipDeviceSynchronize() : hipStreamSynchronize(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::SYNCHRONIZATION_FAILED;
  }

  // Stream management
  eIcicleError create_stream(icicleStreamHandle* stream) const override
  {
    hipStream_t cudaStream;
    hipError_t err = hipStreamCreate(&cudaStream);
    *stream = reinterpret_cast<icicleStreamHandle>(cudaStream);
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_CREATION_FAILED;
  }

  eIcicleError destroy_stream(icicleStreamHandle stream) const override
  {
    hipError_t err = hipStreamDestroy(reinterpret_cast<hipStream_t>(stream));
    return (err == hipSuccess) ? eIcicleError::SUCCESS : eIcicleError::STREAM_DESTRUCTION_FAILED;
  }

  eIcicleError get_device_properties(DeviceProperties& properties) const override
  {
    properties.using_host_memory = false;
    properties.num_memory_regions = 1;
    properties.supports_pinned_memory = false; // TODO support it for compatible devices
    return eIcicleError::SUCCESS;
  }
};

REGISTER_DEVICE_API("CUDA", CudaDeviceAPI);
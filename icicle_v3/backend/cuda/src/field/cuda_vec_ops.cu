#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

#include "icicle/errors.h"
#include "icicle/vec_ops.h"
#include "gpu-utils/error_handler.h"
#include "error_translation.h"

#define MAX_THREADS_PER_BLOCK 256

template <typename E>
__global__ void mul_kernel(const E* scalar_vec, const E* element_vec, int n, E* result)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < n) { result[tid] = scalar_vec[tid] * element_vec[tid]; }
}

template <typename E, typename S>
__global__ void mul_scalar_kernel(const E* element_vec, const S scalar, int n, E* result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) { result[tid] = element_vec[tid] * (scalar); }
}

template <typename E>
__global__ void div_element_wise_kernel(const E* element_vec1, const E* element_vec2, int n, E* result)
{
  // TODO:implement better based on https://eprint.iacr.org/2008/199
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) { result[tid] = element_vec1[tid] * E::inverse(element_vec2[tid]); }
}

template <typename E>
__global__ void add_kernel(const E* element_vec1, const E* element_vec2, int n, E* result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) { result[tid] = element_vec1[tid] + element_vec2[tid]; }
}

template <typename E>
__global__ void sub_kernel(const E* element_vec1, const E* element_vec2, int n, E* result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) { result[tid] = element_vec1[tid] - element_vec2[tid]; }
}

template <typename E>
__global__ void transpose_kernel(const E* in, E* out, uint32_t row_size, uint32_t column_size)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= row_size * column_size) return;
  out[(tid % row_size) * column_size + (tid / row_size)] = in[tid];
}

template <typename E, void (*Kernel)(const E*, const E*, int, E*)>
hipError_t vec_op(const E* vec_a, const E* vec_b, int n, const VecOpsConfig& config, E* result)
{
  CHK_INIT_IF_RETURN();

  hipStream_t hip_stream = reinterpret_cast<hipStream_t>(config.stream);

  // Set the grid and block dimensions
  int num_threads = MAX_THREADS_PER_BLOCK;
  int num_blocks = (n + num_threads - 1) / num_threads;

  E *d_result, *d_alloc_vec_a, *d_alloc_vec_b;
  const E *d_vec_a, *d_vec_b;
  if (!config.is_a_on_device) {
    CHK_IF_RETURN(hipMallocAsync(&d_alloc_vec_a, n * sizeof(E), hip_stream));
    CHK_IF_RETURN(hipMemcpyAsync(d_alloc_vec_a, vec_a, n * sizeof(E), hipMemcpyHostToDevice, hip_stream));
    d_vec_a = d_alloc_vec_a;
  } else {
    d_vec_a = vec_a;
  }

  if (!config.is_b_on_device) {
    CHK_IF_RETURN(hipMallocAsync(&d_alloc_vec_b, n * sizeof(E), hip_stream));
    CHK_IF_RETURN(hipMemcpyAsync(d_alloc_vec_b, vec_b, n * sizeof(E), hipMemcpyHostToDevice, hip_stream));
    d_vec_b = d_alloc_vec_b;
  } else {
    d_vec_b = vec_b;
  }

  if (!config.is_result_on_device) {
    CHK_IF_RETURN(hipMallocAsync(&d_result, n * sizeof(E), hip_stream));
  } else {
    d_result = result;
  }

  // Call the kernel to perform element-wise operation
  Kernel<<<num_blocks, num_threads, 0, hip_stream>>>(d_vec_a, d_vec_b, n, d_result);

  if (!config.is_a_on_device) { CHK_IF_RETURN(hipFreeAsync(d_alloc_vec_a, hip_stream)); }
  if (!config.is_b_on_device) { CHK_IF_RETURN(hipFreeAsync(d_alloc_vec_b, hip_stream)); }

  if (!config.is_result_on_device) {
    CHK_IF_RETURN(hipMemcpyAsync(result, d_result, n * sizeof(E), hipMemcpyDeviceToHost, hip_stream));
    CHK_IF_RETURN(hipFreeAsync(d_result, hip_stream));
  }

  if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(hip_stream));

  return CHK_LAST();
}

template <typename E>
eIcicleError mul(const E* vec_a, const E* vec_b, int n, const VecOpsConfig& config, E* result)
{
  hipError_t err = vec_op<E, mul_kernel>(vec_a, vec_b, n, config, result);
  return translateCudaError(err);
}

template <typename E>
eIcicleError add(const E* vec_a, const E* vec_b, int n, const VecOpsConfig& config, E* result)
{
  hipError_t err = vec_op<E, add_kernel>(vec_a, vec_b, n, config, result);
  return translateCudaError(err);
}

template <typename E>
eIcicleError sub(const E* vec_a, const E* vec_b, int n, const VecOpsConfig& config, E* result)
{
  hipError_t err = vec_op<E, sub_kernel>(vec_a, vec_b, n, config, result);
  return translateCudaError(err);
}

template <typename E>
hipError_t transpose_matrix(
  const E* mat_in,
  E* mat_out,
  uint32_t row_size,
  uint32_t column_size,
  hipStream_t stream,
  bool on_device,
  bool is_async)
{
  int number_of_threads = MAX_THREADS_PER_BLOCK;
  int number_of_blocks = (row_size * column_size + number_of_threads - 1) / number_of_threads;

  const E* d_mat_in;
  E* d_allocated_input = nullptr;
  E* d_mat_out;
  if (!on_device) {
    CHK_IF_RETURN(hipMallocAsync(&d_allocated_input, row_size * column_size * sizeof(E), stream));
    CHK_IF_RETURN(
      hipMemcpyAsync(d_allocated_input, mat_in, row_size * column_size * sizeof(E), hipMemcpyHostToDevice, stream));

    CHK_IF_RETURN(hipMallocAsync(&d_mat_out, row_size * column_size * sizeof(E), stream));
    d_mat_in = d_allocated_input;
  } else {
    d_mat_in = mat_in;
    d_mat_out = mat_out;
  }

  transpose_kernel<<<number_of_blocks, number_of_threads, 0, stream>>>(d_mat_in, d_mat_out, row_size, column_size);

  if (!on_device) {
    CHK_IF_RETURN(
      hipMemcpyAsync(mat_out, d_mat_out, row_size * column_size * sizeof(E), hipMemcpyDeviceToHost, stream));
    CHK_IF_RETURN(hipFreeAsync(d_mat_out, stream));
    CHK_IF_RETURN(hipFreeAsync(d_allocated_input, stream));
  }
  if (!is_async) return CHK_STICKY(hipStreamSynchronize(stream));

  return CHK_LAST();
}

template <typename E>
__global__ void bit_reverse_kernel(const E* input, uint64_t n, unsigned shift, E* output)
{
  uint64_t tid = uint64_t(blockIdx.x) * blockDim.x + threadIdx.x;
  // Handling arbitrary vector size
  if (tid < n) {
    int reversed_index = __brevll(tid) >> shift;
    output[reversed_index] = input[tid];
  }
}
template <typename E>
__global__ void bit_reverse_inplace_kernel(E* input, uint64_t n, unsigned shift)
{
  uint64_t tid = uint64_t(blockIdx.x) * blockDim.x + threadIdx.x;
  // Handling arbitrary vector size
  if (tid < n) {
    int reversed_index = __brevll(tid) >> shift;
    if (reversed_index > tid) {
      E temp = input[tid];
      input[tid] = input[reversed_index];
      input[reversed_index] = temp;
    }
  }
}

template <typename E>
hipError_t bit_reverse_cuda_impl(const E* input, uint64_t size, const VecOpsConfig& cfg, E* output)
{
  hipStream_t hip_stream = reinterpret_cast<hipStream_t>(cfg.stream);

  if (size & (size - 1)) THROW_ICICLE_ERR(eIcicleError::INVALID_ARGUMENT, "bit_reverse: size must be a power of 2");
  if ((input == output) & (cfg.is_a_on_device != cfg.is_result_on_device))
    THROW_ICICLE_ERR(
      eIcicleError::INVALID_ARGUMENT, "bit_reverse: equal devices should have same is_on_device parameters");

  E* d_output;
  if (cfg.is_result_on_device) {
    d_output = output;
  } else {
    // allocate output on gpu
    CHK_IF_RETURN(hipMallocAsync(&d_output, sizeof(E) * size, hip_stream));
  }

  uint64_t shift = __builtin_clzll(size) + 1;
  uint64_t num_blocks = (size + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

  if ((input != output) & cfg.is_a_on_device) {
    bit_reverse_kernel<<<num_blocks, MAX_THREADS_PER_BLOCK, 0, hip_stream>>>(input, size, shift, d_output);
  } else {
    if (!cfg.is_a_on_device) {
      CHK_IF_RETURN(hipMemcpyAsync(d_output, input, sizeof(E) * size, hipMemcpyHostToDevice, hip_stream));
    }
    bit_reverse_inplace_kernel<<<num_blocks, MAX_THREADS_PER_BLOCK, 0, hip_stream>>>(d_output, size, shift);
  }
  if (!cfg.is_result_on_device) {
    CHK_IF_RETURN(hipMemcpyAsync(output, d_output, sizeof(E) * size, hipMemcpyDeviceToHost, hip_stream));
    CHK_IF_RETURN(hipFreeAsync(d_output, hip_stream));
  }
  if (!cfg.is_async) CHK_IF_RETURN(hipStreamSynchronize(hip_stream));
  return CHK_LAST();
}

/************************************ REGISTRATION ************************************/

#include "icicle/fields/field_config.h"
using namespace field_config;

template <typename F>
eIcicleError
add_cuda(const Device& device, const F* vec_a, const F* vec_b, int n, const VecOpsConfig& config, F* result)
{
  return add<F>(vec_a, vec_b, n, config, result);
}

template <typename F>
eIcicleError
sub_cuda(const Device& device, const F* vec_a, const F* vec_b, int n, const VecOpsConfig& config, F* result)
{
  return sub<F>(vec_a, vec_b, n, config, result);
}

template <typename F>
eIcicleError
mul_cuda(const Device& device, const F* vec_a, const F* vec_b, int n, const VecOpsConfig& config, F* result)
{
  return mul<F>(vec_a, vec_b, n, config, result);
}

template <typename F>
eIcicleError matrix_transpose_cuda(
  const Device& device, const F* in, uint32_t nof_rows, uint32_t nof_cols, const VecOpsConfig& config, F* out)
{
  // TODO relax this limitation
  ICICLE_ASSERT(config.is_a_on_device == config.is_result_on_device)
    << "CUDA matrix transpose expects both input and output on host or on device";

  // assert that it is not an inplace computation
  const bool is_on_device = config.is_a_on_device;
  const bool is_inplace = in == out;
  ICICLE_ASSERT(!is_on_device || !is_inplace) << "(CUDA) matrix-transpose-inplace not implemented";
  hipStream_t hip_stream = reinterpret_cast<hipStream_t>(config.stream);
  auto err = transpose_matrix(in, out, nof_cols, nof_rows, hip_stream, config.is_a_on_device, config.is_async);
  return translateCudaError(err);
}

template <typename T>
eIcicleError bit_reverse_cuda(const Device& device, const T* in, uint64_t size, const VecOpsConfig& config, T* out)
{
  auto err = bit_reverse_cuda_impl<T>(in, size, config, out);
  return translateCudaError(err);
}

REGISTER_VECTOR_ADD_BACKEND("CUDA", add_cuda<scalar_t>);
REGISTER_VECTOR_SUB_BACKEND("CUDA", sub_cuda<scalar_t>);
REGISTER_VECTOR_MUL_BACKEND("CUDA", mul_cuda<scalar_t>);
REGISTER_MATRIX_TRANSPOSE_BACKEND("CUDA", matrix_transpose_cuda<scalar_t>);
REGISTER_BIT_REVERSE_BACKEND("CUDA", bit_reverse_cuda<scalar_t>);

#ifdef EXT_FIELD
REGISTER_VECTOR_ADD_EXT_FIELD_BACKEND("CUDA", add_cuda<extension_t>);
REGISTER_VECTOR_SUB_EXT_FIELD_BACKEND("CUDA", sub_cuda<extension_t>);
REGISTER_VECTOR_MUL_EXT_FIELD_BACKEND("CUDA", mul_cuda<extension_t>);
REGISTER_MATRIX_TRANSPOSE_EXT_FIELD_BACKEND("CUDA", matrix_transpose_cuda<extension_t>);
REGISTER_BIT_REVERSE_EXT_FIELD_BACKEND("CUDA", bit_reverse_cuda<extension_t>);
#endif // EXT_FIELD

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

#include "icicle/errors.h"
#include "icicle/vec_ops.h"
#include "gpu-utils/error_handler.h"
#include "error_translation.h"

namespace icicle {

#define MAX_THREADS_PER_BLOCK 256

  template <typename E>
  __global__ void MontgomeryKernel(const E* input, int n, bool is_into, E* output)
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) { output[tid] = is_into ? E::to_montgomery(input[tid]) : E::from_montgomery(input[tid]); }
  }

  template <typename E>
  hipError_t ConvertMontgomery(const E* input, int n, bool is_into, const VecOpsConfig& config, E* output)
  {
    hipStream_t hip_stream = reinterpret_cast<hipStream_t>(config.stream);

    E *d_alloc_out = nullptr, *d_alloc_in = nullptr, *d_out;
    const E* d_in;
    if (!config.is_a_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_alloc_in, n * sizeof(E), hip_stream));
      CHK_IF_RETURN(hipMemcpyAsync(d_alloc_in, input, n * sizeof(E), hipMemcpyHostToDevice, hip_stream));
      d_in = d_alloc_in;
    } else {
      d_in = input;
    }

    if (!config.is_result_on_device) {
      CHK_IF_RETURN(hipMallocAsync(&d_alloc_out, n * sizeof(E), hip_stream));
      d_out = d_alloc_out;
    } else {
      d_out = output;
    }

    int num_threads = MAX_THREADS_PER_BLOCK;
    int num_blocks = (n + num_threads - 1) / num_threads;
    MontgomeryKernel<E><<<num_blocks, num_threads, 0, hip_stream>>>(d_in, n, is_into, d_out);

    if (d_alloc_in) { CHK_IF_RETURN(hipFreeAsync(d_alloc_in, hip_stream)); }
    if (d_alloc_out) {
      CHK_IF_RETURN(hipMemcpyAsync(output, d_out, n * sizeof(E), hipMemcpyDeviceToHost, hip_stream));
      CHK_IF_RETURN(hipFreeAsync(d_out, hip_stream));
    }
    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(hip_stream));

    return CHK_LAST();
  }

  /************************************ REGISTRATION ************************************/

#include "icicle/fields/field_config.h"
  using namespace field_config;

  template <typename F>
  eIcicleError convert_montgomery_cuda(
    const Device& device, const F* input, uint64_t n, bool is_into, const VecOpsConfig& config, F* output)
  {
    auto err = ConvertMontgomery<F>(input, n, is_into, config, output);
    return translateCudaError(err);
  }

  REGISTER_CONVERT_MONTGOMERY_BACKEND("CUDA", convert_montgomery_cuda<scalar_t>);

#ifdef EXT_FIELD
  REGISTER_CONVERT_MONTGOMERY_EXT_FIELD_BACKEND("CUDA", convert_montgomery_cuda<extension_t>);
#endif // EXT_FIELD

} // namespace icicle

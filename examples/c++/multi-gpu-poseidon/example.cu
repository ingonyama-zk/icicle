#include <iostream>
#include <thread>
#include <chrono>
#include <nvml.h>

#include "api/bn254.h"
#include "gpu-utils/error_handler.cuh"

using namespace poseidon;
using namespace bn254;

void checkCudaError(hipError_t error) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        // Handle the error, e.g., exit the program or throw an exception.
    }
}

// these global constants go into template calls
const int size_col = 11;

// this function executes the Poseidon thread
void threadPoseidon(device_context::DeviceContext ctx, unsigned size_partition, scalar_t * layers, scalar_t * column_hashes, PoseidonConstants<scalar_t> * constants) {
    hipError_t err_result =  CHK_STICKY(hipSetDevice(ctx.device_id));
    if (err_result != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err_result) << std::endl;
        return; 
    }
    // CHK_IF_RETURN(); I can't use it in a standard thread function
    PoseidonConfig column_config = {
        ctx,   // ctx
        false, // are_inputes_on_device
        false, // are_outputs_on_device
        false, // input_is_a_state
        false, // aligned
        false, // loop_state
        false, // is_async
        };
    hipError_t err = bn254_poseidon_hash_cuda(layers, column_hashes, (size_t) size_partition, size_col, *constants, column_config);
    checkCudaError(err);
}

using FpMilliseconds = std::chrono::duration<float, std::chrono::milliseconds::period>;
#define START_TIMER(timer) auto timer##_start = std::chrono::high_resolution_clock::now();
#define END_TIMER(timer, msg) printf("%s: %.0f ms\n", msg, FpMilliseconds(std::chrono::high_resolution_clock::now() - timer##_start).count());


#define CHECK_ALLOC(ptr) if ((ptr) == nullptr) { \
    std::cerr << "Memory allocation for '" #ptr "' failed." << std::endl; \
    exit(EXIT_FAILURE); \
}

int main() {
    const unsigned size_row = (1<<30);
    const unsigned nof_partitions = 64;
    const unsigned size_partition = size_row / nof_partitions;
    // layers is allocated only for one partition, need to reuse for different partitions
    const uint32_t size_layers = size_col * size_partition;
    
    nvmlInit();
    unsigned int deviceCount;
    nvmlDeviceGetCount(&deviceCount);
    std::cout << "Available GPUs: " << deviceCount << std::endl;

    for (unsigned int i = 0; i < deviceCount; ++i) {
        nvmlDevice_t device;
        nvmlMemory_t memory;
        char name[NVML_DEVICE_NAME_BUFFER_SIZE];
        nvmlDeviceGetHandleByIndex(i, &device);
        nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE);
        nvmlDeviceGetMemoryInfo(device, &memory);
        std::cout << "Device ID: " << i << ", Type: " << name << ", Memory Total/Free (MiB) " << memory.total/1024/1024 << "/"  << memory.free/1024/1024 << std::endl;
    }

    const unsigned memory_partition = sizeof(scalar_t)*(size_col+1)*size_partition/1024/1024;
    std::cout << "Required Memory (MiB) " << memory_partition << std::endl;

    //===============================================================================
    // Key: multiple devices are supported by device context
    //===============================================================================

    device_context::DeviceContext ctx0 = device_context::get_default_device_context();
    ctx0.device_id=0;
    device_context::DeviceContext ctx1 = device_context::get_default_device_context();
    ctx1.device_id=1;
    
    std::cout << "Allocate and initialize the memory for layers and hashes" << std::endl;
    scalar_t* layers0 = static_cast<scalar_t*>(malloc(size_layers * sizeof(scalar_t)));
    CHECK_ALLOC(layers0);
    scalar_t s = scalar_t::zero();
    for (unsigned i = 0; i < size_col*size_partition ; i++) {
        layers0[i] = s;
        s = s + scalar_t::one();
    }
    scalar_t* layers1 = static_cast<scalar_t*>(malloc(size_layers * sizeof(scalar_t)));
    CHECK_ALLOC(layers1);
    s = scalar_t::zero() + scalar_t::one();
    for (unsigned i = 0; i < size_col*size_partition ; i++) {
        layers1[i] = s;
        s = s + scalar_t::one();
    }

    scalar_t* column_hash0 = static_cast<scalar_t*>(malloc(size_partition * sizeof(scalar_t)));
    CHECK_ALLOC(column_hash0);
    scalar_t* column_hash1 = static_cast<scalar_t*>(malloc(size_partition * sizeof(scalar_t)));
    CHECK_ALLOC(column_hash1);

    PoseidonConstants<scalar_t> column_constants0, column_constants1;
    bn254_init_optimized_poseidon_constants_cuda(size_col, ctx0, &column_constants0);
    hipError_t err_result =  CHK_STICKY(hipSetDevice(ctx1.device_id));
    if (err_result != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err_result) << std::endl;
        return; 
    }
    bn254_init_optimized_poseidon_constants_cuda(size_col, ctx1, &column_constants1);

    std::cout << "Parallel execution of Poseidon threads" << std::endl;
    START_TIMER(parallel);
    std::thread thread0(threadPoseidon, ctx0, size_partition, layers0, column_hash0, &column_constants0);
    std::thread thread1(threadPoseidon, ctx1, size_partition, layers1, column_hash1, &column_constants1);

    // Wait for the threads to finish
    thread0.join();
    thread1.join();
    END_TIMER(parallel,"2 GPUs");
    std::cout << "Output Data from Thread 0: ";
    std::cout << column_hash0[0] << std::endl;
    std::cout << "Output Data from Thread 1: ";
    std::cout << column_hash1[0] << std::endl;

    std::cout << "Sequential execution of Poseidon threads" << std::endl;
    START_TIMER(sequential);
    std::thread thread2(threadPoseidon, ctx0, size_partition, layers0, column_hash0, &column_constants0);
    thread2.join();
    std::thread thread3(threadPoseidon, ctx0, size_partition, layers1, column_hash1, &column_constants0);
    thread3.join();
    END_TIMER(sequential,"1 GPU");
    std::cout << "Output Data from Thread 2: ";
    std::cout << column_hash0[0] << std::endl;
    std::cout << "Output Data from Thread 3: ";
    std::cout << column_hash1[0] << std::endl;

    nvmlShutdown();
    return 0;
}

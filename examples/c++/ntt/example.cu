#include <chrono>
#include <iostream>

// include NTT template

#include "curves/params/bn254.cuh"
#include "api/bn254.h"
using namespace bn254;
using namespace ntt;

// Operate on scalars
typedef scalar_t S;
typedef scalar_t E;

void print_elements(const unsigned n, E* elements)
{
  for (unsigned i = 0; i < n; i++) {
    std::cout << i << ": " << elements[i] << std::endl;
  }
}

void initialize_input(const unsigned ntt_size, const unsigned nof_ntts, E* elements)
{
  // Lowest Harmonics
  for (unsigned i = 0; i < ntt_size; i = i + 1) {
    elements[i] = E::one();
  }
  // print_elements(ntt_size, elements );
  // Highest Harmonics
  for (unsigned i = 1 * ntt_size; i < 2 * ntt_size; i = i + 2) {
    elements[i] = E::one();
    elements[i + 1] = E::neg(scalar_t::one());
  }
  // print_elements(ntt_size, &elements[1*ntt_size] );
}

int validate_output(const unsigned ntt_size, const unsigned nof_ntts, E* elements)
{
  int nof_errors = 0;
  E amplitude = E::from((uint32_t)ntt_size);
  // std::cout << "Amplitude: " << amplitude << std::endl;
  // Lowest Harmonics
  if (elements[0] != amplitude) {
    ++nof_errors;
    std::cout << "Error in lowest harmonics 0! " << std::endl;
    // print_elements(ntt_size, elements );
  } else {
    std::cout << "Validated lowest harmonics" << std::endl;
  }
  // Highest Harmonics
  if (elements[1 * ntt_size + ntt_size / 2] != amplitude) {
    ++nof_errors;
    std::cout << "Error in highest harmonics! " << std::endl;
    // print_elements(ntt_size, &elements[1*ntt_size] );
  } else {
    std::cout << "Validated highest harmonics" << std::endl;
  }
  return nof_errors;
}

using FpMilliseconds = std::chrono::duration<float, std::chrono::milliseconds::period>;
#define START_TIMER(timer) auto timer##_start = std::chrono::high_resolution_clock::now();
#define END_TIMER(timer, msg) printf("%s: %.0f ms\n", msg, FpMilliseconds(std::chrono::high_resolution_clock::now() - timer##_start).count());


int main(int argc, char* argv[])
{
  std::cout << "Icicle Examples: Number Theoretical Transform (NTT)" << std::endl;
  std::cout << "Example parameters" << std::endl;
  const unsigned log_ntt_size = 20;
  std::cout << "Log2(NTT size): " << log_ntt_size << std::endl;
  const unsigned ntt_size = 1 << log_ntt_size;
  std::cout << "NTT size: " << ntt_size << std::endl;
  const unsigned nof_ntts = 2;
  std::cout << "Number of NTTs: " << nof_ntts << std::endl;
  const unsigned batch_size = nof_ntts * ntt_size;

  std::cout << "Generating input data for lowest and highest harmonics" << std::endl;
  E* input;
  input = (E*)malloc(sizeof(E) * batch_size);
  initialize_input(ntt_size, nof_ntts, input);
  E* output;
  output = (E*)malloc(sizeof(E) * batch_size);

  std::cout << "Running NTT with on-host data" << std::endl;
  // Create a device context
  auto ctx = device_context::get_default_device_context();
  S basic_root = S::omega(log_ntt_size /*NTT_LOG_SIZE*/);
  bn254_initialize_domain(&basic_root, ctx, true);
  // Create an NTTConfig instance
  NTTConfig<S> config = default_ntt_config<S>();
  config.ntt_algorithm = NttAlgorithm::MixedRadix; 
  config.batch_size = nof_ntts;
  START_TIMER(MixedRadix);
  hipError_t err = bn254_ntt_cuda(input, ntt_size, NTTDir::kForward, config, output);
  END_TIMER(MixedRadix, "MixedRadix NTT");
  
  std::cout << "Validating output" << std::endl;
  validate_output(ntt_size, nof_ntts, output);

  config.ntt_algorithm = NttAlgorithm::Radix2; 
  START_TIMER(Radix2);
  err = bn254_ntt_cuda(input, ntt_size, NTTDir::kForward, config, output);
  END_TIMER(Radix2, "Radix2 NTT");

  std::cout << "Validating output" << std::endl;
  validate_output(ntt_size, nof_ntts, output);

  std::cout << "Cleaning-up memory" << std::endl;
  free(input);
  free(output);
  return 0;
}

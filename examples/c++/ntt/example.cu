#include <chrono>
#include <iostream>

// select the curve
#define CURVE_ID 1
// include NTT template
#include "appUtils/ntt/ntt.cu"
using namespace curve_config;

// Operate on scalars
typedef scalar_t S;
typedef scalar_t E;

void print_elements(const unsigned n, E * elements ) {
  for (unsigned i = 0; i < n; i++) {
    std::cout << i << ": " << elements[i] << std::endl;   
  }
}

void initialize_input(const unsigned ntt_size, const unsigned nof_ntts, E * elements ) {
  // Lowest Harmonics
  for (unsigned i = 0; i < ntt_size; i=i+1) {
    elements[i] = E::one();
  }
  // print_elements(ntt_size, elements );
  // Highest Harmonics
  for (unsigned i = 1*ntt_size; i < 2*ntt_size; i=i+2) {
    elements[i] =  E::one();
    elements[i+1] = E::neg(scalar_t::one());
  }
  // print_elements(ntt_size, &elements[1*ntt_size] );
}

int validate_output(const unsigned ntt_size, const unsigned nof_ntts, E* elements)
{
  int nof_errors = 0;
  E amplitude = E::from((uint32_t) ntt_size);
  // std::cout << "Amplitude: " << amplitude << std::endl;
  // Lowest Harmonics
  if (elements[0] != amplitude) {
    ++nof_errors;
    std::cout << "Error in lowest harmonics 0! " << std::endl;
    // print_elements(ntt_size, elements );
  } else {
    std::cout << "Validated lowest harmonics" << std::endl;
  }
  // Highest Harmonics 
  if (elements[1*ntt_size+ntt_size/2] != amplitude) {
    ++nof_errors;
    std::cout << "Error in highest harmonics! " << std::endl;
    // print_elements(ntt_size, &elements[1*ntt_size] );
  } else {
    std::cout << "Validated highest harmonics" << std::endl;
  }
  return nof_errors;
}

int main(int argc, char* argv[])
{
  std::cout << "Icicle Examples: Number Theoretical Transform (NTT)" << std::endl;
  std::cout << "Example parameters" << std::endl;
  const unsigned log_ntt_size = 20;
  std::cout << "Log2(NTT size): " << log_ntt_size << std::endl;
  const unsigned ntt_size = 1 << log_ntt_size;
  std::cout << "NTT size: " << ntt_size << std::endl;
  const unsigned nof_ntts = 2;
  std::cout << "Number of NTTs: " << nof_ntts << std::endl;
  const unsigned batch_size = nof_ntts * ntt_size;
  
  std::cout << "Generating input data for lowest and highest harmonics" << std::endl;
  E* input;
  input = (E*) malloc(sizeof(E) * batch_size);
  initialize_input(ntt_size, nof_ntts, input );
  E* output;
  output = (E*) malloc(sizeof(E) * batch_size);
  
  std::cout << "Running NTT with on-host data" << std::endl;
  hipStream_t stream;
  hipStreamCreate(&stream);
  // Create a device context
  auto ctx = device_context::get_default_device_context();
  // the next line is valid only for CURVE_ID 1 (will add support for other curves soon)
  S rou = S{ {0x53337857, 0x53422da9, 0xdbed349f, 0xac616632, 0x6d1e303, 0x27508aba, 0xa0ed063, 0x26125da1} };
  ntt::InitDomain(rou, ctx);
  // Create an NTTConfig instance
  ntt::NTTConfig<S> config=ntt::DefaultNTTConfig<S>();
  config.batch_size = nof_ntts;
  config.ctx.stream = stream;
  auto begin0 = std::chrono::high_resolution_clock::now();
  hipError_t err = ntt::NTT<S, E>(input, ntt_size, ntt::NTTDir::kForward, config, output);
  auto end0 = std::chrono::high_resolution_clock::now();
  auto elapsed0 = std::chrono::duration_cast<std::chrono::nanoseconds>(end0 - begin0);
  printf("On-device runtime: %.3f seconds\n", elapsed0.count() * 1e-9);
  validate_output(ntt_size, nof_ntts, output );
  hipStreamDestroy(stream);
  free(input);
  free(output);
  return 0;
}

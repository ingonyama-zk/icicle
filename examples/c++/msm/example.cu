#include <fstream>
#include <iostream>
#include <iomanip>

// include MSM template
#define CURVE_ID 1
#include "icicle/appUtils/msm/msm.cu"
using namespace curve_config;

int main(int argc, char* argv[])
{
  std::cout << "Icicle example: Muli-Scalar Multiplication (MSM)" << std::endl;
  std::cout << "Example parameters" << std::endl;
  int batch_size = 1;
  std::cout << "Batch size: " << batch_size << std::endl;
  unsigned msm_size = 1048576;
  std::cout << "MSM size: " << msm_size << std::endl;
  int N = batch_size * msm_size;
  
  std::cout << "Generating random inputs on-host" << std::endl;
  scalar_t* scalars = new scalar_t[N];
  affine_t* points = new affine_t[N];
  projective_t result;
  scalar_t::RandHostMany(scalars, N);
  projective_t::RandHostManyAffine(points, N);

  std::cout << "Using default MSM configuration with on-host inputs" << std::endl;
  auto config = msm::DefaultMSMConfig();
  config.batch_size = batch_size;
  
  std::cout << "Running MSM kernel" << std::endl;
  // Create two events to time the MSM kernel
  hipStream_t stream = config.ctx.stream;
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Record the start event on the stream
  hipEventRecord(start, stream);
  // Execute the MSM kernel
  msm::MSM<scalar_t, affine_t, projective_t>(scalars, points, msm_size, config, &result);
  // Record the stop event on the stream
  hipEventRecord(stop, stream);
  // Wait for the stop event to complete
  hipEventSynchronize(stop);
  // Calculate the elapsed time between the start and stop events
  hipEventElapsedTime(&time, start, stop);
  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Print the elapsed time
  std::cout << "Kernel runtime: " << std::fixed << std::setprecision(3) << time * 1e-3 << " sec." << std::endl;
  // Print the result
  std::cout << projective_t::to_affine(result) << std::endl;

  std::cout << "Copying inputs on-device" << std::endl;
  scalar_t* scalars_d;
  affine_t* points_d;
  projective_t* result_d;
  hipMalloc(&scalars_d, sizeof(scalar_t) * N);
  hipMalloc(&points_d, sizeof(affine_t) * N);
  hipMalloc(&result_d, sizeof(projective_t));
  hipMemcpy(scalars_d, scalars, sizeof(scalar_t) * N, hipMemcpyHostToDevice);
  hipMemcpy(points_d, points, sizeof(affine_t) * N, hipMemcpyHostToDevice);

  std::cout << "Reconfiguring MSM to use on-device inputs" << std::endl;
  config.are_results_on_device = true;
  config.are_scalars_on_device = true;
  config.are_points_on_device = true;

  std::cout << "Running MSM kernel with on-device inputs" << std::endl;
  // Create two events to time the MSM kernel
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Record the start event on the stream
  hipEventRecord(start, stream);
  // Execute the MSM kernel
  msm::MSM<scalar_t, affine_t, projective_t>(scalars_d, points_d, msm_size, config, result_d);
  // Record the stop event on the stream
  hipEventRecord(stop, stream);
  // Wait for the stop event to complete
  hipEventSynchronize(stop);
  // Calculate the elapsed time between the start and stop events
  hipEventElapsedTime(&time, start, stop);
  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Print the elapsed time
  std::cout << "Kernel runtime: " << std::fixed << std::setprecision(3) << time * 1e-3 << " sec." << std::endl;
  // Copy the result back to the host
  hipMemcpy(&result, result_d, sizeof(projective_t), hipMemcpyDeviceToHost);
  // Print the result
  std::cout << projective_t::to_affine(result) << std::endl;
  // Free the device memory
  hipFree(scalars_d);
  hipFree(points_d);
  hipFree(result_d);
  hipStreamDestroy(stream);
  return 0;
}
